#include "hip/hip_runtime.h"
// The file dbl_polynomials_kernels.cu defines the kernels with prototypes
// in dbl_polynomials_kernels.h.

#include <iostream>
#include <iomanip>
#ifdef winwalltime
#include "gettimeofday4win.h"
#else
#include <sys/time.h>
#endif
#include "job_coordinates.h"
#include "write_gpu_timings.h"
#include "dbl_polynomials_kernels.h"

// The constant d_shmemsize is the bound on the shared memory size.

#define d_shmemsize 256

using namespace std;

__global__ void dbl_padded_convjobs
 ( double *data, int *in1idx, int *in2idx, int *outidx, int dim )
{
   const int bdx = blockIdx.x;           // index to the convolution job
   const int tdx = threadIdx.x;          // index to the output of the job
   const int idx1 = in1idx[bdx] + tdx;
   const int idx2 = in2idx[bdx] + tdx;
   const int idx3 = outidx[bdx] + tdx;

   __shared__ double xv[d_shmemsize];
   __shared__ double yv[2*d_shmemsize];
   __shared__ double zv[d_shmemsize];

   int ydx = dim + tdx;

   xv[tdx] = data[idx1];  // loading first input
   yv[tdx] = 0.0;         // padded with zeros
   yv[ydx] = data[idx2];  // loading second input

   zv[tdx] = xv[0]*yv[ydx];

   for(int i=1; i<dim; i++)
   {
      ydx = dim + tdx - i;
      zv[tdx] = zv[tdx] + xv[i]*yv[ydx];
   }
   data[idx3] = zv[tdx]; // storing the output
}

__global__ void cmplx_padded_convjobs
 ( double *datare, double *dataim,
   int *in1idx, int *in2idx, int *outidx, int dim )
{
   const int bdx = blockIdx.x;           // index to the convolution job
   const int tdx = threadIdx.x;          // index to the output of the job
   const int idx1 = in1idx[bdx] + tdx;
   const int idx2 = in2idx[bdx] + tdx;
   const int idx3 = outidx[bdx] + tdx;

   __shared__ double xvre[d_shmemsize];
   __shared__ double xvim[d_shmemsize];
   __shared__ double yvre[2*d_shmemsize];
   __shared__ double yvim[2*d_shmemsize];
   __shared__ double zvre[d_shmemsize];
   __shared__ double zvim[d_shmemsize];

   int ydx = dim + tdx;

   xvre[tdx] = datare[idx1];  // loading first input
   xvim[tdx] = dataim[idx1];
   yvre[tdx] = 0.0;           // padded with zeros
   yvim[tdx] = 0.0; 
   yvre[ydx] = datare[idx2];  // loading second input
   yvim[ydx] = dataim[idx2];

   zvre[tdx] = xvre[0]*yvre[ydx] - xvim[0]*yvim[ydx];
   zvim[tdx] = xvre[0]*yvim[ydx] + xvim[0]*yvre[ydx];

   for(int i=1; i<dim; i++)
   {
      ydx = dim + tdx - i;
      zvre[tdx] += xvre[i]*yvre[ydx] - xvim[i]*yvim[ydx];
      zvim[tdx] += xvre[i]*yvim[ydx] + xvim[i]*yvre[ydx];
   }
   datare[idx3] = zvre[tdx]; // storing the output
   dataim[idx3] = zvim[tdx];
}

__global__ void dbl_update_addjobs
 ( double *data, int *in1idx, int *in2idx, int *outidx, int dim )
{
   const int bdx = blockIdx.x;           // index to the addition job
   const int tdx = threadIdx.x;          // index to the output of the job
   const int idx1 = in1idx[bdx] + tdx;
   const int idx2 = in2idx[bdx] + tdx;
   const int idx3 = outidx[bdx] + tdx;

   __shared__ double xv[d_shmemsize];
   __shared__ double yv[d_shmemsize];
   __shared__ double zv[d_shmemsize];

   xv[tdx] = data[idx1];  // loading first input
   yv[tdx] = data[idx2];  // loading second input

   zv[tdx] = xv[tdx] + yv[tdx];

   data[idx3] = zv[tdx]; // storing the output
}

__global__ void cmplx_update_addjobs
 ( double *datare, double *dataim,
   int *in1idx, int *in2idx, int *outidx, int dim )
{
   const int bdx = blockIdx.x;           // index to the addition job
   const int tdx = threadIdx.x;          // index to the output of the job
   const int idx1 = in1idx[bdx] + tdx;
   const int idx2 = in2idx[bdx] + tdx;
   const int idx3 = outidx[bdx] + tdx;

   __shared__ double xvre[d_shmemsize];
   __shared__ double xvim[d_shmemsize];
   __shared__ double yvre[d_shmemsize];
   __shared__ double yvim[d_shmemsize];
   __shared__ double zvre[d_shmemsize];
   __shared__ double zvim[d_shmemsize];

   xvre[tdx] = datare[idx1];  // loading first input
   xvim[tdx] = dataim[idx1];
   yvre[tdx] = datare[idx2];  // loading second input
   yvim[tdx] = dataim[idx2];

   zvre[tdx] = xvre[tdx] + yvre[tdx]; // adding real parts
   zvim[tdx] = xvim[tdx] + yvim[tdx]; // adding imaginary parts

   datare[idx3] = zvre[tdx]; // storing the output
   dataim[idx3] = zvim[tdx];
}

void dbl_convoluted_data_to_output
 ( double *data, double **output, int dim, int nbr, int deg, int *nvr,
   int **idx, int *fstart, int *bstart, int *cstart, bool verbose )
{
   const int deg1 = deg+1;
   int ix0,ix1,ix2;

   for(int i=0; i<=deg; i++) output[dim][i] = data[i];
   for(int i=0; i<dim; i++)
      for(int j=0; j<=deg; j++) output[i][j] = 0.0;

   for(int k=0; k<nbr; k++)
   {
      ix1 = fstart[k] + (nvr[k]-1)*deg1;
      
      if(verbose)
         cout << "monomial " << k << " update starts at " << ix1 << endl;

      for(int i=0; i<=deg; i++) output[dim][i] += data[ix1++];

      ix0 = idx[k][0];
      if(nvr[k] == 1)
      {
         ix1 = (1 + k)*deg1;
            
         for(int i=0; i<=deg; i++) output[ix0][i] += data[ix1++];
      }
      else if(nvr[k] > 1)
      {                               // update first and last derivative
         ix2 = nvr[k]-3;
         if(ix2 < 0) ix2 = 0;
         ix1 = bstart[k] + ix2*deg1;

         for(int i=0; i<=deg; i++) output[ix0][i] += data[ix1++];

         ix2 = nvr[k]-2;
         ix1 = fstart[k] + ix2*deg1;
         ix0 = idx[k][ix2+1];

         for(int i=0; i<=deg; i++) output[ix0][i] += data[ix1++];

         if(nvr[k] > 2)                   // update all other derivatives
         {
            for(int j=1; j<nvr[k]-1; j++)
            {
               ix0 = idx[k][j];            // j-th variable in monomial k
               ix1 = cstart[k] + (j-1)*deg1;

               if(verbose)
                  cout << "monomial " << k << " derivative " << ix0
                       << " update starts at " << ix1 << endl;

               for(int i=0; i<=deg; i++) output[ix0][i] += data[ix1++];
            }
         }
      }
   }
}

void cmplx_convoluted_data_to_output
 ( double *datare, double *dataim, double **outputre, double **outputim,
   int dim, int nbr, int deg, int *nvr,
   int **idx, int *fstart, int *bstart, int *cstart, bool verbose )
{
   const int deg1 = deg+1;
   int ix0,ix1,ix2;

   for(int i=0; i<=deg; i++)
   {
      outputre[dim][i] = datare[i]; outputim[dim][i] = dataim[i];
   }
   for(int i=0; i<dim; i++)
      for(int j=0; j<=deg; j++)
      {
         outputre[i][j] = 0.0; outputim[i][j] = 0.0;
      }

   for(int k=0; k<nbr; k++)
   {
      ix1 = fstart[k] + (nvr[k]-1)*deg1;
      
      if(verbose)
         cout << "monomial " << k << " update starts at " << ix1 << endl;

      for(int i=0; i<=deg; i++)
      {
         outputre[dim][i] += datare[ix1];
         outputim[dim][i] += dataim[ix1++];
      }
      ix0 = idx[k][0];
      if(nvr[k] == 1)
      {
         ix1 = (1 + k)*deg1;
            
         for(int i=0; i<=deg; i++)
         {
            outputre[ix0][i] += datare[ix1];
            outputim[ix0][i] += dataim[ix1++];
         }
      }
      else if(nvr[k] > 1)
      {                               // update first and last derivative
         ix2 = nvr[k]-3;
         if(ix2 < 0) ix2 = 0;
         ix1 = bstart[k] + ix2*deg1;

         for(int i=0; i<=deg; i++)
         {
            outputre[ix0][i] += datare[ix1];
            outputim[ix0][i] += dataim[ix1++];
         }
         ix2 = nvr[k]-2;
         ix1 = fstart[k] + ix2*deg1;
         ix0 = idx[k][ix2+1];

         for(int i=0; i<=deg; i++)
         {
            outputre[ix0][i] += datare[ix1];
            outputim[ix0][i] += dataim[ix1++];
         }
         if(nvr[k] > 2)                   // update all other derivatives
         {
            for(int j=1; j<nvr[k]-1; j++)
            {
               ix0 = idx[k][j];            // j-th variable in monomial k
               ix1 = cstart[k] + (j-1)*deg1;

               if(verbose)
                  cout << "monomial " << k << " derivative " << ix0
                       << " update starts at " << ix1 << endl;

               for(int i=0; i<=deg; i++)
               {
                  outputre[ix0][i] += datare[ix1];
                  outputim[ix0][i] += dataim[ix1++];
               }
            }
         }
      }
   }
}

void dbl_added_data_to_output
 ( double *data, double **output, int dim, int nbr, int deg, int *nvr,
   int **idx, int *fstart, int *bstart, int *cstart, AdditionJobs jobs,
   bool verbose )
{
   const int deg1 = deg + 1;
   const int lastmon = nbr-1;
   const int lastidx = nvr[lastmon]-1;
   int ix;

   ix = fstart[lastmon] + lastidx*deg1;

   if(verbose)
      cout << "Updating value starting at " << ix << " in data." << endl;

   for(int i=0; i<=deg; i++) output[dim][i] = data[ix++];

   int cnt = jobs.get_differential_count(0);
   if(cnt == 0) // it could be there is no first variable anywhere ...
   {
      for(int i=0; i<=deg; i++) output[0][i] = 0.0;
   }
   else
   {
      int ix0 = jobs.get_differential_index(0,cnt);
      int ix2 = nvr[ix0]-3;
      if(ix2 < 0) ix2 = 0; // on GPU, one backward item less

      ix = bstart[ix0] + ix2*deg1;
      
      if(verbose)
         cout << "Updating derivative 0 at " << ix << " in data." << endl;

      for(int i=0; i<=deg; i++) output[0][i] = data[ix++];

      for(int k=1; k<dim; k++) // updating all other derivatives
      {
         int cnt = jobs.get_differential_count(k);
         if(cnt == 0) // it could be there is no variable k anywhere ...
         {
            for(int i=0; i<=deg; i++) output[k][i] = 0.0;
         }
         else
         {
            int ix0 = jobs.get_differential_index(k,cnt);
   
            if(idx[ix0][0] == k) // k is first variable of monomial
            {
               int ix2 = nvr[ix0]-3;
               if(ix2 < 0) ix2 = 0;

               if(verbose)
                  cout << "Updating derivative " << k 
                       << " at " << ix << " in data." << endl;

               ix = bstart[ix0] + ix2*deg1;

               for(int i=0; i<=deg; i++) output[k][i] = data[ix++];
            }
            else if(idx[ix0][nvr[ix0]-1] == k) // k is last variable
            {
               int ix2 = nvr[ix0]-2;
   
               if(verbose)
                  cout << "Updating derivative " << k 
                       << " at " << ix << " in data." << endl;

               ix = fstart[ix0] + ix2*deg1;

               for(int i=0; i<=deg; i++) output[k][i] = data[ix++];
            }
            else // derivative is in some cross product
            {
               int ix2 = jobs.position(nvr[ix0],idx[ix0],k) - 1;
   
               if(verbose)
                  cout << "Updating derivative " << k 
                       << " at " << ix << " in data." << endl;

               ix = cstart[ix0] + ix2*deg1;

               for(int i=0; i<=deg; i++) output[k][i] = data[ix++];
            }
         }
      }
   }
}

void cmplx_added_data_to_output
 ( double *datare, double *dataim, double **outputre, double **outputim,
   int dim, int nbr, int deg, int *nvr, int **idx,
   int *fstart, int *bstart, int *cstart, AdditionJobs jobs, bool verbose )
{
   const int deg1 = deg + 1;
   const int lastmon = nbr-1;
   const int lastidx = nvr[lastmon]-1;
   int ix;

   ix = fstart[lastmon] + lastidx*deg1;

   if(verbose)
      cout << "Updating value starting at " << ix << " in data." << endl;

   for(int i=0; i<=deg; i++)
   {
      outputre[dim][i] = datare[ix]; outputim[dim][i] = dataim[ix++];
   }
   int cnt = jobs.get_differential_count(0);
   if(cnt == 0) // it could be there is no first variable anywhere ...
   {
      for(int i=0; i<=deg; i++)
      {
         outputre[0][i] = 0.0; outputim[0][i] = 0.0;
      }
   }
   else
   {
      int ix0 = jobs.get_differential_index(0,cnt);
      int ix2 = nvr[ix0]-3;
      if(ix2 < 0) ix2 = 0; // on GPU, one backward item less

      ix = bstart[ix0] + ix2*deg1;
      
      if(verbose)
         cout << "Updating derivative 0 at " << ix << " in data." << endl;

      for(int i=0; i<=deg; i++)
      {
         outputre[0][i] = datare[ix]; outputim[0][i] = dataim[ix++];
      }
      for(int k=1; k<dim; k++) // updating all other derivatives
      {
         int cnt = jobs.get_differential_count(k);
         if(cnt == 0) // it could be there is no variable k anywhere ...
         {
            for(int i=0; i<=deg; i++)
            {
               outputre[k][i] = 0.0; outputim[k][i] = 0.0;
            }
         }
         else
         {
            int ix0 = jobs.get_differential_index(k,cnt);
   
            if(idx[ix0][0] == k) // k is first variable of monomial
            {
               int ix2 = nvr[ix0]-3;
               if(ix2 < 0) ix2 = 0;

               if(verbose)
                  cout << "Updating derivative " << k 
                       << " at " << ix << " in data." << endl;

               ix = bstart[ix0] + ix2*deg1;

               for(int i=0; i<=deg; i++)
               {
                  outputre[k][i] = datare[ix]; outputim[k][i] = dataim[ix++];
               }
            }
            else if(idx[ix0][nvr[ix0]-1] == k) // k is last variable
            {
               int ix2 = nvr[ix0]-2;
   
               if(verbose)
                  cout << "Updating derivative " << k 
                       << " at " << ix << " in data." << endl;

               ix = fstart[ix0] + ix2*deg1;

               for(int i=0; i<=deg; i++)
               {
                  outputre[k][i] = datare[ix]; outputim[k][i] = dataim[ix++];
               }
            }
            else // derivative is in some cross product
            {
               int ix2 = jobs.position(nvr[ix0],idx[ix0],k) - 1;
   
               if(verbose)
                  cout << "Updating derivative " << k 
                       << " at " << ix << " in data." << endl;

               ix = cstart[ix0] + ix2*deg1;

               for(int i=0; i<=deg; i++)
               {
                  outputre[k][i] = datare[ix]; outputim[k][i] = dataim[ix++];
               }
            }
         }
      }
   }
}

void GPU_dbl_poly_evaldiff
 ( int BS, int dim, int nbr, int deg, int *nvr, int **idx,
   double *cst, double **cff, double **input, double **output,
   ConvolutionJobs cnvjobs, AdditionJobs addjobs,
   double *cnvlapms, double *addlapms, double *elapsedms,
   double *walltimesec, bool verbose )
{
   const int deg1 = deg+1;
   const int totalcff = coefficient_count(dim,nbr,deg,nvr);

   int *fstart = new int[nbr];
   int *bstart = new int[nbr];
   int *cstart = new int[nbr];
   int *fsums = new int[nbr];
   int *bsums = new int[nbr];
   int *csums = new int[nbr];

   coefficient_indices
      (dim,nbr,deg,nvr,fsums,bsums,csums,fstart,bstart,cstart);

   if(verbose)
      write_coefficient_indices
         (totalcff,nbr,fsums,fstart,bsums,bstart,csums,cstart);

   double *data_h = new double[totalcff];        // data on host
   int ix = 0;
   for(int i=0; i<deg1; i++) data_h[ix++] = cst[i];
   for(int i=0; i<nbr; i++)
      for(int j=0; j<deg1; j++) data_h[ix++] = cff[i][j];
   for(int i=0; i<dim; i++)
      for(int j=0; j<deg1; j++) data_h[ix++] = input[i][j];

   double *data_d;                               // device data
   const size_t szdata = totalcff*sizeof(double);
   hipMalloc((void**)&data_d,szdata);
   hipMemcpy(data_d,data_h,szdata,hipMemcpyHostToDevice);

   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   *cnvlapms = 0.0;
   *addlapms = 0.0;
   float milliseconds;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);
   for(int k=0; k<cnvjobs.get_depth(); k++)
   {
      const int jobnbr = cnvjobs.get_layer_count(k);
      int *in1ix_h = new int[jobnbr];
      int *in2ix_h = new int[jobnbr];
      int *outix_h = new int[jobnbr];

      if(verbose) cout << "preparing convolution jobs at layer "
                       << k << " ..." << endl;

      convjobs_coordinates(cnvjobs,k,in1ix_h,in2ix_h,outix_h,dim,nbr,deg,nvr,
                           fstart,bstart,cstart,verbose);
      if(deg1 == BS)
      {
         int *in1ix_d; // first input on device
         int *in2ix_d; // second input on device
         int *outix_d; // output indices on device
         const size_t szjobidx = jobnbr*sizeof(int);
         hipMalloc((void**)&in1ix_d,szjobidx);
         hipMalloc((void**)&in2ix_d,szjobidx);
         hipMalloc((void**)&outix_d,szjobidx);
         hipMemcpy(in1ix_d,in1ix_h,szjobidx,hipMemcpyHostToDevice);
         hipMemcpy(in2ix_d,in2ix_h,szjobidx,hipMemcpyHostToDevice);
         hipMemcpy(outix_d,outix_h,szjobidx,hipMemcpyHostToDevice);

         if(verbose)
            cout << "launching " << jobnbr << " blocks of " << BS
                 << " threads ..." << endl;
         
         hipEventRecord(start);
         dbl_padded_convjobs<<<jobnbr,BS>>>
            (data_d,in1ix_d,in2ix_d,outix_d,deg1);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *cnvlapms += milliseconds;
      }
      free(in1ix_h); free(in2ix_h); free(outix_h);
   }
   for(int k=0; k<addjobs.get_depth(); k++)
   {
      const int jobnbr = addjobs.get_layer_count(k);
      int *in1ix_h = new int[jobnbr];
      int *in2ix_h = new int[jobnbr];
      int *outix_h = new int[jobnbr];

      if(verbose) cout << "preparing addition jobs at layer "
                       << k << " ..." << endl;

      addjobs_coordinates(addjobs,k,in1ix_h,in2ix_h,outix_h,dim,nbr,deg,nvr,
                          fstart,bstart,cstart,verbose);
      if(deg1 == BS)
      {
         int *in1ix_d; // first input on device
         int *in2ix_d; // second input on device
         int *outix_d; // output indices on device
         const size_t szjobidx = jobnbr*sizeof(int);
         hipMalloc((void**)&in1ix_d,szjobidx);
         hipMalloc((void**)&in2ix_d,szjobidx);
         hipMalloc((void**)&outix_d,szjobidx);
         hipMemcpy(in1ix_d,in1ix_h,szjobidx,hipMemcpyHostToDevice);
         hipMemcpy(in2ix_d,in2ix_h,szjobidx,hipMemcpyHostToDevice);
         hipMemcpy(outix_d,outix_h,szjobidx,hipMemcpyHostToDevice);

         if(verbose)
            cout << "launching " << jobnbr << " blocks of " << BS
                 << " threads ..." << endl;

         hipEventRecord(start);
         dbl_update_addjobs<<<jobnbr,BS>>>
            (data_d,in1ix_d,in2ix_d,outix_d,deg1);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *addlapms += milliseconds;
      }
      free(in1ix_h); free(in2ix_h); free(outix_h);
   }
   gettimeofday(&endtime,0);
   hipMemcpy(data_h,data_d,szdata,hipMemcpyDeviceToHost);
   *elapsedms = *cnvlapms + *addlapms;
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   // dbl_convoluted_data_to_output
   //    (data_h,output,dim,nbr,deg,nvr,idx,fstart,bstart,cstart,verbose);

   dbl_added_data_to_output
      (data_h,output,dim,nbr,deg,nvr,idx,fstart,bstart,cstart,
       addjobs,verbose);

   if(verbose) write_GPU_timings(*cnvlapms,*addlapms,*elapsedms,*walltimesec);
}

void GPU_cmplx_poly_evaldiff
 ( int BS, int dim, int nbr, int deg, int *nvr, int **idx,
   double *cstre, double *cstim, double **cffre, double **cffim,
   double **inputre, double **inputim, double **outputre, double **outputim,
   ConvolutionJobs cnvjobs, AdditionJobs addjobs,
   double *cnvlapms, double *addlapms, double *elapsedms,
   double *walltimesec, bool verbose )
{
   const int deg1 = deg+1;
   const int totalcff = coefficient_count(dim,nbr,deg,nvr);

   int *fstart = new int[nbr];
   int *bstart = new int[nbr];
   int *cstart = new int[nbr];
   int *fsums = new int[nbr];
   int *bsums = new int[nbr];
   int *csums = new int[nbr];

   coefficient_indices
      (dim,nbr,deg,nvr,fsums,bsums,csums,fstart,bstart,cstart);

   if(verbose)
      write_coefficient_indices
         (totalcff,nbr,fsums,fstart,bsums,bstart,csums,cstart);

   double *datare_h = new double[totalcff];        // data on host
   double *dataim_h = new double[totalcff];
   int ix = 0;
   for(int i=0; i<deg1; i++)
   {
      datare_h[ix] = cstre[i]; dataim_h[ix++] = cstim[i];
   }
   for(int i=0; i<nbr; i++)
      for(int j=0; j<deg1; j++)
      {
         datare_h[ix] = cffre[i][j]; dataim_h[ix++] = cffim[i][j];
      }
   for(int i=0; i<dim; i++)
      for(int j=0; j<deg1; j++)
      {
         datare_h[ix] = inputre[i][j]; dataim_h[ix++] = inputim[i][j];
      }

   double *datare_d;                               // device data
   double *dataim_d;
   const size_t szdata = totalcff*sizeof(double);
   hipMalloc((void**)&datare_d,szdata);
   hipMalloc((void**)&dataim_d,szdata);
   hipMemcpy(datare_d,datare_h,szdata,hipMemcpyHostToDevice);
   hipMemcpy(dataim_d,dataim_h,szdata,hipMemcpyHostToDevice);

   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   *cnvlapms = 0.0;
   *addlapms = 0.0;
   float milliseconds;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);
   for(int k=0; k<cnvjobs.get_depth(); k++)
   {
      const int jobnbr = cnvjobs.get_layer_count(k);
      int *in1ix_h = new int[jobnbr];
      int *in2ix_h = new int[jobnbr];
      int *outix_h = new int[jobnbr];

      if(verbose) cout << "preparing convolution jobs at layer "
                       << k << " ..." << endl;

      convjobs_coordinates(cnvjobs,k,in1ix_h,in2ix_h,outix_h,dim,nbr,deg,nvr,
                           fstart,bstart,cstart,verbose);
      if(deg1 == BS)
      {
         int *in1ix_d; // first input on device
         int *in2ix_d; // second input on device
         int *outix_d; // output indices on device
         const size_t szjobidx = jobnbr*sizeof(int);
         hipMalloc((void**)&in1ix_d,szjobidx);
         hipMalloc((void**)&in2ix_d,szjobidx);
         hipMalloc((void**)&outix_d,szjobidx);
         hipMemcpy(in1ix_d,in1ix_h,szjobidx,hipMemcpyHostToDevice);
         hipMemcpy(in2ix_d,in2ix_h,szjobidx,hipMemcpyHostToDevice);
         hipMemcpy(outix_d,outix_h,szjobidx,hipMemcpyHostToDevice);

         if(verbose)
            cout << "launching " << jobnbr << " blocks of " << BS
                 << " threads ..." << endl;
         
         hipEventRecord(start);
         cmplx_padded_convjobs<<<jobnbr,BS>>>
            (datare_d,dataim_d,in1ix_d,in2ix_d,outix_d,deg1);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *cnvlapms += milliseconds;
      }
      free(in1ix_h); free(in2ix_h); free(outix_h);
   }
   for(int k=0; k<addjobs.get_depth(); k++)
   {
      const int jobnbr = addjobs.get_layer_count(k);
      int *in1ix_h = new int[jobnbr];
      int *in2ix_h = new int[jobnbr];
      int *outix_h = new int[jobnbr];

      if(verbose) cout << "preparing addition jobs at layer "
                       << k << " ..." << endl;

      addjobs_coordinates(addjobs,k,in1ix_h,in2ix_h,outix_h,dim,nbr,deg,nvr,
                          fstart,bstart,cstart,verbose);
      if(deg1 == BS)
      {
         int *in1ix_d; // first input on device
         int *in2ix_d; // second input on device
         int *outix_d; // output indices on device
         const size_t szjobidx = jobnbr*sizeof(int);
         hipMalloc((void**)&in1ix_d,szjobidx);
         hipMalloc((void**)&in2ix_d,szjobidx);
         hipMalloc((void**)&outix_d,szjobidx);
         hipMemcpy(in1ix_d,in1ix_h,szjobidx,hipMemcpyHostToDevice);
         hipMemcpy(in2ix_d,in2ix_h,szjobidx,hipMemcpyHostToDevice);
         hipMemcpy(outix_d,outix_h,szjobidx,hipMemcpyHostToDevice);

         if(verbose)
            cout << "launching " << jobnbr << " blocks of " << BS
                 << " threads ..." << endl;

         hipEventRecord(start);
         cmplx_update_addjobs<<<jobnbr,BS>>>
            (datare_d,dataim_d,in1ix_d,in2ix_d,outix_d,deg1);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *addlapms += milliseconds;
      }
      free(in1ix_h); free(in2ix_h); free(outix_h);
   }
   gettimeofday(&endtime,0);
   hipMemcpy(datare_h,datare_d,szdata,hipMemcpyDeviceToHost);
   hipMemcpy(dataim_h,dataim_d,szdata,hipMemcpyDeviceToHost);
   *elapsedms = *cnvlapms + *addlapms;
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   // cmplx_convoluted_data_to_output
   //    (datare_h,dataim_h,outputre,outputim,
   //     dim,nbr,deg,nvr,idx,fstart,bstart,cstart,verbose);

   cmplx_added_data_to_output
      (datare_h,dataim_h,outputre,outputim,dim,nbr,deg,nvr,idx,
       fstart,bstart,cstart,addjobs,verbose);

   if(verbose) write_GPU_timings(*cnvlapms,*addlapms,*elapsedms,*walltimesec);
}
