#include "hip/hip_runtime.h"
/* The file dbl2_tabs_kernels.cu defines the functions specified in
 * the file dbl2_tabs_kernels.h. */

#include <iostream>
#ifdef winwalltime
#include "gettimeofday4win.h"
#else
#include <sys/time.h>
#endif
#ifdef gpufun
#include "double_double_gpufun.cu"
#endif
#include "dbl2_tabs_kernels.h"
#include "dbl_tabs_flopcounts.h"

using namespace std;

__global__ void dbl2_small_invert_upper 
( int dim, double *Uhi, double *Ulo, double *invUhi, double *invUlo )
{
   const int k = threadIdx.x; // thread k computes k-th column of inverse

   __shared__ double Ucolhi[tabsdd_shmemsize];
   __shared__ double Ucollo[tabsdd_shmemsize];
   __shared__ double invUrowshi[tabsdd_shmemsize];
   __shared__ double invUrowslo[tabsdd_shmemsize];

   double rhshi,rhslo,xvalhi,xvallo,acchi,acclo;

   int colidx = dim*(dim-1);          // start with the last column

   Ucolhi[k] = Uhi[colidx+k];         // load the last column
   Ucollo[k] = Ulo[colidx+k];
   rhshi = ((double) int(k == dim-1));  // right hand side for each thread
   rhslo = 0.0;
   int rowidx = (dim - 1)*dim + k;      // the row index in the inverse

   __syncthreads();
   // invUrows[rowidx] = rhs/Ucol[k]; // last row of the inverse
   ddg_div(rhshi,rhslo,Ucolhi[k],Ucollo[k],
           &invUrowshi[rowidx],&invUrowslo[rowidx]);

   for(int i=dim-2; i>=0; i--)        // compute row with index i
   {
      rhshi = ((double) int(k == i)); // set rhs for i-th unit vector
      rhslo = 0.0;

      for(int j=i+1; j<dim; j++)
      {
         colidx = dim*j;              // need column j of U

         Ucolhi[k] = Uhi[colidx+k];
         Ucollo[k] = Ulo[colidx+k];

         rowidx = j*dim + k;          // need solution value

         xvalhi = invUrowshi[rowidx];
         xvallo = invUrowslo[rowidx];

         __syncthreads();
         // rhs = rhs - Ucol[i]*xval; // update right hand side
         ddg_mul(Ucolhi[i],Ucollo[i],xvalhi,xvallo,&acchi,&acclo);
         ddg_dec(&rhshi,&rhslo,acchi,acclo);
      }
      rowidx = i*dim + k;             // save in i-th row of inverse

      colidx = dim*i;                 // need column i of U
      Ucolhi[k] = Uhi[colidx+k];
      Ucollo[k] = Ulo[colidx+k];

      __syncthreads();
      // invUrows[rowidx] = rhs/Ucol[i];
      ddg_div(rhshi,rhslo,Ucolhi[i],Ucollo[i],
              &invUrowshi[rowidx],&invUrowslo[rowidx]);
   }
   rowidx = 0;
   for(int i=0; i<dim; i++)
   {
      __syncthreads();
      invUhi[rowidx+k] = invUrowshi[rowidx+k];
      invUlo[rowidx+k] = invUrowslo[rowidx+k];
      rowidx = rowidx + dim;
   }
}

__global__ void cmplx2_small_invert_upper
 ( int dim, double *Urehi, double *Urelo, double *Uimhi, double *Uimlo,
   double *invUrehi, double *invUrelo, double *invUimhi, double *invUimlo )
{
   const int k = threadIdx.x; // thread k computes k-th column of inverse

   __shared__ double Ucolrehi[tabsdd_shmemsize];
   __shared__ double Ucolrelo[tabsdd_shmemsize];
   __shared__ double Ucolimhi[tabsdd_shmemsize];
   __shared__ double Ucolimlo[tabsdd_shmemsize];
   __shared__ double invUrowsrehi[tabsdd_shmemsize];
   __shared__ double invUrowsrelo[tabsdd_shmemsize];
   __shared__ double invUrowsimhi[tabsdd_shmemsize];
   __shared__ double invUrowsimlo[tabsdd_shmemsize];

   double rhsrehi,rhsrelo,rhsimhi,rhsimlo;
   double xvalrehi,xvalrelo,xvalimhi,xvalimlo;
   double acc1hi,acc1lo,acc2hi,acc2lo;
   double acc3hi,acc3lo,acc4hi,acc4lo;
   double invrehi,invrelo,invimhi,invimlo,denhi,denlo;

   int colidx = dim*(dim-1);             // start with the last column

   Ucolrehi[k] = Urehi[colidx+k];        // load the last column
   Ucolrelo[k] = Urelo[colidx+k];
   Ucolimhi[k] = Uimhi[colidx+k];
   Ucolimlo[k] = Uimlo[colidx+k];
   rhsrehi = ((double) int(k == dim-1)); // right hand side for each thread
   rhsrelo = 0.0;
   rhsimhi = 0.0;
   rhsimlo = 0.0;
   int rowidx = (dim - 1)*dim + k;       // the row index in the inverse

   __syncthreads();
   // invUrows[rowidx] = rhs/Ucol[k];    // last row of the inverse
   ddg_mul(Ucolrehi[k],Ucolrelo[k],Ucolrehi[k],Ucolrelo[k],&denhi,&denlo);
   ddg_mul(Ucolimhi[k],Ucolimlo[k],Ucolimhi[k],Ucolimlo[k],&acc1hi,&acc1lo);
   ddg_inc(&denhi,&denlo,acc1hi,acc1lo);
   ddg_div(Ucolrehi[k],Ucolrelo[k],denhi,denlo,&invrehi,&invrelo);
   ddg_div(Ucolimhi[k],Ucolimlo[k],denhi,denlo,&invimhi,&invimlo);
   ddg_minus(&invimhi,&invimlo);
   ddg_mul(rhsrehi,rhsrelo,invrehi,invrelo,&acc1hi,&acc1lo);
   ddg_mul(rhsimhi,rhsimlo,invimhi,invimlo,&acc2hi,&acc2lo);
   ddg_mul(rhsimhi,rhsimlo,invrehi,invrelo,&acc3hi,&acc3lo);
   ddg_mul(rhsrehi,rhsrelo,invimhi,invimlo,&acc4hi,&acc4lo);
   ddg_dec(&acc1hi,&acc1lo,acc2hi,acc2lo);
   invUrowsrehi[rowidx] = acc1hi;
   invUrowsrelo[rowidx] = acc1lo;
   ddg_inc(&acc3hi,&acc3lo,acc4hi,acc4lo);
   invUrowsimhi[rowidx] = acc3hi;
   invUrowsimlo[rowidx] = acc3lo;

   for(int i=dim-2; i>=0; i--)        // compute row with index i
   {
      rhsrehi = ((double) int(k == i)); // set rhs for i-th unit vector
      rhsrelo = 0.0;
      rhsimhi = 0.0;
      rhsimlo = 0.0;

      for(int j=i+1; j<dim; j++)
      {
         colidx = dim*j;              // need column j of U

         Ucolrehi[k] = Urehi[colidx+k];
         Ucolrelo[k] = Urelo[colidx+k];
         Ucolimhi[k] = Uimhi[colidx+k];
         Ucolimlo[k] = Uimlo[colidx+k];

         rowidx = j*dim + k;          // need solution value

         xvalrehi = invUrowsrehi[rowidx];
         xvalrelo = invUrowsrelo[rowidx];
         xvalimhi = invUrowsimhi[rowidx];
         xvalimlo = invUrowsimlo[rowidx];

         __syncthreads();
         // rhs = rhs - Ucol[i]*xval; // update right hand side
         ddg_mul(Ucolrehi[i],Ucolrelo[i],xvalrehi,xvalrelo,&acc1hi,&acc1lo);
         ddg_mul(Ucolimhi[i],Ucolimlo[i],xvalimhi,xvalimlo,&acc2hi,&acc2lo);
         ddg_mul(Ucolimhi[i],Ucolimlo[i],xvalrehi,xvalrelo,&acc3hi,&acc3lo);
         ddg_mul(Ucolrehi[i],Ucolrelo[i],xvalimhi,xvalimlo,&acc4hi,&acc4lo);
         ddg_dec(&rhsrehi,&rhsrelo,acc1hi,acc1lo);
         ddg_inc(&rhsrehi,&rhsrelo,acc2hi,acc2lo);
         ddg_dec(&rhsimhi,&rhsimlo,acc3hi,acc3lo);
         ddg_dec(&rhsimhi,&rhsimlo,acc4hi,acc4lo);
      }
      rowidx = i*dim + k;             // save in i-th row of inverse

      colidx = dim*i;                 // need column i of U
      Ucolrehi[k] = Urehi[colidx+k];
      Ucolrelo[k] = Urelo[colidx+k];
      Ucolimhi[k] = Uimhi[colidx+k];
      Ucolimlo[k] = Uimlo[colidx+k];

      __syncthreads();
      // invUrows[rowidx] = rhs/Ucol[i];
      ddg_mul(Ucolrehi[i],Ucolrelo[i],Ucolrehi[i],Ucolrelo[i],&denhi,&denlo);
      ddg_mul(Ucolimhi[i],Ucolimlo[i],Ucolimhi[i],Ucolimlo[i],&acc1hi,&acc1lo);
      __syncthreads();
      ddg_inc(&denhi,&denlo,acc1hi,acc1lo);
      ddg_div(Ucolrehi[i],Ucolrelo[i],denhi,denlo,&invrehi,&invrelo);
      ddg_div(Ucolimhi[i],Ucolimlo[i],denhi,denlo,&invimhi,&invimlo);
      ddg_minus(&invimhi,&invimlo);
      ddg_mul(rhsrehi,rhsrelo,invrehi,invrelo,&acc1hi,&acc1lo);
      ddg_mul(rhsimhi,rhsimlo,invimhi,invimlo,&acc2hi,&acc2lo);
      ddg_mul(rhsimhi,rhsimlo,invrehi,invrelo,&acc3hi,&acc3lo);
      ddg_mul(rhsrehi,rhsrelo,invimhi,invimlo,&acc4hi,&acc4lo);
      __syncthreads();
      ddg_dec(&acc1hi,&acc1lo,acc2hi,acc2lo);
      invUrowsrehi[rowidx] = acc1hi;
      invUrowsrelo[rowidx] = acc1lo;
      ddg_inc(&acc3hi,&acc3lo,acc4hi,acc4lo);
      invUrowsimhi[rowidx] = acc3hi;
      invUrowsimlo[rowidx] = acc3lo;
   }
   rowidx = 0;
   for(int i=0; i<dim; i++)
   {
      __syncthreads();
      invUrehi[rowidx+k] = invUrowsrehi[rowidx+k];
      invUrelo[rowidx+k] = invUrowsrelo[rowidx+k];
      invUimhi[rowidx+k] = invUrowsimhi[rowidx+k];
      invUimlo[rowidx+k] = invUrowsimlo[rowidx+k];
      rowidx = rowidx + dim;
   }
}

__global__ void dbl2_medium_invert_upper
 ( int dim, double *Uhi, double *Ulo, double *invUhi, double *invUlo)
{
   const int k = threadIdx.x;  // thread k computes k-th column of inverse

   __shared__ double Ucolhi[tabsdd_shmemsize];      // one column of U
   __shared__ double Ucollo[tabsdd_shmemsize];      // one column of U
   __shared__ double invUrowhi[tabsdd_shmemsize];   // one row of invU
   __shared__ double invUrowlo[tabsdd_shmemsize];   // one row of invU

   double rhshi,rhslo,xvalhi,xvallo,acchi,acclo;

   int colidx = dim*(dim-1);           // start with the last column

   Ucolhi[k] = Uhi[colidx+k];          // load the last column
   Ucollo[k] = Ulo[colidx+k];
   rhshi = ((double) int(k == dim-1)); // right hand side for each thread
   rhslo = 0.0;
   int rowidx = (dim - 1)*dim + k;     // the row index in the inverse

   // invUrow[k] = rhs/Ucol[k];          // last row of the inverse
   ddg_div(rhshi,rhslo,Ucolhi[k],Ucollo[k],&invUrowhi[k],&invUrowlo[k]);
   invUhi[rowidx] = invUrowhi[k];     // store the last row into invU
   invUlo[rowidx] = invUrowlo[k]; 

   for(int i=dim-2; i>=0; i--)        // compute row with index i
   {
      rhshi = ((double) int(k == i)); // set rhs for i-th unit vector
      rhslo = 0.0;

      for(int j=i+1; j<dim; j++)
      {
         colidx = dim*j;              // need column j of U
         Ucolhi[k] = Uhi[colidx+k];
         Ucollo[k] = Ulo[colidx+k];

         rowidx = j*dim + k;            // need solution value
         invUrowhi[k] = invUhi[rowidx]; // load invU row into invUrow
         invUrowlo[k] = invUlo[rowidx];
         xvalhi = invUrowhi[k];
         xvallo = invUrowlo[k];

         __syncthreads();
         // rhs = rhs - Ucol[i]*xval;    // update right hand side
         ddg_mul(Ucolhi[i],Ucollo[i],xvalhi,xvallo,&acchi,&acclo);
         ddg_dec(&rhshi,&rhslo,acchi,acclo);
      }
      colidx = dim*i;                 // need column i of U
      Ucolhi[k] = Uhi[colidx+k];
      Ucollo[k] = Ulo[colidx+k];
      rowidx = i*dim + k;             // save in i-th row of inverse

      __syncthreads();
      // invUrow[k] = rhs/Ucol[i];
      ddg_div(rhshi,rhslo,Ucolhi[i],Ucollo[i],&invUrowhi[k],&invUrowlo[k]);
      invUhi[rowidx] = invUrowhi[k];
      invUlo[rowidx] = invUrowlo[k];
   }
}

__global__ void cmplx2_medium_invert_upper
 ( int dim, double *Urehi, double *Urelo, double *Uimhi, double *Uimlo,
   double *invUrehi, double *invUrelo, double *invUimhi, double *invUimlo )
{
   const int k = threadIdx.x;  // thread k computes k-th column of inverse

   __shared__ double Ucolrehi[tabsdd_shmemsize];    // one column of U
   __shared__ double Ucolrelo[tabsdd_shmemsize]; 
   __shared__ double Ucolimhi[tabsdd_shmemsize];
   __shared__ double Ucolimlo[tabsdd_shmemsize]; 
   __shared__ double invUrowrehi[tabsdd_shmemsize]; // one row of invU
   __shared__ double invUrowrelo[tabsdd_shmemsize]; 
   __shared__ double invUrowimhi[tabsdd_shmemsize]; 
   __shared__ double invUrowimlo[tabsdd_shmemsize]; 

   double rhsrehi,rhsrelo,rhsimhi,rhsimlo;
   double xvalrehi,xvalrelo,xvalimhi,xvalimlo;
   double acc1hi,acc1lo,acc2hi,acc2lo;
   double acc3hi,acc3lo,acc4hi,acc4lo;
   double invrehi,invrelo,invimhi,invimlo,denhi,denlo;

   int colidx = dim*(dim-1);           // start with the last column

   Ucolrehi[k] = Urehi[colidx+k];      // load the last column
   Ucolrelo[k] = Urelo[colidx+k];
   Ucolimhi[k] = Uimhi[colidx+k];
   Ucolimlo[k] = Uimlo[colidx+k];
   rhsrehi = ((double) int(k == dim-1)); // right hand side for each thread
   rhsrelo = 0.0;
   rhsimhi = 0.0;
   rhsimlo = 0.0;
   int rowidx = (dim - 1)*dim + k;     // the row index in the inverse

   __syncthreads();
   // invUrow[k] = rhs/Ucol[k];          // last row of the inverse
   ddg_mul(Ucolrehi[k],Ucolrelo[k],Ucolrehi[k],Ucolrelo[k],&denhi,&denlo);
   ddg_mul(Ucolimhi[k],Ucolimlo[k],Ucolimhi[k],Ucolimlo[k],&acc1hi,&acc1lo);
   ddg_inc(&denhi,&denlo,acc1hi,acc1lo);
   ddg_div(Ucolrehi[k],Ucolrelo[k],denhi,denlo,&invrehi,&invrelo);
   ddg_div(Ucolimhi[k],Ucolimlo[k],denhi,denlo,&invimhi,&invimlo);
   ddg_minus(&invimhi,&invimlo);
   ddg_mul(rhsrehi,rhsrelo,invrehi,invrelo,&acc1hi,&acc1lo);
   ddg_mul(rhsimhi,rhsimlo,invimhi,invimlo,&acc2hi,&acc2lo);
   ddg_mul(rhsimhi,rhsimlo,invrehi,invrelo,&acc3hi,&acc3lo);
   ddg_mul(rhsrehi,rhsrelo,invimhi,invimlo,&acc4hi,&acc4lo);
   ddg_dec(&acc1hi,&acc1lo,acc2hi,acc2lo);
   invUrowrehi[k] = acc1hi;
   invUrowrelo[k] = acc1lo;
   ddg_inc(&acc3hi,&acc3lo,acc4hi,acc4lo);
   invUrowimhi[k] = acc3hi;
   invUrowimlo[k] = acc3lo;
   invUrehi[rowidx] = invUrowrehi[k];     // store the last row into invU
   invUrelo[rowidx] = invUrowrelo[k]; 
   invUimhi[rowidx] = invUrowimhi[k];
   invUimlo[rowidx] = invUrowimlo[k]; 

   for(int i=dim-2; i>=0; i--)        // compute row with index i
   {
      rhsrehi = ((double) int(k == i)); // set rhs for i-th unit vector
      rhsrelo = 0.0;
      rhsimhi = 0.0;
      rhsimlo = 0.0;

      for(int j=i+1; j<dim; j++)
      {
         colidx = dim*j;              // need column j of U
         Ucolrehi[k] = Urehi[colidx+k];
         Ucolrelo[k] = Urelo[colidx+k];
         Ucolimhi[k] = Uimhi[colidx+k];
         Ucolimlo[k] = Uimlo[colidx+k];

         rowidx = j*dim + k;            // need solution value
         invUrowrehi[k] = invUrehi[rowidx]; // load invU row into invUrow
         invUrowrelo[k] = invUrelo[rowidx];
         invUrowimhi[k] = invUimhi[rowidx];
         invUrowimlo[k] = invUimlo[rowidx];
         xvalrehi = invUrowrehi[k];
         xvalrelo = invUrowrelo[k];
         xvalimhi = invUrowimhi[k];
         xvalimlo = invUrowimlo[k];

         __syncthreads();
         // rhs = rhs - Ucol[i]*xval;    // update right hand side
         ddg_mul(Ucolrehi[i],Ucolrelo[i],xvalrehi,xvalrelo,&acc1hi,&acc1lo);
         ddg_mul(Ucolimhi[i],Ucolimlo[i],xvalimhi,xvalimlo,&acc2hi,&acc2lo);
         ddg_mul(Ucolimhi[i],Ucolimlo[i],xvalrehi,xvalrelo,&acc3hi,&acc3lo);
         ddg_mul(Ucolrehi[i],Ucolrelo[i],xvalimhi,xvalimlo,&acc4hi,&acc4lo);
         ddg_dec(&rhsrehi,&rhsrelo,acc1hi,acc1lo);
         ddg_inc(&rhsrehi,&rhsrelo,acc2hi,acc2lo);
         ddg_dec(&rhsimhi,&rhsimlo,acc3hi,acc3lo);
         ddg_dec(&rhsimhi,&rhsimlo,acc4hi,acc4lo);
      }
      colidx = dim*i;                 // need column i of U
      Ucolrehi[k] = Urehi[colidx+k];
      Ucolrelo[k] = Urelo[colidx+k];
      Ucolimhi[k] = Uimhi[colidx+k];
      Ucolimlo[k] = Uimlo[colidx+k];
      rowidx = i*dim + k;             // save in i-th row of inverse

      __syncthreads();
      // invUrow[k] = rhs/Ucol[i];
      ddg_mul(Ucolrehi[i],Ucolrelo[i],Ucolrehi[i],Ucolrelo[i],&denhi,&denlo);
      ddg_mul(Ucolimhi[i],Ucolimlo[i],Ucolimhi[i],Ucolimlo[i],&acc1hi,&acc1lo);
      ddg_inc(&denhi,&denlo,acc1hi,acc1lo);
      ddg_div(Ucolrehi[i],Ucolrelo[i],denhi,denlo,&invrehi,&invrelo);
      ddg_div(Ucolimhi[i],Ucolimlo[i],denhi,denlo,&invimhi,&invimlo);
      ddg_minus(&invimhi,&invimlo);
      ddg_mul(rhsrehi,rhsrelo,invrehi,invrelo,&acc1hi,&acc1lo);
      ddg_mul(rhsimhi,rhsimlo,invimhi,invimlo,&acc2hi,&acc2lo);
      ddg_mul(rhsimhi,rhsimlo,invrehi,invrelo,&acc3hi,&acc3lo);
      ddg_mul(rhsrehi,rhsrelo,invimhi,invimlo,&acc4hi,&acc4lo);
      ddg_dec(&acc1hi,&acc1lo,acc2hi,acc2lo);
      invUrowrehi[k] = acc1hi;
      invUrowrelo[k] = acc1lo;
      ddg_inc(&acc3hi,&acc3lo,acc4hi,acc4lo);
      invUrowimhi[k] = acc3hi;
      invUrowimlo[k] = acc3lo;
      invUrehi[rowidx] = invUrowrehi[k];
      invUrelo[rowidx] = invUrowrelo[k];
      invUimhi[rowidx] = invUrowimhi[k];
      invUimlo[rowidx] = invUrowimlo[k];
   }
}

__global__ void  dbl2_invert_tiles
 ( int dim, double *Uhi, double *Ulo, double *invUhi, double *invUlo )
{
   const int B = blockIdx.x;   // block index
   const int k = threadIdx.x;  // thread k computes k-th column of inverse
   const int offset = dim*dim*B; // offset in U and invU

   __shared__ double Ucolhi[tabsdd_shmemsize];      // one column of U
   __shared__ double Ucollo[tabsdd_shmemsize];
   __shared__ double invUrowhi[tabsdd_shmemsize];   // one row of invU
   __shared__ double invUrowlo[tabsdd_shmemsize]; 

   double rhshi,rhslo,xvalhi,xvallo,acchi,acclo;

   int colidx = offset + dim*(dim-1); // start with the last column

   Ucolhi[k] = Uhi[colidx+k];         // load the last column
   Ucollo[k] = Ulo[colidx+k];
   rhshi = ((double) int(k == dim-1));  // right hand side for each thread
   rhslo = 0.0;
   int rowidx = offset + (dim - 1)*dim + k; // row index in the inverse

   // invUrow[k] = rhs/Ucol[k];       // last row of the inverse
   invUhi[rowidx] = 0.0;      // initialize in case of zero divisor
   invUlo[rowidx] = 0.0;
   if(1.0 + Ucolhi[k] != 1.0)
   {
      ddg_div(rhshi,rhslo,Ucolhi[k],Ucollo[k],&invUrowhi[k],&invUrowlo[k]);
      invUhi[rowidx] = invUrowhi[k];     // store the last row into invU
      invUlo[rowidx] = invUrowlo[k];
   }
   for(int i=dim-2; i>=0; i--)        // compute row with index i
   {
      rhshi = ((double) int(k == i));   // set rhs for i-th unit vector
      rhslo = 0.0;

      for(int j=i+1; j<dim; j++)
      {
         colidx = offset + dim*j;     // need column j of U
         Ucolhi[k] = Uhi[colidx+k];
         Ucollo[k] = Ulo[colidx+k];

         rowidx = offset + j*dim + k; // need solution value
         invUrowhi[k] = invUhi[rowidx]; // load invU row into invUrow
         invUrowlo[k] = invUlo[rowidx]; // load invU row into invUrow
         xvalhi = invUrowhi[k];
         xvallo = invUrowlo[k];

         __syncthreads();
         // rhs = rhs - Ucol[i]*xval;    // update right hand side
         ddg_mul(Ucolhi[i],Ucollo[i],xvalhi,xvallo,&acchi,&acclo);
         ddg_dec(&rhshi,&rhslo,acchi,acclo);
      }
      colidx = offset + dim*i;        // need column i of U
      Ucolhi[k] = Uhi[colidx+k];
      Ucollo[k] = Ulo[colidx+k];
      rowidx = offset + i*dim + k;    // save in i-th row of inverse

      __syncthreads();
      // invUrow[k] = rhs/Ucol[i];
      invUhi[rowidx] = 0.0;
      invUlo[rowidx] = 0.0;
      if(1.0 + Ucolhi[i] != 1.0)
      {
         ddg_div(rhshi,rhslo,Ucolhi[i],Ucollo[i],&invUrowhi[k],&invUrowlo[k]);
         invUhi[rowidx] = invUrowhi[k];
         invUlo[rowidx] = invUrowlo[k];
      }
   }
}

__global__ void  cmplx2_invert_tiles
 ( int dim, double *Urehi, double *Urelo, double *Uimhi, double *Uimlo,
   double *invUrehi, double *invUrelo, double *invUimhi, double *invUimlo )
{
   const int B = blockIdx.x;   // block index
   const int k = threadIdx.x;  // thread k computes k-th column of inverse
   const int offset = dim*dim*B; // offset in U and invU

   __shared__ double Ucolrehi[tabsdd_shmemsize];    // one column of U
   __shared__ double Ucolrelo[tabsdd_shmemsize];
   __shared__ double Ucolimhi[tabsdd_shmemsize]; 
   __shared__ double Ucolimlo[tabsdd_shmemsize];
   __shared__ double invUrowrehi[tabsdd_shmemsize];   // one row of invU
   __shared__ double invUrowrelo[tabsdd_shmemsize]; 
   __shared__ double invUrowimhi[tabsdd_shmemsize];
   __shared__ double invUrowimlo[tabsdd_shmemsize]; 

   double rhsrehi,rhsrelo,rhsimhi,rhsimlo;
   double xvalrehi,xvalrelo,xvalimhi,xvalimlo;
   double acc1hi,acc1lo,acc2hi,acc2lo;
   double acc3hi,acc3lo,acc4hi,acc4lo;
   double invrehi,invrelo,invimhi,invimlo,denhi,denlo;

   int colidx = offset + dim*(dim-1); // start with the last column

   Ucolrehi[k] = Urehi[colidx+k];       // load the last column
   Ucolrelo[k] = Urelo[colidx+k];
   Ucolimhi[k] = Uimhi[colidx+k];
   Ucolimlo[k] = Uimlo[colidx+k];
   rhsrehi = ((double) int(k == dim-1));  // right hand side for each thread
   rhsrelo = 0.0;
   rhsimhi = 0.0;
   rhsimlo = 0.0;
   int rowidx = offset + (dim - 1)*dim + k; // row index in the inverse

   // invUrow[k] = rhs/Ucol[k];       // last row of the inverse
   ddg_mul(Ucolrehi[k],Ucolrelo[k],Ucolrehi[k],Ucolrelo[k],&denhi,&denlo);
   ddg_mul(Ucolimhi[k],Ucolimlo[k],Ucolimhi[k],Ucolimlo[k],&acc1hi,&acc1lo);
   ddg_inc(&denhi,&denlo,acc1hi,acc1lo);

   invUrehi[rowidx] = 0.0;  // initialize in case of zero denominator
   invUrelo[rowidx] = 0.0;
   invUimhi[rowidx] = 0.0;
   invUimlo[rowidx] = 0.0;
   
   if(1.0 + denhi != 1.0)
   {
      ddg_div(Ucolrehi[k],Ucolrelo[k],denhi,denlo,&invrehi,&invrelo);
      ddg_div(Ucolimhi[k],Ucolimlo[k],denhi,denlo,&invimhi,&invimlo);

      ddg_minus(&invimhi,&invimlo);
      ddg_mul(rhsrehi,rhsrelo,invrehi,invrelo,&acc1hi,&acc1lo);
      ddg_mul(rhsimhi,rhsimlo,invimhi,invimlo,&acc2hi,&acc2lo);
      ddg_mul(rhsimhi,rhsimlo,invrehi,invrelo,&acc3hi,&acc3lo);
      ddg_mul(rhsrehi,rhsrelo,invimhi,invimlo,&acc4hi,&acc4lo);
      ddg_dec(&acc1hi,&acc1lo,acc2hi,acc2lo);
      invUrowrehi[k] = acc1hi;
      invUrowrelo[k] = acc1lo;
      ddg_inc(&acc3hi,&acc3lo,acc4hi,acc4lo);
      invUrowimhi[k] = acc3hi;
      invUrowimlo[k] = acc3lo;
      invUrehi[rowidx] = invUrowrehi[k];     // store the last row into invU
      invUrelo[rowidx] = invUrowrelo[k];
      invUimhi[rowidx] = invUrowimhi[k];
      invUimlo[rowidx] = invUrowimlo[k];
   }
   __syncthreads();
   for(int i=dim-2; i>=0; i--)        // compute row with index i
   {
      rhsrehi = ((double) int(k == i));   // set rhs for i-th unit vector
      rhsrelo = 0.0;
      rhsimhi = 0.0;
      rhsimlo = 0.0;

      for(int j=i+1; j<dim; j++)
      {
         colidx = offset + dim*j;        // need column j of U
         Ucolrehi[k] = Urehi[colidx+k];
         Ucolrelo[k] = Urelo[colidx+k];
         Ucolimhi[k] = Uimhi[colidx+k];
         Ucolimlo[k] = Uimlo[colidx+k];

         rowidx = offset + j*dim + k;       // need solution value
         invUrowrehi[k] = invUrehi[rowidx]; // load invU row into invUrow
         invUrowrelo[k] = invUrelo[rowidx];
         invUrowimhi[k] = invUimhi[rowidx];
         invUrowimlo[k] = invUimlo[rowidx];
         xvalrehi = invUrowrehi[k];
         xvalrelo = invUrowrelo[k];
         xvalimhi = invUrowimhi[k];
         xvalimlo = invUrowimlo[k];

         __syncthreads();
         // rhs = rhs - Ucol[i]*xval;    // update right hand side
         ddg_mul(Ucolrehi[i],Ucolrelo[i],xvalrehi,xvalrelo,&acc1hi,&acc1lo);
         ddg_mul(Ucolimhi[i],Ucolimlo[i],xvalimhi,xvalimlo,&acc2hi,&acc2lo);
         ddg_mul(Ucolimhi[i],Ucolimlo[i],xvalrehi,xvalrelo,&acc3hi,&acc3lo);
         ddg_mul(Ucolrehi[i],Ucolrelo[i],xvalimhi,xvalimlo,&acc4hi,&acc4lo);
         ddg_dec(&rhsrehi,&rhsrelo,acc1hi,acc1lo);
         ddg_inc(&rhsrehi,&rhsrelo,acc2hi,acc2lo);
         ddg_dec(&rhsimhi,&rhsimlo,acc3hi,acc3lo);
         ddg_dec(&rhsimhi,&rhsimlo,acc4hi,acc4lo);
      }
      colidx = offset + dim*i;        // need column i of U
      Ucolrehi[k] = Urehi[colidx+k];
      Ucolrelo[k] = Urelo[colidx+k];
      Ucolimhi[k] = Uimhi[colidx+k];
      Ucolimlo[k] = Uimlo[colidx+k];
      rowidx = offset + i*dim + k;    // save in i-th row of inverse

      __syncthreads();
      // invUrow[k] = rhs/Ucol[i];
      ddg_mul(Ucolrehi[i],Ucolrelo[i],Ucolrehi[i],Ucolrelo[i],&denhi,&denlo);
      ddg_mul(Ucolimhi[i],Ucolimlo[i],Ucolimhi[i],Ucolimlo[i],&acc1hi,&acc1lo);
      ddg_inc(&denhi,&denlo,acc1hi,acc1lo);

      invUrehi[rowidx] = 0.0; // initialize in case of zero denominator
      invUrelo[rowidx] = 0.0;
      invUimhi[rowidx] = 0.0;
      invUimlo[rowidx] = 0.0;

      if(1.0 + denhi != 1.0)
      {
         ddg_div(Ucolrehi[i],Ucolrelo[i],denhi,denlo,&invrehi,&invrelo);
         ddg_div(Ucolimhi[i],Ucolimlo[i],denhi,denlo,&invimhi,&invimlo);
         ddg_minus(&invimhi,&invimlo);
         ddg_mul(rhsrehi,rhsrelo,invrehi,invrelo,&acc1hi,&acc1lo);
         ddg_mul(rhsimhi,rhsimlo,invimhi,invimlo,&acc2hi,&acc2lo);
         ddg_mul(rhsimhi,rhsimlo,invrehi,invrelo,&acc3hi,&acc3lo);
         ddg_mul(rhsrehi,rhsrelo,invimhi,invimlo,&acc4hi,&acc4lo);
         ddg_dec(&acc1hi,&acc1lo,acc2hi,acc2lo);
         invUrowrehi[k] = acc1hi;
         invUrowrelo[k] = acc1lo;
         ddg_inc(&acc3hi,&acc3lo,acc4hi,acc4lo);
         invUrowimhi[k] = acc3hi;
         invUrowimlo[k] = acc3lo;
         invUrehi[rowidx] = invUrowrehi[k];
         invUrelo[rowidx] = invUrowrelo[k];
         invUimhi[rowidx] = invUrowimhi[k];
         invUimlo[rowidx] = invUrowimlo[k];
      }
   }
}

__global__ void dbl2_multiply_inverse
 ( int dim, int idx, double *invUhi, double *invUlo,
   double *whi, double *wlo )
{
   const int k = threadIdx.x;     // thread k computes k-th product
   const int rhsoff = dim*idx;    // offset for the right hand size
   const int offset = dim*rhsoff; // offset for diagonal tile

   __shared__ double workhi[tabsdd_shmemsize];      // copy of w
   __shared__ double worklo[tabsdd_shmemsize];      // copy of w

   workhi[k] = whi[rhsoff+k];
   worklo[k] = wlo[rhsoff+k];

   double resulthi = 0.0; // each thread stores its product in result
   double resultlo = 0.0;
   double coeffhi,coefflo,acchi,acclo;

   for(int j=0; j<dim; j++)  // column j of the inverse diagonal tile
   {
      coeffhi = invUhi[offset+k*dim+j]; // thread k does row k
      coefflo = invUlo[offset+k*dim+j];
      // result = result + coeff*work[j];
      ddg_mul(coeffhi,coefflo,workhi[j],worklo[j],&acchi,&acclo);
      ddg_inc(&resulthi,&resultlo,acchi,acclo);
   }
   whi[rhsoff+k] = resulthi;
   wlo[rhsoff+k] = resultlo;
}

__global__ void cmplx2_multiply_inverse
 ( int dim, int idx,
   double *invUrehi, double *invUrelo, double *invUimhi, double *invUimlo,
   double *wrehi, double *wrelo, double *wimhi, double *wimlo )
{
   const int k = threadIdx.x;     // thread k computes k-th product
   const int rhsoff = dim*idx;    // offset for the right hand size
   const int offset = dim*rhsoff; // offset for diagonal tile

   __shared__ double workrehi[tabsdd_shmemsize];      // copy of w
   __shared__ double workrelo[tabsdd_shmemsize]; 
   __shared__ double workimhi[tabsdd_shmemsize];
   __shared__ double workimlo[tabsdd_shmemsize];

   workrehi[k] = wrehi[rhsoff+k];
   workrelo[k] = wrelo[rhsoff+k];
   workimhi[k] = wimhi[rhsoff+k];
   workimlo[k] = wimlo[rhsoff+k];

   double resultrehi = 0.0; // each thread stores its product in result
   double resultrelo = 0.0;
   double resultimhi = 0.0;
   double resultimlo = 0.0;
   double coeffrehi,coeffrelo,coeffimhi,coeffimlo;
   double acc1hi,acc1lo,acc2hi,acc2lo;

   for(int j=0; j<dim; j++)  // column j of the inverse diagonal tile
   {
      coeffrehi = invUrehi[offset+k*dim+j]; // thread k does row k
      coeffrelo = invUrelo[offset+k*dim+j];
      coeffimhi = invUimhi[offset+k*dim+j];
      coeffimlo = invUimlo[offset+k*dim+j];
      // result = result + coeff*work[j];
      ddg_mul(coeffrehi,coeffrelo,workrehi[j],workrelo[j],&acc1hi,&acc1lo);
      ddg_mul(coeffimhi,coeffimlo,workimhi[j],workimlo[j],&acc2hi,&acc2lo);
      ddg_inc(&resultrehi,&resultrelo,acc1hi,acc1lo);
      ddg_dec(&resultrehi,&resultrelo,acc2hi,acc2lo);
      ddg_mul(coeffimhi,coeffimlo,workrehi[j],workrelo[j],&acc1hi,&acc1lo);
      ddg_mul(coeffrehi,coeffrelo,workimhi[j],workimlo[j],&acc2hi,&acc2lo);
      ddg_inc(&resultimhi,&resultimlo,acc1hi,acc1lo);
      ddg_inc(&resultimhi,&resultimlo,acc2hi,acc2lo);
   }
   wrehi[rhsoff+k] = resultrehi; wrelo[rhsoff+k] = resultrelo;
   wimhi[rhsoff+k] = resultimhi; wimlo[rhsoff+k] = resultimlo;
}

__global__ void dbl2_back_substitute
 ( int dim, int idx, double *Uhi, double *Ulo, double *whi, double *wlo )
{
   const int B = blockIdx.x;     // block index
   const int k = threadIdx.x;    // thread k computes k-th product
   const int offset = B*dim*dim; // numbers to skip

   __shared__ double wrkhi[tabsdd_shmemsize];   // copy of w
   __shared__ double wrklo[tabsdd_shmemsize]; 
   __shared__ double solhi[tabsdd_shmemsize];    // solution to update with
   __shared__ double sollo[tabsdd_shmemsize];

   wrkhi[k] = whi[B*dim+k];    // block B updates B-th slice of w
   wrklo[k] = wlo[B*dim+k];
   solhi[k] = whi[idx*dim+k];  // solution that is back substituted
   sollo[k] = wlo[idx*dim+k];

   double resulthi = 0.0; // each thread stores its product in result
   double resultlo = 0.0;
   double coeffhi,coefflo,acchi,acclo;

   for(int j=0; j<dim; j++)  // column j of the inverse diagonal tile
   {
      coeffhi = Uhi[offset+k*dim+j];
      coefflo = Ulo[offset+k*dim+j];
      // result = result + coeff*sol[j];
      ddg_mul(coeffhi,coefflo,solhi[j],sollo[j],&acchi,&acclo);
      ddg_inc(&resulthi,&resultlo,acchi,acclo);
   }
   // wrk[k] = wrk[k] - result; // subtract product
   ddg_dec(&wrkhi[k],&wrklo[k],resulthi,resultlo);
   whi[B*dim+k] = wrkhi[k];
   wlo[B*dim+k] = wrklo[k];
}

__global__ void cmplx2_back_substitute
 ( int dim, int idx,
   double *Urehi, double *Urelo, double *Uimhi, double *Uimlo,
   double *wrehi, double *wrelo, double *wimhi, double *wimlo )
{
   const int B = blockIdx.x;     // block index
   const int k = threadIdx.x;    // thread k computes k-th product
   const int offset = B*dim*dim; // numbers to skip

   __shared__ double wrkrehi[tabsdd_shmemsize];   // copy of w
   __shared__ double wrkrelo[tabsdd_shmemsize]; 
   __shared__ double wrkimhi[tabsdd_shmemsize];
   __shared__ double wrkimlo[tabsdd_shmemsize]; 
   __shared__ double solrehi[tabsdd_shmemsize];    // solution to update with
   __shared__ double solrelo[tabsdd_shmemsize];
   __shared__ double solimhi[tabsdd_shmemsize];
   __shared__ double solimlo[tabsdd_shmemsize];

   wrkrehi[k] = wrehi[B*dim+k];    // block B updates B-th slice of w
   wrkrelo[k] = wrelo[B*dim+k];
   wrkimhi[k] = wimhi[B*dim+k];
   wrkimlo[k] = wimlo[B*dim+k];
   solrehi[k] = wrehi[idx*dim+k];  // solution that is back substituted
   solrelo[k] = wrelo[idx*dim+k];
   solimhi[k] = wimhi[idx*dim+k];
   solimlo[k] = wimlo[idx*dim+k];

   double resultrehi = 0.0; // each thread stores its product in result
   double resultrelo = 0.0;
   double resultimhi = 0.0;
   double resultimlo = 0.0;
   double coeffrehi,coeffrelo,coeffimhi,coeffimlo;
   double acc1hi,acc1lo,acc2hi,acc2lo;

   for(int j=0; j<dim; j++)  // column j of the inverse diagonal tile
   {
      coeffrehi = Urehi[offset+k*dim+j];
      coeffrelo = Urelo[offset+k*dim+j];
      coeffimhi = Uimhi[offset+k*dim+j];
      coeffimlo = Uimlo[offset+k*dim+j];
      // result = result + coeff*sol[j];
      ddg_mul(coeffrehi,coeffrelo,solrehi[j],solrelo[j],&acc1hi,&acc1lo);
      ddg_mul(coeffimhi,coeffimlo,solimhi[j],solimlo[j],&acc2hi,&acc2lo);
      ddg_inc(&resultrehi,&resultrelo,acc1hi,acc1lo);
      ddg_dec(&resultrehi,&resultrelo,acc2hi,acc2lo);
      ddg_mul(coeffimhi,coeffimlo,solrehi[j],solrelo[j],&acc1hi,&acc1lo);
      ddg_mul(coeffrehi,coeffrelo,solimhi[j],solimlo[j],&acc2hi,&acc2lo);
      ddg_inc(&resultimhi,&resultimlo,acc1hi,acc1lo);
      ddg_inc(&resultimhi,&resultimlo,acc2hi,acc2lo);
   }
   // wrk[k] = wrk[k] - result; // subtract product
   ddg_dec(&wrkrehi[k],&wrkrelo[k],resultrehi,resultrelo);
   ddg_dec(&wrkimhi[k],&wrkimlo[k],resultimhi,resultimlo);
   wrehi[B*dim+k] = wrkrehi[k];
   wrelo[B*dim+k] = wrkrelo[k];
   wimhi[B*dim+k] = wrkimhi[k];
   wimlo[B*dim+k] = wrkimlo[k];
}

void GPU_dbl2_upper_inverse
 ( int dim, double **Uhi, double **Ulo, double **invUhi, double **invUlo,
   double *lapms, double *walltimesec )
{
   const int szU = dim*dim;

   double *Uhi_h = new double[szU];     // Uhi_h stores the columns of Uhi
   double *Ulo_h = new double[szU];     // Ulo_h stores the columns of Ulo 
   double *Uhi_d;                       // Uhi_d is Uhi_h on the device
   double *Ulo_d;                       // Ulo_d is Ulo_h on the device
   double *invUhi_h = new double[szU];  // high doubles of the inverse
   double *invUlo_h = new double[szU];  // low doubles of the inverse
   double *invUhi_d;                    // invUhi_d is invUhi_h on the device
   double *invUlo_d;                    // invUlo_d is invUlo_h on the device

   int ix = 0;
   for(int j=0; j<dim; j++)
      for(int i=0; i<dim; i++)
      {
         Uhi_h[ix]   = Uhi[i][j];
         Ulo_h[ix++] = Ulo[i][j];
      }

   // only for debugging
   // test_dbl2_small_invert_upper(dim,Uhi_h,Ulo_h,invUhi,invUlo_h);

   size_t szmat = szU*sizeof(double);
   hipMalloc((void**)&Uhi_d,szmat);
   hipMalloc((void**)&Ulo_d,szmat);
   hipMalloc((void**)&invUhi_d,szmat);
   hipMalloc((void**)&invUlo_d,szmat);
   hipMemcpy(Uhi_d,Uhi_h,szmat,hipMemcpyHostToDevice);
   hipMemcpy(Ulo_d,Ulo_h,szmat,hipMemcpyHostToDevice);

   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   *lapms = 0.0;
   float milliseconds;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   hipEventRecord(start);

   if(dim <= 16)
      dbl2_small_invert_upper<<<1,dim>>>(dim,Uhi_d,Ulo_d,invUhi_d,invUlo_d);
   else
      dbl2_medium_invert_upper<<<1,dim>>>(dim,Uhi_d,Ulo_d,invUhi_d,invUlo_d);

   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;

   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   hipMemcpy(invUhi_h,invUhi_d,szmat,hipMemcpyDeviceToHost);
   hipMemcpy(invUlo_h,invUlo_d,szmat,hipMemcpyDeviceToHost);

   ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<dim; j++)
      {
         invUhi[i][j] = invUhi_h[ix];
         invUlo[i][j] = invUlo_h[ix++];
      }

   free(Uhi_h); free(invUhi_h);
   free(Ulo_h); free(invUlo_h);
}

void GPU_cmplx2_upper_inverse
 ( int dim, double **Urehi, double **Urelo, double **Uimhi, double **Uimlo,
   double **invUrehi, double **invUrelo, double **invUimhi, double **invUimlo,
   double *lapms, double *walltimesec )
{
   const int szU = dim*dim;

   double *Urehi_h = new double[szU];    // Urehi_h has high real parts
   double *Urelo_h = new double[szU];    // Urelo_h has low real parts
   double *Uimhi_h = new double[szU];    // Uimhi_h has high imag parts
   double *Uimlo_h = new double[szU];    // Uimlo_h has low imag parts
   double *Urehi_d;                      // Urehi_d is Urehi_h on the device
   double *Urelo_d;                      // Urelo_d is Urelo_h on the device
   double *Uimhi_d;                      // Uimhi_d is Uimhi_h on the device
   double *Uimlo_d;                      // Uimlo_d is Uimlo_h on the device
   double *invUrehi_h = new double[szU]; // high real parts of the inverse
   double *invUrelo_h = new double[szU]; // low real parts of the inverse
   double *invUimhi_h = new double[szU]; // high imag parts of the inverse
   double *invUimlo_h = new double[szU]; // low imag parts of the inverse
   double *invUrehi_d;                   // invUrehi_d ~ invUrehi_h on device
   double *invUrelo_d;                   // invUrelo_d ~ invUrelo_h on device
   double *invUimhi_d;                   // invUimhi_d ~ invUimhi_h on device
   double *invUimlo_d;                   // invUimlo_d ~ invUimlo_h on device

   int ix = 0;
   for(int j=0; j<dim; j++)
      for(int i=0; i<dim; i++)
      {
         Urehi_h[ix] = Urehi[i][j]; Urelo_h[ix]   = Urelo[i][j];
         Uimhi_h[ix] = Uimhi[i][j]; Uimlo_h[ix++] = Uimlo[i][j];
      }

   size_t szmat = szU*sizeof(double);
   hipMalloc((void**)&Urehi_d,szmat);
   hipMalloc((void**)&Urelo_d,szmat);
   hipMalloc((void**)&Uimhi_d,szmat);
   hipMalloc((void**)&Uimlo_d,szmat);
   hipMalloc((void**)&invUrehi_d,szmat);
   hipMalloc((void**)&invUrelo_d,szmat);
   hipMalloc((void**)&invUimhi_d,szmat);
   hipMalloc((void**)&invUimlo_d,szmat);
   hipMemcpy(Urehi_d,Urehi_h,szmat,hipMemcpyHostToDevice);
   hipMemcpy(Urelo_d,Urelo_h,szmat,hipMemcpyHostToDevice);
   hipMemcpy(Uimhi_d,Uimhi_h,szmat,hipMemcpyHostToDevice);
   hipMemcpy(Uimlo_d,Uimlo_h,szmat,hipMemcpyHostToDevice);

   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   *lapms = 0.0;
   float milliseconds;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   hipEventRecord(start);

   if(dim <= 16)
      cmplx2_small_invert_upper<<<1,dim>>>
         (dim,   Urehi_d,   Urelo_d,   Uimhi_d,   Uimlo_d,
              invUrehi_d,invUrelo_d,invUimhi_d,invUimlo_d);
   else
      cmplx2_medium_invert_upper<<<1,dim>>>
         (dim,   Urehi_d,   Urelo_d,   Uimhi_d,   Uimlo_d,
              invUrehi_d,invUrelo_d,invUimhi_d,invUimlo_d);

   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;

   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   hipMemcpy(invUrehi_h,invUrehi_d,szmat,hipMemcpyDeviceToHost);
   hipMemcpy(invUrelo_h,invUrelo_d,szmat,hipMemcpyDeviceToHost);
   hipMemcpy(invUimhi_h,invUimhi_d,szmat,hipMemcpyDeviceToHost);
   hipMemcpy(invUimlo_h,invUimlo_d,szmat,hipMemcpyDeviceToHost);

   ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<dim; j++)
      {
         invUrehi[i][j] = invUrehi_h[ix];
         invUrelo[i][j] = invUrelo_h[ix];
         invUimhi[i][j] = invUimhi_h[ix];
         invUimlo[i][j] = invUimlo_h[ix++];
      }

   free(Urehi_h); free(Urelo_h); free(invUrehi_h); free(invUrelo_h);
   free(Uimhi_h); free(Uimlo_h); free(invUimhi_h); free(invUimlo_h);
}

void GPU_dbl2_upper_tiled_solver
 ( int dim, int szt, int nbt, double **Uhi, double **Ulo,
   double *bhi, double *blo, double *xhi, double *xlo,
   double *invlapms, double *mullapms, double *sublapms, double *totlapms,
   double *walltimesec,
   long long int *addcnt, long long int *mulcnt, long long int *divcnt )
{
   const int nbr = nbt*szt*szt;   // number of doubles on diagonal tiles
   double *Dhi_h = new double[nbr];    // the diagonal tiles on the host
   double *Dlo_h = new double[nbr];    // low doubles of diagonal tiles
   double *Dhi_d;                      // diagonal tiles on the device
   double *Dlo_d;                      // low doubles of diagonal tiles
   double *invDhi_h = new double[nbr]; // inverse of diagonal tiles on host 
   double *invDlo_h = new double[nbr]; // low doubles of inverse tiles
   double *invDhi_d;                   // invDhi_d is invDhi_h on device
   double *invDlo_d;                   // invDlo_d is invDlo_h on device
   int offset;
   int ix = 0;

   for(int k=0; k<nbt; k++) // copy columns of the k-th tile
   {
      offset = k*szt;
      for(int j=0; j<szt; j++)
         for(int i=0; i<szt; i++)
         {
            Dhi_h[ix]   = Uhi[offset+i][offset+j];
            Dlo_h[ix++] = Ulo[offset+i][offset+j];
         }
   }
   const size_t sznum = nbr*sizeof(double);
   hipMalloc((void**)&Dhi_d,sznum);
   hipMalloc((void**)&Dlo_d,sznum);
   hipMalloc((void**)&invDhi_d,sznum);
   hipMalloc((void**)&invDlo_d,sznum);
   hipMemcpy(Dhi_d,Dhi_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Dlo_d,Dlo_h,sznum,hipMemcpyHostToDevice);

   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   *invlapms = 0.0;
   *mullapms = 0.0;
   *sublapms = 0.0;
   *totlapms = 0.0;
   float milliseconds;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   hipEventRecord(start);
   dbl2_invert_tiles<<<nbt,szt>>>(szt,Dhi_d,Dlo_d,invDhi_d,invDlo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *invlapms += milliseconds;
   *totlapms += milliseconds;
   flopcount_dbl_invert_tiles(nbt,szt,addcnt,mulcnt,divcnt);

   double *rhshi_d;                    // right hand side on device
   double *rhslo_d;
   const size_t szrhs = dim*sizeof(double);
   hipMalloc((void**)&rhshi_d,szrhs);
   hipMalloc((void**)&rhslo_d,szrhs);
   hipMemcpy(rhshi_d,bhi,szrhs,hipMemcpyHostToDevice);
   hipMemcpy(rhslo_d,blo,szrhs,hipMemcpyHostToDevice);

   hipEventRecord(start);
   dbl2_multiply_inverse<<<1,szt>>>
      (szt,nbt-1,invDhi_d,invDlo_d,rhshi_d,rhslo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *mullapms += milliseconds;
   *totlapms += milliseconds;
   flopcount_dbl_multiply_inverse(szt,addcnt,mulcnt);

   int nbrUcol = (nbt-1)*szt*szt;           // #doubles in column of U
   double *Ucolhi_h = new double[nbrUcol];  // column of U on host
   double *Ucollo_h = new double[nbrUcol];  // column of U on host
   double *Ucolhi_d;
   double *Ucollo_d;
   const size_t szUcol = nbrUcol*sizeof(double);
   hipMalloc((void**)&Ucolhi_d,szUcol);
   hipMalloc((void**)&Ucollo_d,szUcol);

   int coloff,rowoff;

   for(int k=nbt-1; k>0; k--)      // update with solution tile k
   {
      coloff = k*szt;      // column offset to update with solution tile k
      ix = 0;
      for(int L=0; L<k; L++)       // copy k tiles of U
      {
         rowoff = L*szt;           // row offset for update data
         for(int i=0; i<szt; i++)
            for(int j=0; j<szt; j++)
            {
               Ucolhi_h[ix]   = Uhi[rowoff+i][coloff+j];
               Ucollo_h[ix++] = Ulo[rowoff+i][coloff+j];
            }
      }
      hipMemcpy(Ucolhi_d,Ucolhi_h,nbrUcol*sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(Ucollo_d,Ucollo_h,nbrUcol*sizeof(double),
                 hipMemcpyHostToDevice);

      hipEventRecord(start);
      dbl2_back_substitute<<<k,szt>>>
         (szt,k,Ucolhi_d,Ucollo_d,rhshi_d,rhslo_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *sublapms += milliseconds;
      *totlapms += milliseconds;
      flopcount_dbl_back_substitute(k,szt,addcnt,mulcnt);

      // (k-1)-th solution tile is ready for inverse multiplication
      hipEventRecord(start);
      dbl2_multiply_inverse<<<1,szt>>>
         (szt,k-1,invDhi_d,invDlo_d,rhshi_d,rhslo_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *mullapms += milliseconds;
      *totlapms += milliseconds;
      flopcount_dbl_multiply_inverse(szt,addcnt,mulcnt);

      nbrUcol = nbrUcol - szt*szt; // one tile less used in update
   }
   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   hipMemcpy(xhi,rhshi_d,szrhs,hipMemcpyDeviceToHost);
   hipMemcpy(xlo,rhslo_d,szrhs,hipMemcpyDeviceToHost);

   // copy of invD_d is needed only for testing purposes
   hipMemcpy(invDhi_h,invDhi_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(invDlo_h,invDlo_d,sznum,hipMemcpyDeviceToHost);

   ix = 0;
   for(int k=0; k<nbt; k++) // copy rows of the inverse of the k-th tile
   {
      offset = k*szt;
      for(int i=0; i<szt; i++)
         for(int j=0; j<szt; j++)
         {
            Uhi[offset+i][offset+j] = invDhi_h[ix];
            Ulo[offset+i][offset+j] = invDlo_h[ix++];
         }
   }
   free(Dhi_h); free(invDhi_h); free(Ucolhi_h);
   free(Dlo_h); free(invDlo_h); free(Ucollo_h);
}

void GPU_cmplx2_upper_tiled_solver
 ( int dim, int szt, int nbt,
   double **Urehi, double **Urelo, double **Uimhi, double **Uimlo,
   double *brehi, double *brelo, double *bimhi, double *bimlo,
   double *xrehi, double *xrelo, double *ximhi, double *ximlo,
   double *invlapms, double *mullapms, double *sublapms, double *totlapms,
   double *walltimesec,
   long long int *addcnt, long long int *mulcnt, long long int *divcnt )
{
   const int nbr = nbt*szt*szt;       // number of doubles on diagonal tiles
   double *Drehi_h = new double[nbr];    // the diagonal tiles on the host
   double *Drelo_h = new double[nbr];    // low doubles of real parts
   double *Dimhi_h = new double[nbr];    // high doubles of imaginary parts
   double *Dimlo_h = new double[nbr];    // low doubles of imaginary parts
   double *Drehi_d;                      // diagonal tiles on the device
   double *Drelo_d;                      // low doubles of real parts
   double *Dimhi_d;                      // high doubles of imaginary parts
   double *Dimlo_d;                      // low doubles of imaginary parts
   double *invDrehi_h = new double[nbr]; // inverse of tiles on host 
   double *invDrelo_h = new double[nbr]; // low doubles of inverse tiles
   double *invDimhi_h = new double[nbr]; // high doubles of imaginary parts
   double *invDimlo_h = new double[nbr]; // low doubles of imaginary parts
   double *invDrehi_d;                   // invDrehi_d ~ invDrehi_h on device
   double *invDrelo_d;                   // invDrelo_d ~ invDrelo_h on device
   double *invDimhi_d;                   // invDimhi_d ~ invDimhi_h on device
   double *invDimlo_d;                   // invDimlo_d ~ invDimlo_h on device
   int offset;
   int ix = 0;

   for(int k=0; k<nbt; k++) // copy columns of the k-th tile
   {
      offset = k*szt;
      for(int j=0; j<szt; j++)
         for(int i=0; i<szt; i++)
         {
            Drehi_h[ix]   = Urehi[offset+i][offset+j];
            Drelo_h[ix]   = Urelo[offset+i][offset+j];
            Dimhi_h[ix]   = Uimhi[offset+i][offset+j];
            Dimlo_h[ix++] = Uimlo[offset+i][offset+j];
         }
   }
   const size_t sznum = nbr*sizeof(double);
   hipMalloc((void**)&Drehi_d,sznum);
   hipMalloc((void**)&Drelo_d,sznum);
   hipMalloc((void**)&Dimhi_d,sznum);
   hipMalloc((void**)&Dimlo_d,sznum);
   hipMalloc((void**)&invDrehi_d,sznum);
   hipMalloc((void**)&invDrelo_d,sznum);
   hipMalloc((void**)&invDimhi_d,sznum);
   hipMalloc((void**)&invDimlo_d,sznum);
   hipMemcpy(Drehi_d,Drehi_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Drelo_d,Drelo_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Dimhi_d,Dimhi_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Dimlo_d,Dimlo_h,sznum,hipMemcpyHostToDevice);

   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   *invlapms = 0.0;
   *mullapms = 0.0;
   *sublapms = 0.0;
   *totlapms = 0.0;
   float milliseconds;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   hipEventRecord(start);
   cmplx2_invert_tiles<<<nbt,szt>>>
      (szt,   Drehi_d,   Drelo_d,   Dimhi_d,   Dimlo_d,
           invDrehi_d,invDrelo_d,invDimhi_d,invDimlo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *invlapms += milliseconds;
   *totlapms += milliseconds;
   flopcount_cmplx_invert_tiles(nbt,szt,addcnt,mulcnt,divcnt);

   double *rhsrehi_d;                    // right hand side on device
   double *rhsrelo_d;
   double *rhsimhi_d;
   double *rhsimlo_d;
   const size_t szrhs = dim*sizeof(double);
   hipMalloc((void**)&rhsrehi_d,szrhs);
   hipMalloc((void**)&rhsrelo_d,szrhs);
   hipMalloc((void**)&rhsimhi_d,szrhs);
   hipMalloc((void**)&rhsimlo_d,szrhs);
   hipMemcpy(rhsrehi_d,brehi,szrhs,hipMemcpyHostToDevice);
   hipMemcpy(rhsrelo_d,brelo,szrhs,hipMemcpyHostToDevice);
   hipMemcpy(rhsimhi_d,bimhi,szrhs,hipMemcpyHostToDevice);
   hipMemcpy(rhsimlo_d,bimlo,szrhs,hipMemcpyHostToDevice);

   hipEventRecord(start);
   cmplx2_multiply_inverse<<<1,szt>>>
      (szt,nbt-1,invDrehi_d,invDrelo_d,invDimhi_d,invDimlo_d,
                  rhsrehi_d, rhsrelo_d, rhsimhi_d, rhsimlo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *mullapms += milliseconds;
   *totlapms += milliseconds;
   flopcount_cmplx_multiply_inverse(szt,addcnt,mulcnt);

   int nbrUcol = (nbt-1)*szt*szt;             // #doubles in column of U
   double *Ucolrehi_h = new double[nbrUcol];  // column of U on host
   double *Ucolrelo_h = new double[nbrUcol];
   double *Ucolimhi_h = new double[nbrUcol];
   double *Ucolimlo_h = new double[nbrUcol];
   double *Ucolrehi_d;
   double *Ucolrelo_d;
   double *Ucolimhi_d;
   double *Ucolimlo_d;
   const size_t szUcol = nbrUcol*sizeof(double);
   hipMalloc((void**)&Ucolrehi_d,szUcol);
   hipMalloc((void**)&Ucolrelo_d,szUcol);
   hipMalloc((void**)&Ucolimhi_d,szUcol);
   hipMalloc((void**)&Ucolimlo_d,szUcol);

   int coloff,rowoff;

   for(int k=nbt-1; k>0; k--)      // update with solution tile k
   {
      coloff = k*szt;      // column offset to update with solution tile k
      ix = 0;
      for(int L=0; L<k; L++)       // copy k tiles of U
      {
         rowoff = L*szt;           // row offset for update data
         for(int i=0; i<szt; i++)
            for(int j=0; j<szt; j++)
            {
               Ucolrehi_h[ix]   = Urehi[rowoff+i][coloff+j];
               Ucolrelo_h[ix]   = Urelo[rowoff+i][coloff+j];
               Ucolimhi_h[ix]   = Uimhi[rowoff+i][coloff+j];
               Ucolimlo_h[ix++] = Uimlo[rowoff+i][coloff+j];
            }
      }
      hipMemcpy(Ucolrehi_d,Ucolrehi_h,nbrUcol*sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(Ucolrelo_d,Ucolrelo_h,nbrUcol*sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(Ucolimhi_d,Ucolimhi_h,nbrUcol*sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(Ucolimlo_d,Ucolimlo_h,nbrUcol*sizeof(double),
                 hipMemcpyHostToDevice);

      hipEventRecord(start);
      cmplx2_back_substitute<<<k,szt>>>
         (szt,k ,Ucolrehi_d,Ucolrelo_d,Ucolimhi_d,Ucolimlo_d,
                  rhsrehi_d, rhsrelo_d, rhsimhi_d, rhsimlo_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *sublapms += milliseconds;
      *totlapms += milliseconds;
      flopcount_cmplx_back_substitute(k,szt,addcnt,mulcnt);

      // (k-1)-th solution tile is ready for inverse multiplication
      hipEventRecord(start);
      cmplx2_multiply_inverse<<<1,szt>>>
         (szt,k-1,invDrehi_d,invDrelo_d,invDimhi_d,invDimlo_d,
                   rhsrehi_d, rhsrelo_d, rhsimhi_d, rhsimlo_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *mullapms += milliseconds;
      *totlapms += milliseconds;
      flopcount_cmplx_multiply_inverse(szt,addcnt,mulcnt);

      nbrUcol = nbrUcol - szt*szt; // one tile less used in update
   }
   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   hipMemcpy(xrehi,rhsrehi_d,szrhs,hipMemcpyDeviceToHost);
   hipMemcpy(xrelo,rhsrelo_d,szrhs,hipMemcpyDeviceToHost);
   hipMemcpy(ximhi,rhsimhi_d,szrhs,hipMemcpyDeviceToHost);
   hipMemcpy(ximlo,rhsimlo_d,szrhs,hipMemcpyDeviceToHost);

   // copy of invD_d is needed only for testing purposes
   hipMemcpy(invDrehi_h,invDrehi_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(invDrelo_h,invDrelo_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(invDimhi_h,invDimhi_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(invDimlo_h,invDimlo_d,sznum,hipMemcpyDeviceToHost);

   ix = 0;
   for(int k=0; k<nbt; k++) // copy rows of the inverse of the k-th tile
   {
      offset = k*szt;
      for(int i=0; i<szt; i++)
         for(int j=0; j<szt; j++)
         {
            Urehi[offset+i][offset+j] = invDrehi_h[ix];
            Urelo[offset+i][offset+j] = invDrelo_h[ix];
            Uimhi[offset+i][offset+j] = invDimhi_h[ix];
            Uimlo[offset+i][offset+j] = invDimlo_h[ix++];
         }
   }
   free(Drehi_h); free(invDrehi_h); free(Ucolrehi_h);
   free(Drelo_h); free(invDrelo_h); free(Ucolrelo_h);
   free(Dimhi_h); free(invDimhi_h); free(Ucolimhi_h);
   free(Dimlo_h); free(invDimlo_h); free(Ucolimlo_h);
}
