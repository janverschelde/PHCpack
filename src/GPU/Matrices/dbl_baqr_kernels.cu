#include "hip/hip_runtime.h"
/* The file dbl_baqr_kernels.cu defines the functions with prototypes in
 * the file dbl_baqr_kernels.h. */

#include <iostream>
#include <iomanip>
#ifdef winwalltime
#include "wingettimeofday.h"
#else
#include <sys/time.h>
#endif
#include "dbl_baqr_kernels.h"

using namespace std;

__global__ void dbl_small_house
 ( double *x0, double *x1, int dim, int dimLog2, double *v, double *beta )
{
   int j = threadIdx.x;

   __shared__ double shv[d_shmemsize];
   __shared__ double prd[d_shmemsize];

   bool stopflag = false;
   double mu,v0,v0p2;

   shv[j] = x1[j];              // reading of vector into shared memory
   prd[j] = shv[j]*shv[j];      // for the 2-norm computation

   v[j+1] = shv[j];             // copies x to v, in case beta is zero
   if(j == 0) v[0] = 1.0;

   __syncthreads();
   int powTwo = 1;                          // sum reduction
   for(int k=0; k < dimLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < dim) prd[j] = prd[j] + prd[j+powTwo];
      powTwo = powTwo*2;
      __syncthreads();
   }
   // thread 0 computes the sqrt of the inner product, others wait
   if(j == 0)
   {
      if(prd[0] == 0.0)                    // prd[0] is sigma of house
      {
         *beta = 0.0; stopflag = true;
      }
   }
   __syncthreads();
   if(stopflag) return;                    // case when sigma is zero
   if(j == 0)                              // thread zero sets beta
   {
      mu = sqrt((*x0)*(*x0) + prd[0]);
      if(*x0 <= 0.0)
         v0 = *x0 - mu;
      else
         v0 = -prd[0]/(*x0 + mu);

      v0p2 = v0*v0;
      *beta = 2.0*v0p2/(prd[0] + v0p2);
      prd[0] = v0;                         // v0 needed for normalization
   }
   __syncthreads();
   shv[j] = shv[j]/prd[0];
   v[j+1] = shv[j];
   if(j == 0) v[0] = 1.0;
}

__global__ void dbl_factors_leftRupdate
 ( int nrows, int ncols, int szt, int k, double *R, double *v, double *beta )
{
   int bdx = blockIdx.x;           // index of block
   int tdx = threadIdx.x;          // index of thread in block
   int idx = bdx*szt + tdx;        // global thread index
   int Roffset = (k+bdx)*szt + k;
   int Rcolidx;
   double w,Rtdx;

   __shared__ double shv[d_shmemsize]; // slice of v

   if(idx < nrows - k)   // nrows - k threads in all blocks work
   {
      shv[tdx] = v[tdx];
      __syncthreads();
      w = 0.0;

      for(int i=0; i<nrows-k; i++)   // loop through rows of R
      {
         Rtdx = R[Roffset + i + tdx*nrows];
         w = w + Rtdx*shv[i];
      }
      w = (*beta)*w;
      __syncthreads();
      for(int i=0; i<nrows-k; i++)   // update i-th row of R
      {
         Rcolidx = Roffset + i + tdx*nrows;
         Rtdx = R[Rcolidx];
         Rtdx = Rtdx - shv[i]*w;
         __syncthreads();
         R[Rcolidx] = Rtdx;
      }
   }
}

__global__ void dbl_small_leftRupdate
 ( int nrows, int ncols, int szt, int k, double *R, double *v, double *beta )
{
   int tdx = threadIdx.x;          // index of thread in block
   int Roffset = k*nrows + k;
   int Rcolidx;
   double w,Rtdx;

   __shared__ double shv[d_shmemsize]; // slice of v

   shv[tdx] = v[tdx];
   __syncthreads();
   w = 0.0;

   for(int i=0; i<nrows-k; i++)   // loop through rows of R
   {
      Rtdx = R[Roffset + i + tdx*nrows];
      w = w + Rtdx*shv[i];
   }
   w = (*beta)*w;
   __syncthreads();
   for(int i=0; i<nrows-k; i++)   // update i-th row of R
   {
      Rcolidx = Roffset + i + tdx*nrows;
      Rtdx = R[Rcolidx];
      Rtdx = Rtdx - shv[i]*w;
      __syncthreads();
      if(tdx < ncols-k) R[Rcolidx] = Rtdx;
   }
}

void GPU_dbl_blocked_houseqr
 ( int nrows, int ncols, int szt, int nbt,
   double **A, double **Q, double **R,
   double *houselapms, double *tileRlapms,
   double *walltimesec, bool verbose )
{
   const int dim = nrows*ncols;     // total number of doubles
   double *A_h = new double[dim];   // matrix A on the host
   double *A_d;                     // matrix on the device
   double *v_h = new double[nrows]; // Householder vector on host
   double *v_d;                     // Householder vector on device
   double *x0_d;                    // first element for house on device
   double beta_h;                   // beta on the host
   double *beta_d;                  // beta on the device

   int ix=0;                        // copy the columns of A to A_h
   for(int j=0; j<ncols; j++)   
      for(int i=0; i<nrows; i++) A_h[ix++] = A[i][j];

   const size_t sznum = dim*sizeof(double);
   hipMalloc((void**)&A_d,sznum);
   hipMemcpy(A_d,A_h,sznum,hipMemcpyHostToDevice);
   const size_t szhouse = nrows*sizeof(double);
   hipMalloc((void**)&v_d,szhouse);
   hipMalloc((void**)&x0_d,sizeof(double));
   hipMalloc((void**)&beta_d,sizeof(double));

   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   *houselapms = 0.0;
   *tileRlapms = 0.0;
   float milliseconds;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   for(int k=0; k<nbt; k++)       // k runs over the number of blocks
   {
      int colidx,nrows1,nrLog2,rowidx;
      // int nbrblocks;

      for(int L=0; L<szt; L++)  // L runs over the columns in one block
      {
         colidx = k*szt + L;              // index of the current column
         nrows1 = nrows - colidx - 1;     // #rows in Householder vector - 1
         if(nrows1 > 0)
         {
            nrLog2 = ceil(log2((double) nrows1));
            rowidx = colidx*(nrows+1);       // start of number in A_h

            if(verbose)
            {
               cout << "nrows : " << nrows
                    << "  ncols : " << ncols
                    << "  szt : " << szt
                    << "  nbt : " << nbt << endl;
               cout << "k : " << k 
                    << "  L : " << L
                    << "  nrows1 : " << nrows1
                    << "  colidx : " << colidx
                    << "  rowidx : " << rowidx << endl;
            }
            hipMemcpy(x0_d,&A_h[rowidx],sizeof(double),hipMemcpyHostToDevice);
            hipEventRecord(start);
            dbl_small_house<<<1,nrows1>>>
               (x0_d,&A_d[rowidx+1],nrows1,nrLog2,v_d,beta_d);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds,start,stop);
            *houselapms += milliseconds;
 
            if(verbose)
            {
               hipMemcpy(&beta_h,beta_d,sizeof(double),hipMemcpyDeviceToHost);
               hipMemcpy(v_h,v_d,szhouse,hipMemcpyDeviceToHost);
               cout << scientific << setprecision(16)
                    << "beta[" << L+k*szt << "] : " << beta_h << endl;
               for(int i=0; i<nrows1+1; i++)
                  cout << "v[" << i << "] : " << v_h[i] << endl;
            }
       /*
         nbrblocks = (nrows - colidx)/szt;
         if(((nrows - colidx) % szt) > 0) nbrblocks = nbrblocks + 1;

         if(verbose)
         {
            cout << "launching " << nbrblocks 
                 << " blocks of " << szt << " threads, ";
            cout << nbrblocks*szt << " threads to process "
                 << nrows - colidx << " rows ..." << endl;
         }
         hipEventRecord(start);
         dbl_factors_leftRupdate<<<nbrblocks,szt>>>
            (nrows,ncols,szt,colidx,A_d,v_d,beta_d);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *tileRlapms += milliseconds;
        */
            hipEventRecord(start);
            dbl_small_leftRupdate<<<1,nrows-colidx>>>
               (nrows,ncols,szt,colidx,A_d,v_d,beta_d);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds,start,stop);
            *tileRlapms += milliseconds;
            if(verbose)
            {
               hipMemcpy(A_h,A_d,sznum,hipMemcpyDeviceToHost);
               cout << "the matrix after the update :" << endl;
               for(int i=0; i<nrows; i++)
                  for(int j=0; j<ncols; j++)
                     cout << "A_d[" << i << "][" << j << "] : "
                          << A_h[j*nrows+i] << endl;
            }
         }
      }
   }
   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   free(A_h); free(v_h);
}
