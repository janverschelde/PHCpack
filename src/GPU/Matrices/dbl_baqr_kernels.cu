#include "hip/hip_runtime.h"
/* The file dbl_baqr_kernels.cu defines the functions with prototypes in
 * the file dbl_baqr_kernels.h. */

#include <iostream>
#include <iomanip>
#ifdef winwalltime
#include "wingettimeofday.h"
#else
#include <sys/time.h>
#endif
#include "dbl_baqr_kernels.h"

using namespace std;

__global__ void dbl_small_house
 ( double *x0, double *x1, int dim, int dimLog2, double *v, double *beta )
{
   int j = threadIdx.x;

   __shared__ double shv[d_shmemsize];
   __shared__ double prd[d_shmemsize];

   bool stopflag = false;
   double mu,v0,v0p2;

   shv[j] = x1[j];              // reading of vector into shared memory
   prd[j] = shv[j]*shv[j];      // for the 2-norm computation

   v[j+1] = shv[j];             // copies x to v, in case beta is zero
   if(j == 0) v[0] = 1.0;

   __syncthreads();
   int powTwo = 1;                          // sum reduction
   for(int k=0; k < dimLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < dim) prd[j] = prd[j] + prd[j+powTwo];
      powTwo = powTwo*2;
      __syncthreads();
   }
   // thread 0 computes the sqrt of the inner product, others wait
   if(j == 0)
   {
      if(prd[0] == 0.0)                    // prd[0] is sigma of house
      {
         *beta = 0.0; stopflag = true;
      }
   }
   __syncthreads();
   if(stopflag) return;                    // case when sigma is zero
   if(j == 0)                              // thread zero sets beta
   {
      mu = sqrt((*x0)*(*x0) + prd[0]);
      if(*x0 <= 0.0)
         v0 = *x0 - mu;
      else
         v0 = -prd[0]/(*x0 + mu);

      v0p2 = v0*v0;
      *beta = 2.0*v0p2/(prd[0] + v0p2);
      prd[0] = v0;                         // v0 needed for normalization
   }
   __syncthreads();
   shv[j] = shv[j]/prd[0];
   v[j+1] = shv[j];
   if(j == 0) v[0] = 1.0;
}

__global__ void dbl_factors_leftRupdate
 ( int nrows, int ncols, int szt, int k, double *R, double *v, double *beta )
{
   int bdx = blockIdx.x;           // index of block
   int tdx = threadIdx.x;          // index of thread in block
   int idx = bdx*szt + tdx;        // global thread index
   int Roffset = (k+bdx)*szt + k;
   int Rcolidx;
   double w,Rtdx;

   __shared__ double shv[d_shmemsize]; // slice of v

   if(idx < nrows - k)   // nrows - k threads in all blocks work
   {
      shv[tdx] = v[tdx];
      __syncthreads();
      w = 0.0;

      for(int i=0; i<nrows-k; i++)   // loop through rows of R
      {
         Rtdx = R[Roffset + i + tdx*nrows];
         w = w + Rtdx*shv[i];
      }
      w = (*beta)*w;
      __syncthreads();
      for(int i=0; i<nrows-k; i++)   // update i-th row of R
      {
         Rcolidx = Roffset + i + tdx*nrows;
         Rtdx = R[Rcolidx];
         Rtdx = Rtdx - shv[i]*w;
         __syncthreads();
         R[Rcolidx] = Rtdx;
      }
   }
}

__global__ void dbl_small_leftRupdate
 ( int nrows, int ncols, int szt, int k, double *R, double *v, double *beta )
{
   int tdx = threadIdx.x;          // index of thread in block
   int Roffset = k*nrows + k;
   int Rcolidx;
   double w,Rtdx;

   __shared__ double shv[d_shmemsize]; // slice of v

   shv[tdx] = v[tdx];
   __syncthreads();
   w = 0.0;

   for(int i=0; i<nrows-k; i++)   // loop through rows of R
   {
      Rtdx = R[Roffset + i + tdx*nrows];
      w = w + Rtdx*shv[i];
   }
   w = (*beta)*w;
   __syncthreads();
   for(int i=0; i<nrows-k; i++)   // update i-th row of R
   {
      Rcolidx = Roffset + i + tdx*nrows;
      Rtdx = R[Rcolidx];
      Rtdx = Rtdx - shv[i]*w;
      __syncthreads();
      if(tdx < ncols-k) R[Rcolidx] = Rtdx;
   }
}

__global__ void dbl_VB_to_W
 ( int nrows, int ncols, double *B, double *V, double *W )
{
   int tdx = threadIdx.x;              // index of thread in block
   double wrk,pk,mypk,zi;

   __shared__ double shv[d_shmemsize]; // one work vector
   __shared__ double shw[d_shmemsize]; // the other work vector
   __shared__ double shp[d_shmemsize]; // to share Y^T*v

   shv[tdx] = V[tdx];
   wrk = -B[0]*shv[tdx];               // first column of W
   W[tdx] = wrk;

   for(int j=1; j<ncols; j++)          // compute column j of W
   {
      shv[tdx] = V[j*nrows + tdx];     // j-th Householder vector
      for(int k=0; k<j; k++)
      {
         pk = 0.0;                     // k-th component of Y^T*v
         shw[tdx] = V[k*nrows + tdx];  // load V[k][i]
         shp[tdx] = shw[tdx]*shv[tdx]; // V[k][i]*v[i]
         __syncthreads();
         for(int i=0; i<nrows; i++) pk = pk + shp[i];
         if(tdx == k) mypk = pk;
      }
      __syncthreads();
      shp[tdx] = mypk;                 // share p[k]
      __syncthreads();
      zi = 0.0;                        // i-th component of W*p
      for(int k=0; k<j; k++)
      {
         shw[tdx] = W[k*nrows + tdx];  // load W[k][i]
         zi = zi + shw[tdx]*shp[k];
      }
      zi = zi + shv[tdx];
      wrk = -B[j]*zi;
      W[j*nrows + tdx] = wrk;          // wrk is assigned to W[j][tdx]
      __syncthreads();
   }
}

void GPU_dbl_blocked_houseqr
 ( int nrows, int ncols, int szt, int nbt,
   double **A, double **Q, double **R,
   double *houselapms, double *tileRlapms, double *vb2Wlapms,
   double *walltimesec, bool verbose )
{
   const int dim = nrows*ncols;         // total number of doubles
   double *A_h = new double[dim];       // matrix A on the host
   double *A_d;                         // matrix on the device
   double *v_h = new double[nrows];     // Householder vector on host
   double *x0_d;                        // first element for house on device
   double *beta_h = new double[szt];    // beta on the host
   double *beta_d;                      // beta on the device
   double *V_h = new double[nrows*szt]; // matrix of Householder vectors
   double *V_d;                         // Householder vectors on device
   double *W_h = new double[nrows*szt]; // the W matrix on the host
   double *W_d;                         // the W matrix 

   int ix=0;                            // copy the columns of A to A_h
   for(int j=0; j<ncols; j++)   
      for(int i=0; i<nrows; i++) A_h[ix++] = A[i][j];

   const size_t sznum = dim*sizeof(double);
   hipMalloc((void**)&A_d,sznum);
   hipMemcpy(A_d,A_h,sznum,hipMemcpyHostToDevice);
   const size_t szhouse = nrows*sizeof(double);
   hipMalloc((void**)&x0_d,sizeof(double));
   const size_t szbeta = szt*sizeof(double);
   hipMalloc((void**)&beta_d,szbeta);
   for(int i=0; i<szt; i++) beta_h[i] = 0.0;
   // beta_h[szt-1] = 1.0; // initialize last beta for square tiles
   hipMemcpy(beta_d,beta_h,szbeta,hipMemcpyHostToDevice);
   const size_t szVandW = szt*szhouse;
   hipMalloc((void**)&V_d,szVandW);
   ix = 0;
   for(int i=0; i<nrows*szt; i++) V_h[ix++] = 0.0; 
   V_h[--ix] = 1.0; // initialize last vector for square tiles
   hipMemcpy(V_d,V_h,szVandW,hipMemcpyHostToDevice);
   hipMalloc((void**)&W_d,szVandW);

   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   *houselapms = 0.0;
   *tileRlapms = 0.0;
   *vb2Wlapms = 0.0;
   float milliseconds;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   for(int k=0; k<nbt; k++)       // k runs over the number of blocks
   {
      int colidx,nrows1,nrLog2,rowidx;
      // int nbrblocks;

      for(int L=0; L<szt; L++)  // L runs over the columns in one block
      {
         colidx = k*szt + L;              // index of the current column
         nrows1 = nrows - colidx - 1;     // #rows in Householder vector - 1
         if(nrows1 > 0)
         {
            nrLog2 = ceil(log2((double) nrows1));
            rowidx = colidx*(nrows+1);       // start of number in A_h

            if(verbose)
            {
               cout << "nrows : " << nrows
                    << "  ncols : " << ncols
                    << "  szt : " << szt
                    << "  nbt : " << nbt << endl;
               cout << "k : " << k 
                    << "  L : " << L
                    << "  nrows1 : " << nrows1
                    << "  colidx : " << colidx
                    << "  rowidx : " << rowidx << endl;
            }
            hipMemcpy
               (x0_d,&A_h[rowidx],sizeof(double),hipMemcpyHostToDevice);
            hipEventRecord(start);
            dbl_small_house<<<1,nrows1>>>
               (x0_d,&A_d[rowidx+1],nrows1,nrLog2,&V_d[L*nrows+L],&beta_d[L]);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds,start,stop);
            *houselapms += milliseconds;
 
            if(verbose)
            {
               hipMemcpy(&beta_h[L],&beta_d[L],sizeof(double),
                          hipMemcpyDeviceToHost);
               hipMemcpy(v_h,&V_d[L*nrows],szhouse,hipMemcpyDeviceToHost);
               cout << scientific << setprecision(16)
                    << "beta[" << L << "] : " << beta_h[L] << endl;
               for(int i=0; i<nrows; i++)
                  cout << "v[" << i << "] : " << v_h[i] << endl;
            }
       /*
         nbrblocks = (nrows - colidx)/szt;
         if(((nrows - colidx) % szt) > 0) nbrblocks = nbrblocks + 1;

         if(verbose)
         {
            cout << "launching " << nbrblocks 
                 << " blocks of " << szt << " threads, ";
            cout << nbrblocks*szt << " threads to process "
                 << nrows - colidx << " rows ..." << endl;
         }
         hipEventRecord(start);
         dbl_factors_leftRupdate<<<nbrblocks,szt>>>
            (nrows,ncols,szt,colidx,A_d,v_d,beta_d);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *tileRlapms += milliseconds;
        */
            hipEventRecord(start);
            dbl_small_leftRupdate<<<1,nrows-colidx>>>
               (nrows,ncols,szt,colidx,A_d,&V_d[L*nrows+L],&beta_d[L]);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds,start,stop);
            *tileRlapms += milliseconds;
            if(verbose)
            {
               hipMemcpy(A_h,A_d,sznum,hipMemcpyDeviceToHost);
               cout << "the matrix after the update :" << endl;
               for(int i=0; i<nrows; i++)
                  for(int j=0; j<ncols; j++)
                     cout << "A_d[" << i << "][" << j << "] : "
                          << A_h[j*nrows+i] << endl;
            }
         } // end if(nrows1 > 0)
      } // end for(int L=0; L<szt; L++) loop
      hipEventRecord(start);
      dbl_VB_to_W<<<1,nrows>>>(nrows,ncols,beta_d,V_d,W_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *vb2Wlapms += milliseconds;
      if(verbose)
      {
         hipMemcpy(beta_h,beta_d,szbeta,hipMemcpyDeviceToHost);
         cout << "the betas :" << endl;
         for(int j=0; j<szt; j++)
            cout << "beta[" << j << "] : " << beta_h[j] << endl;
         hipMemcpy(V_h,V_d,szVandW,hipMemcpyDeviceToHost);
         cout << "the columns of the V matrix :" << endl;
         ix = 0;
         for(int j=0; j<szt; j++) 
            for(int i=0; i<nrows; i++) 
               cout << "V[" << i << "][" << j << "] : " << V_h[ix++] << endl;
         hipMemcpy(W_h,W_d,szVandW,hipMemcpyDeviceToHost);
         cout << "the columns of the W matrix :" << endl;
         ix = 0;
         for(int j=0; j<szt; j++) 
            for(int i=0; i<nrows; i++) 
               cout << "W[" << i << "][" << j << "] : " << W_h[ix++] << endl;
      }
   }
   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   free(A_h); free(v_h); free(V_h); free(W_h);
}
