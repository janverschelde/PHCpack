#include "hip/hip_runtime.h"
/* The file dbl_baqr_kernels.cu defines the functions with prototypes in
 * the file dbl_baqr_kernels.h. */

#include <iostream>
#include <iomanip>
#ifdef winwalltime
#include "gettimeofday4win.h"
#else
#include <sys/time.h>
#endif
#include "dbl_baqr_kernels.h"
#include "dbl_baqr_flopcounts.h"

using namespace std;

__global__ void dbl_small_house
 ( double *x0, double *x1, int dim, int dimLog2, double *v, double *beta )
{
   const int j = threadIdx.x;

   __shared__ double shv[d_shmemsize];
   __shared__ double prd[d_shmemsize];

   bool stopflag = false;
   double mu,v0,v0p2;

   shv[j] = x1[j];              // reading of vector into shared memory
   prd[j] = shv[j]*shv[j];      // for the 2-norm computation

   v[j+1] = shv[j];             // copies x to v, in case beta is zero
   if(j == 0) v[0] = 1.0;

   __syncthreads();
   int powTwo = 1;                          // sum reduction
   for(int k=0; k < dimLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < dim) prd[j] = prd[j] + prd[j+powTwo];
      powTwo = powTwo*2;
      __syncthreads();
   }
   // thread 0 computes the sqrt of the inner product, others wait
   if(j == 0)
   {
      if(prd[0] == 0.0)                    // prd[0] is sigma of house
      {
         *beta = 0.0; stopflag = true;
      }
   }
   __syncthreads();
   if(stopflag) return;                    // case when sigma is zero
   if(j == 0)                              // thread zero sets beta
   {
      mu = sqrt((*x0)*(*x0) + prd[0]);
      if(*x0 <= 0.0)
         v0 = *x0 - mu;
      else
         v0 = -prd[0]/(*x0 + mu);

      v0p2 = v0*v0;
      *beta = 2.0*v0p2/(prd[0] + v0p2);
      prd[0] = v0;                         // v0 needed for normalization
   }
   __syncthreads();
   if(*beta != 0.0) shv[j] = shv[j]/prd[0];
   __syncthreads();
   v[j+1] = shv[j];
   if(j == 0) v[0] = 1.0;
}

__global__ void cmplx_small_house
 ( double *x0re, double *x0im, double *x1re, double *x1im,
   int dim, int dimLog2, double *vre, double *vim, double *beta )
{
   const int j = threadIdx.x;

   __shared__ double shvre[cd_shmemsize];
   __shared__ double shvim[cd_shmemsize];
   __shared__ double prd[cd_shmemsize];
   __shared__ double v0parts[2];

   bool stopflag = false;
   double mu,v0re,v0im,x0rad,sqrx0,sqrv0,inv0re,inv0im,zre,zim;

   shvre[j] = x1re[j];          // reading of vector into shared memory
   shvim[j] = x1im[j];
   // prd[j] = shv[j]*shv[j];   // for the 2-norm computation
   prd[j] = shvre[j]*shvre[j] + shvim[j]*shvim[j];

   vre[j+1] = shvre[j];         // copies x to v, in case beta is zero
   vim[j+1] = shvim[j];
   if(j == 0) vre[0] = 1.0;
   if(j == 0) vim[0] = 0.0;

   __syncthreads();
   int powTwo = 1;                          // sum reduction
   for(int k=0; k < dimLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < dim) prd[j] = prd[j] + prd[j+powTwo];
      powTwo = powTwo*2;
      __syncthreads();
   }
   // thread 0 computes the sqrt of the inner product, others wait
   if(j == 0)
   {
      if(prd[0] == 0.0)                    // prd[0] is sigma of house
      {
         *beta = 0.0; stopflag = true;
      }
   }
   __syncthreads();
   if(stopflag) return;                    // case when sigma is zero
   if(j == 0)                              // thread zero sets beta
   {
      sqrx0 = (*x0re)*(*x0re) + (*x0im)*(*x0im);
      x0rad = sqrt(sqrx0);
      mu = sqrt(sqrx0 + prd[0]);

      if(x0rad == 0.0)
      {
         v0re = -mu;
         v0im = 0.0;
      }
      else
      {
         mu = mu/x0rad;
         v0re = (*x0re) - mu*(*x0re);
         v0im = (*x0im) - mu*(*x0im);
      }
      sqrv0 = v0re*v0re + v0im*v0im;
      *beta = 2.0*sqrv0/(prd[0] + sqrv0);

      prd[0] = sqrv0;                     // sqrv0 needed for normalization
      v0parts[0] = v0re;                  // share v0re with all threads
      v0parts[1] = v0im;                  // share v0im with all threads
   }
   __syncthreads();
   if(prd[0] != 0.0)
   {
      inv0re = v0parts[0]/prd[0];               // real part of 1/v[0]
      inv0im = -v0parts[1]/prd[0];              // imag part of 1/v[0]
      zre = shvre[j]*inv0re - shvim[j]*inv0im;  // real part of v[j]/v[0]
      zim = shvim[j]*inv0re + shvre[j]*inv0im;  // imag part of v[j]/v[0]
      vre[j+1] = zre;
      vim[j+1] = zim;
   }
   __syncthreads();
   if(j == 0) vre[0] = 1.0;
   if(j == 0) vim[0] = 0.0;
}

__global__ void dbl_small_leftRupdate
 ( int nrows, int ncols, int szt, int k, double *R, double *v, double *beta )
{
   const int tdx = threadIdx.x;          // index of thread in block
   const int Roffset = k*nrows + k;
   int Rcolidx;
   double w,Rtdx;

   __shared__ double shv[d_shmemsize]; // slice of v

   shv[tdx] = v[tdx];
   __syncthreads();
   w = 0.0;

   for(int i=0; i<nrows-k; i++)   // loop through rows of R
   {
      Rtdx = R[Roffset + i + tdx*nrows];
      w = w + Rtdx*shv[i];
   }
   w = (*beta)*w;
   __syncthreads();
   for(int i=0; i<nrows-k; i++)   // update i-th row of R
   {
      Rcolidx = Roffset + i + tdx*nrows;
      Rtdx = R[Rcolidx];
      Rtdx = Rtdx - shv[i]*w;
      __syncthreads();
      // changed nrows-k into ncols-k, where ncols = endcol
      if(tdx < ncols-k) R[Rcolidx] = Rtdx;
   }
}

__global__ void cmplx_small_leftRupdate
 ( int nrows, int ncols, int szt, int k,
   double *Rre, double *Rim, double *vre, double *vim, double *beta )
{
   const int tdx = threadIdx.x;          // index of thread in block
   const int Roffset = k*nrows + k;
   int Rcolidx;
   double w_re,w_im,Rtdx_re,Rtdx_im,acc;

   __shared__ double shvre[cd_shmemsize]; // slice of vre
   __shared__ double shvim[cd_shmemsize]; // slice of vim

   shvre[tdx] = vre[tdx];
   shvim[tdx] = vim[tdx];
   __syncthreads();
   w_re = 0.0;
   w_im = 0.0;

   for(int i=0; i<nrows-k; i++)   // loop through rows of R
   {
      Rcolidx = Roffset + i + tdx*nrows;
      Rtdx_re = Rre[Rcolidx];
      Rtdx_im = Rim[Rcolidx];
      // w = w + Rtdx*shv[i]; beware of the Hermitian transpose!
      w_re = w_re + Rtdx_re*shvre[i] + Rtdx_im*shvim[i];
      w_im = w_im - Rtdx_im*shvre[i] + Rtdx_re*shvim[i];
   }
   acc = *beta;
   w_re = acc*w_re;
   w_im = acc*w_im;
   __syncthreads();
   for(int i=0; i<nrows-k; i++)   // update i-th row of R
   {
      Rcolidx = Roffset + i + tdx*nrows;
      Rtdx_re = Rre[Rcolidx];
      Rtdx_im = Rim[Rcolidx];
      // Rtdx = Rtdx - shv[i]*w; beware of the Hermitian transpose!
      Rtdx_re = Rtdx_re - (shvre[i]*w_re + shvim[i]*w_im);
      Rtdx_im = Rtdx_im - (shvim[i]*w_re - shvre[i]*w_im);
      __syncthreads();
      // changed nrows-k into ncols-k, where ncols = endcol
      if(tdx < ncols-k)
      {
         Rre[Rcolidx] = Rtdx_re;
         Rim[Rcolidx] = Rtdx_im;
      }
   }
}

__global__ void dbl_small_betaRTv
 ( int nrows, int ncols, int szt, int k,
   double *R, double *v, double *beta, double *w )
{
   const int tdx = threadIdx.x;          // index of thread in block
   const int Roffset = k*nrows + k;
   double result = 0.0;
   const double mybeta = *beta;
   double Rtdx;

   __shared__ double shv[d_shmemsize]; // slice of v

   shv[tdx] = v[tdx];
   __syncthreads();

   for(int i=0; i<nrows-k; i++)   // loop through rows of R
   {
      Rtdx = R[Roffset + i + tdx*nrows];
      result = result + Rtdx*shv[i];
   }
   result = mybeta*result;
   w[tdx] = result;
}

__global__ void cmplx_small_betaRHv
 ( int nrows, int ncols, int szt, int k,
   double *Rre, double *Rim, double *vre, double *vim, double *beta,
   double *wre, double *wim )
{
   const int tdx = threadIdx.x;          // index of thread in block
   const int Roffset = k*nrows + k;
   int Rcolidx;
   double resultre = 0.0;
   double resultim = 0.0;
   const double mybeta = *beta;
   double Rtdx_re;
   double Rtdx_im;

   __shared__ double shvre[cd_shmemsize]; // slice of v
   __shared__ double shvim[cd_shmemsize]; 

   shvre[tdx] = vre[tdx];
   shvim[tdx] = vim[tdx];
   __syncthreads();

   for(int i=0; i<nrows-k; i++)   // loop through rows of R
   {
      Rcolidx = Roffset + i + tdx*nrows;
      Rtdx_re = Rre[Rcolidx];
      Rtdx_im = Rim[Rcolidx];
      // do not forget about the Hermitian transpose of R
      resultre = resultre + (  Rtdx_re*shvre[i] + Rtdx_im*shvim[i]);
      resultim = resultim + (- Rtdx_im*shvre[i] + Rtdx_re*shvim[i]);
   }
   resultre = mybeta*resultre;
   resultim = mybeta*resultim;
   wre[tdx] = resultre;
   wim[tdx] = resultim;
}

__global__ void dbl_medium_betaRTv
 ( int nrows, int ncols, int szt, int k,
   double *R, double *v, double *beta, double *w )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;       // index of thread in block
   const int Roffset = k*nrows + k;
   const int widx = bdx*szt + tdx;    // thread tdx computes w[widx]
   const int endrow = nrows - k;
   const int nbr = endrow/szt;
   int vidx = 0;
   double result = 0.0;
   double Rtdx;

   __shared__ double shv[d_shmemsize]; // slice of v

   shv[tdx] = v[tdx];
   for(int i=0; i<nbr; i++) 
   {
      vidx = vidx + szt;
      shv[vidx] = v[vidx];
   }
   __syncthreads();

   for(int i=0; i<endrow; i++)   // loop through rows of R
   {
      Rtdx = R[Roffset + i + widx*nrows];  // instead of tdx, use widx
      result = result + Rtdx*shv[i];
   }
   result = (*beta)*result;
   w[widx] = result;
}

__global__ void dbl_RTdotv
 ( int nrows, int szt, int colidx, int Roffset, int dim,
   double *R, double *v, double *RTdotv )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int idx = bdx*szt + tdx;        // thread tdx computes RTv[idx]

   const int vdx = idx % nrows;          // index in v is column in R^T
   const int row = idx / nrows;          // R is stored column-by-column

   const int Rdx = Roffset + idx + (row+1)*colidx;

   const double Vval = v[vdx];
   const double Rval = R[Rdx];
   double result = Rval*Vval;

   RTdotv[idx] = result;
}

__global__ void cmplx_RHdotv
 ( int nrows, int szt, int colidx, int Roffset, int dim,
   double *Rre, double *Rim, double *vre, double *vim,
   double *RHdotvre, double *RHdotvim )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int idx = bdx*szt + tdx;        // thread tdx computes RTv[idx]

   const int vdx = idx % nrows;          // index in v is column in R^T
   const int row = idx / nrows;          // R is stored column-by-column

   const int Rdx = Roffset + idx + (row+1)*colidx;

   const double Vvalre = vre[vdx];
   const double Vvalim = vim[vdx];
   const double Rvalre = Rre[Rdx];
   const double Rvalim = Rim[Rdx];
   // Hermitian transpose of R
   double resultre =   Rvalre*Vvalre + Rvalim*Vvalim;
   double resultim = - Rvalim*Vvalre + Rvalre*Vvalim;

   RHdotvre[idx] = resultre;
   RHdotvim[idx] = resultim;
}

__global__ void dbl_sum_betaRTdotv
 ( int nrows, double *beta, double *RTdotv, double *w )
{
   const int tdx = threadIdx.x;  // tdx sums elements on row tdx
   const int offset = tdx*nrows; // number of rows before current row

   double result = 0.0;
   double Rval;

   for(int i=0; i<nrows; i++)
   {
      Rval = RTdotv[offset + i];
      result = result + Rval;
   }
   Rval = *beta;
   w[tdx] = Rval*result;
}

__global__ void cmplx_sum_betaRHdotv
 ( int nrows, double *beta, double *RHdotvre, double *RHdotvim,
   double *wre, double *wim )
{
   const int tdx = threadIdx.x;  // tdx sums elements on row tdx
   const int offset = tdx*nrows; // number of rows before current row
   int idx;

   double resultre = 0.0;
   double resultim = 0.0;
   double Rvalre,Rvalim;

   for(int i=0; i<nrows; i++)
   {
      idx = offset + i;
      Rvalre = RHdotvre[idx];
      Rvalim = RHdotvim[idx];
      resultre = resultre + Rvalre;
      resultim = resultim + Rvalim;
   }
   Rvalre = *beta;
   wre[tdx] = Rvalre*resultre;
   wim[tdx] = Rvalre*resultim;
}

__global__ void dbl_medium_subvbetaRTv
 ( int nrows, int ncols, int szt, int k,
   double *R, double *v, double *beta, double *w )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int Roffset = k*nrows + k;    // start in R
   const int widx = bdx*szt + tdx;     // global thread index 

   const int coldim = ncols - k;       // number of columns in R
   const int bound = coldim*(nrows-k); // bound on Ridx
   const int rowidx = widx / coldim;   // row index
   const int colidx = widx % coldim;   // column index

   const int Ridx = Roffset + nrows*colidx + rowidx;

   __shared__ double shw[d_shmemsize];  // values in beta*R^T*v
   shw[tdx] = w[tdx];                   // are less in number than szt
   __syncthreads();

   double Rwidx = R[Ridx];             // number that tdx updates
   double vValue = v[rowidx];          // value in Householder vector
   double wValue = shw[colidx];        // value in beta*R^T*v
 
   Rwidx = Rwidx - vValue*wValue;      // update R[rowidx,colidx]

   if(widx < bound) R[Ridx] = Rwidx;   // if() takes care of padding
}

__global__ void cmplx_medium_subvbetaRHv
 ( int nrows, int ncols, int szt, int k,
   double *Rre, double *Rim, double *vre, double *vim, double *beta,
   double *wre, double *wim )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int Roffset = k*nrows + k;    // start in R
   const int widx = bdx*szt + tdx;     // global thread index 

   const int coldim = ncols - k;       // number of columns in R
   const int bound = coldim*(nrows-k); // bound on Ridx
   const int rowidx = widx / coldim;   // row index
   const int colidx = widx % coldim;   // column index

   const int Ridx = Roffset + nrows*colidx + rowidx;

   __shared__ double shwre[cd_shmemsize];  // values in beta*R^T*v
   __shared__ double shwim[cd_shmemsize];  // are less in number than szt
   shwre[tdx] = wre[tdx];
   shwim[tdx] = wim[tdx];
   __syncthreads();

   double Rwidxre = Rre[Ridx];         // number that tdx updates
   double Rwidxim = Rim[Ridx];
   double vValre = vre[rowidx];        // value in Householder vector
   double vValim = vim[rowidx];
   double wValre = shwre[colidx];      // value in beta*R^T*v
   double wValim = shwim[colidx];
 
   // Rwidx = Rwidx - vValue*wValue;   // update R[rowidx,colidx]
   // take the Hermitian transpose of w
   Rwidxre = Rwidxre - (vValre*wValre + vValim*wValim);
   Rwidxim = Rwidxim - (vValim*wValre - vValre*wValim);

   if(widx < bound)                    // if() takes care of padding
   {
      Rre[Ridx] = Rwidxre;
      Rim[Ridx] = Rwidxim;
   }
}

__global__ void dbl_VB_to_W
 ( int nrows, int ncols, double *B, double *V, double *W )
{
   const int tdx = threadIdx.x;        // index of thread in block
   double wrk,pk,mypk,zi;

   __shared__ double shv[d_shmemsize]; // one work vector
   __shared__ double shw[d_shmemsize]; // the other work vector
   __shared__ double shp[d_shmemsize]; // to share Y^T*v

   shv[tdx] = V[tdx];
   wrk = -B[0]*shv[tdx];               // first column of W
   W[tdx] = wrk;

   for(int j=1; j<ncols; j++)          // compute column j of W
   {
      shv[tdx] = V[j*nrows + tdx];     // j-th Householder vector
      for(int k=0; k<j; k++)
      {
         pk = 0.0;                     // k-th component of Y^T*v
         shw[tdx] = V[k*nrows + tdx];  // load V[k][i]
         shp[tdx] = shw[tdx]*shv[tdx]; // V[k][i]*v[i]
         __syncthreads();
         for(int i=0; i<nrows; i++) pk = pk + shp[i];
         __syncthreads();              // critical synchronization
         if(tdx == k) mypk = pk;
      }
      __syncthreads();
      shp[tdx] = mypk;                 // share p[k]
      __syncthreads();
      zi = 0.0;                        // i-th component of W*p
      for(int k=0; k<j; k++)
      {
         shw[tdx] = W[k*nrows + tdx];  // load W[k][i]
         zi = zi + shw[tdx]*shp[k];
      }
      zi = zi + shv[tdx];
      wrk = -B[j]*zi;
      W[j*nrows + tdx] = wrk;          // wrk is assigned to W[j][tdx]
      __syncthreads();
   }
}

__global__ void cmplx_VB_to_W
 ( int nrows, int ncols, double *B,
   double *Vre, double *Vim, double *Wre, double *Wim )
{
   const int tdx = threadIdx.x;        // index of thread in block
   double wrk_re,wrk_im,pk_re,pk_im,mypk_re,mypk_im,zi_re,zi_im;
   int VWidx;

   __shared__ double shvre[cd_shmemsize]; // one work vector
   __shared__ double shvim[cd_shmemsize];
   __shared__ double shwre[cd_shmemsize]; // the other work vector
   __shared__ double shwim[cd_shmemsize];
   __shared__ double shpre[cd_shmemsize]; // to share Y^T*v
   __shared__ double shpim[cd_shmemsize];

   shvre[tdx] = Vre[tdx];
   shvim[tdx] = Vim[tdx];
   wrk_re = -B[0]*shvre[tdx];            // first column of W
   wrk_im = -B[0]*shvim[tdx];
   Wre[tdx] = wrk_re;
   Wim[tdx] = wrk_im;

   for(int j=1; j<ncols; j++)          // compute column j of W
   {
      VWidx = j*nrows + tdx;
      shvre[tdx] = Vre[VWidx];         // j-th Householder vector
      shvim[tdx] = Vim[VWidx];

      for(int k=0; k<j; k++)
      {
         pk_re = 0.0;                  // k-th component of Y^H*v
         pk_im = 0.0;
         VWidx = k*nrows + tdx;
         shwre[tdx] = Vre[VWidx];      // load V[k][i]
         shwim[tdx] = Vim[VWidx];
         // shp[tdx] = shw[tdx]*shv[tdx]; V[k][i]*v[i], Hermitian transpose!
         shpre[tdx] =   shwre[tdx]*shvre[tdx] + shwim[tdx]*shvim[tdx];
         shpim[tdx] = - shwim[tdx]*shvre[tdx] + shwre[tdx]*shvim[tdx];
         __syncthreads();
         for(int i=0; i<nrows; i++)
         {
            pk_re = pk_re + shpre[i];
            pk_im = pk_im + shpim[i];
         }
         __syncthreads();              // important synchronization
         if(tdx == k)
         {
            mypk_re = pk_re;
            mypk_im = pk_im;
         }
      }
      __syncthreads();
      shpre[tdx] = mypk_re;            // share p[k]
      shpim[tdx] = mypk_im;
      __syncthreads();
      zi_re = 0.0;                     // i-th component of W*p
      zi_im = 0.0;
      for(int k=0; k<j; k++)
      {
         VWidx = k*nrows + tdx;
         shwre[tdx] = Wre[VWidx];      // load W[k][i]
         shwim[tdx] = Wim[VWidx];
         // zi = zi + shw[tdx]*shp[k];
         zi_re = zi_re + shwre[tdx]*shpre[k] - shwim[tdx]*shpim[k];
         zi_im = zi_im + shwim[tdx]*shpre[k] + shwre[tdx]*shpim[k];
      }
      zi_re = zi_re + shvre[tdx];
      zi_im = zi_im + shvim[tdx];
      wrk_re = -B[j]*zi_re;
      wrk_im = -B[j]*zi_im;
      VWidx = j*nrows + tdx;
      Wre[VWidx] = wrk_re;             // wrk is assigned to W[j][tdx]
      Wim[VWidx] = wrk_im;
      __syncthreads();
   }
}

__global__ void dbl_beta_times_V
 ( int nrows, int szt, double *B, double *V, double *W )
{
   const int bdx = blockIdx.x;        // index of block
   const int tdx = threadIdx.x;       // index of thread in block
   const int idx = bdx*szt + tdx;     // thread tdx computes W[idx]
   double result;

   __shared__ double shv[d_shmemsize]; // to store a slice of V

   shv[tdx] = V[idx]; // thread tdx loads the data at the global index

   result = -B[0]*shv[tdx];

   if(idx < nrows) W[idx] = result;
}

__global__ void cmplx_beta_times_V
 ( int nrows, int szt, double *B,
   double *Vre, double *Vim, double *Wre, double *Wim )
{
   const int bdx = blockIdx.x;        // index of block
   const int tdx = threadIdx.x;       // index of thread in block
   const int idx = bdx*szt + tdx;     // thread tdx computes W[idx]
   double resultre,resultim;

   __shared__ double shvre[cd_shmemsize]; // to store a slice of V,
   __shared__ double shvim[cd_shmemsize]; // imaginary parts

   shvre[tdx] = Vre[idx]; // thread tdx loads the data
   shvim[tdx] = Vim[idx]; // at the global index

   resultre = -B[0]*shvre[tdx];
   resultim = -B[0]*shvim[tdx];

   if(idx < nrows)
   {  
      Wre[idx] = resultre;
      Wim[idx] = resultim;
   }
}

__global__ void dbl_initialize_WYT
 ( int dim, int szt, double *V, double *W, double *WYT )
{
   const int bdx = blockIdx.x;        // index of block
   const int tdx = threadIdx.x;       // index of thread in block
   const int idx = bdx*szt + tdx;     // global index of the thread
   const int row = idx / dim;         // row index in YWT
   const int col = idx % dim;         // column index in YWT

   const double Vval = V[col];
   const double Wval = W[row];
   const double result = Vval*Wval;

   if(idx < dim*dim) WYT[idx] = result;
}

__global__ void cmplx_initialize_WYH
 ( int dim, int szt, double *Vre, double *Vim, double *Wre, double *Wim,
   double *WYHre, double *WYHim )
{
   const int bdx = blockIdx.x;        // index of block
   const int tdx = threadIdx.x;       // index of thread in block
   const int idx = bdx*szt + tdx;     // global index of the thread
   const int row = idx / dim;         // row index in YWT
   const int col = idx % dim;         // column index in YWT

   const double Vvalre = Vre[col];
   const double Vvalim = Vim[col];
   const double Wvalre = Wre[row];
   const double Wvalim = Wim[row];
   // beware of the Hermitian transpose of W, must be V instead!
   const double resultre =   Vvalre*Wvalre + Vvalim*Wvalim;
   const double resultim = - Vvalim*Wvalre + Vvalre*Wvalim;

   if(idx < dim*dim)
   {
      WYHre[idx] = resultre;
      WYHim[idx] = resultim;
   }
}

__global__ void dbl_update_WYT
 ( int dim, int szt, double *V, double *W, double *WYT )
{
   const int bdx = blockIdx.x;        // index of block
   const int tdx = threadIdx.x;       // index of thread in block
   const int idx = bdx*szt + tdx;     // global index of the thread
   const int row = idx / dim;         // row index in YWT
   const int col = idx % dim;         // column index in YWT

   const double Vval = V[col];
   const double Wval = W[row];
   double result = WYT[idx];

   result = result + Vval*Wval;

   if(idx < dim*dim) WYT[idx] = result;
}

__global__ void cmplx_update_WYH
 ( int dim, int szt, double *Vre, double *Vim, double *Wre, double *Wim,
   double *WYHre, double *WYHim )
{
   const int bdx = blockIdx.x;        // index of block
   const int tdx = threadIdx.x;       // index of thread in block
   const int idx = bdx*szt + tdx;     // global index of the thread
   const int row = idx / dim;         // row index in YWT
   const int col = idx % dim;         // column index in YWT

   const double Vvalre = Vre[col];
   const double Vvalim = Vim[col];
   const double Wvalre = Wre[row];
   const double Wvalim = Wim[row];

   double resultre = WYHre[idx];
   double resultim = WYHim[idx];

   // beware of the Hermitian transpose of W, must be V instead!
   resultre = resultre + Vvalre*Wvalre + Vvalim*Wvalim;
   resultim = resultim - Vvalim*Wvalre + Vvalre*Wvalim;

   if(idx < dim*dim)
   {
      WYHre[idx] = resultre;
      WYHim[idx] = resultim;
   }
}

__global__ void dbl_beta_next_W
 ( int nrows, int szt, double *B, double *V, double *W, double *WYT )
{
   const int bdx = blockIdx.x;        // index of block
   const int tdx = threadIdx.x;       // index of thread in block
   const int idx = bdx*szt + tdx;     // global index of the thread
   const int WYToff = idx*nrows;      // start of idx row in YWT
   const double mybeta = B[0];
   int vdx;
   double result,WYTval,Vvalue;

   __shared__ double shV[d_shmemsize];   // to store a slice of V

   shV[tdx] = V[idx]; // thread tdx loads the data at the global index

   __syncthreads();
   result = shV[tdx]; // thread tdx computes the value at index idx

   for(int i=0; i<nrows/szt; i++)
   {
      vdx = i*szt + tdx;                 // index in V and in YWT
      shV[tdx] = V[vdx];                 // threads load next szt values

      __syncthreads();
      for(int j=0; j<szt; j++)           // multiply szt values with YWT
      {
         WYTval = WYT[WYToff+i*szt+j];   // YWT is stored row by row
         Vvalue = shV[j];
         result = result + WYTval*Vvalue;
      }
      __syncthreads();
   }
   int quot = nrows/szt;
   int rest = nrows - quot*szt;          // remainder to compute

   vdx = quot*szt + tdx;                 // next index to compute
   shV[tdx] = V[vdx];

   for(int j=0; j<rest; j++)            // rest < szt prevents overflow
   {
      __syncthreads();
      WYTval = WYT[WYToff+quot*szt+j];
      Vvalue = shV[j];
      result = result + WYTval*Vvalue;
   }
   result = -mybeta*result;

   if(idx < nrows) W[idx] = result;
}

__global__ void cmplx_beta_next_W
 ( int nrows, int szt, double *B, double *Vre, double *Vim,
   double *Wre, double *Wim, double *WYHre, double *WYHim )
{
   const int bdx = blockIdx.x;        // index of block
   const int tdx = threadIdx.x;       // index of thread in block
   const int idx = bdx*szt + tdx;     // global index of the thread
   const int WYHoff = idx*nrows;      // start of idx row in YWT
   const double mybeta = B[0];
   int vdx,ydx;
   double resultre,resultim,WYHvre,WYHvim,Vvalre,Vvalim;

   __shared__ double shVre[cd_shmemsize];   // to store a slice of V
   __shared__ double shVim[cd_shmemsize];

   shVre[tdx] = Vre[idx]; // thread tdx loads data at the global index
   shVim[tdx] = Vim[idx];

   __syncthreads();
   resultre = shVre[tdx]; // thread tdx computes the value at index idx
   resultim = shVim[tdx];

   for(int i=0; i<nrows/szt; i++)
   {
      vdx = i*szt + tdx;                 // index in V and in YWT
      shVre[tdx] = Vre[vdx];             // threads load next szt values
      shVim[tdx] = Vim[vdx];

      __syncthreads();
      for(int j=0; j<szt; j++)           // multiply szt values with YWT
      {
         ydx = WYHoff + i*szt + j;       // YWT is stored row by row
         WYHvre = WYHre[ydx];
         WYHvim = WYHim[ydx];
         Vvalre = shVre[j];
         Vvalim = shVim[j];
         // result = result + YWTval*Vvalue;
         resultre = resultre + WYHvre*Vvalre - WYHvim*Vvalim;
         resultim = resultim + WYHvim*Vvalre + WYHvre*Vvalim;
      }
      __syncthreads();
   }
   int quot = nrows/szt;
   int rest = nrows - quot*szt;          // remainder to compute

   vdx = quot*szt + tdx;                 // next index to compute
   shVre[tdx] = Vre[vdx];
   shVim[tdx] = Vim[vdx];

   for(int j=0; j<rest; j++)            // rest < szt prevents overflow
   {
      __syncthreads();
      ydx = WYHoff + quot*szt + j;
      WYHvre = WYHre[ydx];
      WYHvim = WYHim[ydx];
      Vvalre = shVre[j];
      Vvalim = shVim[j];
      // result = result + YWTval*Vvalue;
      resultre = resultre + WYHvre*Vvalre - WYHvim*Vvalim;
      resultim = resultim + WYHvim*Vvalre + WYHvre*Vvalim;
   }
   resultre = -mybeta*resultre;
   resultim = -mybeta*resultim;

   if(idx < nrows)
   {
      Wre[idx] = resultre;
      Wim[idx] = resultim;
   }
}

__global__ void dbl_small_WYT
 ( int nrows, int szt, double *W, double *Y, double *WYT )
{
   const int bdx = blockIdx.x;           // index of block
   const int tdx = threadIdx.x;          // index of thread in block
   const int offset = bdx*szt + tdx;     // offset in result
   const int row = offset / nrows;
   const int col = offset % nrows;       // thread 0 computes WYT[row][col]

   double result = 0.0;
   double a,b;

   for(int k=0; k<szt; k++)
   {
      a = W[k*nrows + row];   // if(nrows == szt) then row = bdx
      b = Y[k*nrows + col];   // if(nrows == szt) then col = tdx
      result = result + a*b;
   }
   __syncthreads();
   WYT[offset] = result;
}

__global__ void cmplx_small_WYH
 ( int nrows, int szt, double *Wre, double *Wim,
   double *Yre, double *Yim, double *WYHre, double *WYHim )
{
   const int bdx = blockIdx.x;           // index of block
   const int tdx = threadIdx.x;          // index of thread in block
   const int offset = bdx*szt + tdx;     // offset in result
   const int row = offset / nrows;
   const int col = offset % nrows;       // thread 0 computes WYT[row][col]

   double resultre = 0.0;
   double resultim = 0.0;
   double a_re,a_im,b_re,b_im;
   int Widx,Yidx;

   for(int k=0; k<szt; k++)
   {
      Widx = k*nrows + row;
      a_re = Wre[Widx];            // if(nrows == szt) then row = bdx
      a_im = Wim[Widx]; 
      Yidx = k*nrows + col;
      b_re = Yre[Yidx];            // if(nrows == szt) then col = tdx
      b_im = Yim[Yidx];
      // result = result + a*b; with Hermitian transpose of Y
      resultre = resultre + a_re*b_re + a_im*b_im;
      resultim = resultim + a_im*b_re - a_re*b_im;
   }
   __syncthreads();
   WYHre[offset] = resultre;
   WYHim[offset] = resultim;
}

__global__ void dbl_small_QWYT
 ( int dim, int rowdim, int szt, int coloff,
   double *Q, double *WYT, double *QWYT )
{
   const int bdx = blockIdx.x;         // index of block
   const int tdx = threadIdx.x;        // index of thread in block
   const int offset = bdx*szt + tdx;   // offset in result
   const int row = offset / rowdim;
   const int col = offset % rowdim;    // thread 0 computes QWYT[row][col]

   double result = 0.0;
   double a,b;

   for(int k=0; k<rowdim; k++)       // run over rowdim, not just szt
   {                                 // coloff shifts by col*row elements
      a = Q[row*dim + coloff + k];   // row = bdx, if dim == szt, coloff == 0
      b = WYT[k*rowdim + col];       // if(dim == szt) then col = tdx
      result = result + a*b;
   }
   __syncthreads();
   QWYT[offset] = result;            // no column offset in saving QWYT
}

__global__ void cmplx_small_QWYH
 ( int dim, int rowdim, int szt, int coloff,
   double *Qre, double *Qim, double *WYHre, double *WYHim,
   double *QWYHre, double *QWYHim )
{
   const int bdx = blockIdx.x;         // index of block
   const int tdx = threadIdx.x;        // index of thread in block
   const int offset = bdx*szt + tdx;   // offset in result
   const int row = offset / rowdim;
   const int col = offset % rowdim;    // thread 0 computes QWYT[row][col]

   double resultre = 0.0;
   double resultim = 0.0;
   double a_re,a_im,b_re,b_im;
   int Qidx,WYTidx;

   for(int k=0; k<rowdim; k++)          // run over rowdim, not just szt
   {                                    // coloff shifts by col*row elements
      Qidx = row*dim + coloff + k;
      a_re = Qre[Qidx];                 // row = bdx,
      a_im = Qim[Qidx];                 // if dim == szt, coloff == 0
      WYTidx = k*rowdim + col;
      b_re = WYHre[WYTidx];             // if(dim == szt) then col = tdx
      b_im = WYHim[WYTidx];
      // result = result + a*b;
      resultre = resultre + a_re*b_re - a_im*b_im;
      resultim = resultim + a_im*b_re + a_re*b_im;
   }
   __syncthreads();
   QWYHre[offset] = resultre;           // no column offset in saving QWYT
   QWYHim[offset] = resultim;
}

__global__ void dbl_small_YWTC
 ( int nrows, int ncols, int rowdim, int coldim, int szt,
   int rowoff, int coloff, double *YWT, double *C, double *YWTC )
{
   const int bdx = blockIdx.x;         // bdx*szt done by previous blocks
   const int tdx = threadIdx.x;        // index of thread in block
   const int offset = bdx*szt + tdx;   // offset in result
   const int row = offset / coldim;    // 1st thread does YWTC[row][col]
   const int col = offset % coldim;
   const int colCoff0 = (coloff+col)*nrows + rowoff; // 1st element in C

   double result = 0.0;
   double a,b;

   for(int k=0; k<rowdim; k++)         // innermost loop runs over rowdim
   {
      a = YWT[row*rowdim + k];         // YWT is stored row by row
      b = C[colCoff0 + k];             // but C is stored column by column
      result = result + a*b;
   }
   __syncthreads();
   YWTC[(coloff + col)*nrows + (rowoff + row)] = result;
}

__global__ void cmplx_small_YWHC
 ( int nrows, int ncols, int rowdim, int coldim, int szt,
   int rowoff, int coloff, double *YWHre, double *YWHim,
   double *Cre, double *Cim, double *YWHCre, double *YWHCim )
{
   const int bdx = blockIdx.x;         // bdx*szt done by previous blocks
   const int tdx = threadIdx.x;        // index of thread in block
   const int offset = bdx*szt + tdx;   // offset in result
   const int row = offset / coldim;    // 1st thread does YWTC[row][col]
   const int col = offset % coldim;
   const int colCoff0 = (coloff+col)*nrows + rowoff; // 1st element in C

   double resultre = 0.0;
   double resultim = 0.0;
   double a_re,a_im,b_re,b_im;
   int YWHidx,Cidx;

   for(int k=0; k<rowdim; k++)         // innermost loop runs over rowdim
   {
      YWHidx = row*rowdim + k;
      a_re = YWHre[YWHidx];           // YWH is stored row by row
      a_im = YWHim[YWHidx];
      Cidx = colCoff0 + k;
      b_re = Cre[Cidx];               // but C is stored column by column
      b_im = Cim[Cidx];
      // result = result + a*b;
      resultre = resultre + a_re*b_re - a_im*b_im;
      resultim = resultim + a_im*b_re + a_re*b_im;
   }
   __syncthreads();
   YWHCre[(coloff + col)*nrows + (rowoff + row)] = resultre;
   YWHCim[(coloff + col)*nrows + (rowoff + row)] = resultim;
}

__global__ void dbl_small_Qupdate
 ( int dim, int rowdim, int szt, int coloff, double *Q, double *QWYT )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int offset = bdx*szt + tdx;   // offset in result
   const int row = offset / rowdim;
   const int col = offset % rowdim;
   const int idx1 = row*dim + coloff + col;

   double a,b;

   a = Q[idx1];       // row = bdx, if dim == szt, coloff == 0
   b = QWYT[offset];  // if(dim == szt) then col = tdx
   a = a + b;

   __syncthreads();
   Q[idx1] = a;
}

__global__ void cmplx_small_Qupdate
 ( int dim, int rowdim, int szt, int coloff,
   double *Qre, double *Qim, double *QWYHre, double *QWYHim )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int offset = bdx*szt + tdx;   // offset in result
   const int row = offset / rowdim;
   const int col = offset % rowdim;
   const int idx1 = row*dim + coloff + col;

   double a_re,a_im,b_re,b_im;

   a_re = Qre[idx1];       // row = bdx, if dim == szt, coloff == 0
   a_im = Qim[idx1];
   b_re = QWYHre[offset];  // if(dim == szt) then col = tdx
   b_im = QWYHim[offset];
   a_re = a_re + b_re;
   a_im = a_im + b_im;

   __syncthreads();
   Qre[idx1] = a_re;
   Qim[idx1] = a_im;
}

__global__ void dbl_small_R_add_YWTC
 ( int nrows, int coldim, int szt, int rowoff, int coloff,
   double *R, double *YWTC )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int offset = bdx*szt + tdx;   // offset in result
   const int row = offset / coldim;    // thread updates R[row][col]
   const int col = offset % coldim;
   const int idx = (coloff + col)*nrows + (rowoff + row);
 
   double a,b;
   
   a = R[idx];
   b = YWTC[idx];
   a = a + b;
  
   __syncthreads();
   R[idx] = a;
}

__global__ void cmplx_small_R_add_YWHC
 ( int nrows, int coldim, int szt, int rowoff, int coloff,
   double *Rre, double *Rim, double *YWHCre, double *YWHCim )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int offset = bdx*szt + tdx;   // offset in result
   const int row = offset / coldim;    // thread updates R[row][col]
   const int col = offset % coldim;
   const int idx = (coloff + col)*nrows + (rowoff + row);
 
   double a_re,a_im,b_re,b_im;
   
   a_re = Rre[idx];
   a_im = Rim[idx];
   b_re = YWHCre[idx];
   b_im = YWHCim[idx];
   a_re = a_re + b_re;
   a_im = a_im + b_im;
  
   __syncthreads();
   Rre[idx] = a_re;
   Rim[idx] = a_im;
}

void GPU_dbl_small_house
 ( int nrows, int ncols, int szt, int nbt,
   int colidx, int nrows1, int k, int L,
   double *A_h, double *A_d,
   double *v_h, double *V_d, double *beta_h, double *beta_d,
   double *lapms, long long int *add, long long int *mul, long long int *div,
   long long int *sqrtfun, bool verbose )
{
   const int nrLog2 = ceil(log2((double) nrows1));
   const int rowidx = colidx*(nrows+1);       // start of number in A_h
   const int nVrows = nrows - k*szt;          // dimension of V matrix

   if(verbose)
   {
      cout << "nrows : " << nrows
           << "  nVrows : " << nVrows
           << "  ncols : " << ncols
           << "  szt : " << szt
           << "  nbt : " << nbt << endl;
      cout << "k : " << k 
           << "  L : " << L
           << "  nrows1 : " << nrows1
           << "  colidx : " << colidx
           << "  rowidx : " << rowidx << endl;
   }
   if(L > 0)
   {
      for(int i=0; i<L; i++) v_h[i] = 0.0; // insert zeros
      hipMemcpy(&V_d[L*nVrows],v_h,L*sizeof(double),
                 hipMemcpyHostToDevice);
   }
   if(nrows1 == 0)
   {
      beta_h[L] = 0.0; v_h[0] = 1.0;
      hipMemcpy(&beta_d[L],&beta_h[L],sizeof(double),hipMemcpyHostToDevice);
      hipMemcpy(&V_d[L*nVrows+L],v_h,sizeof(double),hipMemcpyHostToDevice);
   }
   else
   {
      hipEvent_t start,stop;           // to measure time spent by kernels 
      hipEventCreate(&start);
      hipEventCreate(&stop);
      float milliseconds;

      hipEventRecord(start);
      dbl_small_house<<<1,nrows1>>>
         (&A_d[rowidx],&A_d[rowidx+1],nrows1,nrLog2,
          &V_d[L*nVrows+L],&beta_d[L]);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *lapms += milliseconds;
      flopcount_dbl_small_house(nrows1,nrLog2,add,mul,div,sqrtfun);
   }
   hipMemcpy(&beta_h[L],&beta_d[L],sizeof(double),hipMemcpyDeviceToHost);
   if(verbose)
   {
      const size_t szhouse = nVrows*sizeof(double);

      hipMemcpy(v_h,&V_d[L*nVrows],szhouse,hipMemcpyDeviceToHost);
      cout << scientific << setprecision(16)
           << "beta[" << colidx << "] : " << beta_h[L] << endl;
      for(int i=0; i<nVrows; i++)
         cout << "v[" << i << "] : " << v_h[i] << endl;
   }
}

void GPU_cmplx_small_house
 ( int nrows, int ncols, int szt, int nbt,
   int colidx, int nrows1, int k, int L,
   double *Are_h, double *Aim_h, double *Are_d, double *Aim_d,
   double *vre_h, double *vim_h, double *Vre_d, double *Vim_d,
   double *beta_h, double *beta_d,
   double *lapms, long long int *add, long long int *mul, long long int *div,
   long long int *sqrtfun, bool verbose )
{
   const int nrLog2 = ceil(log2((double) nrows1));
   const int rowidx = colidx*(nrows+1);       // start of number in A_h
   const int nVrows = nrows - k*szt;          // dimension of V matrix

   if(verbose)
   {
      cout << "nrows : " << nrows
           << "  nVrows : " << nVrows
           << "  ncols : " << ncols
           << "  szt : " << szt
           << "  nbt : " << nbt << endl;
      cout << "k : " << k 
           << "  L : " << L
           << "  nrows1 : " << nrows1
           << "  colidx : " << colidx
           << "  rowidx : " << rowidx << endl;
   }
   if(L > 0)
   {
      for(int i=0; i<L; i++)   // insert zeros
      {
         vre_h[i] = 0.0;
         vim_h[i] = 0.0;
      }
      hipMemcpy(&Vre_d[L*nVrows],vre_h,L*sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(&Vim_d[L*nVrows],vim_h,L*sizeof(double),
                 hipMemcpyHostToDevice);
   }
   if(nrows1 == 0)
   {
      beta_h[L] = 0.0; vre_h[0] = 1.0; vim_h[0] = 0.0;
      hipMemcpy(&beta_d[L],&beta_h[L],sizeof(double),hipMemcpyHostToDevice);
      hipMemcpy(&Vre_d[L*nVrows+L],vre_h,sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(&Vim_d[L*nVrows+L],vim_h,sizeof(double),
                 hipMemcpyHostToDevice);
   }
   else
   {
      hipEvent_t start,stop;           // to measure time spent by kernels 
      hipEventCreate(&start);
      hipEventCreate(&stop);
      float milliseconds;

      hipEventRecord(start);
      cmplx_small_house<<<1,nrows1>>>
         (&Are_d[rowidx],&Aim_d[rowidx],&Are_d[rowidx+1],&Aim_d[rowidx+1],
          nrows1,nrLog2,&Vre_d[L*nVrows+L],&Vim_d[L*nVrows+L],&beta_d[L]);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *lapms += milliseconds;
      flopcount_cmplx_small_house(nrows1,nrLog2,add,mul,div,sqrtfun);
   }
   hipMemcpy(&beta_h[L],&beta_d[L],sizeof(double),hipMemcpyDeviceToHost);
   if(verbose)
   {
      const size_t szhouse = nVrows*sizeof(double);

      hipMemcpy(vre_h,&Vre_d[L*nVrows],szhouse,hipMemcpyDeviceToHost);
      hipMemcpy(vim_h,&Vim_d[L*nVrows],szhouse,hipMemcpyDeviceToHost);
      cout << scientific << setprecision(16)
           << "beta[" << colidx << "] : " << beta_h[L] << endl;
      for(int i=0; i<nVrows; i++)
         cout << "v[" << i << "] : "
              << vre_h[i] << "  " << vim_h[i] << endl;
   }
}

void GPU_dbl_small_leftRupdate
 ( int nrows, int ncols, int szt, int colidx, int k, int L,
   double *A_h, double *A_d, double *V_d, double *beta_h, double *beta_d,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int endcol = (k+1)*szt;     // 1 + last column index in tile
   const int nVrows = nrows - k*szt;          // dimension of V matrix

   hipEventRecord(start);           // 2nd argument: ncols -> endcol
   // changed second argument ncols into endcol
   // to avoid updating the next tile
   // must use nrows - colidx instead of ncols - colidx
   dbl_small_leftRupdate<<<1,nrows-colidx>>>
      (nrows,endcol,szt,colidx,A_d,&V_d[L*nVrows+L],&beta_d[L]);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_small_leftRupdate(nrows,ncols,szt,colidx,add,mul);

   if(verbose)
   {
      const int dim = nrows*ncols;
      const size_t sznum = dim*sizeof(double);

      hipMemcpy(A_h,A_d,sznum,hipMemcpyDeviceToHost);
      cout << "the matrix after the update :" << endl;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++)
            cout << "A_d[" << i << "][" << j << "] : "
                 << A_h[j*nrows+i] << endl;
   }
}

void GPU_cmplx_small_leftRupdate
 ( int nrows, int ncols, int szt, int colidx, int k, int L,
   double *Are_h, double *Aim_h, double *Are_d, double *Aim_d,
   double *Vre_d, double *Vim_d, double *beta_h, double *beta_d,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int endcol = (k+1)*szt;     // 1 + last column index in tile
   const int nVrows = nrows - k*szt;          // dimension of V matrix

   hipEventRecord(start);
   cmplx_small_leftRupdate<<<1,nrows-colidx>>>
      (nrows,endcol,szt,colidx,Are_d,Aim_d,
       &Vre_d[L*nVrows+L],&Vim_d[L*nVrows+L],&beta_d[L]);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_small_leftRupdate(nrows,ncols,szt,colidx,add,mul);

   if(verbose)
   {
      const int dim = nrows*ncols;
      const size_t sznum = dim*sizeof(double);

      hipMemcpy(Are_h,Are_d,sznum,hipMemcpyDeviceToHost);
      hipMemcpy(Aim_h,Aim_d,sznum,hipMemcpyDeviceToHost);
      cout << "the matrix after the update :" << endl;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++)
            cout << "A_d[" << i << "][" << j << "] : "
                 << Are_h[j*nrows+i] << "  "
                 << Aim_h[j*nrows+i] << endl;
   }
}

void GPU_dbl_medium_leftRupdate
 ( int nrows, int ncols, int szt, int colidx, int k, int L,
   double *A_h, double *A_d, double *V_d, double *beta_h, double *beta_d,
   double *RTdotv_h, double *RTdotv_d, double *w_h, double *w_d,
   double *RTvlapms, double *redlapms,
   long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int endcol = (k+1)*szt;     // 1 + last column index in tile
   const int nVrows = nrows - k*szt;          // dimension of V matrix
   const int nhouse = nrows - colidx;  // length of Householder vector
   // total number of entries in R that will be modified
   const int RToffset = colidx*nrows;
   const int dimRTdotv = endcol - colidx;
   const int sizenum = (nrows - colidx)*dimRTdotv;
   const int nbrblocks = (int) ceil(sizenum/((double) szt));

   if(verbose)
   {
      cout << "-> launching " << nbrblocks << " blocks of " << szt
           << " threads to compute RTdotv ..." << endl;
      cout << "   nhouse : " << nhouse << "  RToffset : " << RToffset
           << "  dimRTdotv : " << dimRTdotv << endl;
   }

   hipEventRecord(start);
   // 2nd argument: ncols -> endcol
   // changed second argument ncols into endcol
   // to avoid updating the next tile
   // dbl_medium_betaRTv<<<nbrblocks,szt>>>
   //   (nrows,endcol,szt,colidx,A_d,&V_d[L*nVrows+L],&beta_d[L],w_d);
   // number of threads must be ncols - colidx, not endcol - colidx
   // dbl_small_betaRTv<<<1,nrows-colidx>>> // nrows ...
   //   (nrows,endcol,szt,colidx,A_d,&V_d[L*nVrows+L],&beta_d[L],w_d);
   dbl_RTdotv<<<nbrblocks,szt>>>
      (nhouse,szt,colidx,RToffset,dimRTdotv,A_d,&V_d[L*nVrows+L],RTdotv_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *RTvlapms += milliseconds;
   hipEventRecord(start);
   dbl_sum_betaRTdotv<<<1,dimRTdotv>>>(nhouse,&beta_d[L],RTdotv_d,w_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *RTvlapms += milliseconds;
   flopcount_dbl_RTdotv(nhouse,szt,mul);
   flopcount_dbl_sum_betaRTdotv(nhouse,dimRTdotv,add,mul);
   // flopcount_dbl_small_betaRTv(nrows,endcol,szt,colidx,add,mul);

   if(verbose)
   {
      cout << "-> launching " << nbrblocks << " blocks of " << szt
           << " threads to update " << sizenum << " numbers ..." << endl;
      cout << "   nrows : " << nrows << "  endcol : " << endcol
           << "  szt : " << szt << "  colidx : " << colidx << endl;
   }
   hipEventRecord(start);
   dbl_medium_subvbetaRTv<<<nbrblocks,szt>>>
      (nrows,endcol,szt,colidx,A_d,&V_d[L*nVrows+L],&beta_d[L],w_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *redlapms += milliseconds;
   flopcount_dbl_medium_subvbetaRTv(nrows,endcol,szt,colidx,add,mul);

   if(verbose)
   {
      const int dim = nrows*ncols;
      const size_t sznum = dim*sizeof(double);
      const size_t szbRTv = dimRTdotv*sizeof(double);
      const size_t szRTdotv = nVrows*szbRTv;

      hipMemcpy(RTdotv_h,RTdotv_d,szRTdotv,hipMemcpyDeviceToHost);
      cout << "the matrix R^T dot v : " << endl;
      int ix = 0;
      for(int i=0; i<endcol-colidx; i++)
      {
         w_h[i] = 0.0;                    // take the row sum
         for(int j=0; j<nhouse; j++)      // must use nhouse
         {
            w_h[i] = w_h[i] + RTdotv_h[ix];
            cout << "RTdotv[" << i << "][" << j << "] : "
                 << RTdotv_h[ix++] << endl;
         }
         w_h[i] = beta_h[L]*w_h[i];
      }
      hipMemcpy(&beta_h[L],&beta_d[L],sizeof(double),hipMemcpyDeviceToHost);
      cout << "row sum of R^T dot v times beta : " << endl;
      for(int i=0; i<endcol-colidx; i++)
         cout << "w[" << i << "] : " << w_h[i] << endl;

      hipMemcpy(w_h,w_d,szbRTv,hipMemcpyDeviceToHost);
      cout << "the vector w = beta*R^T*v : " << endl;
      for(int i=0; i<endcol-colidx; i++)
         cout << "w[" << i << "] : " << w_h[i] << endl;

      hipMemcpy(A_h,A_d,sznum,hipMemcpyDeviceToHost);
      cout << "the matrix after the update :" << endl;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++)
            cout << "A_d[" << i << "][" << j << "] : "
                 << A_h[j*nrows+i] << endl;
   }
}

void GPU_cmplx_medium_leftRupdate
 ( int nrows, int ncols, int szt, int colidx, int k, int L,
   double *Are_h, double *Aim_h, double *Are_d, double *Aim_d,
   double *Vre_d, double *Vim_d, double *beta_h, double *beta_d,
   double *RHdotvre_h, double *RHdotvim_h,
   double *RHdotvre_d, double *RHdotvim_d,
   double *wre_h, double *wim_h, double *wre_d, double *wim_d,
   double *RHvlapms, double *redlapms,
   long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int endcol = (k+1)*szt;     // 1 + last column index in tile
   const int nVrows = nrows - k*szt;          // dimension of V matrix
   const int nhouse = nrows - colidx;  // length of Householder vector
   // total number of entries in R that will be modified
   const int RToffset = colidx*nrows;
   const int dimRTdotv = endcol - colidx;
   const int sizenum = (nrows - colidx)*dimRTdotv;
   const int nbrblocks = (int) ceil(sizenum/((double) szt));

   hipEventRecord(start);
   // 2nd argument: ncols -> endcol
   // changed second argument ncols into endcol
   // to avoid updating the next tile
   // dbl_medium_betaRTv<<<nbrblocks,szt>>>
   //   (nrows,endcol,szt,colidx,A_d,&V_d[L*nVrows+L],&beta_d[L],w_d);
   // number of threads must be ncols - colidx, not endcol - colidx
   // cmplx_small_betaRHv<<<1,nrows-colidx>>> // nrows ...
   //    (nrows,endcol,szt,colidx,Are_d,Aim_d,
   //     &Vre_d[L*nVrows+L],&Vim_d[L*nVrows+L],&beta_d[L],wre_d,wim_d);
   cmplx_RHdotv<<<nbrblocks,szt>>>
      (nhouse,szt,colidx,RToffset,dimRTdotv,Are_d,Aim_d,
       &Vre_d[L*nVrows+L],&Vim_d[L*nVrows+L],RHdotvre_d,RHdotvim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *RHvlapms += milliseconds;
   hipEventRecord(start);
   cmplx_sum_betaRHdotv<<<1,dimRTdotv>>>
      (nhouse,&beta_d[L],RHdotvre_d,RHdotvim_d,wre_d,wim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *RHvlapms += milliseconds;
   flopcount_cmplx_RHdotv(nhouse,szt,add,mul);
   flopcount_cmplx_sum_betaRHdotv(nhouse,dimRTdotv,add,mul);
   // flopcount_cmplx_small_betaRHv(nrows,endcol,szt,colidx,add,mul);

   if(verbose)
   {
      cout << "-> launching " << nbrblocks << " blocks of " << szt
           << " threads to update " << sizenum << " numbers ..." << endl;
      cout << "   nrows : " << nrows << "  endcol : " << endcol
           << "  szt : " << szt << "  colidx : " << colidx << endl;
   }
   hipEventRecord(start);
   cmplx_medium_subvbetaRHv<<<nbrblocks,szt>>>
      (nrows,endcol,szt,colidx,Are_d,Aim_d,
       &Vre_d[L*nVrows+L],&Vim_d[L*nVrows+L],&beta_d[L],wre_d,wim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *redlapms += milliseconds;
   flopcount_cmplx_medium_subvbetaRHv(nrows,endcol,szt,colidx,add,mul);

   if(verbose)
   {
      const int dim = nrows*ncols;
      const size_t sznum = dim*sizeof(double);
      const size_t szbRTv = (endcol-colidx)*sizeof(double);

      hipMemcpy(wre_h,wre_d,szbRTv,hipMemcpyDeviceToHost);
      hipMemcpy(wim_h,wim_d,szbRTv,hipMemcpyDeviceToHost);
      cout << "the vector w = beta*R^T*v : " << endl;
      for(int i=0; i<endcol-colidx; i++)
         cout << "w[" << i << "] : "
              << wre_h[i] << "  " << wim_h[i] << endl;

      hipMemcpy(Are_h,Are_d,sznum,hipMemcpyDeviceToHost);
      hipMemcpy(Aim_h,Aim_d,sznum,hipMemcpyDeviceToHost);
      cout << "the matrix after the update :" << endl;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++)
            cout << "A_d[" << i << "][" << j << "] : "
                 << Are_h[j*nrows+i] << "  "
                 << Aim_h[j*nrows+i] << endl;
   }
}

void GPU_dbl_VB_to_W
 ( int nrows, int ncols, int szt,
   double *V_h, double *V_d, double *W_h, double *W_d,
   double *beta_h, double *beta_d, double *lapms,
   long long int *add, long long int *mul, long long int *div, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;

   hipEventRecord(start);
   dbl_VB_to_W<<<1,nrows>>>(nrows,ncols,beta_d,V_d,W_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_VB_to_W(nrows,ncols,add,mul);

   if(verbose)
   {
      const size_t szbeta = szt*sizeof(double);
      const size_t szhouse = nrows*sizeof(double);
      const size_t szVandW = szt*szhouse;

      hipMemcpy(beta_h,beta_d,szbeta,hipMemcpyDeviceToHost);
      cout << "the betas :" << endl;
      for(int j=0; j<szt; j++)
         cout << "beta[" << j << "] : " << beta_h[j] << endl;
      hipMemcpy(V_h,V_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the V matrix :" << endl;
      int ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<nrows; i++) 
            cout << "V[" << i << "][" << j << "] : " << V_h[ix++] << endl;

      hipMemcpy(W_h,W_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the W matrix :" << endl;
      ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<nrows; i++) 
            cout << "W[" << i << "][" << j << "] : " << W_h[ix++] << endl;
   }
}

void GPU_cmplx_VB_to_W
 ( int nrows, int ncols, int szt,
   double *Vre_h, double *Vim_h, double *Vre_d, double *Vim_d,
   double *Wre_h, double *Wim_h, double *Wre_d, double *Wim_d,
   double *beta_h, double *beta_d, double *lapms,
   long long int *add, long long int *mul, long long int *div, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;

   if(verbose)
   {
      const size_t szhouse = nrows*sizeof(double);
      const size_t szVandW = szt*szhouse;

      hipMemcpy(Vre_h,Vre_d,szVandW,hipMemcpyDeviceToHost);
      hipMemcpy(Vim_h,Vim_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the V matrix :" << endl;
      int ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<nrows; i++) 
         {
            cout << "V[" << i << "][" << j << "] : "
                 << Vre_h[ix] << "  " << Vim_h[ix] << endl;
            ix = ix + 1;
         }
   }
   hipEventRecord(start);
   cmplx_VB_to_W<<<1,nrows>>>(nrows,ncols,beta_d,Vre_d,Vim_d,Wre_d,Wim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_VB_to_W(nrows,ncols,add,mul);

   if(verbose)
   {
      const size_t szbeta = szt*sizeof(double);
      const size_t szhouse = nrows*sizeof(double);
      const size_t szVandW = szt*szhouse;

      hipMemcpy(beta_h,beta_d,szbeta,hipMemcpyDeviceToHost);
      cout << "the betas :" << endl;
      for(int j=0; j<szt; j++)
         cout << "beta[" << j << "] : " << beta_h[j] << endl;

      hipMemcpy(Vre_h,Vre_d,szVandW,hipMemcpyDeviceToHost);
      hipMemcpy(Vim_h,Vim_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the V matrix :" << endl;
      int ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<nrows; i++) 
         {
            cout << "V[" << i << "][" << j << "] : "
                 << Vre_h[ix] << "  " << Vim_h[ix] << endl;
            ix = ix + 1;
         }

      hipMemcpy(Wre_h,Wre_d,szVandW,hipMemcpyDeviceToHost);
      hipMemcpy(Wim_h,Wim_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the W matrix :" << endl;
      ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<nrows; i++) 
         {
            cout << "W[" << i << "][" << j << "] : "
                 << Wre_h[ix] << "  " << Wim_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl_medium_VB_to_W
 ( int nrows, int ncols, int szt, int idx,
   double *V_h, double *V_d, double *W_h, double *W_d,
   double *WYT_h, double *WYT_d, double *beta_h, double *beta_d,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowdim = nrows - idx*szt;
   const int nbrblocks1 = (int) ceil(rowdim/((double) szt));

   hipEventRecord(start);
   dbl_beta_times_V<<<nbrblocks1,szt>>>(rowdim,szt,beta_d,V_d,W_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_beta_times_V(rowdim,mul);

   const int nbrblocks2 = (int) ceil(rowdim*rowdim/((double) szt));

   hipEventRecord(start);
   dbl_initialize_WYT<<<nbrblocks2,szt>>>(rowdim,szt,V_d,W_d,WYT_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_initialize_WYT(rowdim,mul);

   for(int j=1; j<szt; j++)
   {
      hipEventRecord(start);
      dbl_beta_next_W<<<nbrblocks1,szt>>>
         (rowdim,szt,&beta_d[j],&V_d[j*rowdim],&W_d[j*rowdim],WYT_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *lapms += milliseconds;
      flopcount_dbl_beta_next_W(rowdim,add,mul);

      hipEventRecord(start);
      dbl_update_WYT<<<nbrblocks2,szt>>>
         (rowdim,szt,&V_d[j*rowdim],&W_d[j*rowdim],WYT_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *lapms += milliseconds;
      flopcount_dbl_update_WYT(rowdim,add,mul);
   }

   if(verbose)
   {
      const size_t szbeta = szt*sizeof(double);
      const size_t szhouse = rowdim*sizeof(double);
      const size_t szVandW = szt*szhouse;
      const size_t szmat = rowdim*rowdim*sizeof(double);

      hipMemcpy(beta_h,beta_d,szbeta,hipMemcpyDeviceToHost);
      cout << "the betas :" << endl;
      for(int j=0; j<szt; j++)
         cout << "beta[" << j << "] : " << beta_h[j] << endl;

      hipMemcpy(V_h,V_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the V matrix :" << endl;
      int ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<rowdim; i++) 
            cout << "V[" << i << "][" << j << "] : " << V_h[ix++] << endl;

      hipMemcpy(W_h,W_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the W matrix :" << endl;
      ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<rowdim; i++) 
            cout << "W[" << i << "][" << j << "] : " << W_h[ix++] << endl;

      hipMemcpy(WYT_h,WYT_d,szmat,hipMemcpyDeviceToHost);
      cout << "the WYT matrix :" << endl;
      ix = 0;
      for(int i=0; i<rowdim; i++) 
         for(int j=0; j<rowdim; j++) 
            cout << "WYT[" << i << "][" << j << "] : "
                 << WYT_h[ix++] << endl;
   }
}

void GPU_cmplx_medium_VB_to_W
 ( int nrows, int ncols, int szt, int idx,
   double *Vre_h, double *Vim_h, double *Vre_d, double *Vim_d,
   double *Wre_h, double *Wim_h, double *Wre_d, double *Wim_d,
   double *WYHre_h, double *WYHim_h, double *WYHre_d, double *WYHim_d,
   double *beta_h, double *beta_d,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowdim = nrows - idx*szt;
   const int nbrblocks1 = (int) ceil(rowdim/((double) szt));

   if(verbose)
   {
      cout << "rowdim : " << rowdim << endl;
      cout << "-> launching " << nbrblocks1 << " blocks of " << szt
           << " threads to compute the first W column ... " << endl;
   }
   hipEventRecord(start);
   cmplx_beta_times_V<<<nbrblocks1,szt>>>
      (rowdim,szt,beta_d,Vre_d,Vim_d,Wre_d,Wim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_beta_times_V(rowdim,mul);

   const int nbrblocks2 = (int) ceil(rowdim*rowdim/((double) szt));

   if(verbose)
      cout << "-> launching " << nbrblocks2 << " blocks of " << szt
           << " threads to initialize WYH ... " << endl;

   hipEventRecord(start);
   cmplx_initialize_WYH<<<nbrblocks2,szt>>>
      (rowdim,szt,Vre_d,Vim_d,Wre_d,Wim_d,WYHre_d,WYHim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_initialize_WYH(rowdim,add,mul);

   for(int j=1; j<szt; j++)
   {
      if(verbose)
         cout << "-> launching " << nbrblocks1 << " blocks of " << szt
              << " threads to compute the next W column ... " << endl;

      hipEventRecord(start);
      cmplx_beta_next_W<<<nbrblocks1,szt>>>
         (rowdim,szt,&beta_d[j],&Vre_d[j*rowdim],&Vim_d[j*rowdim],
          &Wre_d[j*rowdim],&Wim_d[j*rowdim],WYHre_d,WYHim_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *lapms += milliseconds;
      flopcount_cmplx_beta_next_W(rowdim,add,mul);

      if(verbose)
         cout << "-> launching " << nbrblocks2 << " blocks of " << szt
              << " threads to update WYH ... " << endl;

      hipEventRecord(start);
      cmplx_update_WYH<<<nbrblocks2,szt>>>
         (rowdim,szt,&Vre_d[j*rowdim],&Vim_d[j*rowdim],
          &Wre_d[j*rowdim],&Wim_d[j*rowdim],WYHre_d,WYHim_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *lapms += milliseconds;
      flopcount_cmplx_update_WYH(rowdim,add,mul);
   }

   if(verbose)
   {
      const size_t szbeta = szt*sizeof(double);
      const size_t szhouse = rowdim*sizeof(double);
      const size_t szVandW = szt*szhouse;
      const size_t szmat = rowdim*rowdim*sizeof(double);

      hipMemcpy(beta_h,beta_d,szbeta,hipMemcpyDeviceToHost);
      cout << "the betas :" << endl;
      for(int j=0; j<szt; j++)
         cout << "beta[" << j << "] : " << beta_h[j] << endl;

      hipMemcpy(Vre_h,Vre_d,szVandW,hipMemcpyDeviceToHost);
      hipMemcpy(Vim_h,Vim_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the V matrix :" << endl;
      int ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<rowdim; i++) 
         {
            cout << "V[" << i << "][" << j << "] : "
                 << Vre_h[ix] << "  " << Vim_h[ix] << endl;
            ix = ix + 1;
         }

      hipMemcpy(Wre_h,Wre_d,szVandW,hipMemcpyDeviceToHost);
      hipMemcpy(Wim_h,Wim_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the W matrix :" << endl;
      ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<rowdim; i++) 
         {
            cout << "W[" << i << "][" << j << "] : "
                 << Wre_h[ix] << "  " << Wim_h[ix] << endl;
            ix = ix + 1;
         }

      hipMemcpy(WYHre_h,WYHre_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(WYHim_h,WYHim_d,szmat,hipMemcpyDeviceToHost);
      cout << "the WYH matrix :" << endl;
      ix = 0;
      for(int i=0; i<rowdim; i++) 
         for(int j=0; j<rowdim; j++) 
         {
            cout << "WYH[" << i << "][" << j << "] : "
                 << WYHre_h[ix] << "  " << WYHim_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl_small_WYT
 ( int nrows, int szt, double *W_d, double *Y_d, double *WYT_d,
   double *WYT_h, double *lapms, long long int *add, long long int *mul,
   long long int *div, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int nbrblocks = (int) ceil(nrows*nrows/((double) szt));

   hipEventRecord(start);
   dbl_small_WYT<<<nbrblocks,szt>>>(nrows,szt,W_d,Y_d,WYT_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_small_WYT(nrows,szt,add,mul);

   if(verbose)
   {
      const size_t szmat = nrows*nrows*sizeof(double);

      hipMemcpy(WYT_h,WYT_d,szmat,hipMemcpyDeviceToHost);

      cout << "the WYT matrix :" << endl;
      int ix = 0;
      for(int i=0; i<nrows; i++) 
         for(int j=0; j<nrows; j++) 
            cout << "WYT[" << i << "][" << j << "] : "
                 << WYT_h[ix++] << endl;
   }
}

void GPU_cmplx_small_WYH
 ( int nrows, int szt, double *Wre_d, double *Wim_d,
   double *Yre_d, double *Yim_d, double *WYHre_d, double *WYHim_d,
   double *WYHre_h, double *WYHim_h, double *lapms,
   long long int *add, long long int *mul, long long int *div, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int nbrblocks = (int) ceil(nrows*nrows/((double) szt));

   hipEventRecord(start);
   cmplx_small_WYH<<<nbrblocks,szt>>>
      (nrows,szt,Wre_d,Wim_d,Yre_d,Yim_d,WYHre_d,WYHim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_small_WYH(nrows,szt,add,mul);

   if(verbose)
   {
      const size_t szmat = nrows*nrows*sizeof(double);

      hipMemcpy(WYHre_h,WYHre_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(WYHim_h,WYHim_d,szmat,hipMemcpyDeviceToHost);

      cout << "the WYH matrix :" << endl;
      int ix = 0;
      for(int i=0; i<nrows; i++) 
         for(int j=0; j<nrows; j++) 
         {
            cout << "WYH[" << i << "][" << j << "] : "
                 << WYHre_h[ix] << "  " << WYHim_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl_small_YWT
 ( int nrows, int szt, int idx, double *Y_d, double *W_d, double *YWT_d,
   double *YWT_h, double *lapms, long long int *add, long long int *mul,
   bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowdim = nrows - idx*szt;
   int nbrblocks = (int) ceil(rowdim*rowdim/((double) szt));

   hipEventRecord(start);
   dbl_small_WYT<<<nbrblocks,szt>>>(rowdim,szt,Y_d,W_d,YWT_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_small_WYT(rowdim,szt,add,mul);

   if(verbose)
   {
      const size_t szmat = rowdim*rowdim*sizeof(double);

      hipMemcpy(YWT_h,YWT_d,szmat,hipMemcpyDeviceToHost);

      cout << "the YWT matrix :" << endl;
      int ix = 0;
      for(int i=0; i<rowdim; i++) 
         for(int j=0; j<rowdim; j++) 
            cout << "YWT[" << i << "][" << j << "] : "
                 << YWT_h[ix++] << endl;
   }
}

void GPU_cmplx_small_YWH
 ( int nrows, int szt, int idx,
   double *Yre_d, double *Yim_d, double *Wre_d, double *Wim_d,
   double *YWHre_d, double *YWHim_d, double *YWHre_h, double *YWHim_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowdim = nrows - idx*szt;
   int nbrblocks = (int) ceil(rowdim*rowdim/((double) szt));

   hipEventRecord(start);
   cmplx_small_WYH<<<nbrblocks,szt>>>
      (rowdim,szt,Yre_d,Yim_d,Wre_d,Wim_d,YWHre_d,YWHim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_small_WYH(rowdim,szt,add,mul);

   if(verbose)
   {
      const size_t szmat = rowdim*rowdim*sizeof(double);

      hipMemcpy(YWHre_h,YWHre_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(YWHim_h,YWHim_d,szmat,hipMemcpyDeviceToHost);

      cout << "the YWH matrix :" << endl;
      int ix = 0;
      for(int i=0; i<rowdim; i++) 
         for(int j=0; j<rowdim; j++) 
         {
            cout << "YWH[" << i << "][" << j << "] : "
                 << YWHre_h[ix] << "  " << YWHim_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl_small_QWYT
 ( int dim, int szt, int idx, double *Q_d, double *WYT_d, double *QWYT_d,
   double *QWYT_h, double *Q_h, double *lapms,
   long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int coloff = idx*szt;
   const int rowdim = dim - coloff;
   const int nbrblocks = (int) ceil(dim*rowdim/((double) szt));

   if(verbose)
   {
      const size_t szmat = dim*dim*sizeof(double);

      hipMemcpy(Q_h,Q_d,szmat,hipMemcpyDeviceToHost);

      cout << "the Q matrix :" << endl;
      int ix = 0;
      for(int i=0; i<dim; i++) 
         for(int j=0; j<dim; j++) 
            cout << "Q[" << i << "][" << j << "] : "
                 << Q_h[ix++] << endl;
   }

   hipEventRecord(start);
   dbl_small_QWYT<<<nbrblocks,szt>>>(dim,rowdim,szt,coloff,Q_d,WYT_d,QWYT_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_small_QWYT(dim,rowdim,szt,coloff,add,mul);

   if(verbose)
   {
      const size_t szmat = dim*rowdim*sizeof(double);

      hipMemcpy(QWYT_h,QWYT_d,szmat,hipMemcpyDeviceToHost);

      cout << "the QWYT matrix :" << endl;
      int ix = 0;
      for(int i=0; i<dim; i++) 
         for(int j=0; j<rowdim; j++) 
            cout << "QWYT[" << i << "][" << j << "] : "
                 << QWYT_h[ix++] << endl;
   }
}

void GPU_cmplx_small_QWYH
 ( int dim, int szt, int idx, double *Qre_d, double *Qim_d,
   double *WYHre_d, double *WYHim_d, double *QWYHre_d, double *QWYHim_d,
   double *QWYHre_h, double *QWYHim_h, double *Qre_h, double *Qim_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int coloff = idx*szt;
   const int rowdim = dim - coloff;
   const int nbrblocks = (int) ceil(dim*rowdim/((double) szt));

   if(verbose)
   {
      const size_t szmat = dim*dim*sizeof(double);

      hipMemcpy(Qre_h,Qre_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(Qim_h,Qim_d,szmat,hipMemcpyDeviceToHost);

      cout << "the Q matrix :" << endl;
      int ix = 0;
      for(int i=0; i<dim; i++) 
         for(int j=0; j<dim; j++) 
         {
            cout << "Q[" << i << "][" << j << "] : "
                 << Qre_h[ix] << "  " << Qim_h[ix] << endl;
            ix = ix + 1;
         }
   }

   hipEventRecord(start);
   cmplx_small_QWYH<<<nbrblocks,szt>>>
      (dim,rowdim,szt,coloff,Qre_d,Qim_d,WYHre_d,WYHim_d,
       QWYHre_d,QWYHim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_small_QWYH(dim,rowdim,szt,coloff,add,mul);

   if(verbose)
   {
      const size_t szmat = dim*rowdim*sizeof(double);

      hipMemcpy(QWYHre_h,QWYHre_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(QWYHim_h,QWYHim_d,szmat,hipMemcpyDeviceToHost);

      cout << "the QWYH matrix :" << endl;
      int ix = 0;
      for(int i=0; i<dim; i++) 
         for(int j=0; j<rowdim; j++) 
         {
            cout << "QWYH[" << i << "][" << j << "] : "
                 << QWYHre_h[ix] << "  " << QWYHim_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl_small_YWTC
 ( int nrows, int ncols, int szt, int idx,
   double *YWT_d, double *C_d, double *YWTC_d, double *YWTC_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowoff = idx*szt;
   const int rowdim = nrows - rowoff;
   const int coloff = (idx+1)*szt;
   const int coldim = ncols - coloff;
   const int nbrblocks = (int) ceil(rowdim*coldim/((double) szt));

   if(verbose)
   {
      cout << "in GPU_dbl_small_YWTC ..." << endl;
      cout << "-> nrows : " << nrows
           << "  ncols : " << ncols
           << "  szt : " << szt
           << "  idx : " << idx << endl;
      cout << "   rowdim : " << rowdim
           << "  coldim : " << coldim
           << "  rowoff : " << rowoff
           << "  coloff : " << coloff
           << "  nbrblocks : " << nbrblocks << endl;

      double *C_h = new double[nrows*ncols];
      const size_t szmat = nrows*ncols*sizeof(double);

      hipMemcpy(C_h,C_d,szmat,hipMemcpyDeviceToHost);

      cout << "the matrix C : " << endl;
      for(int i=rowoff; i<nrows; i++)
         for(int j=coloff; j<ncols; j++)
            cout << "C_h[" << i << "][" << j << "] : "
                 << C_h[j*nrows+i] << endl;

      free(C_h);
   }

   hipEventRecord(start);
   dbl_small_YWTC<<<nbrblocks,szt>>>
      (nrows,ncols,rowdim,coldim,szt,rowoff,coloff,YWT_d,C_d,YWTC_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_small_YWTC(rowdim,coldim,add,mul);

   if(verbose)
   {
      const size_t szmat = nrows*ncols*sizeof(double);

      hipMemcpy(YWTC_h,YWTC_d,szmat,hipMemcpyDeviceToHost);

      cout << "the YWTC matrix :" << endl;
      for(int i=rowoff; i<nrows; i++) 
         for(int j=coloff; j<ncols; j++)
            cout << "YWTC[" << i << "][" << j << "] : "
                 << YWTC_h[j*nrows + i] << endl;
   }
}

void GPU_cmplx_small_YWHC
 ( int nrows, int ncols, int szt, int idx,
   double *YWHre_d, double *YWHim_d, double *Cre_d, double *Cim_d,
   double *YWHCre_d, double *YWHCim_d, double *YWHCre_h, double *YWHCim_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowoff = idx*szt;
   const int rowdim = nrows - rowoff;
   const int coloff = (idx+1)*szt;
   const int coldim = ncols - coloff;
   const int nbrblocks = (int) ceil(rowdim*coldim/((double) szt));

   if(verbose)
   {
      cout << "in GPU_cmplx_small_YWHC ..." << endl;
      cout << "-> nrows : " << nrows
           << "  ncols : " << ncols
           << "  szt : " << szt
           << "  idx : " << idx << endl;
      cout << "   rowdim : " << rowdim
           << "  coldim : " << coldim
           << "  rowoff : " << rowoff
           << "  coloff : " << coloff
           << "  nbrblocks : " << nbrblocks << endl;

      double *Cre_h = new double[nrows*ncols];
      double *Cim_h = new double[nrows*ncols];
      const size_t szmat = nrows*ncols*sizeof(double);

      hipMemcpy(Cre_h,Cre_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(Cim_h,Cim_d,szmat,hipMemcpyDeviceToHost);

      cout << "the matrix C : " << endl;
      for(int i=rowoff; i<nrows; i++)
         for(int j=coloff; j<ncols; j++)
            cout << "C_h[" << i << "][" << j << "] : "
                 << Cre_h[j*nrows+i] << "  "
                 << Cim_h[j*nrows+i] << endl;

      free(Cre_h); free(Cim_h);
   }
   hipEventRecord(start);
   cmplx_small_YWHC<<<nbrblocks,szt>>>
      (nrows,ncols,rowdim,coldim,szt,rowoff,coloff,
       YWHre_d,YWHim_d,Cre_d,Cim_d,YWHCre_d,YWHCim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_small_YWHC(rowdim,coldim,add,mul);

   if(verbose)
   {
      const size_t szmat = nrows*ncols*sizeof(double);

      hipMemcpy(YWHCre_h,YWHCre_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(YWHCim_h,YWHCim_d,szmat,hipMemcpyDeviceToHost);

      cout << "the YWHC matrix :" << endl;
      for(int i=rowoff; i<nrows; i++) 
         for(int j=coloff; j<ncols; j++)
            cout << "YWHC[" << i << "][" << j << "] : "
                 << YWHCre_h[j*nrows + i] << "  "
                 << YWHCim_h[j*nrows + i] << endl;
   }
}

void GPU_dbl_small_Qupdate
 ( int dim, int szt, int idx, double *Q_d, double *QWYT_d, double *Q_h,
   double *lapms, long long int *add, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int coloff = idx*szt;
   const int rowdim = dim - coloff;
   const int nbrblocks = (int) ceil(dim*rowdim/((double) szt));

   hipEventRecord(start);
   dbl_small_Qupdate<<<nbrblocks,szt>>>(dim,rowdim,szt,coloff,Q_d,QWYT_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_small_Qupdate(dim,rowdim,add);

   if(verbose)
   {
      const size_t szmat = dim*dim*sizeof(double);

      hipMemcpy(Q_h,Q_d,szmat,hipMemcpyDeviceToHost);

      cout << "the Q matrix :" << endl;
      int ix = 0;
      for(int i=0; i<dim; i++) 
         for(int j=0; j<dim; j++) 
            cout << "Q[" << i << "][" << j << "] : "
                 << Q_h[ix++] << endl;
   }
}

void GPU_cmplx_small_Qupdate
 ( int dim, int szt, int idx, double *Qre_d, double *Qim_d,
   double *QWYHre_d, double *QWYHim_d, double *Qre_h, double *Qim_h,
   double *lapms, long long int *add, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int coloff = idx*szt;
   const int rowdim = dim - coloff;
   const int nbrblocks = (int) ceil(dim*rowdim/((double) szt));

   hipEventRecord(start);
   cmplx_small_Qupdate<<<nbrblocks,szt>>>
      (dim,rowdim,szt,coloff,Qre_d,Qim_d,QWYHre_d,QWYHim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_small_Qupdate(dim,rowdim,add);

   if(verbose)
   {
      const size_t szmat = dim*dim*sizeof(double);

      hipMemcpy(Qre_h,Qre_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(Qim_h,Qim_d,szmat,hipMemcpyDeviceToHost);

      cout << "the Q matrix :" << endl;
      int ix = 0;
      for(int i=0; i<dim; i++) 
         for(int j=0; j<dim; j++) 
         {
            cout << "Q[" << i << "][" << j << "] : "
                 << Qre_h[ix] << "  " << Qim_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl_small_R_add_YWTC
 ( int nrows, int ncols, int szt, int idx, double *R_d, double *YWTC_d,
   double *R_h, double *lapms, long long int *add, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowoff = idx*szt;
   const int rowdim = nrows - rowoff;
   const int coloff = (idx+1)*szt;
   const int coldim = ncols - coloff;
   const int nbrblocks = (int) ceil(rowdim*coldim/((double) szt));

   hipEventRecord(start);
   dbl_small_R_add_YWTC<<<nbrblocks,szt>>>
      (nrows,coldim,szt,rowoff,coloff,R_d,YWTC_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_dbl_small_R_add_YWTC(nrows,coldim,szt,rowoff,coloff,add);

   if(verbose)
   {
      const size_t szmat = nrows*ncols*sizeof(double);

      hipMemcpy(R_h,R_d,szmat,hipMemcpyDeviceToHost);

      cout << "the R matrix :" << endl;
      for(int i=rowoff; i<nrows; i++) 
         for(int j=coloff; j<ncols; j++)
            cout << "R[" << i << "][" << j << "] : "
                 << R_h[j*nrows + i] << endl;
   }
}

void GPU_cmplx_small_R_add_YWHC
 ( int nrows, int ncols, int szt, int idx,
   double *Rre_d, double *Rim_d, double *YWHCre_d, double *YWHCim_d,
   double *Rre_h, double *Rim_h, double *lapms,
   long long int *add, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowoff = idx*szt;
   const int rowdim = nrows - rowoff;
   const int coloff = (idx+1)*szt;
   const int coldim = ncols - coloff;
   const int nbrblocks = (int) ceil(rowdim*coldim/((double) szt));

   hipEventRecord(start);
   cmplx_small_R_add_YWHC<<<nbrblocks,szt>>>
      (nrows,coldim,szt,rowoff,coloff,Rre_d,Rim_d,YWHCre_d,YWHCim_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;
   flopcount_cmplx_small_R_add_YWHC(nrows,coldim,szt,rowoff,coloff,add);

   if(verbose)
   {
      const size_t szmat = nrows*ncols*sizeof(double);

      hipMemcpy(Rre_h,Rre_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(Rim_h,Rim_d,szmat,hipMemcpyDeviceToHost);

      cout << "the R matrix :" << endl;
      for(int i=rowoff; i<nrows; i++) 
         for(int j=coloff; j<ncols; j++)
            cout << "R[" << i << "][" << j << "] : "
                 << Rre_h[j*nrows + i] << "  "
                 << Rim_h[j*nrows + i] << endl;
   }
}

void GPU_dbl_blocked_houseqr
 ( int nrows, int ncols, int szt, int nbt,
   double **A, double **Q, double **R,
   double *houselapms, double *RTvlapms, double *tileRlapms,
   double *vb2Wlapms, double *WYTlapms, double *QWYTlapms, double *Qaddlapms,
   double *YWTlapms, double *YWTClapms, double *Raddlapms,
   double *walltimesec, long long int *addcnt, long long int *mulcnt,
   long long int *divcnt, long long int *sqrtcnt, bool verbose )
{
   const int dim = nrows*ncols;         // total number of doubles
   const int nrows2 = nrows*nrows;
   double *A_h = new double[dim];       // matrix A on the host
   double *A_d;                         // matrix on the device
   double *Q_h = new double[nrows2];    // orthogonal Q on host
   double *Q_d;                         // orthogonal Q on device
   double *v_h = new double[nrows];     // Householder vector on host
   double *beta_h = new double[szt];    // beta on the host
   double *beta_d;                      // beta on the device
   double *V_h = new double[nrows*szt]; // matrix of Householder vectors
   double *V_d;                         // Householder vectors on device
   double *W_h = new double[nrows*szt]; // the W matrix on the host
   double *W_d;                         // the W matrix on the device
   double *WYT_h = new double[nrows2];  // W*Y^T on the host
   double *WYT_d;                       // W*Y^T on the device
   double *YWT_h = new double[nrows2];  // Y*W^T on the host
   double *YWT_d;                       // Y*W^T on the device
   double *QWYT_h = new double[nrows2]; // Q*WY^T on the host
   double *QWYT_d;                      // Q*WY^T on the device
   double *YWTC_h = new double[dim];    // YWT*C on the host
   double *YWTC_d;                      // YWT*C on the device
   double *RTdotv_h = new double[nrows2]; // R^T dotted with v
   double *RTdotv_d;                      // RTdotv on the device
   double *bRTv_h = new double[nrows];  // beta*R^T*v
   double *bRTv_d;                      // beta*R^T*v on the device

   int ix = 0;                          // copy the columns of A to A_h
   for(int j=0; j<ncols; j++)   
      for(int i=0; i<nrows; i++) A_h[ix++] = A[i][j];

   ix = 0;                              // initialize Q with identity
   for(int i=0; i<nrows; i++)
   {
      for(int j=0; j<nrows; j++)
      {
         if(i == j)
            Q_h[ix++] = 1.0;
         else
            Q_h[ix++] = 0.0;
      }
   }
   const size_t sznum = dim*sizeof(double);
   hipMalloc((void**)&A_d,sznum);
   hipMemcpy(A_d,A_h,sznum,hipMemcpyHostToDevice);

   const size_t szbeta = szt*sizeof(double);
   hipMalloc((void**)&beta_d,szbeta);
   for(int i=0; i<szt; i++) beta_h[i] = 0.0;
   hipMemcpy(beta_d,beta_h,szbeta,hipMemcpyHostToDevice);

   const size_t szhouse = nrows*sizeof(double);
   const size_t szpad = szt*sizeof(double);  // padding for nonsquare tiles
   const size_t szVandW = szt*szhouse;
   hipMalloc((void**)&V_d,szVandW + szpad); // padding only in allocation
   ix = 0;
   for(int i=0; i<nrows*szt; i++) V_h[ix++] = 0.0; 
   V_h[--ix] = 1.0; // initialize last vector for square tiles
   hipMemcpy(V_d,V_h,szVandW,hipMemcpyHostToDevice);
   hipMalloc((void**)&W_d,szVandW + szpad); // padding only in allocation

   hipMalloc((void**)&RTdotv_d,szVandW + szpad);
   hipMalloc((void**)&bRTv_d,szhouse + szpad);

   const size_t szWYT = nrows2*sizeof(double);
   hipMalloc((void**)&WYT_d,szWYT + szpad); // padding for W*Y^T product
   hipMalloc((void**)&Q_d,szWYT + szpad);
   hipMemcpy(Q_d,Q_h,szWYT,hipMemcpyHostToDevice);
   hipMalloc((void**)&QWYT_d,szWYT + szpad);

   const size_t szYWT = nrows2*sizeof(double);
   hipMalloc((void**)&YWT_d,szYWT + szpad); // padding for Y*W^T product
   hipMalloc((void**)&YWTC_d,sznum + szpad);

   *houselapms = 0.0; *RTvlapms = 0.0; *tileRlapms = 0.0; *vb2Wlapms = 0.0;
   *WYTlapms = 0.0; *QWYTlapms = 0.0; *Qaddlapms = 0.0;
   *YWTlapms = 0.0; *YWTClapms = 0.0; *Raddlapms = 0.0;
   *addcnt = 0; *mulcnt = 0; *divcnt = 0; *sqrtcnt = 0;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   for(int k=0; k<nbt; k++)       // k runs over the number of blocks
   {
      if(verbose)
         cout << "Tile k = " << k << " out of " << nbt << " ..." << endl;

      int colidx,nrows1;

      for(int L=0; L<szt; L++)  // L runs over the columns in one block
      {
         colidx = k*szt + L;              // index of the current column
         nrows1 = nrows - colidx - 1;     // #rows in Householder vector - 1
         GPU_dbl_small_house
            (nrows,ncols,szt,nbt,colidx,nrows1,k,L,
             A_h,A_d,v_h,V_d,beta_h,beta_d,
             houselapms,addcnt,mulcnt,divcnt,sqrtcnt,verbose);
         if(beta_h[L] == 0.0)
         {
            if(verbose) cout << "Zero beta detected." << endl;
         }
         else
         {
            if(nrows - colidx <= szt)
            {
               GPU_dbl_small_leftRupdate
                  (nrows,ncols,szt,colidx,k,L,A_h,A_d,V_d,beta_h,beta_d,
                   tileRlapms,addcnt,mulcnt,verbose);
            }
            else
            {
               GPU_dbl_medium_leftRupdate
                  (nrows,ncols,szt,colidx,k,L,A_h,A_d,V_d,beta_h,beta_d,
                   RTdotv_h,RTdotv_d,bRTv_h,bRTv_d,
                   RTvlapms,tileRlapms,addcnt,mulcnt,verbose);
            }
         }
      }
/*
      GPU_dbl_VB_to_W   // changed nrows into nrows - k*szt and ncols into szt
         (nrows-k*szt,szt,szt,V_h,V_d,W_h,W_d,beta_h,beta_d,
          vb2Wlapms,addcnt,mulcnt,divcnt,verbose);
 */
      GPU_dbl_medium_VB_to_W
         (nrows,szt,szt,k,V_h,V_d,W_h,W_d,WYT_h,WYT_d,beta_h,beta_d,
          vb2Wlapms,addcnt,mulcnt,verbose);
      // update Q, WYT matrix has nrows - k*szt instead of nrows
/*
      GPU_dbl_small_WYT
         (nrows-k*szt,szt,W_d,V_d,WYT_d,WYT_h,
          WYTlapms,addcnt,mulcnt,divcnt,verbose);
 */
      GPU_dbl_small_QWYT
         (nrows,szt,k,Q_d,WYT_d,QWYT_d,QWYT_h,Q_h,
          QWYTlapms,addcnt,mulcnt,verbose);
      GPU_dbl_small_Qupdate
         (nrows,szt,k,Q_d,QWYT_d,Q_h,Qaddlapms,addcnt,verbose);
      if(k < nbt-1)                                           // update R
      {
         GPU_dbl_small_YWT
            (nrows,szt,k,V_d,W_d,YWT_d,YWT_h,YWTlapms,addcnt,mulcnt,verbose);
         GPU_dbl_small_YWTC
            (nrows,ncols,szt,k,YWT_d,A_d,YWTC_d,YWTC_h,
             YWTClapms,addcnt,mulcnt,verbose);
         GPU_dbl_small_R_add_YWTC
            (nrows,ncols,szt,k,A_d,YWTC_d,A_h,Raddlapms,addcnt,verbose);
      }
   }
   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   hipMemcpy(Q_h,Q_d,szWYT,hipMemcpyDeviceToHost);
   ix = 0;                                           // copy rows of Q
   for(int i=0; i<nrows; i++)
      for(int j=0; j<nrows; j++) Q[i][j] = Q_h[ix++];

   hipMemcpy(A_h,A_d,sznum,hipMemcpyDeviceToHost);
   for(int i=0; i<nrows; i++)                       // copy columns of R
      for(int j=0; j<ncols; j++)
         R[i][j] = A_h[j*nrows+i];

   free(A_h); free(Q_h); free(v_h); free(V_h);
   free(RTdotv_h); free(bRTv_h); free(W_h);
   free(WYT_h); free(QWYT_h); free(YWT_h); free(YWTC_h);
}

void GPU_cmplx_blocked_houseqr
 ( int nrows, int ncols, int szt, int nbt,
   double **Are, double **Aim, double **Qre, double **Qim,
   double **Rre, double **Rim,
   double *houselapms, double *RHvlapms, double *tileRlapms,
   double *vb2Wlapms, double *WYHlapms, double *QWYHlapms, double *Qaddlapms,
   double *YWHlapms, double *YWHClapms, double *Raddlapms,
   double *walltimesec, long long int *addcnt, long long int *mulcnt,
   long long int *divcnt, long long int *sqrtcnt, bool verbose )
{
   const int dim = nrows*ncols;           // total number of doubles
   const int nrows2 = nrows*nrows;
   double *Are_h = new double[dim];       // real parts of A on the host
   double *Aim_h = new double[dim];       // imaginary parts of A on the host
   double *Are_d;                         // Are on the device
   double *Aim_d;                         // Aim on the device
   double *Qre_h = new double[nrows2];    // real parts of Q on host
   double *Qim_h = new double[nrows2];    // imaginary parts of Q on host
   double *Qre_d;                         // Qre on device
   double *Qim_d;                         // Qim on device
   double *vre_h = new double[nrows];     // real parts of Householder vector
   double *vim_h = new double[nrows];     // imaginary parts on host
   double *beta_h = new double[szt];      // beta on the host
   double *beta_d;                        // beta on the device
   double *Vre_h = new double[nrows*szt]; // real parts of Householder vectors
   double *Vim_h = new double[nrows*szt]; // imaginary parts
   double *Vre_d;                         // Vre on device
   double *Vim_d;                         // Vim on device
   double *Wre_h = new double[nrows*szt]; // real parts of the W matrix
   double *Wim_h = new double[nrows*szt]; // imaginary parts of the W matrix
   double *Wre_d;                         // Wre on the device
   double *Wim_d;                         // Wim on the device
   double *WYTre_h = new double[nrows2];  // real parts of W*Y^T on the host
   double *WYTim_h = new double[nrows2];  // imaginary parts of W*Y^T
   double *WYTre_d;                       // WYTre on the device 
   double *WYTim_d;                       // WYTim on the device
   double *YWTre_h = new double[nrows2];  // real parts of Y*W^T on the host
   double *YWTim_h = new double[nrows2];  // imginary parts of Y*W^T
   double *YWTre_d;                       // YWTre on the device
   double *YWTim_d;                       // YWTim on the device
   double *QWYTre_h = new double[nrows2]; // real parts of Q*WY^T on the host
   double *QWYTim_h = new double[nrows2]; // imaginary parts of Q*WY^T
   double *QWYTre_d;                      // QWYTre on the device
   double *QWYTim_d;                      // QWYTim on the device
   double *YWTCre_h = new double[dim];    // real parts of YWT*C on the host
   double *YWTCim_h = new double[dim];    // imaginary parts of YWT*C
   double *YWTCre_d;                      // YWTCre on the device
   double *YWTCim_d;                      // YWTCim on the device
   double *RHdotvre_h = new double[nrows2]; // real part of R^H dotted with v
   double *RHdotvim_h = new double[nrows2]; // imag part of R^H dotted with v
   double *RHdotvre_d;                      // RHdotvre on the device
   double *RHdotvim_d;                      // RHdotvim on the device
   double *bRHvre_h = new double[nrows];  // real parts of beta*R^H*v
   double *bRHvim_h = new double[nrows];  // imaginary parts of beta*R^H*v
   double *bRHvre_d;                      // bRHvre_h on the device
   double *bRHvim_d;                      // bRHvim_d on the device

   int ix = 0;                            // copy the columns of A to A_h
   for(int j=0; j<ncols; j++)   
      for(int i=0; i<nrows; i++)
      {
         Are_h[ix]   = Are[i][j];
         Aim_h[ix++] = Aim[i][j];
      }

   ix = 0;                                // initialize Q with identity
   for(int i=0; i<nrows; i++)
   {
      for(int j=0; j<nrows; j++)
      {
         if(i == j)
         {
            Qre_h[ix]   = 1.0;
            Qim_h[ix++] = 0.0;
         }
         else
         {
            Qre_h[ix]   = 0.0;
            Qim_h[ix++] = 0.0;
         }
         // cout << "Q[" << ix-1 << "] : "
         //      << Qre_h[ix-1] << "  " << Qim_h[ix-1] << endl;
      }
   }
   if(verbose)
   {
      ix = 0;
      cout << "The identity matrix :" << endl;
      cout << scientific << setprecision(16);
      for(int i=0; i<nrows; i++)
         for(int j=0; j<nrows; j++)
         {
            cout << "Q[" << i << "][" << j << "] : "
                 << Qre_h[ix] << "  " << Qim_h[ix] << endl;
            ix = ix + 1;
            // cout << "Q[" << i << "][" << j << "] : "
            //      << Qre_h[j*nrows+i] << "  " << Qim_h[j*nrows+i] << endl;
         }
   }
   const size_t sznum = dim*sizeof(double);
   hipMalloc((void**)&Are_d,sznum);
   hipMalloc((void**)&Aim_d,sznum);
   hipMemcpy(Are_d,Are_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Aim_d,Aim_h,sznum,hipMemcpyHostToDevice);

   const size_t szbeta = szt*sizeof(double);
   hipMalloc((void**)&beta_d,szbeta);
   for(int i=0; i<szt; i++) beta_h[i] = 0.0;
   hipMemcpy(beta_d,beta_h,szbeta,hipMemcpyHostToDevice);

   const size_t szhouse = nrows*sizeof(double);
   const size_t szpad = szt*sizeof(double);    // padding for nonsquare tiles
   const size_t szVandW = szt*szhouse;
   hipMalloc((void**)&Vre_d,szVandW + szpad); // padding added
   hipMalloc((void**)&Vim_d,szVandW + szpad); // padding added
   ix = 0;
   for(int i=0; i<nrows*szt; i++)
   {
      Vre_h[ix]   = 0.0; 
      Vim_h[ix++] = 0.0; 
   }
   Vre_h[--ix] = 1.0; // initialize last vector for square tiles
   hipMemcpy(Vre_d,Vre_h,szVandW,hipMemcpyHostToDevice);
   hipMemcpy(Vim_d,Vim_h,szVandW,hipMemcpyHostToDevice);
   hipMalloc((void**)&Wre_d,szVandW + szpad); // padding added
   hipMalloc((void**)&Wim_d,szVandW + szpad); // padding added

   hipMalloc((void**)&RHdotvre_d,szVandW + szpad);
   hipMalloc((void**)&RHdotvim_d,szVandW + szpad);
   hipMalloc((void**)&bRHvre_d,szhouse + szpad);
   hipMalloc((void**)&bRHvim_d,szhouse + szpad);

   const size_t szWYT = nrows2*sizeof(double);
   hipMalloc((void**)&WYTre_d,szWYT + szpad); // padding for W*Y^T 
   hipMalloc((void**)&WYTim_d,szWYT + szpad);
   hipMalloc((void**)&Qre_d,szWYT + szpad); // needed for 129-by-128
   hipMalloc((void**)&Qim_d,szWYT + szpad); // and one tile of size 128
   hipMemcpy(Qre_d,Qre_h,szWYT,hipMemcpyHostToDevice);
   hipMemcpy(Qim_d,Qim_h,szWYT,hipMemcpyHostToDevice);
   hipMalloc((void**)&QWYTre_d,szWYT + szpad); // padding also here needed
   hipMalloc((void**)&QWYTim_d,szWYT + szpad); // for correct Q computation

   const size_t szYWT = nrows2*sizeof(double);
   hipMalloc((void**)&YWTre_d,szYWT + szpad); // padding for Y*W^T
   hipMalloc((void**)&YWTim_d,szYWT + szpad);
   hipMalloc((void**)&YWTCre_d,sznum + szpad);
   hipMalloc((void**)&YWTCim_d,sznum + szpad);

   *houselapms = 0.0; *RHvlapms = 0.0; *tileRlapms = 0.0; *vb2Wlapms = 0.0;
   *WYHlapms = 0.0; *QWYHlapms = 0.0; *Qaddlapms = 0.0;
   *YWHlapms = 0.0; *YWHClapms = 0.0; *Raddlapms = 0.0;
   *addcnt = 0; *mulcnt = 0; *divcnt = 0; *sqrtcnt = 0;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   for(int k=0; k<nbt; k++)       // k runs over the number of blocks
   {
      if(verbose)
         cout << "Tile k = " << k << " out of " << nbt << " ..." << endl;

      int colidx,nrows1;

      for(int L=0; L<szt; L++)  // L runs over the columns in one block
      {
         colidx = k*szt + L;              // index of the current column
         nrows1 = nrows - colidx - 1;     // #rows in Householder vector - 1
         GPU_cmplx_small_house
            (nrows,ncols,szt,nbt,colidx,nrows1,k,L,
             Are_h,Aim_h,Are_d,Aim_d,vre_h,vim_h,Vre_d,Vim_d,
             beta_h,beta_d,houselapms,addcnt,mulcnt,divcnt,sqrtcnt,verbose);

         if(beta_h[L] == 0.0)
         {
            if(verbose) cout << "Zero beta detected." << endl;
         }
         else
         {
            if(nrows - colidx <= szt)
            {
               GPU_cmplx_small_leftRupdate
                  (nrows,ncols,szt,colidx,k,L,Are_h,Aim_h,Are_d,Aim_d,
                   Vre_d,Vim_d,beta_h,beta_d,tileRlapms,
                   addcnt,mulcnt,verbose);
            }
            else
            {
               GPU_cmplx_medium_leftRupdate
                  (nrows,ncols,szt,colidx,k,L,Are_h,Aim_h,Are_d,Aim_d,
                   Vre_d,Vim_d,beta_h,beta_d,
                   RHdotvre_h,RHdotvim_h,RHdotvre_d,RHdotvim_d,
                   bRHvre_h,bRHvim_h,bRHvre_d,bRHvim_d,
                   RHvlapms,tileRlapms,addcnt,mulcnt,verbose);
            }
         }
      }
/*
      GPU_cmplx_VB_to_W
         (nrows-k*szt,szt,szt,Vre_h,Vim_h,Vre_d,Vim_d,Wre_h,Wim_h,
          Wre_d,Wim_d,beta_h,beta_d,vb2Wlapms,
          addcnt,mulcnt,divcnt,verbose);
 */
      GPU_cmplx_medium_VB_to_W
         (nrows,szt,szt,k,Vre_h,Vim_h,Vre_d,Vim_d,Wre_h,Wim_h,Wre_d,Wim_d,
          WYTre_h,WYTim_h,WYTre_d,WYTim_d,beta_h,beta_d,
          vb2Wlapms,addcnt,mulcnt,verbose);
/*
      GPU_cmplx_small_WYT
         (nrows-k*szt,szt,Wre_d,Wim_d,Vre_d,Vim_d,WYTre_d,WYTim_d,
          WYTre_h,WYTim_h,WYHlapms,addcnt,mulcnt,divcnt,verbose);
 */
      GPU_cmplx_small_QWYH
         (nrows,szt,k,Qre_d,Qim_d,WYTre_d,WYTim_d,QWYTre_d,QWYTim_d,
          QWYTre_h,QWYTim_h,Qre_h,Qim_h,QWYHlapms,addcnt,mulcnt,verbose);
      GPU_cmplx_small_Qupdate
         (nrows,szt,k,Qre_d,Qim_d,QWYTre_d,QWYTim_d,Qre_h,Qim_h,
          Qaddlapms,addcnt,verbose);
      if(k < nbt-1)                              // update R
      {
         GPU_cmplx_small_YWH
            (nrows,szt,k,Vre_d,Vim_d,Wre_d,Wim_d,YWTre_d,YWTim_d,
             YWTre_h,YWTim_h,YWHlapms,addcnt,mulcnt,verbose);
         GPU_cmplx_small_YWHC
            (nrows,ncols,szt,k,YWTre_d,YWTim_d,Are_d,Aim_d,YWTCre_d,
             YWTCim_d,YWTCre_h,YWTCim_h,YWHClapms,
             addcnt,mulcnt,verbose);
         GPU_cmplx_small_R_add_YWHC
            (nrows,ncols,szt,k,Are_d,Aim_d,YWTCre_d,YWTCim_d,Are_h,Aim_h,
             Raddlapms,addcnt,verbose);
      }
   }
   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   hipMemcpy(Qre_h,Qre_d,szWYT,hipMemcpyDeviceToHost);
   hipMemcpy(Qim_h,Qim_d,szWYT,hipMemcpyDeviceToHost);
   ix = 0;                                           // copy rows of Q
   for(int i=0; i<nrows; i++)
      for(int j=0; j<nrows; j++)
      {
         Qre[i][j] = Qre_h[ix];
         Qim[i][j] = Qim_h[ix++];
      }

   hipMemcpy(Are_h,Are_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(Aim_h,Aim_d,sznum,hipMemcpyDeviceToHost);
   for(int i=0; i<nrows; i++)                       // copy columns of R
      for(int j=0; j<ncols; j++)
      {
         Rre[i][j] = Are_h[j*nrows+i];
         Rim[i][j] = Aim_h[j*nrows+i];
      }

   free(Are_h); free(Aim_h); free(Qre_h); free(Qim_h);
   free(vre_h); free(vim_h); free(Vre_h); free(Vim_h);
   free(Wre_h); free(Wim_h);
   free(RHdotvre_h); free(RHdotvim_h); free(bRHvre_h); free(bRHvim_h);
   free(WYTre_h); free(QWYTre_h); free(YWTre_h); free(YWTCre_h);
   free(WYTim_h); free(QWYTim_h); free(YWTim_h); free(YWTCim_h);
}
