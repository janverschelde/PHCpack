#include "hip/hip_runtime.h"
/* The file dbl2_baqr_kernels.cu defines the functions with prototypes in
 * the file dbl2_baqr_kernels.h. */

#include <iostream>
#include <iomanip>
#ifdef winwalltime
#include "wingettimeofday.h"
#else
#include <sys/time.h>
#endif
#include "dbl2_baqr_kernels.h"

using namespace std;

__global__ void dbl2_small_house
 ( double *x0hi, double *x0lo, double *x1hi, double *x1lo,
   int dim, int dimLog2,
   double *vhi, double *vlo, double *betahi, double *betalo )
{
   const int j = threadIdx.x;
}

__global__ void dbl2_small_leftRupdate
 ( int nrows, int ncols, int szt, int k, double *Rhi, double *Rlo,
   double *vhi, double *vlo, double *betahi, double *betalo )
{
   const int j = threadIdx.x;
}

__global__ void dbl2_VB_to_W
 ( int nrows, int ncols, double *Bhi, double *Blo,
   double *Vhi, double *Vlo, double *Whi, double *Wlo )
{
   const int j = threadIdx.x;
}

__global__ void dbl2_small_WYT
 ( int nrows, int szt, double *Whi, double *Wlo, double *Yhi, double *Ylo,
   double *WYThi, double *WYTlo )
{
   const int j = threadIdx.x;
}

__global__ void dbl2_small_QWYT
 ( int dim, int rowdim, int szt, int coloff,
   double *Qhi, double *Qlo, double *WYThi, double *WYTlo,
   double *QWYThi, double *QWYTlo )
{
   const int j = threadIdx.x;
}

__global__ void dbl2_small_YWTC
 ( int nrows, int ncols, int rowdim, int coldim, int szt,
   int rowoff, int coloff, double *YWThi, double *YWTlo,
   double *Chi, double *Clo, double *YWTChi, double *YWTClo )
{
   const int j = threadIdx.x;
}

__global__ void dbl2_small_Qupdate
 ( int dim, int szt, int coloff,
   double *Qhi, double *Qlo, double *QWYThi, double *QWYTlo )
{
   const int j = threadIdx.x;
}

__global__ void dbl2_small_R_add_YWTC
 ( int nrows, int coldim, int szt, int rowoff, int coloff,
   double *Rhi, double *Rlo, double *YWTChi, double *YWTClo )
{
   const int j = threadIdx.x;
}

void GPU_dbl2_small_house
 ( int nrows, int ncols, int szt, int nbt,
   int colidx, int nrows1, int k, int L,
   double *Ahi_h, double *Alo_h, double *Ahi_d, double *Alo_d,
   double *vhi_h, double *vlo_h, double *Vhi_d, double *Vlo_d,
   double *betahi_h, double *betalo_h, double *betahi_d, double *betalo_d,
   double *lapms, bool verbose )
{
   const int nrLog2 = ceil(log2((double) nrows1));
   const int rowidx = colidx*(nrows+1);       // start of number in A_h
   const int nVrows = nrows - k*szt;          // dimension of V matrix

   if(verbose)
   {
      cout << "nrows : " << nrows
           << "  nVrows : " << nVrows
           << "  ncols : " << ncols
           << "  szt : " << szt
           << "  nbt : " << nbt << endl;
      cout << "k : " << k 
           << "  L : " << L
           << "  nrows1 : " << nrows1
           << "  colidx : " << colidx
           << "  rowidx : " << rowidx << endl;
   }
   if(L > 0)
   {
      for(int i=0; i<L; i++)             // insert zeros
      {
         vhi_h[i] = 0.0;
         vlo_h[i] = 0.0;
      }
      hipMemcpy(&Vhi_d[L*nVrows],vhi_h,L*sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(&Vlo_d[L*nVrows],vlo_h,L*sizeof(double),
                 hipMemcpyHostToDevice);
   }
   if(nrows1 == 0)
   {
      betahi_h[L] = 0.0; vhi_h[0] = 1.0;
      betalo_h[L] = 0.0; vlo_h[0] = 0.0;
      hipMemcpy(&betahi_d[L],&betahi_h[L],sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(&betalo_d[L],&betalo_h[L],sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(&Vhi_d[L*nVrows+L],vhi_h,sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(&Vlo_d[L*nVrows+L],vlo_h,sizeof(double),
                 hipMemcpyHostToDevice);
   }
   else
   {
      hipEvent_t start,stop;           // to measure time spent by kernels 
      hipEventCreate(&start);
      hipEventCreate(&stop);
      float milliseconds;

      hipEventRecord(start);
      dbl2_small_house<<<1,nrows1>>>
         (&Ahi_d[rowidx],&Alo_d[rowidx],&Ahi_d[rowidx+1],&Alo_d[rowidx+1],
          nrows1,nrLog2,&Vhi_d[L*nVrows+L],&Vlo_d[L*nVrows+L],
          &betahi_d[L],&betalo_d[L]);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);
      *lapms += milliseconds;
   }
   if(verbose)
   {
      const size_t szhouse = nVrows*sizeof(double);

      hipMemcpy(&betahi_h[L],&betahi_d[L],sizeof(double),
                 hipMemcpyDeviceToHost);
      hipMemcpy(&betalo_h[L],&betalo_d[L],sizeof(double),
                 hipMemcpyDeviceToHost);
      hipMemcpy(vhi_h,&Vhi_d[L*nVrows],szhouse,hipMemcpyDeviceToHost);
      hipMemcpy(vlo_h,&Vlo_d[L*nVrows],szhouse,hipMemcpyDeviceToHost);
      cout << scientific << setprecision(16)
           << "beta[" << colidx << "] : "
           << betahi_h[L] << "  " << betalo_h[L] << endl;
      for(int i=0; i<nVrows; i++)
         cout << "v[" << i << "] : " << vhi_h[i] << "  " << vlo_h[i] << endl;
   }
}

void GPU_dbl2_small_leftRupdate
 ( int nrows, int ncols, int szt, int colidx, int k, int L,
   double *Ahi_h, double *Alo_h, double *Ahi_d, double *Alo_d,
   double *Vhi_d, double *Vlo_d,
   double *betahi_h, double *betalo_h, double *betahi_d, double *betalo_d,
   double *lapms, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int endcol = (k+1)*szt;     // 1 + last column index in tile
   const int nVrows = nrows - k*szt;          // dimension of V matrix

   hipEventRecord(start);           // 2nd argument: ncols -> szt
   // changed second argument ncols into szt
   // to avoid updating the next tile
   dbl2_small_leftRupdate<<<1,nrows-colidx>>>
      (nrows,endcol,szt,colidx,Ahi_d,Alo_d,
       &Vhi_d[L*nVrows+L],&Vlo_d[L*nVrows+L],&betahi_d[L],&betalo_d[L]);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;

   if(verbose)
   {
      const int dim = nrows*ncols;
      const size_t sznum = dim*sizeof(double);

      hipMemcpy(Ahi_h,Ahi_d,sznum,hipMemcpyDeviceToHost);
      hipMemcpy(Ahi_h,Alo_d,sznum,hipMemcpyDeviceToHost);
      cout << "the matrix after the update :" << endl;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++)
            cout << "A_d[" << i << "][" << j << "] : "
                 << Ahi_h[j*nrows+i] << "  "
                 << Alo_h[j*nrows+i] << endl;
   }
}

void GPU_dbl2_VB_to_W
 ( int nrows, int ncols, int szt,
   double *Vhi_h, double *Vlo_h, double *Vhi_d, double *Vlo_d,
   double *Whi_h, double *Wlo_h, double *Whi_d, double *Wlo_d,
   double *betahi_h, double *betalo_h, double *betahi_d, double *betalo_d,
   double *lapms, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;

   hipEventRecord(start);
   dbl2_VB_to_W<<<1,nrows>>>
      (nrows,ncols,betahi_d,betalo_d,Vhi_d,Vlo_d,Whi_d,Wlo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;

   if(verbose)
   {
      const size_t szbeta = szt*sizeof(double);
      const size_t szhouse = nrows*sizeof(double);
      const size_t szVandW = szt*szhouse;

      hipMemcpy(betahi_h,betahi_d,szbeta,hipMemcpyDeviceToHost);
      hipMemcpy(betalo_h,betalo_d,szbeta,hipMemcpyDeviceToHost);
      cout << "the betas :" << endl;
      for(int j=0; j<szt; j++)
         cout << "beta[" << j << "] : "
              << betahi_h[j] << "  " << betalo_h[j] << endl;

      hipMemcpy(Vhi_h,Vhi_d,szVandW,hipMemcpyDeviceToHost);
      hipMemcpy(Vlo_h,Vlo_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the V matrix :" << endl;
      int ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<nrows; i++) 
         {
            cout << "V[" << i << "][" << j << "] : "
                 << Vhi_h[ix] << "  " << Vlo_h[ix] << endl;
            ix = ix + 1;
         }

      hipMemcpy(Whi_h,Whi_d,szVandW,hipMemcpyDeviceToHost);
      hipMemcpy(Wlo_h,Wlo_d,szVandW,hipMemcpyDeviceToHost);
      cout << "the columns of the W matrix :" << endl;
      ix = 0;
      for(int j=0; j<szt; j++) 
         for(int i=0; i<nrows; i++) 
         {
            cout << "W[" << i << "][" << j << "] : "
                 << Whi_h[ix] << "  " << Wlo_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl2_small_WYT
 ( int nrows, int szt,
   double *Whi_d, double *Wlo_d, double *Yhi_d, double *Ylo_d,
   double *WYThi_d, double *WYTlo_d, double *WYThi_h, double *WYTlo_h,
   double *lapms, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int nbrblocks = (int) ceil(nrows*nrows/((double) szt));

   hipEventRecord(start);
   dbl2_small_WYT<<<nbrblocks,szt>>>
      (nrows,szt,Whi_d,Wlo_d,Yhi_d,Ylo_d,WYThi_d,WYTlo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;

   if(verbose)
   {
      const size_t szmat = nrows*nrows*sizeof(double);

      hipMemcpy(WYThi_h,WYThi_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(WYTlo_h,WYTlo_d,szmat,hipMemcpyDeviceToHost);

      cout << "the WYT matrix :" << endl;
      int ix = 0;
      for(int i=0; i<nrows; i++) 
         for(int j=0; j<nrows; j++) 
         {
            cout << "WYT[" << i << "][" << j << "] : "
                 << WYThi_h[ix] << "  " << WYTlo_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl2_small_YWT
 ( int nrows, int szt, int idx,
   double *Yhi_d, double *Ylo_d, double *Whi_d, double *Wlo_d,
   double *YWThi_d, double *YWTlo_d, double *YWThi_h, double *YWTlo_h,
   double *lapms, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowdim = nrows - idx*szt;
   int nbrblocks = (int) ceil(rowdim*rowdim/((double) szt));

   hipEventRecord(start);
   dbl2_small_WYT<<<nbrblocks,szt>>>
      (rowdim,szt,Yhi_d,Ylo_d,Whi_d,Wlo_d,YWThi_d,YWTlo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;

   if(verbose)
   {
      const size_t szmat = rowdim*rowdim*sizeof(double);

      hipMemcpy(YWThi_h,YWThi_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(YWTlo_h,YWTlo_d,szmat,hipMemcpyDeviceToHost);

      cout << "the YWT matrix :" << endl;
      int ix = 0;
      for(int i=0; i<rowdim; i++) 
         for(int j=0; j<rowdim; j++) 
         {
            cout << "YWT[" << i << "][" << j << "] : "
                 << YWThi_h[ix] << "  " << YWTlo_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl2_small_QWYT
 ( int dim, int szt, int idx, double *Qhi_d, double *Qlo_d,
   double *WYThi_d, double *WYTlo_d, double *QWYThi_d, double *QWYTlo_d,
   double *QWYThi_h, double *QWYTlo_h, double *Qhi_h, double *Qlo_h,
   double *lapms, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int coloff = idx*szt;
   const int rowdim = dim - coloff;
   const int nbrblocks = (int) ceil(dim*rowdim/((double) szt));

   if(verbose)
   {
      const size_t szmat = dim*dim*sizeof(double);

      hipMemcpy(Qhi_h,Qhi_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(Qlo_h,Qlo_d,szmat,hipMemcpyDeviceToHost);

      cout << "the Q matrix :" << endl;
      int ix = 0;
      for(int i=0; i<dim; i++) 
         for(int j=0; j<dim; j++) 
         {
            cout << "Q[" << i << "][" << j << "] : "
                 << Qhi_h[ix] << "  " << Qlo_h[ix] << endl;
            ix = ix + 1;
         }
   }

   hipEventRecord(start);
   dbl2_small_QWYT<<<nbrblocks,szt>>>
      (dim,rowdim,szt,coloff,Qhi_d,Qlo_d,WYThi_d,WYTlo_d,QWYThi_d,QWYTlo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;

   if(verbose)
   {
      const size_t szmat = dim*rowdim*sizeof(double);

      hipMemcpy(QWYThi_h,QWYThi_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(QWYTlo_h,QWYTlo_d,szmat,hipMemcpyDeviceToHost);

      cout << "the QWYT matrix :" << endl;
      int ix = 0;
      for(int i=0; i<dim; i++) 
         for(int j=0; j<rowdim; j++) 
         {
            cout << "QWYT[" << i << "][" << j << "] : "
                 << QWYThi_h[ix] << "  " << QWYTlo_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl2_small_YWTC
 ( int nrows, int ncols, int szt, int idx, double *YWThi_d, double *YWTlo_d,
   double *Chi_d, double *Clo_d, double *YWTChi_d, double *YWTClo_d,
   double *YWTChi_h, double *YWTClo_h, double *lapms, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
}

void GPU_dbl2_small_Qupdate
 ( int dim, int szt, int idx,
   double *Qhi_d, double *Qlo_d, double *QWYThi_d, double *QWYTlo_d,
   double *Qhi_h, double *Qlo_h, double *lapms, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int coloff = idx*szt;
   const int rowdim = dim - coloff;
   const int nbrblocks = (int) ceil(dim*rowdim/((double) szt));

   hipEventRecord(start);
   dbl2_small_Qupdate<<<nbrblocks,szt>>>
      (dim,szt,coloff,Qhi_d,Qlo_d,QWYThi_d,QWYTlo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;

   if(verbose)
   {
      const size_t szmat = dim*dim*sizeof(double);

      hipMemcpy(Qhi_h,Qhi_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(Qlo_h,Qlo_d,szmat,hipMemcpyDeviceToHost);

      cout << "the Q matrix :" << endl;
      int ix = 0;
      for(int i=0; i<dim; i++) 
         for(int j=0; j<dim; j++) 
         {
            cout << "Q[" << i << "][" << j << "] : "
                 << Qhi_h[ix] << "  " << Qlo_h[ix] << endl;
            ix = ix + 1;
         }
   }
}

void GPU_dbl2_small_R_add_YWTC
 ( int nrows, int ncols, int szt, int idx, double *Rhi_d, double *Rlo_d,
   double *YWTChi_d, double *YWTClo_d, double *Rhi_h, double *Rlo_h,
   double *lapms, bool verbose )
{
   hipEvent_t start,stop;           // to measure time spent by kernels 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float milliseconds;
   const int rowoff = idx*szt;
   const int rowdim = nrows - rowoff;
   const int coloff = (idx+1)*szt;
   const int coldim = ncols - coloff;
   const int nbrblocks = (int) ceil(rowdim*coldim/((double) szt));

   hipEventRecord(start);
   dbl2_small_R_add_YWTC<<<nbrblocks,szt>>>
      (nrows,coldim,szt,rowoff,coloff,Rhi_d,Rlo_d,YWTChi_d,YWTClo_d);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds,start,stop);
   *lapms += milliseconds;

   if(verbose)
   {
      const size_t szmat = nrows*ncols*sizeof(double);

      hipMemcpy(Rhi_h,Rhi_d,szmat,hipMemcpyDeviceToHost);
      hipMemcpy(Rlo_h,Rlo_d,szmat,hipMemcpyDeviceToHost);

      cout << "the R matrix :" << endl;
      for(int i=rowoff; i<nrows; i++) 
         for(int j=coloff; j<ncols; j++)
            cout << "R[" << i << "][" << j << "] : "
                 << Rhi_h[j*nrows + i] << "  "
                 << Rlo_h[j*nrows + i] << endl;
   }
}

void GPU_dbl2_blocked_houseqr
 ( int nrows, int ncols, int szt, int nbt,
   double **Ahi, double **Alo, double **Qhi, double **Qlo,
   double **Rhi, double **Rlo,
   double *houselapms, double *tileRlapms, double *vb2Wlapms,
   double *WYTlapms, double *QWYTlapms, double *Qaddlapms,
   double *YWTlapms, double *YWTClapms, double *Raddlapms,
   double *walltimesec, bool verbose )
{
   const int dim = nrows*ncols;         // total number of doubles
   const int nrows2 = nrows*nrows;
   double *Ahi_h = new double[dim];     // high doubles of A on the host
   double *Alo_h = new double[dim];     // low doubles of A on the host
   double *Ahi_d;                       // Ahi on the device
   double *Alo_d;                       // Alo on the device
   double *Qhi_h = new double[nrows2];  // high doubles of Q on the host
   double *Qlo_h = new double[nrows2];  // low doubles of Q on the host
   double *Qhi_d;                       // Qhi on the device
   double *Qlo_d;                       // Qlo on the device
   double *vhi_h = new double[nrows];   // high doubles of Householder vector
   double *vlo_h = new double[nrows];   // low doubles of Householder vector
   double *betahi_h = new double[szt];  // high doubles of beta on the host
   double *betalo_h = new double[szt];  // low doubles of beta on the host
   double *betahi_d;                      // betahi on the device
   double *betalo_d;                      // betalo on the device
   double *Vhi_h = new double[nrows*szt]; // high doubles of V matrix
   double *Vlo_h = new double[nrows*szt]; // low doubles of V matrix
   double *Vhi_d;                         // Vhi on the device
   double *Vlo_d;                         // Vlo on the device
   double *Whi_h = new double[nrows*szt]; // high doubes of W on the host
   double *Wlo_h = new double[nrows*szt]; // low doules of W on the host
   double *Whi_d;                         // Whi on the device
   double *Wlo_d;                         // Wlo on the device
   double *WYThi_h = new double[nrows2];  // high doubles of W*Y^T 
   double *WYTlo_h = new double[nrows2];  // low doubles of W*Y^T
   double *WYThi_d;                       // WYThi on the device
   double *WYTlo_d;                       // WYTlo on the device
   double *YWThi_h = new double[nrows2];  // high doubles of Y*W^T
   double *YWTlo_h = new double[nrows2];  // low doubles of Y*W^T 
   double *YWThi_d;                       // YWThi on the device
   double *YWTlo_d;                       // YWTlo on the device
   double *QWYThi_h = new double[nrows2]; // high doubles of Q*WY^T
   double *QWYTlo_h = new double[nrows2]; // low doubles of Q*WY^T
   double *QWYThi_d;                      // QWYThi on the device
   double *QWYTlo_d;                      // QWYTlo on the device
   double *YWTChi_h = new double[dim];    // YWT*C on the host
   double *YWTClo_h = new double[dim];    // YWT*C on the host
   double *YWTChi_d;                      // YWTChi on the device
   double *YWTClo_d;                      // YWTClo on the device

   int ix = 0;                          // copy the columns of A to A_h
   for(int j=0; j<ncols; j++)   
      for(int i=0; i<nrows; i++)
      {
         Ahi_h[ix]   = Ahi[i][j];
         Alo_h[ix++] = Alo[i][j];
      }

   ix = 0;                              // initialize Q with identity
   for(int i=0; i<nrows; i++)
   {
      for(int j=0; j<nrows; j++)
      {
         if(i == j)
         {
            Qhi_h[ix]   = 1.0;
            Qlo_h[ix++] = 0.0;
         }
         else
         {
            Qhi_h[ix]   = 0.0;
            Qlo_h[ix++] = 0.0;
         }
      }
   }
   const size_t sznum = dim*sizeof(double);
   hipMalloc((void**)&Ahi_d,sznum);
   hipMalloc((void**)&Alo_d,sznum);
   hipMemcpy(Ahi_d,Ahi_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Alo_d,Alo_h,sznum,hipMemcpyHostToDevice);

   const size_t szbeta = szt*sizeof(double);
   hipMalloc((void**)&betahi_d,szbeta);
   hipMalloc((void**)&betalo_d,szbeta);
   for(int i=0; i<szt; i++)
   {
      betahi_h[i] = 0.0;
      betalo_h[i] = 0.0;
   }
   hipMemcpy(betahi_d,betahi_h,szbeta,hipMemcpyHostToDevice);
   hipMemcpy(betalo_d,betalo_h,szbeta,hipMemcpyHostToDevice);

   const size_t szhouse = nrows*sizeof(double);
   const size_t szpad = szt*sizeof(double);  // padding for nonsquare tiles
   const size_t szVandW = szt*szhouse;
   hipMalloc((void**)&Vhi_d,szVandW + szpad); // padding only in allocation
   hipMalloc((void**)&Vlo_d,szVandW + szpad);
   ix = 0;
   for(int i=0; i<nrows*szt; i++)
   {
      Vhi_h[ix] = 0.0; 
      Vlo_h[ix++] = 0.0; 
   }
   Vhi_h[--ix] = 1.0; // initialize last vector for square tiles
   hipMemcpy(Vhi_d,Vhi_h,szVandW,hipMemcpyHostToDevice);
   hipMemcpy(Vlo_d,Vlo_h,szVandW,hipMemcpyHostToDevice);
   hipMalloc((void**)&Whi_d,szVandW + szpad); // padding only in allocation
   hipMalloc((void**)&Wlo_d,szVandW + szpad); 

   const size_t szWYT = nrows2*sizeof(double);
   hipMalloc((void**)&WYThi_d,szWYT + szpad); // padding for W*Y^T product
   hipMalloc((void**)&WYTlo_d,szWYT + szpad); 
   hipMalloc((void**)&Qhi_d,szWYT);
   hipMalloc((void**)&Qlo_d,szWYT);
   hipMemcpy(Qhi_d,Qhi_h,szWYT,hipMemcpyHostToDevice);
   hipMemcpy(Qlo_d,Qlo_h,szWYT,hipMemcpyHostToDevice);
   hipMalloc((void**)&QWYThi_d,szWYT);
   hipMalloc((void**)&QWYTlo_d,szWYT);

   const size_t szYWT = nrows2*sizeof(double);
   hipMalloc((void**)&YWThi_d,szYWT + szpad); // padding for Y*W^T product
   hipMalloc((void**)&YWTlo_d,szYWT + szpad);
   hipMalloc((void**)&YWTChi_d,sznum + szpad);
   hipMalloc((void**)&YWTClo_d,sznum + szpad);

   *houselapms = 0.0;
   *tileRlapms = 0.0;
   *vb2Wlapms = 0.0;
   *WYTlapms = 0.0; *QWYTlapms = 0.0; *Qaddlapms = 0.0;
   *YWTlapms = 0.0; *YWTClapms = 0.0; *Raddlapms = 0.0;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   for(int k=0; k<nbt; k++)       // k runs over the number of blocks
   {
      if(verbose)
         cout << "Tile k = " << k << " out of " << nbt << " ..." << endl;

      int colidx,nrows1;

      for(int L=0; L<szt; L++)  // L runs over the columns in one block
      {
         colidx = k*szt + L;              // index of the current column
         nrows1 = nrows - colidx - 1;     // #rows in Householder vector - 1
         GPU_dbl2_small_house
            (nrows,ncols,szt,nbt,colidx,nrows1,k,L,
             Ahi_h,Alo_h,Ahi_d,Alo_d,vhi_h,vlo_h,Vhi_d,Vlo_d,
             betahi_h,betalo_h,betahi_d,betalo_d,houselapms,verbose);
         GPU_dbl2_small_leftRupdate
            (nrows,ncols,szt,colidx,k,L,Ahi_h,Alo_h,Ahi_d,Alo_d,
             Vhi_d,Vlo_d,betahi_h,betalo_d,betahi_d,betalo_d,
             tileRlapms,verbose);
      }
      // changed nrows into nrows - k*szt and ncols into szt
      GPU_dbl2_VB_to_W
         (nrows-k*szt,szt,szt,Vhi_h,Vlo_h,Vhi_d,Vlo_d,Whi_h,Wlo_h,
          Whi_d,Wlo_d,betahi_h,betalo_h,betahi_d,betalo_d,vb2Wlapms,verbose);
      // update Q, WYT matrix has nrows - k*szt instead of nrows
      GPU_dbl2_small_WYT
         (nrows-k*szt,szt,Whi_d,Wlo_d,Vhi_d,Vlo_d,WYThi_d,WYTlo_d,
          WYThi_h,WYTlo_h,WYTlapms,verbose);
      GPU_dbl2_small_QWYT
         (nrows,szt,k,Qhi_d,Qlo_d,WYThi_d,WYThi_d,QWYThi_d,QWYTlo_d,
          QWYThi_h,QWYTlo_h,Qhi_h,Qlo_h,QWYTlapms,verbose);
      GPU_dbl2_small_Qupdate
         (nrows,szt,k,Qhi_d,Qlo_d,QWYThi_d,QWYTlo_d,Qhi_h,Qlo_h,
          Qaddlapms,verbose);
      if(k < nbt-1)                                           // update R
      {
         GPU_dbl2_small_YWT
            (nrows,szt,k,Vhi_d,Vlo_d,Whi_d,Wlo_d,YWThi_d,YWTlo_d,
             YWThi_h,YWTlo_h,YWTlapms,verbose);
         GPU_dbl2_small_YWTC
            (nrows,ncols,szt,k,YWThi_d,YWTlo_d,Ahi_d,Alo_d,
             YWTChi_d,YWTClo_d,YWTChi_h,YWTClo_h,YWTClapms,verbose);
         GPU_dbl2_small_R_add_YWTC
            (nrows,ncols,szt,k,Ahi_d,Alo_d,YWTChi_d,YWTClo_d,
             Ahi_h,Alo_h,Raddlapms,verbose);
      }
   }
   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   hipMemcpy(Qhi_h,Qhi_d,szWYT,hipMemcpyDeviceToHost);
   hipMemcpy(Qlo_h,Qlo_d,szWYT,hipMemcpyDeviceToHost);
   ix = 0;                                           // copy rows of Q
   for(int i=0; i<nrows; i++)
      for(int j=0; j<nrows; j++)
      {
         Qhi[i][j] = Qhi_h[ix];
         Qlo[i][j] = Qlo_h[ix++];
      }

   hipMemcpy(Ahi_h,Ahi_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(Alo_h,Alo_d,sznum,hipMemcpyDeviceToHost);
   for(int i=0; i<nrows; i++)                       // copy columns of R
      for(int j=0; j<ncols; j++)
      {
         Rhi[i][j] = Ahi_h[j*nrows+i];
         Rlo[i][j] = Alo_h[j*nrows+i];
      }

   free(Ahi_h); free(Alo_h); free(Qhi_h); free(Qlo_h); 
   free(vhi_h); free(vlo_h); free(Vhi_h); free(Vlo_h);
   free(Whi_h); free(Wlo_h);
   free(WYThi_h); free(QWYThi_h); free(YWThi_h); free(YWTChi_h);
   free(WYTlo_h); free(QWYTlo_h); free(YWTlo_h); free(YWTClo_h);
}
