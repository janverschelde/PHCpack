#include "hip/hip_runtime.h"
/* The file dbl8_baqr_kernels.cu defines the functions with prototypes in
 * the file dbl8_baqr_kernels.h. */

#include <iostream>
#include <iomanip>
#ifdef winwalltime
#include "gettimeofday4win.h"
#else
#include <sys/time.h>
#endif
#ifdef gpufun
#include "double_double_gpufun.cu"
#include "quad_double_gpufun.cu"
#include "octo_double_gpufun.cu"
#endif
#include "dbl8_baqr_kernels.h"
#include "octo_double_functions.h"
#include "dbl_baqr_flopcounts.h"

using namespace std;

__global__ void dbl8_small_house
 ( double *x0hihihi, double *x0lohihi, double *x0hilohi, double *x0lolohi,
   double *x0hihilo, double *x0lohilo, double *x0hilolo, double *x0lololo,
   double *x1hihihi, double *x1lohihi, double *x1hilohi, double *x1lolohi,
   double *x1hihilo, double *x1lohilo, double *x1hilolo, double *x1lololo,
   int dim, int dimLog2,
   double *vhihihi, double *vlohihi, double *vhilohi, double *vlolohi,
   double *vhihilo, double *vlohilo, double *vhilolo, double *vlololo,
   double *betahihihi, double *betalohihi,
   double *betahilohi, double *betalolohi,
   double *betahihilo, double *betalohilo,
   double *betahilolo, double *betalololo )
{
}

__global__ void cmplx8_small_house
 ( double *x0rehihihi, double *x0relohihi,
   double *x0rehilohi, double *x0relolohi,
   double *x0rehihilo, double *x0relohilo,
   double *x0rehilolo, double *x0relololo,
   double *x0imhihihi, double *x0imlohihi,
   double *x0imhilohi, double *x0imlolohi,
   double *x0imhihilo, double *x0imlohilo,
   double *x0imhilolo, double *x0imlololo,
   double *x1rehihihi, double *x1relohihi,
   double *x1rehilohi, double *x1relolohi,
   double *x1rehihilo, double *x1relohilo,
   double *x1rehilolo, double *x1relololo,
   double *x1imhihihi, double *x1imlohihi,
   double *x1imhilohi, double *x1imlolohi,
   double *x1imhihilo, double *x1imlohilo,
   double *x1imhilolo, double *x1imlololo,
   int dim, int dimLog2,
   double *vrehihihi, double *vrelohihi, double *vrehilohi, double *vrelolohi,
   double *vrehihilo, double *vrelohilo, double *vrehilolo, double *vrelololo,
   double *vimhihihi, double *vimlohihi, double *vimhilohi, double *vimlolohi,
   double *vimhihilo, double *vimlohilo, double *vimhilolo, double *vimlololo,
   double *betahihihi, double *betalohihi,
   double *betahilohi, double *betalolohi,
   double *betahihilo, double *betalohilo,
   double *betahilolo, double *betalololo )
{
}

__global__ void dbl8_large_sum_of_squares
 ( double *vhihihi, double *vlohihi, double *vhilohi, double *vlolohi,
   double *vhihilo, double *vlohilo, double *vhilolo, double *vlololo,
   double *sumshihihi, double *sumshilohi,
   double *sumslohihi, double *sumslolohi,
   double *sumshihilo, double *sumshilolo,
   double *sumslohilo, double *sumslololo, int dim, int BS, int BSLog2 )
{
}

__global__ void cmplx8_large_sum_of_squares
 ( double *vrehihihi, double *vrelohihi, double *vrehilohi, double *vrelolohi,
   double *vrehihilo, double *vrelohilo, double *vrehilolo, double *vrelololo,
   double *vimhihihi, double *vimlohihi, double *vimhilohi, double *vimlolohi,
   double *vimhihilo, double *vimlohilo, double *vimhilolo, double *vimlololo,
   double *sumshihihi, double *sumslohihi,
   double *sumshilohi, double *sumslolohi,
   double *sumshihilo, double *sumslohilo,
   double *sumshilolo, double *sumslololo, int dim, int BS, int BSLog2 )
{
}

__global__ void dbl8_sum_accumulator
 ( double *sumshihihi, double *sumslohihi,
   double *sumshilohi, double *sumslolohi,
   double *sumshihilo, double *sumslohilo,
   double *sumshilolo, double *sumslololo,
   int nbsums, int nbsumsLog2,
   double *acchihihi, double *acclohihi,
   double *acchilohi, double *acclolohi,
   double *acchihilo, double *acclohilo,
   double *acchilolo, double *acclololo )
{
}

__global__ void dbl8_normalize
 ( int dim, int szt,
   double *xhihihi, double *xlohihi, double *xhilohi, double *xlolohi,
   double *xhihilo, double *xlohilo, double *xhilolo, double *xlololo,
   double *v0hihihi, double *v0lohihi, double *v0hilohi, double *v0lolohi,
   double *v0hihilo, double *v0lohilo, double *v0hilolo, double *v0lololo,
   double *vhihihi, double *vlohihi, double *vhilohi, double *vlolohi,
   double *vhihilo, double *vlohilo, double *vhilolo, double *vlololo )
{
}

__global__ void cmplx8_normalize
 ( int dim, int szt,
   double *xrehihihi, double *xrelohihi, double *xrehilohi, double *xrelolohi,
   double *xrehihilo, double *xrelohilo, double *xrehilolo, double *xrelololo,
   double *ximhihihi, double *ximlohihi, double *ximhilohi, double *ximlolohi,
   double *ximhihilo, double *ximlohilo, double *ximhilolo, double *ximlololo,
   double *inv0rehihihi, double *inv0relohihi,
   double *inv0rehilohi, double *inv0relolohi,
   double *inv0rehihilo, double *inv0relohilo,
   double *inv0rehilolo, double *inv0relololo,
   double *inv0imhihihi, double *inv0imlohihi,
   double *inv0imhilohi, double *inv0imlolohi,
   double *inv0imhihilo, double *inv0imlohilo,
   double *inv0imhilolo, double *inv0imlololo,
   double *vrehihihi, double *vrelohihi, double *vrehilohi, double *vrelolohi,
   double *vrehihilo, double *vrelohilo, double *vrehilolo, double *vrelololo,
   double *vimhihihi, double *vimlohihi, double *vimhilohi, double *vimlolohi,
   double *vimhihilo, double *vimlohilo,
   double *vimhilolo, double *vimlololo )
{
}

__global__ void dbl8_small_leftRupdate
 ( int nrows, int ncols, int szt, int k,
   double *Rhihihi, double *Rlohihi, double *Rhilohi, double *Rlolohi,
   double *Rhihilo, double *Rlohilo, double *Rhilolo, double *Rlololo,
   double *vhihihi, double *vlohihi, double *vhilohi, double *vlolohi,
   double *vhihilo, double *vlohilo, double *vhilolo, double *vlololo,
   double *betahihihi, double *betalohihi,
   double *betahilohi, double *betalolohi,
   double *betahihilo, double *betalohilo,
   double *betahilolo, double *betalololo )
{
}

__global__ void cmplx8_small_leftRupdate
 ( int nrows, int ncols, int szt, int k,
   double *Rrehihihi, double *Rrelohihi, double *Rrehilohi, double *Rrelolohi,
   double *Rrehihilo, double *Rrelohilo, double *Rrehilolo, double *Rrelololo,
   double *Rimhihihi, double *Rimlohihi, double *Rimhilohi, double *Rimlolohi,
   double *Rimhihilo, double *Rimlohilo, double *Rimhilolo, double *Rimlololo,
   double *vrehihihi, double *vrelohihi, double *vrehilohi, double *vrelolohi,
   double *vrehihilo, double *vrelohilo, double *vrehilolo, double *vrelololo,
   double *vimhihihi, double *vimlohihi, double *vimhilohi, double *vimlolohi,
   double *vimhihilo, double *vimlohilo, double *vimhilolo, double *vimlololo,
   double *betahihihi, double *betalohihi, 
   double *betahilohi, double *betalolohi,
   double *betahihilo, double *betalohilo,
   double *betahilolo, double *betalololo )
{
}

__global__ void dbl8_RTdotv
 ( int nrows, int szt, int colidx, int Roffset, int dim,
   double *Rhihihi, double *Rlohihi, double *Rhilohi, double *Rlolohi,
   double *Rhihilo, double *Rlohilo, double *Rhilolo, double *Rlololo,
   double *vhihihi, double *vlohihi, double *vhilohi, double *vlolohi,
   double *vhihilo, double *vlohilo, double *vhilolo, double *vlololo,
   double *RTdotvhihihi, double *RTdotvlohihi,
   double *RTdotvhilohi, double *RTdotvlolohi,
   double *RTdotvhihilo, double *RTdotvlohilo,
   double *RTdotvhilolo, double *RTdotvlololo )
{
}

__global__ void cmplx8_RHdotv
 ( int nrows, int szt, int colidx, int Roffset, int dim,
   double *Rrehihihi, double *Rrelohihi, double *Rrehilohi, double *Rrelolohi,
   double *Rrehihilo, double *Rrelohilo, double *Rrehilolo, double *Rrelololo,
   double *Rimhihihi, double *Rimlohihi, double *Rimhilohi, double *Rimlolohi,
   double *Rimhihilo, double *Rimlohilo, double *Rimhilolo, double *Rimlololo,
   double *vrehihihi, double *vrelohihi, double *vrehilohi, double *vrelolohi,
   double *vrehihilo, double *vrelohilo, double *vrehilolo, double *vrelololo,
   double *vimhihihi, double *vimlohihi, double *vimhilohi, double *vimlolohi,
   double *vimhihilo, double *vimlohilo, double *vimhilolo, double *vimlololo,
   double *RHdotvrehihihi, double *RHdotvrelohihi,
   double *RHdotvrehilohi, double *RHdotvrelolohi,
   double *RHdotvrehihilo, double *RHdotvrelohilo,
   double *RHdotvrehilolo, double *RHdotvrelololo,
   double *RHdotvimhihihi, double *RHdotvimlohihi,
   double *RHdotvimhilohi, double *RHdotvimlolohi,
   double *RHdotvimhihilo, double *RHdotvimlohilo,
   double *RHdotvimhilolo, double *RHdotvimlololo )
{
}

__global__ void dbl8_sum_betaRTdotv
 ( int nrows,
   double *betahihihi, double *betalohihi,
   double *betahilohi, double *betalolohi,
   double *betahihilo, double *betalohilo,
   double *betahilolo, double *betalololo,
   double *RTdotvhihihi, double *RTdotvlohihi,
   double *RTdotvhilohi, double *RTdotvlolohi,
   double *RTdotvhihilo, double *RTdotvlohilo,
   double *RTdotvhilolo, double *RTdotvlololo,
   double *whihihi, double *wlohihi, double *whilohi, double *wlolohi,
   double *whihilo, double *wlohilo, double *whilolo, double *wlololo )
{
}

__global__ void cmplx8_sum_betaRHdotv
 ( int nrows,
   double *betahihihi, double *betalohihi,
   double *betahilohi, double *betalolohi,
   double *betahihilo, double *betalohilo,
   double *betahilolo, double *betalololo,
   double *RTdotvrehihihi, double *RTdotvrelohihi,
   double *RTdotvrehilohi, double *RTdotvrelolohi,
   double *RTdotvrehihilo, double *RTdotvrelohilo,
   double *RTdotvrehilolo, double *RTdotvrelololo,
   double *RTdotvimhihihi, double *RTdotvimlohihi,
   double *RTdotvimhilohi, double *RTdotvimlolohi,
   double *RTdotvimhihilo, double *RTdotvimlohilo,
   double *RTdotvimhilolo, double *RTdotvimlololo,
   double *wrehihihi, double *wrelohihi, double *wrehilohi, double *wrelolohi,
   double *wrehihilo, double *wrelohilo, double *wrehilolo, double *wrelololo,
   double *wimhihihi, double *wimlohihi,
   double *wimhilohi, double *wimlolohi,
   double *wimhihilo, double *wimlohilo,
   double *wimhilolo, double *wimlololo )
{
}

__global__ void dbl8_medium_subvbetaRTv
 ( int nrows, int ncols, int szt, int k,
   double *Rhihihi, double *Rlohihi, double *Rhilohi, double *Rlolohi,
   double *Rhihilo, double *Rlohilo, double *Rhilolo, double *Rlololo,
   double *vhihihi, double *vlohihi, double *vhilohi, double *vlolohi,
   double *vhihilo, double *vlohilo, double *vhilolo, double *vlololo,
   double *betahihihi, double *betalohihi,
   double *betahilohi, double *betalolohi,
   double *betahihilo, double *betalohilo,
   double *betahilolo, double *betalololo,
   double *whihihi, double *wlohihi, double *whilohi, double *wlolohi,
   double *whihilo, double *wlohilo, double *whilolo, double *wlololo )
{
}

__global__ void cmplx8_medium_subvbetaRHv
 ( int nrows, int ncols, int szt, int k,
   double *Rrehihihi, double *Rrelohihi, double *Rrehilohi, double *Rrelolohi,
   double *Rrehihilo, double *Rrelohilo, double *Rrehilolo, double *Rrelololo,
   double *Rimhihihi, double *Rimlohihi, double *Rimhilohi, double *Rimlolohi,
   double *Rimhihilo, double *Rimlohilo, double *Rimhilolo, double *Rimlololo,
   double *vrehihihi, double *vrelohihi, double *vrehilohi, double *vrelolohi,
   double *vrehihilo, double *vrelohilo, double *vrehilolo, double *vrelololo,
   double *vimhihihi, double *vimlohihi, double *vimhilohi, double *vimlolohi,
   double *vimhihilo, double *vimlohilo, double *vimhilolo, double *vimlololo,
   double *betahihihi, double *betalohihi,
   double *betahilohi, double *betalolohi,
   double *betahihilo, double *betalohilo,
   double *betahilolo, double *betalololo,
   double *wrehihihi, double *wrelohihi, double *wrehilohi, double *wrelolohi,
   double *wrehihilo, double *wrelohilo, double *wrehilolo, double *wrelololo,
   double *wimhihihi, double *wimlohihi,
   double *wimhilohi, double *wimlolohi,
   double *wimhihilo, double *wimlohilo,
   double *wimhilolo, double *wimlololo )
{
}

__global__ void dbl8_beta_times_V
 ( int nrows, int szt,
   double *Bhihihi, double *Blohihi, double *Bhilohi, double *Blolohi,
   double *Bhihilo, double *Blohilo, double *Bhilolo, double *Blololo,
   double *Vhihihi, double *Vlohihi, double *Vhilohi, double *Vlolohi,
   double *Vhihilo, double *Vlohilo, double *Vhilolo, double *Vlololo,
   double *Whihihi, double *Wlohihi, double *Whilohi, double *Wlolohi,
   double *Whihilo, double *Wlohilo, double *Whilolo, double *Wlololo )
{
}

__global__ void cmplx8_beta_times_V
 ( int nrows, int szt,
   double *Bhihihi, double *Blohihi, double *Bhilohi, double *Blolohi,
   double *Bhihilo, double *Blohilo, double *Bhilolo, double *Blololo,
   double *Vrehihihi, double *Vrelohihi, double *Vrehilohi, double *Vrelolohi,
   double *Vrehihilo, double *Vrelohilo, double *Vrehilolo, double *Vrelololo,
   double *Vimhihihi, double *Vimlohihi, double *Vimhilohi, double *Vimlolohi,
   double *Vimhihilo, double *Vimlohilo, double *Vimhilolo, double *Vimlololo,
   double *Wrehihihi, double *Wrelohihi, double *Wrehilohi, double *Wrelolohi,
   double *Wrehihilo, double *Wrelohilo, double *Wrehilolo, double *Wrelololo,
   double *Wimhihihi, double *Wimlohihi,
   double *Wimhilohi, double *Wimlolohi,
   double *Wimhihilo, double *Wimlohilo,
   double *Wimhilolo, double *Wimlololo )
{
}

__global__ void dbl8_initialize_WYT
 ( int dim, int szt,
   double *Vhihihi, double *Vlohihi, double *Vhilohi, double *Vlolohi,
   double *Vhihilo, double *Vlohilo, double *Vhilolo, double *Vlololo,
   double *Whihihi, double *Wlohihi, double *Whilohi, double *Wlolohi,
   double *Whihilo, double *Wlohilo, double *Whilolo, double *Wlololo,
   double *WYThihihi, double *WYTlohihi,
   double *WYThilohi, double *WYTlolohi,
   double *WYThihilo, double *WYTlohilo,
   double *WYThilolo, double *WYTlololo )
{
}

__global__ void cmplx8_initialize_WYH
 ( int dim, int szt,
   double *Vrehihihi, double *Vrelohihi, double *Vrehilohi, double *Vrelolohi,
   double *Vrehihilo, double *Vrelohilo, double *Vrehilolo, double *Vrelololo,
   double *Vimhihihi, double *Vimlohihi, double *Vimhilohi, double *Vimlolohi,
   double *Vimhihilo, double *Vimlohilo, double *Vimhilolo, double *Vimlololo,
   double *Wrehihihi, double *Wrelohihi, double *Wrehilohi, double *Wrelolohi,
   double *Wrehihilo, double *Wrelohilo, double *Wrehilolo, double *Wrelololo,
   double *Wimhihihi, double *Wimlohihi, double *Wimhilohi, double *Wimlolohi,
   double *Wimhihilo, double *Wimlohilo, double *Wimhilolo, double *Wimlololo,
   double *WYTrehihihi, double *WYTrelohihi,
   double *WYTrehilohi, double *WYTrelolohi,
   double *WYTrehihilo, double *WYTrelohilo,
   double *WYTrehilolo, double *WYTrelololo,
   double *WYTimhihihi, double *WYTimlohihi,
   double *WYTimhilohi, double *WYTimlolohi,
   double *WYTimhihilo, double *WYTimlohilo,
   double *WYTimhilolo, double *WYTimlololo )
{
}

__global__ void dbl8_update_WYT
 ( int dim, int szt,
   double *Vhihihi, double *Vlohihi, double *Vhilohi, double *Vlolohi,
   double *Vhihilo, double *Vlohilo, double *Vhilolo, double *Vlololo,
   double *Whihihi, double *Wlohihi, double *Whilohi, double *Wlolohi,
   double *Whihilo, double *Wlohilo, double *Whilolo, double *Wlololo,
   double *WYThihihi, double *WYTlohihi,
   double *WYThilohi, double *WYTlolohi,
   double *WYThihilo, double *WYTlohilo,
   double *WYThilolo, double *WYTlololo )
{
}

__global__ void cmplx8_update_WYH
 ( int dim, int szt,
   double *Vrehihihi, double *Vrelohihi, double *Vrehilohi, double *Vrelolohi,
   double *Vrehihilo, double *Vrelohilo, double *Vrehilolo, double *Vrelololo,
   double *Vimhihihi, double *Vimlohihi, double *Vimhilohi, double *Vimlolohi,
   double *Vimhihilo, double *Vimlohilo, double *Vimhilolo, double *Vimlololo,
   double *Wrehihihi, double *Wrelohihi, double *Wrehilohi, double *Wrelolohi,
   double *Wrehihilo, double *Wrelohilo, double *Wrehilolo, double *Wrelololo,
   double *Wimhihihi, double *Wimlohihi, double *Wimhilohi, double *Wimlolohi,
   double *Wimhihilo, double *Wimlohilo, double *Wimhilolo, double *Wimlololo,
   double *WYHrehihihi, double *WYHrelohihi,
   double *WYHrehilohi, double *WYHrelolohi,
   double *WYHrehihilo, double *WYHrelohilo,
   double *WYHrehilolo, double *WYHrelololo,
   double *WYHimhihihi, double *WYHimlohihi,
   double *WYHimhilohi, double *WYHimlolohi,
   double *WYHimhihilo, double *WYHimlohilo,
   double *WYHimhilolo, double *WYHimlololo )
{
}

__global__ void dbl8_beta_next_W
 ( int nrows, int szt,
   double *Bhihihi, double *Blohihi, double *Bhilohi, double *Blolohi,
   double *Bhihilo, double *Blohilo, double *Bhilolo, double *Blololo,
   double *Vhihihi, double *Vlohihi, double *Vhilohi, double *Vlolohi,
   double *Vhihilo, double *Vlohilo, double *Vhilolo, double *Vlololo,
   double *Whihihi, double *Wlohihi, double *Whilohi, double *Wlolohi,
   double *Whihilo, double *Wlohilo, double *Whilolo, double *Wlololo,
   double *WYThihihi, double *WYTlohihi,
   double *WYThilohi, double *WYTlolohi,
   double *WYThihilo, double *WYTlohilo,
   double *WYThilolo, double *WYTlololo )
{
}

__global__ void cmplx8_beta_next_W
 ( int nrows, int szt,
   double *Bhihihi, double *Blohihi, double *Bhilohi, double *Blolohi,
   double *Bhihilo, double *Blohilo, double *Bhilolo, double *Blololo,
   double *Vrehihihi, double *Vrelohihi, double *Vrehilohi, double *Vrelolohi,
   double *Vrehihilo, double *Vrelohilo, double *Vrehilolo, double *Vrelololo,
   double *Vimhihihi, double *Vimlohihi, double *Vimhilohi, double *Vimlolohi,
   double *Vimhihilo, double *Vimlohilo, double *Vimhilolo, double *Vimlololo,
   double *Wrehihihi, double *Wrelohihi, double *Wrehilohi, double *Wrelolohi,
   double *Wrehihilo, double *Wrelohilo, double *Wrehilolo, double *Wrelololo,
   double *Wimhihihi, double *Wimlohihi, double *Wimhilohi, double *Wimlolohi,
   double *Wimhihilo, double *Wimlohilo, double *Wimhilolo, double *Wimlololo,
   double *WYHrehihihi, double *WYHrelohihi,
   double *WYHrehilohi, double *WYHrelolohi,
   double *WYHrehihilo, double *WYHrelohilo,
   double *WYHrehilolo, double *WYHrelololo,
   double *WYHimhihihi, double *WYHimlohihi,
   double *WYHimhilohi, double *WYHimlolohi,
   double *WYHimhihilo, double *WYHimlohilo,
   double *WYHimhilolo, double *WYHimlololo )
{
}

__global__ void dbl8_small_WYT
 ( int nrows, int szt,
   double *Whihihi, double *Wlohihi, double *Whilohi, double *Wlolohi,
   double *Whihilo, double *Wlohilo, double *Whilolo, double *Wlololo,
   double *Vhihihi, double *Vlohihi, double *Vhilohi, double *Vlolohi,
   double *Vhihilo, double *Vlohilo, double *Vhilolo, double *Vlololo,
   double *WYThihihi, double *WYTlohihi,
   double *WYThilohi, double *WYTlolohi,
   double *WYThihilo, double *WYTlohilo,
   double *WYThilolo, double *WYTlololo )
{
}

__global__ void cmplx8_small_WYH
 ( int nrows, int szt,
   double *Wrehihihi, double *Wrelohihi, double *Wrehilohi, double *Wrelolohi,
   double *Wrehihilo, double *Wrelohilo, double *Wrehilolo, double *Wrelololo,
   double *Wimhihihi, double *Wimlohihi, double *Wimhilohi, double *Wimlolohi,
   double *Wimhihilo, double *Wimlohilo, double *Wimhilolo, double *Wimlololo,
   double *Yrehihihi, double *Yrelohihi, double *Yrehilohi, double *Yrelolohi,
   double *Yrehihilo, double *Yrelohilo, double *Yrehilolo, double *Yrelololo,
   double *Yimhihihi, double *Yimlohihi, double *Yimhilohi, double *Yimlolohi,
   double *Yimhihilo, double *Yimlohilo, double *Yimhilolo, double *Yimlololo,
   double *WYTrehihihi, double *WYTrelohihi,
   double *WYTrehilohi, double *WYTrelolohi,
   double *WYTrehihilo, double *WYTrelohilo,
   double *WYTrehilolo, double *WYTrelololo,
   double *WYTimhihihi, double *WYTimlohihi,
   double *WYTimhilohi, double *WYTimlolohi,
   double *WYTimhihilo, double *WYTimlohilo,
   double *WYTimhilolo, double *WYTimlololo )
{
}

__global__ void dbl8_small_QWYT
 ( int dim, int rowdim, int szt, int coloff,
   double *Qhihihi, double *Qlohihi, double *Qhilohi, double *Qlolohi,
   double *Qhihilo, double *Qlohilo, double *Qhilolo, double *Qlololo,
   double *WYThihihi, double *WYTlohihi, double *WYThilohi, double *WYTlolohi,
   double *WYThihilo, double *WYTlohilo, double *WYThilolo, double *WYTlololo,
   double *QWYThihihi, double *QWYTlohihi,
   double *QWYThilohi, double *QWYTlolohi,
   double *QWYThihilo, double *QWYTlohilo,
   double *QWYThilolo, double *QWYTlololo )
{
}

__global__ void cmplx8_small_QWYH
 ( int dim, int rowdim, int szt, int coloff,
   double *Qrehihihi, double *Qrelohihi, double *Qrehilohi, double *Qrelolohi,
   double *Qrehihilo, double *Qrelohilo, double *Qrehilolo, double *Qrelololo,
   double *Qimhihihi, double *Qimlohihi, double *Qimhilohi, double *Qimlolohi,
   double *Qimhihilo, double *Qimlohilo, double *Qimhilolo, double *Qimlololo,
   double *WYTrehihihi, double *WYTrelohihi,
   double *WYTrehilohi, double *WYTrelolohi,
   double *WYTrehihilo, double *WYTrelohilo,
   double *WYTrehilolo, double *WYTrelololo,
   double *WYTimhihihi, double *WYTimlohihi,
   double *WYTimhilohi, double *WYTimlolohi,
   double *WYTimhihilo, double *WYTimlohilo,
   double *WYTimhilolo, double *WYTimlololo,
   double *QWYTrehihihi, double *QWYTrelohihi,
   double *QWYTrehilohi, double *QWYTrelolohi,
   double *QWYTrehihilo, double *QWYTrelohilo,
   double *QWYTrehilolo, double *QWYTrelololo,
   double *QWYTimhihihi, double *QWYTimlohihi,
   double *QWYTimhilohi, double *QWYTimlolohi,
   double *QWYTimhihilo, double *QWYTimlohilo,
   double *QWYTimhilolo, double *QWYTimlololo )
{
}

__global__ void dbl8_small_YWTC
 ( int nrows, int ncols, int rowdim, int coldim, int szt,
   int rowoff, int coloff,
   double *YWThihihi, double *YWTlohihi, double *YWThilohi, double *YWTlolohi,
   double *YWThihilo, double *YWTlohilo, double *YWThilolo, double *YWTlololo,
   double *Chihihi, double *Clohihi, double *Chilohi, double *Clolohi,
   double *Chihilo, double *Clohilo, double *Chilolo, double *Clololo,
   double *YWTChihihi, double *YWTClohihi,
   double *YWTChilohi, double *YWTClolohi,
   double *YWTChihilo, double *YWTClohilo,
   double *YWTChilolo, double *YWTClololo )
{
}

__global__ void cmplx8_small_YWHC
 ( int nrows, int ncols, int rowdim, int coldim, int szt,
   int rowoff, int coloff,
   double *YWTrehihihi, double *YWTrelohihi,
   double *YWTrehilohi, double *YWTrelolohi,
   double *YWTrehihilo, double *YWTrelohilo,
   double *YWTrehilolo, double *YWTrelololo,
   double *YWTimhihihi, double *YWTimlohihi,
   double *YWTimhilohi, double *YWTimlolohi,
   double *YWTimhihilo, double *YWTimlohilo,
   double *YWTimhilolo, double *YWTimlololo,
   double *Crehihihi, double *Crelohihi, double *Crehilohi, double *Crelolohi,
   double *Crehihilo, double *Crelohilo, double *Crehilolo, double *Crelololo,
   double *Cimhihihi, double *Cimlohihi, double *Cimhilohi, double *Cimlolohi,
   double *Cimhihilo, double *Cimlohilo, double *Cimhilolo, double *Cimlololo,
   double *YWTCrehihihi, double *YWTCrelohihi,
   double *YWTCrehilohi, double *YWTCrelolohi,
   double *YWTCrehihilo, double *YWTCrelohilo,
   double *YWTCrehilolo, double *YWTCrelololo,
   double *YWTCimhihihi, double *YWTCimlohihi,
   double *YWTCimhilohi, double *YWTCimlolohi,
   double *YWTCimhihilo, double *YWTCimlohilo,
   double *YWTCimhilolo, double *YWTCimlololo )
{
}

__global__ void dbl8_small_Qupdate
 ( int dim, int rowdim, int szt, int coloff,
   double *Qhihihi, double *Qlohihi, double *Qhilohi, double *Qlolohi,
   double *Qhihilo, double *Qlohilo, double *Qhilolo, double *Qlololo,
   double *QWYThihihi, double *QWYTlohihi,
   double *QWYThilohi, double *QWYTlolohi,
   double *QWYThihilo, double *QWYTlohilo,
   double *QWYThilolo, double *QWYTlololo )
{
}

__global__ void cmplx8_small_Qupdate
 ( int dim, int rowdim, int szt, int coloff,
   double *Qrehihihi, double *Qrelohihi, double *Qrehilohi, double *Qrelolohi,
   double *Qrehihilo, double *Qrelohilo, double *Qrehilolo, double *Qrelololo,
   double *Qimhihihi, double *Qimlohihi, double *Qimhilohi, double *Qimlolohi,
   double *Qimhihilo, double *Qimlohilo, double *Qimhilolo, double *Qimlololo,
   double *QWYTrehihihi, double *QWYTrelohihi,
   double *QWYTrehilohi, double *QWYTrelolohi,
   double *QWYTrehihilo, double *QWYTrelohilo,
   double *QWYTrehilolo, double *QWYTrelololo,
   double *QWYTimhihihi, double *QWYTimlohihi,
   double *QWYTimhilohi, double *QWYTimlolohi,
   double *QWYTimhihilo, double *QWYTimlohilo,
   double *QWYTimhilolo, double *QWYTimlololo )
{
}

__global__ void dbl8_small_R_add_YWTC
 ( int nrows, int coldim, int szt, int rowoff, int coloff,
   double *Rhihihi, double *Rlohihi, double *Rhilohi, double *Rlolohi,
   double *Rhihilo, double *Rlohilo, double *Rhilolo, double *Rlololo,
   double *YWTChihihi, double *YWTClohihi,
   double *YWTChilohi, double *YWTClolohi,
   double *YWTChihilo, double *YWTClohilo,
   double *YWTChilolo, double *YWTClololo )
{
}

__global__ void cmplx8_small_R_add_YWHC
 ( int nrows, int coldim, int szt, int rowoff, int coloff,
   double *Rrehihihi, double *Rrelohihi, double *Rrehilohi, double *Rrelolohi,
   double *Rrehihilo, double *Rrelohilo, double *Rrehilolo, double *Rrelololo,
   double *Rimhihihi, double *Rimlohihi, double *Rimhilohi, double *Rimlolohi,
   double *Rimhihilo, double *Rimlohilo, double *Rimhilolo, double *Rimlololo,
   double *YWTCrehihihi, double *YWTCrelohihi,
   double *YWTCrehilohi, double *YWTCrelolohi,
   double *YWTCrehihilo, double *YWTCrelohilo,
   double *YWTCrehilolo, double *YWTCrelololo,
   double *YWTCimhihihi, double *YWTCimlohihi,
   double *YWTCimhilohi, double *YWTCimlolohi,
   double *YWTCimhihilo, double *YWTCimlohilo,
   double *YWTCimhilolo, double *YWTCimlololo )
{
}

void GPU_dbl8_small_house
 ( int nrows, int ncols, int szt, int nbt,
   int colidx, int nrows1, int k, int L,
   double *Ahihihi_h, double *Alohihi_h, double *Ahilohi_h, double *Alolohi_h,
   double *Ahihilo_h, double *Alohilo_h, double *Ahilolo_h, double *Alololo_h,
   double *Ahihihi_d, double *Alohihi_d, double *Ahilohi_d, double *Alolohi_d,
   double *Ahihilo_d, double *Alohilo_d, double *Ahilolo_d, double *Alololo_d,
   double *vhihihi_h, double *vlohihi_h, double *vhilohi_h, double *vlolohi_h,
   double *vhihilo_h, double *vlohilo_h, double *vhilolo_h, double *vlololo_h,
   double *Vhihihi_d, double *Vlohihi_d, double *Vhilohi_d, double *Vlolohi_d,
   double *Vhihilo_d, double *Vlohilo_d, double *Vhilolo_d, double *Vlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *lapms, long long int *add, long long int *mul, long long int *div,
   long long int *sqrtfun, bool verbose )
{
}

void GPU_cmplx8_small_house
 ( int nrows, int ncols, int szt, int nbt,
   int colidx, int nrows1, int k, int L,
   double *Arehihihi_h, double *Arelohihi_h,
   double *Arehilohi_h, double *Arelolohi_h,
   double *Arehihilo_h, double *Arelohilo_h,
   double *Arehilolo_h, double *Arelololo_h,
   double *Aimhihihi_h, double *Aimlohihi_h,
   double *Aimhilohi_h, double *Aimlolohi_h,
   double *Aimhihilo_h, double *Aimlohilo_h,
   double *Aimhilolo_h, double *Aimlololo_h,
   double *Arehihihi_d, double *Arelohihi_d,
   double *Arehilohi_d, double *Arelolohi_d,
   double *Arehihilo_d, double *Arelohilo_d,
   double *Arehilolo_d, double *Arelololo_d,
   double *Aimhihihi_d, double *Aimlohihi_d,
   double *Aimhilohi_d, double *Aimlolohi_d,
   double *Aimhihilo_d, double *Aimlohilo_d,
   double *Aimhilolo_d, double *Aimlololo_d,
   double *vrehihihi_h, double *vrelohihi_h,
   double *vrehilohi_h, double *vrelolohi_h,
   double *vrehihilo_h, double *vrelohilo_h,
   double *vrehilolo_h, double *vrelololo_h,
   double *vimhihihi_h, double *vimlohihi_h,
   double *vimhilohi_h, double *vimlolohi_h,
   double *vimhihilo_h, double *vimlohilo_h,
   double *vimhilolo_h, double *vimlololo_h,
   double *Vrehihihi_d, double *Vrelohihi_d,
   double *Vrehilohi_d, double *Vrelolohi_d,
   double *Vrehihilo_d, double *Vrelohilo_d,
   double *Vrehilolo_d, double *Vrelololo_d,
   double *Vimhihihi_d, double *Vimlohihi_d,
   double *Vimhilohi_d, double *Vimlolohi_d,
   double *Vimhihilo_d, double *Vimlohilo_d,
   double *Vimhilolo_d, double *Vimlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *lapms, long long int *add, long long int *mul, long long int *div,
   long long int *sqrtfun, bool verbose )
{
}

void GPU_dbl8_large_house
 ( int nrows, int ncols, int szt, int nbt,
   int colidx, int nrows1, int k, int L,
   double *Ahihihi_h, double *Alohihi_h, double *Ahilohi_h, double *Alolohi_h,
   double *Ahihilo_h, double *Alohilo_h, double *Ahilolo_h, double *Alololo_h,
   double *Ahihihi_d, double *Alohihi_d, double *Ahilohi_d, double *Alolohi_d,
   double *Ahihilo_d, double *Alohilo_d, double *Ahilolo_d, double *Alololo_d,
   double *vhihihi_h, double *vlohihi_h, double *vhilohi_h, double *vlolohi_h,
   double *vhihilo_h, double *vlohilo_h, double *vhilolo_h, double *vlololo_h,
   double *Vhihihi_d, double *Vlohihi_d, double *Vhilohi_d, double *Vlolohi_d,
   double *Vhihilo_d, double *Vlohilo_d, double *Vhilolo_d, double *Vlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *sumshihihi_h, double *sumslohihi_h,
   double *sumshilohi_h, double *sumslolohi_h,
   double *sumshihilo_h, double *sumslohilo_h,
   double *sumshilolo_h, double *sumslololo_h,
   double *sumshihihi_d, double *sumslohihi_d,
   double *sumshilohi_d, double *sumslolohi_d,
   double *sumshihilo_d, double *sumslohilo_d,
   double *sumshilolo_d, double *sumslololo_d,
   double *sigmahihihi_h, double *sigmalohihi_h, 
   double *sigmahilohi_h, double *sigmalolohi_h, 
   double *sigmahihilo_h, double *sigmalohilo_h, 
   double *sigmahilolo_h, double *sigmalololo_h, 
   double *sigmahihihi_d, double *sigmalohihi_d,
   double *sigmahilohi_d, double *sigmalolohi_d,
   double *sigmahihilo_d, double *sigmalohilo_d,
   double *sigmahilolo_d, double *sigmalololo_d,
   double *lapms, long long int *add, long long int *mul, long long int *div,
   long long int *sqrtfun, bool verbose )
{
}

void GPU_cmplx8_large_house
 ( int nrows, int ncols, int szt, int nbt,
   int colidx, int nrows1, int k, int L,
   double *Arehihihi_h, double *Arelohihi_h,
   double *Arehilohi_h, double *Arelolohi_h,
   double *Arehihilo_h, double *Arelohilo_h,
   double *Arehilolo_h, double *Arelololo_h,
   double *Aimhihihi_h, double *Aimlohihi_h,
   double *Aimhilohi_h, double *Aimlolohi_h,
   double *Aimhihilo_h, double *Aimlohilo_h,
   double *Aimhilolo_h, double *Aimlololo_h,
   double *Arehihihi_d, double *Arelohihi_d,
   double *Arehilohi_d, double *Arelolohi_d,
   double *Arehihilo_d, double *Arelohilo_d,
   double *Arehilolo_d, double *Arelololo_d,
   double *Aimhihihi_d, double *Aimlohihi_d,
   double *Aimhilohi_d, double *Aimlolohi_d,
   double *Aimhihilo_d, double *Aimlohilo_d,
   double *Aimhilolo_d, double *Aimlololo_d,
   double *vrehihihi_h, double *vrelohihi_h,
   double *vrehilohi_h, double *vrelolohi_h,
   double *vrehihilo_h, double *vrelohilo_h,
   double *vrehilolo_h, double *vrelololo_h,
   double *vimhihihi_h, double *vimlohihi_h,
   double *vimhilohi_h, double *vimlolohi_h,
   double *vimhihilo_h, double *vimlohilo_h,
   double *vimhilolo_h, double *vimlololo_h,
   double *Vrehihihi_d, double *Vrelohihi_d,
   double *Vrehilohi_d, double *Vrelolohi_d,
   double *Vrehihilo_d, double *Vrelohilo_d,
   double *Vrehilolo_d, double *Vrelololo_d,
   double *Vimhihihi_d, double *Vimlohihi_d,
   double *Vimhilohi_d, double *Vimlolohi_d,
   double *Vimhihilo_d, double *Vimlohilo_d,
   double *Vimhilolo_d, double *Vimlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *sumshihihi_h, double *sumslohihi_h,
   double *sumshilohi_h, double *sumslolohi_h,
   double *sumshihilo_h, double *sumslohilo_h,
   double *sumshilolo_h, double *sumslololo_h,
   double *sumshihihi_d, double *sumslohihi_d,
   double *sumshilohi_d, double *sumslolohi_d,
   double *sumshihilo_d, double *sumslohilo_d,
   double *sumshilolo_d, double *sumslololo_d,
   double *sigmahihihi_h, double *sigmalohihi_h,
   double *sigmahilohi_h, double *sigmalolohi_h,
   double *sigmahihilo_h, double *sigmalohilo_h,
   double *sigmahilolo_h, double *sigmalololo_h,
   double *sigmahihihi_d, double *sigmalohihi_d,
   double *sigmahilohi_d, double *sigmalolohi_d,
   double *sigmahihilo_d, double *sigmalohilo_d,
   double *sigmahilolo_d, double *sigmalololo_d,
   double *lapms, long long int *add, long long int *mul, long long int *div,
   long long int *sqrtfun, bool verbose )
{
}

void GPU_dbl8_small_leftRupdate
 ( int nrows, int ncols, int szt, int colidx, int k, int L,
   double *Ahihihi_h, double *Alohihi_h, double *Ahilohi_h, double *Alolohi_h,
   double *Ahihilo_h, double *Alohilo_h, double *Ahilolo_h, double *Alololo_h,
   double *Ahihihi_d, double *Alohihi_d, double *Ahilohi_d, double *Alolohi_d,
   double *Ahihilo_d, double *Alohilo_d, double *Ahilolo_d, double *Alololo_d,
   double *Vhihihi_d, double *Vlohihi_d, double *Vhilohi_d, double *Vlolohi_d,
   double *Vhihilo_d, double *Vlohilo_d, double *Vhilolo_d, double *Vlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_cmplx8_small_leftRupdate
 ( int nrows, int ncols, int szt, int colidx, int k, int L,
   double *Arehihihi_h, double *Arelohihi_h,
   double *Arehilohi_h, double *Arelolohi_h,
   double *Arehihilo_h, double *Arelohilo_h,
   double *Arehilolo_h, double *Arelololo_h,
   double *Aimhihihi_h, double *Aimlohihi_h,
   double *Aimhilohi_h, double *Aimlolohi_h,
   double *Aimhihilo_h, double *Aimlohilo_h,
   double *Aimhilolo_h, double *Aimlololo_h,
   double *Arehihihi_d, double *Arelohihi_d,
   double *Arehilohi_d, double *Arelolohi_d,
   double *Arehihilo_d, double *Arelohilo_d,
   double *Arehilolo_d, double *Arelololo_d,
   double *Aimhihihi_d, double *Aimlohihi_d,
   double *Aimhilohi_d, double *Aimlolohi_d,
   double *Aimhihilo_d, double *Aimlohilo_d,
   double *Aimhilolo_d, double *Aimlololo_d,
   double *Vrehihihi_d, double *Vrelohihi_d,
   double *Vrehilohi_d, double *Vrelolohi_d,
   double *Vrehihilo_d, double *Vrelohilo_d,
   double *Vrehilolo_d, double *Vrelololo_d,
   double *Vimhihihi_d, double *Vimlohihi_d,
   double *Vimhilohi_d, double *Vimlolohi_d,
   double *Vimhihilo_d, double *Vimlohilo_d,
   double *Vimhilolo_d, double *Vimlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_dbl8_medium_leftRupdate
 ( int nrows, int ncols, int szt, int colidx, int k, int L,
   double *Ahihihi_h, double *Alohihi_h, double *Ahilohi_h, double *Alolohi_h,
   double *Ahihilo_h, double *Alohilo_h, double *Ahilolo_h, double *Alololo_h,
   double *Ahihihi_d, double *Alohihi_d, double *Ahilohi_d, double *Alolohi_d,
   double *Ahihilo_d, double *Alohilo_d, double *Ahilolo_d, double *Alololo_d,
   double *Vhihihi_d, double *Vlohihi_d, double *Vhilohi_d, double *Vlolohi_d,
   double *Vhihilo_d, double *Vlohilo_d, double *Vhilolo_d, double *Vlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *RTdotvhihihi_h, double *RTdotvlohihi_h,
   double *RTdotvhilohi_h, double *RTdotvlolohi_h,
   double *RTdotvhihilo_h, double *RTdotvlohilo_h,
   double *RTdotvhilolo_h, double *RTdotvlololo_h,
   double *RTdotvhihihi_d, double *RTdotvlohihi_d,
   double *RTdotvhilohi_d, double *RTdotvlolohi_d,
   double *RTdotvhihilo_d, double *RTdotvlohilo_d,
   double *RTdotvhilolo_d, double *RTdotvlololo_d,
   double *whihihi_h, double *wlohihi_h, double *whilohi_h, double *wlolohi_h,
   double *whihilo_h, double *wlohilo_h, double *whilolo_h, double *wlololo_h,
   double *whihihi_d, double *wlohihi_d, double *whilohi_d, double *wlolohi_d,
   double *whihilo_d, double *wlohilo_d, double *whilolo_d, double *wlololo_d,
   double *RTvlapms, double *redlapms,
   long long int *add, long long int *mul, bool verbose )
{
}

void GPU_cmplx8_medium_leftRupdate
 ( int nrows, int ncols, int szt, int colidx, int k, int L,
   double *Arehihihi_h, double *Arelohihi_h,
   double *Arehilohi_h, double *Arelolohi_h,
   double *Arehihilo_h, double *Arelohilo_h,
   double *Arehilolo_h, double *Arelololo_h,
   double *Aimhihihi_h, double *Aimlohihi_h,
   double *Aimhilohi_h, double *Aimlolohi_h,
   double *Aimhihilo_h, double *Aimlohilo_h,
   double *Aimhilolo_h, double *Aimlololo_h,
   double *Arehihihi_d, double *Arelohihi_d,
   double *Arehilohi_d, double *Arelolohi_d,
   double *Arehihilo_d, double *Arelohilo_d,
   double *Arehilolo_d, double *Arelololo_d,
   double *Aimhihihi_d, double *Aimlohihi_d,
   double *Aimhilohi_d, double *Aimlolohi_d,
   double *Aimhihilo_d, double *Aimlohilo_d,
   double *Aimhilolo_d, double *Aimlololo_d,
   double *Vrehihihi_d, double *Vrelohihi_d,
   double *Vrehilohi_d, double *Vrelolohi_d,
   double *Vrehihilo_d, double *Vrelohilo_d,
   double *Vrehilolo_d, double *Vrelololo_d,
   double *Vimhihihi_d, double *Vimlohihi_d,
   double *Vimhilohi_d, double *Vimlolohi_d,
   double *Vimhihilo_d, double *Vimlohilo_d,
   double *Vimhilolo_d, double *Vimlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *RHdotvrehihihi_h, double *RHdotvrelohihi_h,
   double *RHdotvrehilohi_h, double *RHdotvrelolohi_h,
   double *RHdotvrehihilo_h, double *RHdotvrelohilo_h,
   double *RHdotvrehilolo_h, double *RHdotvrelololo_h,
   double *RHdotvimhihihi_h, double *RHdotvimlohihi_h,
   double *RHdotvimhilohi_h, double *RHdotvimlolohi_h,
   double *RHdotvimhihilo_h, double *RHdotvimlohilo_h,
   double *RHdotvimhilolo_h, double *RHdotvimlololo_h,
   double *RHdotvrehihihi_d, double *RHdotvrelohihi_d,
   double *RHdotvrehilohi_d, double *RHdotvrelolohi_d,
   double *RHdotvrehihilo_d, double *RHdotvrelohilo_d,
   double *RHdotvrehilolo_d, double *RHdotvrelololo_d,
   double *RHdotvimhihihi_d, double *RHdotvimlohihi_d,
   double *RHdotvimhilohi_d, double *RHdotvimlolohi_d,
   double *RHdotvimhihilo_d, double *RHdotvimlohilo_d,
   double *RHdotvimhilolo_d, double *RHdotvimlololo_d,
   double *wrehihihi_h, double *wrelohihi_h,
   double *wrehilohi_h, double *wrelolohi_h,
   double *wrehihilo_h, double *wrelohilo_h,
   double *wrehilolo_h, double *wrelololo_h,
   double *wimhihihi_h, double *wimlohihi_h,
   double *wimhilohi_h, double *wimlolohi_h,
   double *wimhihilo_h, double *wimlohilo_h,
   double *wimhilolo_h, double *wimlololo_h,
   double *wrehihihi_d, double *wrelohihi_d,
   double *wrehilohi_d, double *wrelolohi_d,
   double *wrehihilo_d, double *wrelohilo_d,
   double *wrehilolo_d, double *wrelololo_d,
   double *wimhihihi_d, double *wimlohihi_d,
   double *wimhilohi_d, double *wimlolohi_d,
   double *wimhihilo_d, double *wimlohilo_d,
   double *wimhilolo_d, double *wimlololo_d,
   double *RHvlapms, double *redlapms,
   long long int *add, long long int *mul, bool verbose )
{
}

void GPU_dbl8_medium_VB_to_W
 ( int nrows, int ncols, int szt, int idx,
   double *Vhihihi_h, double *Vlohihi_h, double *Vhilohi_h, double *Vlolohi_h,
   double *Vhihilo_h, double *Vlohilo_h, double *Vhilolo_h, double *Vlololo_h,
   double *Vhihihi_d, double *Vlohihi_d, double *Vhilohi_d, double *Vlolohi_d,
   double *Vhihilo_d, double *Vlohilo_d, double *Vhilolo_d, double *Vlololo_d,
   double *Whihihi_h, double *Wlohihi_h, double *Whilohi_h, double *Wlolohi_h,
   double *Whihilo_h, double *Wlohilo_h, double *Whilolo_h, double *Wlololo_h,
   double *Whihihi_d, double *Wlohihi_d, double *Whilohi_d, double *Wlolohi_d,
   double *Whihilo_d, double *Wlohilo_d, double *Whilolo_d, double *Wlololo_d,
   double *WYThihihi_h, double *WYTlohihi_h,
   double *WYThilohi_h, double *WYTlolohi_h,
   double *WYThihilo_h, double *WYTlohilo_h,
   double *WYThilolo_h, double *WYTlololo_h,
   double *WYThihihi_d, double *WYTlohihi_d,
   double *WYThilohi_d, double *WYTlolohi_d,
   double *WYThihilo_d, double *WYTlohilo_d,
   double *WYThilolo_d, double *WYTlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_cmplx8_medium_VB_to_W
 ( int nrows, int ncols, int szt, int idx,
   double *Vrehihihi_h, double *Vrelohihi_h,
   double *Vrehilohi_h, double *Vrelolohi_h,
   double *Vrehihilo_h, double *Vrelohilo_h,
   double *Vrehilolo_h, double *Vrelololo_h,
   double *Vimhihihi_h, double *Vimlohihi_h,
   double *Vimhilohi_h, double *Vimlolohi_h,
   double *Vimhihilo_h, double *Vimlohilo_h,
   double *Vimhilolo_h, double *Vimlololo_h,
   double *Vrehihihi_d, double *Vrelohihi_d,
   double *Vrehilohi_d, double *Vrelolohi_d,
   double *Vrehihilo_d, double *Vrelohilo_d, 
   double *Vrehilolo_d, double *Vrelololo_d,
   double *Vimhihihi_d, double *Vimlohihi_d,
   double *Vimhilohi_d, double *Vimlolohi_d,
   double *Vimhihilo_d, double *Vimlohilo_d,
   double *Vimhilolo_d, double *Vimlololo_d,
   double *Wrehihihi_h, double *Wrelohihi_h,
   double *Wrehilohi_h, double *Wrelolohi_h,
   double *Wrehihilo_h, double *Wrelohilo_h,
   double *Wrehilolo_h, double *Wrelololo_h,
   double *Wimhihihi_h, double *Wimlohihi_h,
   double *Wimhilohi_h, double *Wimlolohi_h,
   double *Wimhihilo_h, double *Wimlohilo_h,
   double *Wimhilolo_h, double *Wimlololo_h,
   double *Wrehihihi_d, double *Wrelohihi_d,
   double *Wrehilohi_d, double *Wrelolohi_d,
   double *Wrehihilo_d, double *Wrelohilo_d,
   double *Wrehilolo_d, double *Wrelololo_d,
   double *Wimhihihi_d, double *Wimlohihi_d,
   double *Wimhilohi_d, double *Wimlolohi_d,
   double *Wimhihilo_d, double *Wimlohilo_d,
   double *Wimhilolo_d, double *Wimlololo_d,
   double *WYHrehihihi_h, double *WYHrelohihi_h,
   double *WYHrehilohi_h, double *WYHrelolohi_h,
   double *WYHrehihilo_h, double *WYHrelohilo_h,
   double *WYHrehilolo_h, double *WYHrelololo_h,
   double *WYHimhihihi_h, double *WYHimlohihi_h,
   double *WYHimhilohi_h, double *WYHimlolohi_h,
   double *WYHimhihilo_h, double *WYHimlohilo_h,
   double *WYHimhilolo_h, double *WYHimlololo_h,
   double *WYHrehihihi_d, double *WYHrelohihi_d,
   double *WYHrehilohi_d, double *WYHrelolohi_d,
   double *WYHrehihilo_d, double *WYHrelohilo_d,
   double *WYHrehilolo_d, double *WYHrelololo_d,
   double *WYHimhihihi_d, double *WYHimlohihi_d,
   double *WYHimhilohi_d, double *WYHimlolohi_d,
   double *WYHimhihilo_d, double *WYHimlohilo_d,
   double *WYHimhilolo_d, double *WYHimlololo_d,
   double *betahihihi_h, double *betalohihi_h,
   double *betahilohi_h, double *betalolohi_h,
   double *betahihilo_h, double *betalohilo_h,
   double *betahilolo_h, double *betalololo_h,
   double *betahihihi_d, double *betalohihi_d,
   double *betahilohi_d, double *betalolohi_d,
   double *betahihilo_d, double *betalohilo_d,
   double *betahilolo_d, double *betalololo_d,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_dbl8_small_WYT
 ( int nrows, int szt,
   double *Whihihi_d, double *Wlohihi_d,
   double *Whilohi_d, double *Wlolohi_d,
   double *Whihilo_d, double *Wlohilo_d,
   double *Whilolo_d, double *Wlololo_d,
   double *Yhihihi_d, double *Ylohihi_d,
   double *Yhilohi_d, double *Ylolohi_d,
   double *Yhihilo_d, double *Ylohilo_d,
   double *Yhilolo_d, double *Ylololo_d,
   double *WYThihihi_d, double *WYTlohihi_d,
   double *WYThilohi_d, double *WYTlolohi_d,
   double *WYThihilo_d, double *WYTlohilo_d,
   double *WYThilolo_d, double *WYTlololo_d,
   double *WYThihihi_h, double *WYTlohihi_h,
   double *WYThilohi_h, double *WYTlolohi_h,
   double *WYThihilo_h, double *WYTlohilo_h,
   double *WYThilolo_h, double *WYTlololo_h,
   double *lapms, bool verbose )
{
}

void GPU_cmplx8_small_WYH
 ( int nrows, int szt,
   double *Wrehihihi_d, double *Wrelohihi_d,
   double *Wrehilohi_d, double *Wrelolohi_d,
   double *Wrehihilo_d, double *Wrelohilo_d,
   double *Wrehilolo_d, double *Wrelololo_d,
   double *Wimhihihi_d, double *Wimlohihi_d,
   double *Wimhilohi_d, double *Wimlolohi_d,
   double *Wimhihilo_d, double *Wimlohilo_d,
   double *Wimhilolo_d, double *Wimlololo_d,
   double *Yrehihihi_d, double *Yrelohihi_d,
   double *Yrehilohi_d, double *Yrelolohi_d,
   double *Yrehihilo_d, double *Yrelohilo_d,
   double *Yrehilolo_d, double *Yrelololo_d,
   double *Yimhihihi_d, double *Yimlohihi_d,
   double *Yimhilohi_d, double *Yimlolohi_d,
   double *Yimhihilo_d, double *Yimlohilo_d,
   double *Yimhilolo_d, double *Yimlololo_d,
   double *WYTrehihihi_d, double *WYTrelohihi_d,
   double *WYTrehilohi_d, double *WYTrelolohi_d,
   double *WYTrehihilo_d, double *WYTrelohilo_d,
   double *WYTrehilolo_d, double *WYTrelololo_d,
   double *WYTimhihihi_d, double *WYTimlohihi_d,
   double *WYTimhilohi_d, double *WYTimlolohi_d,
   double *WYTimhihilo_d, double *WYTimlohilo_d,
   double *WYTimhilolo_d, double *WYTimlololo_d,
   double *WYTrehihihi_h, double *WYTrelohihi_h,
   double *WYTrehilohi_h, double *WYTrelolohi_h,
   double *WYTrehihilo_h, double *WYTrelohilo_h,
   double *WYTrehilolo_h, double *WYTrelololo_h,
   double *WYTimhihihi_h, double *WYTimlohihi_h,
   double *WYTimhilohi_h, double *WYTimlolohi_h,
   double *WYTimhihilo_h, double *WYTimlohilo_h,
   double *WYTimhilolo_h, double *WYTimlololo_h,
   double *lapms, bool verbose )
{
}

void GPU_dbl8_small_YWT
 ( int nrows, int szt, int idx,
   double *Yhihihi_d, double *Ylohihi_d, double *Yhilohi_d, double *Ylolohi_d,
   double *Yhihilo_d, double *Ylohilo_d, double *Yhilolo_d, double *Ylololo_d,
   double *Whihihi_d, double *Wlohihi_d, double *Whilohi_d, double *Wlolohi_d,
   double *Whihilo_d, double *Wlohilo_d, double *Whilolo_d, double *Wlololo_d,
   double *YWThihihi_d, double *YWTlohihi_d,
   double *YWThilohi_d, double *YWTlolohi_d,
   double *YWThihilo_d, double *YWTlohilo_d,
   double *YWThilolo_d, double *YWTlololo_d,
   double *YWThihihi_h, double *YWTlohihi_h,
   double *YWThilohi_h, double *YWTlolohi_h,
   double *YWThihilo_h, double *YWTlohilo_h,
   double *YWThilolo_h, double *YWTlololo_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_cmplx8_small_YWH
 ( int nrows, int szt, int idx,
   double *Yrehihihi_d, double *Yrelohihi_d,
   double *Yrehilohi_d, double *Yrelolohi_d,
   double *Yrehihilo_d, double *Yrelohilo_d,
   double *Yrehilolo_d, double *Yrelololo_d,
   double *Yimhihihi_d, double *Yimlohihi_d,
   double *Yimhilohi_d, double *Yimlolohi_d,
   double *Yimhihilo_d, double *Yimlohilo_d,
   double *Yimhilolo_d, double *Yimlololo_d,
   double *Wrehihihi_d, double *Wrelohihi_d,
   double *Wrehilohi_d, double *Wrelolohi_d,
   double *Wrehihilo_d, double *Wrelohilo_d,
   double *Wrehilolo_d, double *Wrelololo_d,
   double *Wimhihihi_d, double *Wimlohihi_d,
   double *Wimhilohi_d, double *Wimlolohi_d,
   double *Wimhihilo_d, double *Wimlohilo_d,
   double *Wimhilolo_d, double *Wimlololo_d,
   double *YWTrehihihi_d, double *YWTrelohihi_d,
   double *YWTrehilohi_d, double *YWTrelolohi_d,
   double *YWTrehihilo_d, double *YWTrelohilo_d,
   double *YWTrehilolo_d, double *YWTrelololo_d,
   double *YWTimhihihi_d, double *YWTimlohihi_d,
   double *YWTimhilohi_d, double *YWTimlolohi_d,
   double *YWTimhihilo_d, double *YWTimlohilo_d,
   double *YWTimhilolo_d, double *YWTimlololo_d,
   double *YWTrehihihi_h, double *YWTrelohihi_h,
   double *YWTrehilohi_h, double *YWTrelolohi_h,
   double *YWTrehihilo_h, double *YWTrelohilo_h,
   double *YWTrehilolo_h, double *YWTrelololo_h,
   double *YWTimhihihi_h, double *YWTimlohihi_h,
   double *YWTimhilohi_h, double *YWTimlolohi_h,
   double *YWTimhihilo_h, double *YWTimlohilo_h,
   double *YWTimhilolo_h, double *YWTimlololo_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_dbl8_small_QWYT
 ( int dim, int szt, int idx,
   double *Qhihihi_d, double *Qlohihi_d, double *Qhilohi_d, double *Qlolohi_d,
   double *Qhihilo_d, double *Qlohilo_d, double *Qhilolo_d, double *Qlololo_d,
   double *WYThihihi_d, double *WYTlohihi_d,
   double *WYThilohi_d, double *WYTlolohi_d,
   double *WYThihilo_d, double *WYTlohilo_d,
   double *WYThilolo_d, double *WYTlololo_d,
   double *QWYThihihi_d, double *QWYTlohihi_d,
   double *QWYThilohi_d, double *QWYTlolohi_d,
   double *QWYThihilo_d, double *QWYTlohilo_d,
   double *QWYThilolo_d, double *QWYTlololo_d,
   double *QWYThihihi_h, double *QWYTlohihi_h,
   double *QWYThilohi_h, double *QWYTlolohi_h,
   double *QWYThihilo_h, double *QWYTlohilo_h,
   double *QWYThilolo_h, double *QWYTlololo_h,
   double *Qhihihi_h, double *Qlohihi_h, double *Qhilohi_h, double *Qlolohi_h,
   double *Qhihilo_h, double *Qlohilo_h, double *Qhilolo_h, double *Qlololo_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_cmplx8_small_QWYH
 ( int dim, int szt, int idx,
   double *Qrehihihi_d, double *Qrelohihi_d,
   double *Qrehilohi_d, double *Qrelolohi_d,
   double *Qrehihilo_d, double *Qrelohilo_d,
   double *Qrehilolo_d, double *Qrelololo_d,
   double *Qimhihihi_d, double *Qimlohihi_d,
   double *Qimhilohi_d, double *Qimlolohi_d,
   double *Qimhihilo_d, double *Qimlohilo_d,
   double *Qimhilolo_d, double *Qimlololo_d,
   double *WYTrehihihi_d, double *WYTrelohihi_d,
   double *WYTrehilohi_d, double *WYTrelolohi_d,
   double *WYTrehihilo_d, double *WYTrelohilo_d,
   double *WYTrehilolo_d, double *WYTrelololo_d,
   double *WYTimhihihi_d, double *WYTimlohihi_d,
   double *WYTimhilohi_d, double *WYTimlolohi_d,
   double *WYTimhihilo_d, double *WYTimlohilo_d,
   double *WYTimhilolo_d, double *WYTimlololo_d,
   double *QWYTrehihihi_d, double *QWYTrelohihi_d,
   double *QWYTrehilohi_d, double *QWYTrelolohi_d,
   double *QWYTrehihilo_d, double *QWYTrelohilo_d,
   double *QWYTrehilolo_d, double *QWYTrelololo_d,
   double *QWYTimhihihi_d, double *QWYTimlohihi_d,
   double *QWYTimhilohi_d, double *QWYTimlolohi_d,
   double *QWYTimhihilo_d, double *QWYTimlohilo_d,
   double *QWYTimhilolo_d, double *QWYTimlololo_d,
   double *QWYTrehihihi_h, double *QWYTrelohihi_h,
   double *QWYTrehilohi_h, double *QWYTrelolohi_h,
   double *QWYTrehihilo_h, double *QWYTrelohilo_h,
   double *QWYTrehilolo_h, double *QWYTrelololo_h,
   double *QWYTimhihihi_h, double *QWYTimlohihi_h,
   double *QWYTimhilohi_h, double *QWYTimlolohi_h,
   double *QWYTimhihilo_h, double *QWYTimlohilo_h,
   double *QWYTimhilolo_h, double *QWYTimlololo_h,
   double *Qrehihihi_h, double *Qrelohihi_h,
   double *Qrehilohi_h, double *Qrelolohi_h,
   double *Qrehihilo_h, double *Qrelohilo_h,
   double *Qrehilolo_h, double *Qrelololo_h,
   double *Qimhihihi_h, double *Qimlohihi_h,
   double *Qimhilohi_h, double *Qimlolohi_h,
   double *Qimhihilo_h, double *Qimlohilo_h,
   double *Qimhilolo_h, double *Qimlololo_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_dbl8_small_YWTC
 ( int nrows, int ncols, int szt, int idx,
   double *YWThihihi_d, double *YWTlohihi_d,
   double *YWThilohi_d, double *YWTlolohi_d,
   double *YWThihilo_d, double *YWTlohilo_d,
   double *YWThilolo_d, double *YWTlololo_d,
   double *Chihihi_d, double *Clohihi_d, double *Chilohi_d, double *Clolohi_d,
   double *Chihilo_d, double *Clohilo_d, double *Chilolo_d, double *Clololo_d,
   double *YWTChihihi_d, double *YWTClohihi_d,
   double *YWTChilohi_d, double *YWTClolohi_d,
   double *YWTChihilo_d, double *YWTClohilo_d,
   double *YWTChilolo_d, double *YWTClololo_d,
   double *YWTChihihi_h, double *YWTClohihi_h,
   double *YWTChilohi_h, double *YWTClolohi_h,
   double *YWTChihilo_h, double *YWTClohilo_h,
   double *YWTChilolo_h, double *YWTClololo_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_cmplx8_small_YWHC
 ( int nrows, int ncols, int szt, int idx,
   double *YWTrehihihi_d, double *YWTrelohihi_d,
   double *YWTrehilohi_d, double *YWTrelolohi_d,
   double *YWTrehihilo_d, double *YWTrelohilo_d,
   double *YWTrehilolo_d, double *YWTrelololo_d,
   double *YWTimhihihi_d, double *YWTimlohihi_d,
   double *YWTimhilohi_d, double *YWTimlolohi_d,
   double *YWTimhihilo_d, double *YWTimlohilo_d,
   double *YWTimhilolo_d, double *YWTimlololo_d,
   double *Crehihihi_d, double *Crelohihi_d,
   double *Crehilohi_d, double *Crelolohi_d,
   double *Crehihilo_d, double *Crelohilo_d,
   double *Crehilolo_d, double *Crelololo_d,
   double *Cimhihihi_d, double *Cimlohihi_d,
   double *Cimhilohi_d, double *Cimlolohi_d,
   double *Cimhihilo_d, double *Cimlohilo_d,
   double *Cimhilolo_d, double *Cimlololo_d,
   double *YWTCrehihihi_d, double *YWTCrelohihi_d,
   double *YWTCrehilohi_d, double *YWTCrelolohi_d,
   double *YWTCrehihilo_d, double *YWTCrelohilo_d,
   double *YWTCrehilolo_d, double *YWTCrelololo_d,
   double *YWTCimhihihi_d, double *YWTCimlohihi_d,
   double *YWTCimhilohi_d, double *YWTCimlolohi_d,
   double *YWTCimhihilo_d, double *YWTCimlohilo_d,
   double *YWTCimhilolo_d, double *YWTCimlololo_d,
   double *YWTCrehihihi_h, double *YWTCrelohihi_h,
   double *YWTCrehilohi_h, double *YWTCrelolohi_h,
   double *YWTCrehihilo_h, double *YWTCrelohilo_h,
   double *YWTCrehilolo_h, double *YWTCrelololo_h,
   double *YWTCimhihihi_h, double *YWTCimlohihi_h,
   double *YWTCimhilohi_h, double *YWTCimlolohi_h,
   double *YWTCimhihilo_h, double *YWTCimlohilo_h,
   double *YWTCimhilolo_h, double *YWTCimlololo_h,
   double *lapms, long long int *add, long long int *mul, bool verbose )
{
}

void GPU_dbl8_small_Qupdate
 ( int dim, int szt, int idx,
   double *Qhihihi_d, double *Qlohihi_d, double *Qhilohi_d, double *Qlolohi_d,
   double *Qhihilo_d, double *Qlohilo_d, double *Qhilolo_d, double *Qlololo_d,
   double *QWYThihihi_d, double *QWYTlohihi_d,
   double *QWYThilohi_d, double *QWYTlolohi_d,
   double *QWYThihilo_d, double *QWYTlohilo_d,
   double *QWYThilolo_d, double *QWYTlololo_d,
   double *Qhihihi_h, double *Qlohihi_h, double *Qhilohi_h, double *Qlolohi_h,
   double *Qhihilo_h, double *Qlohilo_h, double *Qhilolo_h, double *Qlololo_h,
   double *lapms, long long int *add, bool verbose )
{
}

void GPU_cmplx8_small_Qupdate
 ( int dim, int szt, int idx,
   double *Qrehihihi_d, double *Qrelohihi_d,
   double *Qrehilohi_d, double *Qrelolohi_d,
   double *Qrehihilo_d, double *Qrelohilo_d,
   double *Qrehilolo_d, double *Qrelololo_d,
   double *Qimhihihi_d, double *Qimlohihi_d,
   double *Qimhilohi_d, double *Qimlolohi_d,
   double *Qimhihilo_d, double *Qimlohilo_d,
   double *Qimhilolo_d, double *Qimlololo_d,
   double *QWYTrehihihi_d, double *QWYTrelohihi_d,
   double *QWYTrehilohi_d, double *QWYTrelolohi_d,
   double *QWYTrehihilo_d, double *QWYTrelohilo_d,
   double *QWYTrehilolo_d, double *QWYTrelololo_d,
   double *QWYTimhihihi_d, double *QWYTimlohihi_d,
   double *QWYTimhilohi_d, double *QWYTimlolohi_d,
   double *QWYTimhihilo_d, double *QWYTimlohilo_d,
   double *QWYTimhilolo_d, double *QWYTimlololo_d,
   double *Qrehihihi_h, double *Qrelohihi_h,
   double *Qrehilohi_h, double *Qrelolohi_h,
   double *Qrehihilo_h, double *Qrelohilo_h,
   double *Qrehilolo_h, double *Qrelololo_h,
   double *Qimhihihi_h, double *Qimlohihi_h,
   double *Qimhilohi_h, double *Qimlolohi_h,
   double *Qimhihilo_h, double *Qimlohilo_h,
   double *Qimhilolo_h, double *Qimlololo_h,
   double *lapms, long long int *add, bool verbose )
{
}

void GPU_dbl8_small_R_add_YWTC
 ( int nrows, int ncols, int szt, int idx,
   double *Rhihihi_d, double *Rlohihi_d, double *Rhilohi_d, double *Rlolohi_d,
   double *Rhihilo_d, double *Rlohilo_d, double *Rhilolo_d, double *Rlololo_d,
   double *YWTChihihi_d, double *YWTClohihi_d,
   double *YWTChilohi_d, double *YWTClolohi_d,
   double *YWTChihilo_d, double *YWTClohilo_d,
   double *YWTChilolo_d, double *YWTClololo_d,
   double *Rhihihi_h, double *Rlohihi_h, double *Rhilohi_h, double *Rlolohi_h,
   double *Rhihilo_h, double *Rlohilo_h, double *Rhilolo_h, double *Rlololo_h,
   double *lapms, long long int *add, bool verbose )
{
}

void GPU_cmplx8_small_R_add_YWHC
 ( int nrows, int ncols, int szt, int idx,
   double *Rrehihihi_d, double *Rrelohihi_d,
   double *Rrehilohi_d, double *Rrelolohi_d,
   double *Rrehihilo_d, double *Rrelohilo_d,
   double *Rrehilolo_d, double *Rrelololo_d,
   double *Rimhihihi_d, double *Rimlohihi_d,
   double *Rimhilohi_d, double *Rimlolohi_d,
   double *Rimhihilo_d, double *Rimlohilo_d,
   double *Rimhilolo_d, double *Rimlololo_d,
   double *YWTCrehihihi_d, double *YWTCrelohihi_d,
   double *YWTCrehilohi_d, double *YWTCrelolohi_d,
   double *YWTCrehihilo_d, double *YWTCrelohilo_d,
   double *YWTCrehilolo_d, double *YWTCrelololo_d,
   double *YWTCimhihihi_d, double *YWTCimlohihi_d,
   double *YWTCimhilohi_d, double *YWTCimlolohi_d,
   double *YWTCimhihilo_d, double *YWTCimlohilo_d,
   double *YWTCimhilolo_d, double *YWTCimlololo_d,
   double *Rrehihihi_h, double *Rrelohihi_h,
   double *Rrehilohi_h, double *Rrelolohi_h,
   double *Rrehihilo_h, double *Rrelohilo_h,
   double *Rrehilolo_h, double *Rrelololo_h,
   double *Rimhihihi_h, double *Rimlohihi_h,
   double *Rimhilohi_h, double *Rimlolohi_h,
   double *Rimhihilo_h, double *Rimlohilo_h,
   double *Rimhilolo_h, double *Rimlololo_h,
   double *lapms, long long int *add, bool verbose )
{
}

void GPU_dbl8_blocked_houseqr
 ( int nrows, int ncols, int szt, int nbt,
   double **Ahihihi, double **Alohihi, double **Ahilohi, double **Alolohi,
   double **Ahihilo, double **Alohilo, double **Ahilolo, double **Alololo,
   double **Qhihihi, double **Qlohihi, double **Qhilohi, double **Qlolohi,
   double **Qhihilo, double **Qlohilo, double **Qhilolo, double **Qlololo,
   double **Rhihihi, double **Rlohihi, double **Rhilohi, double **Rlolohi,
   double **Rhihilo, double **Rlohilo, double **Rhilolo, double **Rlololo,
   double *houselapms, double *RTvlapms, double *tileRlapms,
   double *vb2Wlapms, double *WYTlapms, double *QWYTlapms, double *Qaddlapms,
   double *YWTlapms, double *YWTClapms, double *Raddlapms,
   double *walltimesec, long long int *addcnt, long long int *mulcnt,
   long long int *divcnt, long long int *sqrtcnt, bool verbose )
{
   const int dim = nrows*ncols;          // total number of doubles
   const int nrows2 = nrows*nrows;
   double *Ahihihi_h = new double[dim];    // A on the host
   double *Alohihi_h = new double[dim]; 
   double *Ahilohi_h = new double[dim];
   double *Alolohi_h = new double[dim]; 
   double *Ahihilo_h = new double[dim]; 
   double *Alohilo_h = new double[dim]; 
   double *Ahilolo_h = new double[dim];
   double *Alololo_h = new double[dim]; 
   double *Ahihihi_d;                      // A on the device
   double *Alohihi_d; 
   double *Ahilohi_d; 
   double *Alolohi_d; 
   double *Ahihilo_d;
   double *Alohilo_d; 
   double *Ahilolo_d; 
   double *Alololo_d; 
   double *Qhihihi_h = new double[nrows2]; // Q on the host
   double *Qlohihi_h = new double[nrows2]; 
   double *Qhilohi_h = new double[nrows2]; 
   double *Qlolohi_h = new double[nrows2]; 
   double *Qhihilo_h = new double[nrows2];
   double *Qlohilo_h = new double[nrows2]; 
   double *Qhilolo_h = new double[nrows2]; 
   double *Qlololo_h = new double[nrows2]; 
   double *Qhihihi_d;                      // Q on the device
   double *Qlohihi_d;
   double *Qhilohi_d;
   double *Qlolohi_d;
   double *Qhihilo_d;                      // Q on the device
   double *Qlohilo_d;
   double *Qhilolo_d;
   double *Qlololo_d;
   double *vhihihi_h = new double[nrows];  // Householder vector
   double *vlohihi_h = new double[nrows];
   double *vhilohi_h = new double[nrows];
   double *vlolohi_h = new double[nrows];
   double *vhihilo_h = new double[nrows]; 
   double *vlohilo_h = new double[nrows];
   double *vhilolo_h = new double[nrows];
   double *vlololo_h = new double[nrows];
   double *betahihihi_h = new double[szt]; //  beta on the host
   double *betalohihi_h = new double[szt]; 
   double *betahilohi_h = new double[szt]; 
   double *betalolohi_h = new double[szt]; 
   double *betahihilo_h = new double[szt]; 
   double *betalohilo_h = new double[szt]; 
   double *betahilolo_h = new double[szt]; 
   double *betalololo_h = new double[szt]; 
   double *betahihihi_d;                   // beta on the device
   double *betalohihi_d;
   double *betahilohi_d;
   double *betalolohi_d;
   double *betahihilo_d;
   double *betalohilo_d;
   double *betahilolo_d;
   double *betalololo_d;
   double *Vhihihi_h = new double[nrows*szt]; // V matrix
   double *Vlohihi_h = new double[nrows*szt];
   double *Vhilohi_h = new double[nrows*szt];
   double *Vlolohi_h = new double[nrows*szt];
   double *Vhihilo_h = new double[nrows*szt];
   double *Vlohilo_h = new double[nrows*szt];
   double *Vhilolo_h = new double[nrows*szt];
   double *Vlololo_h = new double[nrows*szt];
   double *Vhihihi_d;                         // V on the device
   double *Vlohihi_d;
   double *Vhilohi_d;
   double *Vlolohi_d;
   double *Vhihilo_d;
   double *Vlohilo_d;
   double *Vhilolo_d;
   double *Vlololo_d;
   double *Whihihi_h = new double[nrows*szt]; // W on the host
   double *Wlohihi_h = new double[nrows*szt];
   double *Whilohi_h = new double[nrows*szt];
   double *Wlolohi_h = new double[nrows*szt];
   double *Whihilo_h = new double[nrows*szt];
   double *Wlohilo_h = new double[nrows*szt];
   double *Whilolo_h = new double[nrows*szt];
   double *Wlololo_h = new double[nrows*szt];
   double *Whihihi_d;                         // W on the device
   double *Wlohihi_d;
   double *Whilohi_d;
   double *Wlolohi_d;
   double *Whihilo_d;
   double *Wlohilo_d;
   double *Whilolo_d;
   double *Wlololo_d;
   double *WYThihihi_h = new double[nrows2];  // W*Y^T 
   double *WYTlohihi_h = new double[nrows2];
   double *WYThilohi_h = new double[nrows2];
   double *WYTlolohi_h = new double[nrows2];
   double *WYThihilo_h = new double[nrows2];
   double *WYTlohilo_h = new double[nrows2];
   double *WYThilolo_h = new double[nrows2];
   double *WYTlololo_h = new double[nrows2];
   double *WYThihihi_d;                       // WYT on the device
   double *WYTlohihi_d;
   double *WYThilohi_d;
   double *WYTlolohi_d;
   double *WYThihilo_d;
   double *WYTlohilo_d;
   double *WYThilolo_d;
   double *WYTlololo_d;
   double *YWThihihi_h = new double[nrows2];  // Y*W^T
   double *YWTlohihi_h = new double[nrows2];
   double *YWThilohi_h = new double[nrows2];
   double *YWTlolohi_h = new double[nrows2];
   double *YWThihilo_h = new double[nrows2];
   double *YWTlohilo_h = new double[nrows2];
   double *YWThilolo_h = new double[nrows2];
   double *YWTlololo_h = new double[nrows2];
   double *YWThihihi_d;                       // YWT on the device
   double *YWTlohihi_d;
   double *YWThilohi_d;
   double *YWTlolohi_d;
   double *YWThihilo_d;
   double *YWTlohilo_d;
   double *YWThilolo_d;
   double *YWTlololo_d;
   double *QWYThihihi_h = new double[nrows2]; // Q*WY^T
   double *QWYTlohihi_h = new double[nrows2];
   double *QWYThilohi_h = new double[nrows2];
   double *QWYTlolohi_h = new double[nrows2];
   double *QWYThihilo_h = new double[nrows2];
   double *QWYTlohilo_h = new double[nrows2];
   double *QWYThilolo_h = new double[nrows2];
   double *QWYTlololo_h = new double[nrows2];
   double *QWYThihihi_d;                      // QWYT on the device
   double *QWYTlohihi_d;
   double *QWYThilohi_d;
   double *QWYTlolohi_d;
   double *QWYThihilo_d;
   double *QWYTlohilo_d;
   double *QWYThilolo_d;
   double *QWYTlololo_d;
   double *YWTChihihi_h = new double[dim];    // YWT*C on the host
   double *YWTClohihi_h = new double[dim];
   double *YWTChilohi_h = new double[dim];
   double *YWTClolohi_h = new double[dim];
   double *YWTChihilo_h = new double[dim];
   double *YWTClohilo_h = new double[dim];
   double *YWTChilolo_h = new double[dim];
   double *YWTClololo_h = new double[dim];
   double *YWTChihihi_d;                      // YWTC on the device
   double *YWTClohihi_d;
   double *YWTChilohi_d;
   double *YWTClolohi_d;
   double *YWTChihilo_d;
   double *YWTClohilo_d;
   double *YWTChilolo_d;
   double *YWTClololo_d;
   double *RTdotvhihihi_h = new double[nrows2]; // R^T dotted with v
   double *RTdotvlohihi_h = new double[nrows2];
   double *RTdotvhilohi_h = new double[nrows2];
   double *RTdotvlolohi_h = new double[nrows2];
   double *RTdotvhihilo_h = new double[nrows2];
   double *RTdotvlohilo_h = new double[nrows2];
   double *RTdotvhilolo_h = new double[nrows2];
   double *RTdotvlololo_h = new double[nrows2];
   double *RTdotvhihihi_d;                      // RTdotv on the device
   double *RTdotvlohihi_d;
   double *RTdotvhilohi_d;
   double *RTdotvlolohi_d;
   double *RTdotvhihilo_d;                      // RTdotv on the device
   double *RTdotvlohilo_d;
   double *RTdotvhilolo_d;
   double *RTdotvlololo_d;
   double *bRTvhihihi_h = new double[nrows];  // beta*R^T*v
   double *bRTvlohihi_h = new double[nrows];
   double *bRTvhilohi_h = new double[nrows];
   double *bRTvlolohi_h = new double[nrows];
   double *bRTvhihilo_h = new double[nrows];
   double *bRTvlohilo_h = new double[nrows];
   double *bRTvhilolo_h = new double[nrows];
   double *bRTvlololo_h = new double[nrows];
   double *bRTvhihihi_d;                      // bRTv on the device
   double *bRTvlohihi_d;
   double *bRTvhilohi_d;
   double *bRTvlolohi_d;
   double *bRTvhihilo_d; 
   double *bRTvlohilo_d;
   double *bRTvhilolo_d;
   double *bRTvlololo_d;
   double *sumshihihi_h = new double[nrows];  // subsums for large house
   double *sumslohihi_h = new double[nrows];
   double *sumshilohi_h = new double[nrows];
   double *sumslolohi_h = new double[nrows];
   double *sumshihilo_h = new double[nrows];
   double *sumslohilo_h = new double[nrows];
   double *sumshilolo_h = new double[nrows];
   double *sumslololo_h = new double[nrows];
   double *sumshihihi_d;                      // sums on the device
   double *sumslohihi_d;
   double *sumshilohi_d;
   double *sumslolohi_d;
   double *sumshihilo_d;
   double *sumslohilo_d;
   double *sumshilolo_d;
   double *sumslololo_d;
   double sigmahihihi_h,sigmalohihi_h,sigmahilohi_h,sigmalolohi_h;
   double sigmahihilo_h,sigmalohilo_h,sigmahilolo_h,sigmalololo_h;
   double *sigmahihihi_d;                     // sigma on the device
   double *sigmalohihi_d;
   double *sigmahilohi_d;
   double *sigmalolohi_d;
   double *sigmahihilo_d;
   double *sigmalohilo_d;
   double *sigmahilolo_d;
   double *sigmalololo_d;

   int ix = 0;                          // copy the columns of A to A_h
   for(int j=0; j<ncols; j++)   
      for(int i=0; i<nrows; i++)
      {
         Ahihihi_h[ix]   = Ahihihi[i][j];
         Alohihi_h[ix]   = Alohihi[i][j];
         Ahilohi_h[ix]   = Ahilohi[i][j];
         Alolohi_h[ix]   = Alolohi[i][j];
         Ahihilo_h[ix]   = Ahihilo[i][j];
         Alohilo_h[ix]   = Alohilo[i][j];
         Ahilolo_h[ix]   = Ahilolo[i][j];
         Alololo_h[ix++] = Alololo[i][j];
      }

   ix = 0;                              // initialize Q with identity
   for(int i=0; i<nrows; i++)
   {
      for(int j=0; j<nrows; j++)
      {
         if(i == j)
         {
            Qhihihi_h[ix]   = 1.0;
            Qlohihi_h[ix]   = 0.0;
            Qhilohi_h[ix]   = 0.0;
            Qlolohi_h[ix]   = 0.0;
            Qhihilo_h[ix]   = 1.0;
            Qlohilo_h[ix]   = 0.0;
            Qhilolo_h[ix]   = 0.0;
            Qlololo_h[ix++] = 0.0;
         }
         else
         {
            Qhihihi_h[ix]   = 0.0;
            Qlohihi_h[ix]   = 0.0;
            Qhilohi_h[ix]   = 0.0;
            Qlolohi_h[ix]   = 0.0;
            Qhihilo_h[ix]   = 0.0;
            Qlohilo_h[ix]   = 0.0;
            Qhilolo_h[ix]   = 0.0;
            Qlololo_h[ix++] = 0.0;
         }
      }
   }
   const size_t sznum = dim*sizeof(double);
   hipMalloc((void**)&Ahihihi_d,sznum);
   hipMalloc((void**)&Alohihi_d,sznum);
   hipMalloc((void**)&Ahilohi_d,sznum);
   hipMalloc((void**)&Alolohi_d,sznum);
   hipMalloc((void**)&Ahihilo_d,sznum);
   hipMalloc((void**)&Alohilo_d,sznum);
   hipMalloc((void**)&Ahilolo_d,sznum);
   hipMalloc((void**)&Alololo_d,sznum);
   hipMemcpy(Ahihihi_d,Ahihihi_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Alohihi_d,Alohihi_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Ahilohi_d,Ahilohi_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Alolohi_d,Alolohi_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Ahihilo_d,Ahihilo_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Alohilo_d,Alohilo_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Ahilolo_d,Ahilolo_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Alololo_d,Alololo_h,sznum,hipMemcpyHostToDevice);

   const size_t szbeta = szt*sizeof(double);
   hipMalloc((void**)&betahihihi_d,szbeta);
   hipMalloc((void**)&betalohihi_d,szbeta);
   hipMalloc((void**)&betahilohi_d,szbeta);
   hipMalloc((void**)&betalolohi_d,szbeta);
   hipMalloc((void**)&betahihilo_d,szbeta);
   hipMalloc((void**)&betalohilo_d,szbeta);
   hipMalloc((void**)&betahilolo_d,szbeta);
   hipMalloc((void**)&betalololo_d,szbeta);

   for(int i=0; i<szt; i++)
   {
      betahihihi_h[i] = 0.0;
      betalohihi_h[i] = 0.0;
      betahilohi_h[i] = 0.0;
      betalolohi_h[i] = 0.0;
      betahihilo_h[i] = 0.0;
      betalohilo_h[i] = 0.0;
      betahilolo_h[i] = 0.0;
      betalololo_h[i] = 0.0;
   }
   hipMemcpy(betahihihi_d,betahihihi_h,szbeta,hipMemcpyHostToDevice);
   hipMemcpy(betalohihi_d,betalohihi_h,szbeta,hipMemcpyHostToDevice);
   hipMemcpy(betahilohi_d,betahilohi_h,szbeta,hipMemcpyHostToDevice);
   hipMemcpy(betalolohi_d,betalolohi_h,szbeta,hipMemcpyHostToDevice);
   hipMemcpy(betahihilo_d,betahihilo_h,szbeta,hipMemcpyHostToDevice);
   hipMemcpy(betalohilo_d,betalohilo_h,szbeta,hipMemcpyHostToDevice);
   hipMemcpy(betahilolo_d,betahilolo_h,szbeta,hipMemcpyHostToDevice);
   hipMemcpy(betalololo_d,betalololo_h,szbeta,hipMemcpyHostToDevice);

   const size_t szhouse = nrows*sizeof(double);
   const size_t szpad = szt*sizeof(double);  // padding for nonsquare tiles
   const size_t szVandW = szt*szhouse;
   hipMalloc((void**)&Vhihihi_d,szVandW + szpad); // pad only in allocation
   hipMalloc((void**)&Vlohihi_d,szVandW + szpad);
   hipMalloc((void**)&Vhilohi_d,szVandW + szpad);
   hipMalloc((void**)&Vlolohi_d,szVandW + szpad);
   hipMalloc((void**)&Vhihilo_d,szVandW + szpad);
   hipMalloc((void**)&Vlohilo_d,szVandW + szpad);
   hipMalloc((void**)&Vhilolo_d,szVandW + szpad);
   hipMalloc((void**)&Vlololo_d,szVandW + szpad);

   ix = 0;
   for(int i=0; i<nrows*szt; i++)
   {
      Vhihihi_h[ix] = 0.0;
      Vlohihi_h[ix] = 0.0; 
      Vhilohi_h[ix] = 0.0; 
      Vlolohi_h[ix] = 0.0; 
      Vhihilo_h[ix] = 0.0;
      Vlohilo_h[ix] = 0.0; 
      Vhilolo_h[ix] = 0.0; 
      Vlololo_h[ix++] = 0.0; 
   }
   Vhihihi_h[--ix] = 1.0; // initialize last vector for square tiles

   hipMemcpy(Vhihihi_d,Vhihihi_h,szVandW,hipMemcpyHostToDevice);
   hipMemcpy(Vlohihi_d,Vlohihi_h,szVandW,hipMemcpyHostToDevice);
   hipMemcpy(Vhilohi_d,Vhilohi_h,szVandW,hipMemcpyHostToDevice);
   hipMemcpy(Vlolohi_d,Vlolohi_h,szVandW,hipMemcpyHostToDevice);
   hipMemcpy(Vhihilo_d,Vhihilo_h,szVandW,hipMemcpyHostToDevice);
   hipMemcpy(Vlohilo_d,Vlohilo_h,szVandW,hipMemcpyHostToDevice);
   hipMemcpy(Vhilolo_d,Vhilolo_h,szVandW,hipMemcpyHostToDevice);
   hipMemcpy(Vlololo_d,Vlololo_h,szVandW,hipMemcpyHostToDevice);
   hipMalloc((void**)&Whihihi_d,szVandW + szpad); // pad only in allocation
   hipMalloc((void**)&Wlohihi_d,szVandW + szpad); 
   hipMalloc((void**)&Whilohi_d,szVandW + szpad); 
   hipMalloc((void**)&Wlolohi_d,szVandW + szpad); 
   hipMalloc((void**)&Whihilo_d,szVandW + szpad); 
   hipMalloc((void**)&Wlohilo_d,szVandW + szpad); 
   hipMalloc((void**)&Whilolo_d,szVandW + szpad); 
   hipMalloc((void**)&Wlololo_d,szVandW + szpad); 

   hipMalloc((void**)&RTdotvhihihi_d,szVandW + szpad);
   hipMalloc((void**)&RTdotvlohihi_d,szVandW + szpad);
   hipMalloc((void**)&RTdotvhilohi_d,szVandW + szpad);
   hipMalloc((void**)&RTdotvlolohi_d,szVandW + szpad);
   hipMalloc((void**)&RTdotvhihilo_d,szVandW + szpad);
   hipMalloc((void**)&RTdotvlohilo_d,szVandW + szpad);
   hipMalloc((void**)&RTdotvhilolo_d,szVandW + szpad);
   hipMalloc((void**)&RTdotvlololo_d,szVandW + szpad);
   hipMalloc((void**)&bRTvhihihi_d,szhouse + szpad);
   hipMalloc((void**)&bRTvlohihi_d,szhouse + szpad);
   hipMalloc((void**)&bRTvhilohi_d,szhouse + szpad);
   hipMalloc((void**)&bRTvlolohi_d,szhouse + szpad);
   hipMalloc((void**)&bRTvhihilo_d,szhouse + szpad);
   hipMalloc((void**)&bRTvlohilo_d,szhouse + szpad);
   hipMalloc((void**)&bRTvhilolo_d,szhouse + szpad);
   hipMalloc((void**)&bRTvlololo_d,szhouse + szpad);

   hipMalloc((void**)&sumshihihi_d,szhouse);
   hipMalloc((void**)&sumslohihi_d,szhouse);
   hipMalloc((void**)&sumshilohi_d,szhouse);
   hipMalloc((void**)&sumslolohi_d,szhouse);
   hipMalloc((void**)&sumshihilo_d,szhouse);
   hipMalloc((void**)&sumslohilo_d,szhouse);
   hipMalloc((void**)&sumshilolo_d,szhouse);
   hipMalloc((void**)&sumslololo_d,szhouse);
   hipMalloc((void**)&sigmahihihi_d,sizeof(double));
   hipMalloc((void**)&sigmalohihi_d,sizeof(double));
   hipMalloc((void**)&sigmahilohi_d,sizeof(double));
   hipMalloc((void**)&sigmalolohi_d,sizeof(double));
   hipMalloc((void**)&sigmahihilo_d,sizeof(double));
   hipMalloc((void**)&sigmalohilo_d,sizeof(double));
   hipMalloc((void**)&sigmahilolo_d,sizeof(double));
   hipMalloc((void**)&sigmalololo_d,sizeof(double));

   const size_t szWYT = nrows2*sizeof(double);
   hipMalloc((void**)&WYThihihi_d,szWYT + szpad); // pad for W*Y^T product
   hipMalloc((void**)&WYTlohihi_d,szWYT + szpad); 
   hipMalloc((void**)&WYThilohi_d,szWYT + szpad); 
   hipMalloc((void**)&WYTlolohi_d,szWYT + szpad); 
   hipMalloc((void**)&WYThihilo_d,szWYT + szpad);
   hipMalloc((void**)&WYTlohilo_d,szWYT + szpad); 
   hipMalloc((void**)&WYThilolo_d,szWYT + szpad); 
   hipMalloc((void**)&WYTlololo_d,szWYT + szpad); 
   hipMalloc((void**)&Qhihihi_d,szWYT + szpad);
   hipMalloc((void**)&Qlohihi_d,szWYT + szpad);
   hipMalloc((void**)&Qhilohi_d,szWYT + szpad);
   hipMalloc((void**)&Qlolohi_d,szWYT + szpad);
   hipMalloc((void**)&Qhihilo_d,szWYT + szpad);
   hipMalloc((void**)&Qlohilo_d,szWYT + szpad);
   hipMalloc((void**)&Qhilolo_d,szWYT + szpad);
   hipMalloc((void**)&Qlololo_d,szWYT + szpad);
   hipMemcpy(Qhihihi_d,Qhihihi_h,szWYT,hipMemcpyHostToDevice);
   hipMemcpy(Qlohihi_d,Qlohihi_h,szWYT,hipMemcpyHostToDevice);
   hipMemcpy(Qhilohi_d,Qhilohi_h,szWYT,hipMemcpyHostToDevice);
   hipMemcpy(Qlolohi_d,Qlolohi_h,szWYT,hipMemcpyHostToDevice);
   hipMemcpy(Qhihilo_d,Qhihilo_h,szWYT,hipMemcpyHostToDevice);
   hipMemcpy(Qlohilo_d,Qlohilo_h,szWYT,hipMemcpyHostToDevice);
   hipMemcpy(Qhilolo_d,Qhilolo_h,szWYT,hipMemcpyHostToDevice);
   hipMemcpy(Qlololo_d,Qlololo_h,szWYT,hipMemcpyHostToDevice);
   hipMalloc((void**)&QWYThihihi_d,szWYT + szpad);
   hipMalloc((void**)&QWYTlohihi_d,szWYT + szpad);
   hipMalloc((void**)&QWYThilohi_d,szWYT + szpad);
   hipMalloc((void**)&QWYTlolohi_d,szWYT + szpad);
   hipMalloc((void**)&QWYThihilo_d,szWYT + szpad);
   hipMalloc((void**)&QWYTlohilo_d,szWYT + szpad);
   hipMalloc((void**)&QWYThilolo_d,szWYT + szpad);
   hipMalloc((void**)&QWYTlololo_d,szWYT + szpad);

   const size_t szYWT = nrows2*sizeof(double);
   hipMalloc((void**)&YWThihihi_d,szYWT + szpad); // pad for Y*W^T product
   hipMalloc((void**)&YWTlohihi_d,szYWT + szpad);
   hipMalloc((void**)&YWThilohi_d,szYWT + szpad);
   hipMalloc((void**)&YWTlolohi_d,szYWT + szpad);
   hipMalloc((void**)&YWThihilo_d,szYWT + szpad);
   hipMalloc((void**)&YWTlohilo_d,szYWT + szpad);
   hipMalloc((void**)&YWThilolo_d,szYWT + szpad);
   hipMalloc((void**)&YWTlololo_d,szYWT + szpad);
   hipMalloc((void**)&YWTChihihi_d,sznum + szpad);
   hipMalloc((void**)&YWTClohihi_d,sznum + szpad);
   hipMalloc((void**)&YWTChilohi_d,sznum + szpad);
   hipMalloc((void**)&YWTClolohi_d,sznum + szpad);
   hipMalloc((void**)&YWTChihilo_d,sznum + szpad);
   hipMalloc((void**)&YWTClohilo_d,sznum + szpad);
   hipMalloc((void**)&YWTChilolo_d,sznum + szpad);
   hipMalloc((void**)&YWTClololo_d,sznum + szpad);

   *houselapms = 0.0; *RTvlapms = 0.0; *tileRlapms = 0.0; *vb2Wlapms = 0.0;
   *WYTlapms = 0.0; *QWYTlapms = 0.0; *Qaddlapms = 0.0;
   *YWTlapms = 0.0; *YWTClapms = 0.0; *Raddlapms = 0.0;
   *addcnt = 0; *mulcnt = 0; *divcnt = 0; *sqrtcnt = 0;
   struct timeval begintime,endtime; // wall clock time of computations

   gettimeofday(&begintime,0);

   for(int k=0; k<nbt; k++)       // k runs over the number of blocks
   {
      if(verbose)
         cout << "Tile k = " << k << " out of " << nbt << " ..." << endl;

      int colidx,nrows1;

      for(int L=0; L<szt; L++)  // L runs over the columns in one block
      {
         colidx = k*szt + L;              // index of the current column
         nrows1 = nrows - colidx - 1;     // #rows in Householder vector - 1

         if(verbose)
            cout << "-> current column : " << colidx << endl
                 << "-> #nrows in Householder vector - 1 : "
                 << nrows1 << endl;

         if(nrows1 <= szt)
         {
            GPU_dbl8_small_house
               (nrows,ncols,szt,nbt,colidx,nrows1,k,L,
                   Ahihihi_h,   Alohihi_h,   Ahilohi_h,   Alolohi_h,
                   Ahihilo_h,   Alohilo_h,   Ahilolo_h,   Alololo_h,
                   Ahihihi_d,   Alohihi_d,   Ahilohi_d,   Alolohi_d,
                   Ahihilo_d,   Alohilo_d,   Ahilolo_d,   Alololo_d,
                   vhihihi_h,   vlohihi_h,   vhilohi_h,   vlolohi_h,
                   vhihilo_h,   vlohilo_h,   vhilolo_h,   vlololo_h,
                   Vhihihi_d,   Vlohihi_d,   Vhilohi_d,   Vlolohi_d,
                   Vhihilo_d,   Vlohilo_d,   Vhilolo_d,   Vlololo_d,
                betahihihi_h,betalohihi_h,betahilohi_h,betalolohi_h,
                betahihilo_h,betalohilo_h,betahilolo_h,betalololo_h,
                betahihihi_d,betalohihi_d,betahilohi_d,betalolohi_d,
                betahihilo_d,betalohilo_d,betahilolo_d,betalololo_d,
                houselapms,addcnt,mulcnt,divcnt,sqrtcnt,verbose);

            GPU_dbl8_small_leftRupdate
               (nrows,ncols,szt,colidx,k,L,
                   Ahihihi_h,   Alohihi_h,   Ahilohi_h,   Alolohi_h,
                   Ahihilo_h,   Alohilo_h,   Ahilolo_h,   Alololo_h,
                   Ahihihi_d,   Alohihi_d,   Ahilohi_d,   Alolohi_d,
                   Ahihilo_d,   Alohilo_d,   Ahilolo_d,   Alololo_d,
                   Vhihihi_d,   Vlohihi_d,   Vhilohi_d,   Vlolohi_d,
                   Vhihilo_d,   Vlohilo_d,   Vhilolo_d,   Vlololo_d,
                betahihihi_h,betalohihi_h,betahilohi_h,betalolohi_h,
                betahihilo_h,betalohilo_h,betahilolo_h,betalololo_h,
                betahihihi_d,betalohihi_d,betahilohi_d,betalolohi_d,
                betahihilo_d,betalohilo_d,betahilolo_d,betalololo_d,
                tileRlapms,addcnt,mulcnt,verbose);
         }
         else
         {
            GPU_dbl8_large_house
               (nrows,ncols,szt,nbt,colidx,nrows1,k,L,
                     Ahihihi_h,     Alohihi_h,     Ahilohi_h,     Alolohi_h,
                     Ahihilo_h,     Alohilo_h,     Ahilolo_h,     Alololo_h,
                     Ahihihi_d,     Alohihi_d,     Ahilohi_d,     Alolohi_d,
                     Ahihilo_d,     Alohilo_d,     Ahilolo_d,     Alololo_d,
                     vhihihi_h,     vlohihi_h,     vhilohi_h,     vlolohi_h,
                     vhihilo_h,     vlohilo_h,     vhilolo_h,     vlololo_h,
                     Vhihihi_d,     Vlohihi_d,     Vhilohi_d,     Vlolohi_d,
                     Vhihilo_d,     Vlohilo_d,     Vhilolo_d,     Vlololo_d,
                  betahihihi_h,  betalohihi_h,  betahilohi_h,  betalolohi_h,
                  betahihilo_h,  betalohilo_h,  betahilolo_h,  betalololo_h,
                  betahihihi_d,  betalohihi_d,  betahilohi_d,  betalolohi_d,
                  betahihilo_d,  betalohilo_d,  betahilolo_d,  betalololo_d,
                  sumshihihi_h,  sumslohihi_h,  sumshilohi_h,  sumslolohi_h,
                  sumshihilo_h,  sumslohilo_h,  sumshilolo_h,  sumslololo_h,
                  sumshihihi_d,  sumslohihi_d,  sumshilohi_d,  sumslolohi_d,
                  sumshihilo_d,  sumslohilo_d,  sumshilolo_d,  sumslololo_d,
                &sigmahihihi_h,&sigmalohihi_h,&sigmahilohi_h,&sigmalolohi_h,
                &sigmahihilo_h,&sigmalohilo_h,&sigmahilolo_h,&sigmalololo_h,
                 sigmahihihi_d, sigmalohihi_d, sigmahilohi_d, sigmalolohi_d,
                 sigmahihilo_d, sigmalohilo_d, sigmahilolo_d, sigmalololo_d,
                houselapms,addcnt,mulcnt,divcnt,sqrtcnt,verbose);

            GPU_dbl8_medium_leftRupdate
               (nrows,ncols,szt,colidx,k,L,
                     Ahihihi_h,     Alohihi_h,     Ahilohi_h,     Alolohi_h,
                     Ahihilo_h,     Alohilo_h,     Ahilolo_h,     Alololo_h,
                     Ahihihi_d,     Alohihi_d,     Ahilohi_d,     Alolohi_d,
                     Ahihilo_d,     Alohilo_d,     Ahilolo_d,     Alololo_d,
                     Vhihihi_d,     Vlohihi_d,     Vhilohi_d,     Vlolohi_d,
                     Vhihilo_d,     Vlohilo_d,     Vhilolo_d,     Vlololo_d,
                  betahihihi_h,  betalohihi_h,  betahilohi_h,  betalolohi_h,
                  betahihilo_h,  betalohilo_h,  betahilolo_h,  betalololo_h,
                  betahihihi_d,  betalohihi_d,  betahilohi_d,  betalolohi_d,
                  betahihilo_d,  betalohilo_d,  betahilolo_d,  betalololo_d,
                RTdotvhihihi_h,RTdotvlohihi_h,RTdotvhilohi_h,RTdotvlolohi_h,
                RTdotvhihilo_h,RTdotvlohilo_h,RTdotvhilolo_h,RTdotvlololo_h,
                RTdotvhihihi_d,RTdotvlohihi_d,RTdotvhilohi_d,RTdotvlolohi_d,
                RTdotvhihilo_d,RTdotvlohilo_d,RTdotvhilolo_d,RTdotvlololo_d,
                  bRTvhihihi_h,  bRTvlohihi_h,  bRTvhilohi_h,  bRTvlolohi_h,
                  bRTvhihilo_h,  bRTvlohilo_h,  bRTvhilolo_h,  bRTvlololo_h,
                  bRTvhihihi_d,  bRTvlohihi_d,  bRTvhilohi_d,  bRTvlolohi_d,
                  bRTvhihilo_d,  bRTvlohilo_d,  bRTvhilolo_d,  bRTvlololo_d,
                RTvlapms,tileRlapms,addcnt,mulcnt,verbose);
         }
      }
      GPU_dbl8_medium_VB_to_W
         (nrows,szt,szt,k,
             Vhihihi_h,   Vlohihi_h,   Vhilohi_h,   Vlolohi_h,
             Vhihilo_h,   Vlohilo_h,   Vhilolo_h,   Vlololo_h,
             Vhihihi_d,   Vlohihi_d,   Vhilohi_d,   Vlolohi_d,
             Vhihilo_d,   Vlohilo_d,   Vhilolo_d,   Vlololo_d,
             Whihihi_h,   Wlohihi_h,   Whilohi_h,   Wlolohi_h,
             Whihilo_h,   Wlohilo_h,   Whilolo_h,   Wlololo_h,
             Whihihi_d,   Wlohihi_d,   Whilohi_d,   Wlolohi_d,
             Whihilo_d,   Wlohilo_d,   Whilolo_d,   Wlololo_d,
           WYThihihi_h, WYTlohihi_h, WYThilohi_h, WYTlolohi_h,
           WYThihilo_h, WYTlohilo_h, WYThilolo_h, WYTlololo_h,
           WYThihihi_d, WYTlohihi_d, WYThilohi_d, WYTlolohi_d,
           WYThihilo_d, WYTlohilo_d, WYThilolo_d, WYTlololo_d,
          betahihihi_h,betalohihi_h,betahilohi_h,betalolohi_h,
          betahihilo_h,betalohilo_h,betahilolo_h,betalololo_h,
          betahihihi_d,betalohihi_d,betahilohi_d,betalolohi_d,
          betahihilo_d,betalohilo_d,betahilolo_d,betalololo_d,
          vb2Wlapms,addcnt,mulcnt,verbose);
/*
      GPU_dbl2_small_WYT
         (nrows-k*szt,szt,Whi_d,Wlo_d,Vhi_d,Vlo_d,WYThi_d,WYTlo_d,
          WYThi_h,WYTlo_h,WYTlapms,verbose);
 */
      GPU_dbl8_small_QWYT
         (nrows,szt,k,
             Qhihihi_d,   Qlohihi_d,   Qhilohi_d,   Qlolohi_d,
             Qhihilo_d,   Qlohilo_d,   Qhilolo_d,   Qlololo_d,
           WYThihihi_d, WYTlohihi_d, WYThilohi_d, WYTlolohi_d,
           WYThihilo_d, WYTlohilo_d, WYThilolo_d, WYTlololo_d,
          QWYThihihi_d,QWYTlohihi_d,QWYThilohi_d,QWYTlolohi_d,
          QWYThihilo_d,QWYTlohilo_d,QWYThilolo_d,QWYTlololo_d,
          QWYThihihi_h,QWYTlohihi_h,QWYThilohi_h,QWYTlolohi_h,
          QWYThihilo_h,QWYTlohilo_h,QWYThilolo_h,QWYTlololo_h,
             Qhihihi_h,   Qlohihi_h,   Qhilohi_h,   Qlolohi_h,
             Qhihilo_h,   Qlohilo_h,   Qhilolo_h,   Qlololo_h,
          QWYTlapms,addcnt,mulcnt,verbose);

      GPU_dbl8_small_Qupdate
         (nrows,szt,k,
             Qhihihi_d,   Qlohihi_d,   Qhilohi_d,   Qlolohi_d,
             Qhihilo_d,   Qlohilo_d,   Qhilolo_d,   Qlololo_d,
          QWYThihihi_d,QWYTlohihi_d,QWYThilohi_d,QWYTlolohi_d,
          QWYThihilo_d,QWYTlohilo_d,QWYThilolo_d,QWYTlololo_d,
             Qhihihi_h,   Qlohihi_h,   Qhilohi_h,   Qlolohi_h,
             Qhihilo_h,   Qlohilo_h,   Qhilolo_h,   Qlololo_h,
          Qaddlapms,addcnt,verbose);

      if(k < nbt-1)                                           // update R
      {
         GPU_dbl8_small_YWT
            (nrows,szt,k,
               Vhihihi_d,  Vlohihi_d,  Vhilohi_d,  Vlolohi_d,
               Vhihilo_d,  Vlohilo_d,  Vhilolo_d,  Vlololo_d,
               Whihihi_d,  Wlohihi_d,  Whilohi_d,  Wlolohi_d,
               Whihilo_d,  Wlohilo_d,  Whilolo_d,  Wlololo_d,
             YWThihihi_d,YWTlohihi_d,YWThilohi_d,YWTlolohi_d,
             YWThihilo_d,YWTlohilo_d,YWThilolo_d,YWTlololo_d,
             YWThihihi_h,YWTlohihi_h,YWThilohi_h,YWTlolohi_h,
             YWThihilo_h,YWTlohilo_h,YWThilolo_h,YWTlololo_h,
             YWTlapms,addcnt,mulcnt,verbose);

         GPU_dbl8_small_YWTC
            (nrows,ncols,szt,k,
              YWThihihi_d, YWTlohihi_d, YWThilohi_d, YWTlolohi_d,
              YWThihilo_d, YWTlohilo_d, YWThilolo_d, YWTlololo_d,
                Ahihihi_d,   Alohihi_d,   Ahilohi_d,   Alolohi_d,
                Ahihilo_d,   Alohilo_d,   Ahilolo_d,   Alololo_d,
             YWTChihihi_d,YWTClohihi_d,YWTChilohi_d,YWTClolohi_d,
             YWTChihilo_d,YWTClohilo_d,YWTChilolo_d,YWTClololo_d,
             YWTChihihi_h,YWTClohihi_h,YWTChilohi_h,YWTClolohi_h,
             YWTChihilo_h,YWTClohilo_h,YWTChilolo_h,YWTClololo_h,
             YWTClapms,addcnt,mulcnt,verbose);

         GPU_dbl8_small_R_add_YWTC
            (nrows,ncols,szt,k,
                Ahihihi_d,   Alohihi_d,   Ahilohi_d,   Alolohi_d,
                Ahihilo_d,   Alohilo_d,   Ahilolo_d,   Alololo_d,
             YWTChihihi_d,YWTClohihi_d,YWTChilohi_d,YWTClolohi_d,
             YWTChihilo_d,YWTClohilo_d,YWTChilolo_d,YWTClololo_d,
                Ahihihi_h,   Alohihi_h,   Ahilohi_h,   Alolohi_h,
                Ahihilo_h,   Alohilo_h,   Ahilolo_h,   Alololo_h,
             Raddlapms,addcnt,verbose);
      }
   }
   gettimeofday(&endtime,0);
   long seconds = endtime.tv_sec - begintime.tv_sec;
   long microseconds = endtime.tv_usec - begintime.tv_usec;
   *walltimesec = seconds + microseconds*1.0e-6;

   hipMemcpy(Qhihihi_h,Qhihihi_d,szWYT,hipMemcpyDeviceToHost);
   hipMemcpy(Qlohihi_h,Qlohihi_d,szWYT,hipMemcpyDeviceToHost);
   hipMemcpy(Qhilohi_h,Qhilohi_d,szWYT,hipMemcpyDeviceToHost);
   hipMemcpy(Qlolohi_h,Qlolohi_d,szWYT,hipMemcpyDeviceToHost);
   hipMemcpy(Qhihilo_h,Qhihilo_d,szWYT,hipMemcpyDeviceToHost);
   hipMemcpy(Qlohilo_h,Qlohilo_d,szWYT,hipMemcpyDeviceToHost);
   hipMemcpy(Qhilolo_h,Qhilolo_d,szWYT,hipMemcpyDeviceToHost);
   hipMemcpy(Qlololo_h,Qlololo_d,szWYT,hipMemcpyDeviceToHost);
   ix = 0;                                           // copy rows of Q
   for(int i=0; i<nrows; i++)
      for(int j=0; j<nrows; j++)
      {
         Qhihihi[i][j] = Qhihihi_h[ix];
         Qlohihi[i][j] = Qlohihi_h[ix];
         Qhilohi[i][j] = Qhilohi_h[ix];
         Qlolohi[i][j] = Qlolohi_h[ix];
         Qhihilo[i][j] = Qhihilo_h[ix];
         Qlohilo[i][j] = Qlohilo_h[ix];
         Qhilolo[i][j] = Qhilolo_h[ix];
         Qlololo[i][j] = Qlololo_h[ix++];
      }

   hipMemcpy(Ahihihi_h,Ahihihi_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(Alohihi_h,Alohihi_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(Ahilohi_h,Ahilohi_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(Alolohi_h,Alolohi_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(Ahihilo_h,Ahihilo_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(Alohilo_h,Alohilo_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(Ahilolo_h,Ahilolo_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(Alololo_h,Alololo_d,sznum,hipMemcpyDeviceToHost);

   for(int i=0; i<nrows; i++)                       // copy columns of R
      for(int j=0; j<ncols; j++)
      {
         Rhihihi[i][j] = Ahihihi_h[j*nrows+i];
         Rlohihi[i][j] = Alohihi_h[j*nrows+i];
         Rhilohi[i][j] = Ahilohi_h[j*nrows+i];
         Rlolohi[i][j] = Alolohi_h[j*nrows+i];
         Rhihilo[i][j] = Ahihilo_h[j*nrows+i];
         Rlohilo[i][j] = Alohilo_h[j*nrows+i];
         Rhilolo[i][j] = Ahilolo_h[j*nrows+i];
         Rlololo[i][j] = Alololo_h[j*nrows+i];
      }

   free(Ahihihi_h); free(Alohihi_h); free(Ahilohi_h); free(Alolohi_h);
   free(Ahihilo_h); free(Alohilo_h); free(Ahilolo_h); free(Alololo_h);
   free(Qhihihi_h); free(Qlohihi_h); free(Qhilohi_h); free(Qlolohi_h); 
   free(Qhihilo_h); free(Qlohilo_h); free(Qhilolo_h); free(Qlololo_h); 
   free(vhihihi_h); free(vlohihi_h); free(vhilohi_h); free(vlolohi_h);
   free(vhihilo_h); free(vlohilo_h); free(vhilolo_h); free(vlololo_h);
   free(Vhihihi_h); free(Vlohihi_h); free(Vhilohi_h); free(Vlolohi_h);
   free(Vhihilo_h); free(Vlohilo_h); free(Vhilolo_h); free(Vlololo_h);
   free(Whihihi_h); free(Wlohihi_h); free(Whilohi_h); free(Wlolohi_h);
   free(Whihilo_h); free(Wlohilo_h); free(Whilolo_h); free(Wlololo_h);
   free(sumshihihi_h); free(sumslohihi_h);
   free(sumshilohi_h); free(sumslolohi_h);
   free(sumshihilo_h); free(sumslohilo_h);
   free(sumshilolo_h); free(sumslololo_h);

   free(RTdotvhihihi_h); free(RTdotvlohihi_h);
   free(RTdotvhilohi_h); free(RTdotvlolohi_h);
   free(RTdotvhihilo_h); free(RTdotvlohilo_h);
   free(RTdotvhilolo_h); free(RTdotvlololo_h);
   free(bRTvhihihi_h); free(bRTvlohihi_h);
   free(bRTvhilohi_h); free(bRTvlolohi_h);
   free(bRTvhihilo_h); free(bRTvlohilo_h);
   free(bRTvhilolo_h); free(bRTvlololo_h);
   free(WYThihihi_h); free(QWYThihihi_h);
   free(WYThilohi_h); free(QWYThilohi_h);
   free(WYThihilo_h); free(QWYThihilo_h);
   free(WYThilolo_h); free(QWYThilolo_h);
   free(YWThihihi_h); free(YWTChihihi_h);
   free(YWThilohi_h); free(YWTChilohi_h);
   free(YWThihilo_h); free(YWTChihilo_h);
   free(YWThilolo_h); free(YWTChilolo_h);
   free(WYTlohihi_h); free(QWYTlohihi_h);
   free(WYTlolohi_h); free(QWYTlolohi_h);
   free(WYTlohilo_h); free(QWYTlohilo_h);
   free(WYTlololo_h); free(QWYTlololo_h);
   free(YWTlohihi_h); free(YWTClohihi_h);
   free(YWTlolohi_h); free(YWTClolohi_h);
   free(YWTlohilo_h); free(YWTClohilo_h);
   free(YWTlololo_h); free(YWTClololo_h);
}

void GPU_cmplx8_blocked_houseqr
 ( int nrows, int ncols, int szt, int nbt,
   double **Arehihihi, double **Arelohihi,
   double **Arehilohi, double **Arelolohi,
   double **Arehihilo, double **Arelohilo,
   double **Arehilolo, double **Arelololo,
   double **Aimhihihi, double **Aimlohihi,
   double **Aimhilohi, double **Aimlolohi,
   double **Aimhihilo, double **Aimlohilo,
   double **Aimhilolo, double **Aimlololo,
   double **Qrehihihi, double **Qrelohihi,
   double **Qrehilohi, double **Qrelolohi,
   double **Qrehihilo, double **Qrelohilo,
   double **Qrehilolo, double **Qrelololo,
   double **Qimhihihi, double **Qimlohihi,
   double **Qimhilohi, double **Qimlolohi,
   double **Qimhihilo, double **Qimlohilo,
   double **Qimhilolo, double **Qimlololo,
   double **Rrehihihi, double **Rrelohihi,
   double **Rrehilohi, double **Rrelolohi,
   double **Rrehihilo, double **Rrelohilo,
   double **Rrehilolo, double **Rrelololo,
   double **Rimhihihi, double **Rimlohihi,
   double **Rimhilohi, double **Rimlolohi,
   double **Rimhihilo, double **Rimlohilo,
   double **Rimhilolo, double **Rimlololo,
   double *houselapms, double *RHvlapms, double *tileRlapms,
   double *vb2Wlapms, double *WYHlapms, double *QWYHlapms, double *Qaddlapms,
   double *YWHlapms, double *YWHClapms, double *Raddlapms,
   double *walltimesec, long long int *addcnt, long long int *mulcnt,
   long long int *divcnt, long long int *sqrtcnt, bool verbose )
{
}
