#include "hip/hip_runtime.h"
// The file dbl_tail_kernels.cu defines the functions with prototypes in
// the file dbl_tail_kernels.h.

#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <cmath>
#include <hip/hip_vector_types.h>
#include "dbl_tail_kernels.h"
#include "dbl_bals_flopcounts.h"

using namespace std;

__global__ void dbl_bals_tail
 ( int ncols, int szt, double *A, double *x, double *b )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int idx = bdx*szt + tdx; // thread tdx updates b[idx]

   double Aj;           // register for A[idx][j]
   double xj;           // register for x[j]
   double bi = b[idx];  // register for b[idx]

   int offset = idx*ncols;

   for(int j=0; j<ncols; j++)
   {
      Aj = A[offset+j];
      xj = x[j];
      bi = bi - Aj*xj;
   }
   b[idx] = bi;
}

__global__ void cmplx_bals_tail
 ( int ncols, int szt, double *Are, double *Aim,
   double *xre, double *xim, double *bre, double *bim )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int idx = bdx*szt + tdx; // thread tdx updates b[idx]

   double Ajre;             // register for Are[idx][j]
   double Ajim;             // register for Aim[idx][j]
   double xjre;             // register for xre[j]
   double xjim;             // register for xim[j]
   double bire = bre[idx];  // register for bre[idx]
   double biim = bim[idx];  // register for bim[idx]
   double zre,zim;

   int offset = idx*ncols;

   for(int j=0; j<ncols; j++)
   {
      Ajre = Are[offset+j];
      Ajim = Aim[offset+j];
      xjre = xre[j];
      xjim = xim[j];
      // bi = bi - Aj*xj;
      zre = Ajre*xjre - Ajim*xjim;
      zim = Ajre*xjim + Ajim*xjre;
      bire = bire - zre;
      biim = biim - zim;
   }
   bre[idx] = bire;
   bim[idx] = biim;
}

void GPU_dbl_bals_tail
 ( int nrows, int ncols, int szt, int nbt, int degp1, int stage,
   double ***mat, double **rhs, double **sol, bool verbose )
{
   if(verbose)
   {
      cout << "GPU_dbl_bals_tail input blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : " << rhs[k][i] << endl;
      }
   }

   double *b_d;
   const size_t szrhs = nrows*sizeof(double);
   hipMalloc((void**)&b_d,szrhs);

   double *x_d;
   const size_t szsol = ncols*sizeof(double);
   hipMalloc((void**)&x_d,szsol);
   hipMemcpy(x_d,sol[stage-1],szsol,hipMemcpyHostToDevice);

   double *A_d;
   const size_t szmat = nrows*ncols*sizeof(double);
   hipMalloc((void**)&A_d,szmat);

   double *A_h = new double[szmat];

   for(int k=stage; k<degp1; k++)
   {
      if(verbose)
         cout << "GPU_dbl_bals_tail launches " << nbt
              << " thread blocks in step " << k-stage << endl;

      int idx=0;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++) A_h[idx++] = mat[k-stage+1][i][j];
      
      hipMemcpy(b_d,rhs[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(A_d,A_h,szmat,hipMemcpyHostToDevice);

      if(verbose)
         cout << "nbt = " << nbt << ", szt = " << szt
              << ", ncols = " << ncols << endl;

      dbl_bals_tail<<<nbt,szt>>>(ncols,szt,A_d,x_d,b_d);
      
      if(verbose)
         cout << "copying block " << k << " of right hand side ..." << endl;

      hipMemcpy(rhs[k],b_d,szrhs,hipMemcpyDeviceToHost);
   }
   free(A_h);

   if(verbose)
   {
      cout << "GPU_dbl_bals_tail copied blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : " << rhs[k][i] << endl;
      }
   }
   hipFree(b_d); hipFree(x_d); hipFree(A_d);
}

void GPU_cmplx_bals_tail
 ( int nrows, int ncols, int szt, int nbt, int degp1, int stage,
   double ***matre, double ***matim, double **rhsre, double **rhsim,
   double **solre, double **solim, bool verbose )
{
   if(verbose)
   {
      cout << "GPU_cmplx_bals_tail input blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : "
                 << rhsre[k][i] << "  " << rhsim[k][i] << endl;
      }
   }
   double *bre_d;
   double *bim_d;
   const size_t szrhs = nrows*sizeof(double);
   hipMalloc((void**)&bre_d,szrhs);
   hipMalloc((void**)&bim_d,szrhs);

   double *xre_d;
   double *xim_d;
   const size_t szsol = ncols*sizeof(double);
   hipMalloc((void**)&xre_d,szsol);
   hipMalloc((void**)&xim_d,szsol);
   hipMemcpy(xre_d,solre[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xim_d,solim[stage-1],szsol,hipMemcpyHostToDevice);

   if(verbose)
   {
      cout << "GPU_cmplx_bals_tail solution x :" << endl;
      for(int i=0; i<ncols; i++)
         cout << solre[stage-1][i] << "  " << solim[stage-1][i] << endl;
   }
   double *Are_d;
   double *Aim_d;
   const size_t szmat = nrows*ncols*sizeof(double);
   hipMalloc((void**)&Are_d,szmat);
   hipMalloc((void**)&Aim_d,szmat);

   double *Are_h = new double[szmat];
   double *Aim_h = new double[szmat];

   for(int k=stage; k<degp1; k++)
   {
      if(verbose)
         cout << "GPU_cmplx_bals_tail launches " << nbt
              << " thread blocks in step " << k-stage << endl;

      int idx=0;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++)
         {
            Are_h[idx]   = matre[k-stage+1][i][j];
            Aim_h[idx++] = matim[k-stage+1][i][j];
         }
      
      hipMemcpy(bre_d,rhsre[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bim_d,rhsim[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(Are_d,Are_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Aim_d,Aim_h,szmat,hipMemcpyHostToDevice);

      if(verbose)
         cout << "nbt = " << nbt << ", szt = " << szt
              << ", ncols = " << ncols << endl;

      cmplx_bals_tail<<<nbt,szt>>>
         (ncols,szt,Are_d,Aim_d,xre_d,xim_d,bre_d,bim_d);
      
      if(verbose)
         cout << "copying block " << k << " of right hand side ..." << endl;

      hipMemcpy(rhsre[k],bre_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsim[k],bim_d,szrhs,hipMemcpyDeviceToHost);
   }
   free(Are_h); free(Aim_h);

   if(verbose)
   {
      cout << "GPU_cmplx_bals_tail copied blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : "
                 << rhsre[k][i] << "  " << rhsim[k][i] << endl;
      }
   }
   hipFree(bre_d); hipFree(bim_d);
   hipFree(xre_d); hipFree(xim_d);
   hipFree(Are_d); hipFree(Aim_d);
}

void GPU_dbl_linear_residue
 ( int dim, int degp1, int szt, int nbt,
   double ***mat, double **rhs, double **sol,
   double **resvec, double *resmax,
   double *lapms, long long int *add, long long int *mul,
   int vrblvl )
{
   double *r_d;
   const size_t szrhs = dim*sizeof(double);
   hipMalloc((void**)&r_d,szrhs);

   double *x_d;
   const size_t szsol = dim*sizeof(double);
   hipMalloc((void**)&x_d,szsol);

   double *A_d;
   const size_t szmat = dim*dim*sizeof(double);
   hipMalloc((void**)&A_d,szmat);

   double *A_h = new double[dim*dim];

   *add = 0; // initialize number of additions
   *mul = 0; // initialize number of multiplications

   for(int i=0; i<degp1; i++)  // compute i-th residual vector
   {
      hipMemcpy(r_d,rhs[i],szrhs,hipMemcpyHostToDevice);

      for(int j=0; j<=i; j++)  // multiply mat[j] with sol[i-j]
      {
         int idx=0;
         for(int i1=0; i1<dim; i1++)
            for(int j1=0; j1<dim; j1++) A_h[idx++] = mat[j][i1][j1];
      
         hipMemcpy(A_d,A_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(x_d,sol[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(A_d,A_h,szmat,hipMemcpyHostToDevice);

         if(vrblvl > 0)
            cout << "GPU_dbl_linear_residue launches " << nbt
                 << " thread blocks in step " << i << ", " << j << endl;

         hipEvent_t start,stop;       // to measure time spent by kernels 
         hipEventCreate(&start);
         hipEventCreate(&stop);
         float milliseconds;

         hipEventRecord(start);
         dbl_bals_tail<<<nbt,szt>>>(dim,szt,A_d,x_d,r_d);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *lapms += milliseconds;
         flopcount_dbl_bals_tail(dim,add,mul);
      }
      hipMemcpy(resvec[i],r_d,szrhs,hipMemcpyDeviceToHost);
   }
   if(vrblvl > 1)
   {
      for(int i=0; i<degp1; i++) 
      {
         cout << "Solution vector " << i << " :" << endl;
         for(int j=0; j<dim; j++) cout << sol[i][j] << endl;
         cout << "Residual vector " << i << " :" << endl;
         for(int j=0; j<dim; j++) cout << resvec[i][j] << endl;
      }
   }
   *resmax = 0.0;
   for(int i=0; i<degp1; i++)
   {
      double *ri = resvec[i];
      for(int j=0; j<dim; j++)
         if(abs(ri[j]) > *resmax) *resmax = abs(ri[j]);
   }
   free(A_h);

   hipFree(r_d); hipFree(x_d); hipFree(A_d);
}

void GPU_cmplx_linear_residue
 ( int dim, int degp1, int szt, int nbt,
   double ***matre, double ***matim, double **rhsre, double **rhsim,
   double **solre, double **solim,
   double **resvecre, double **resvecim, double *resmax,
   double *lapms, long long int *add, long long int *mul,
   int vrblvl )
{
   double *rre_d;
   double *rim_d;
   const size_t szrhs = dim*sizeof(double);
   hipMalloc((void**)&rre_d,szrhs);
   hipMalloc((void**)&rim_d,szrhs);

   double *xre_d;
   double *xim_d;
   const size_t szsol = dim*sizeof(double);
   hipMalloc((void**)&xre_d,szsol);
   hipMalloc((void**)&xim_d,szsol);

   double *Are_d;
   double *Aim_d;
   const size_t szmat = dim*dim*sizeof(double);
   hipMalloc((void**)&Are_d,szmat);
   hipMalloc((void**)&Aim_d,szmat);

   double *Are_h = new double[dim*dim];
   double *Aim_h = new double[dim*dim];

   *add = 0; // initialize number of additions
   *mul = 0; // initialize number of multiplications

   for(int i=0; i<degp1; i++)  // compute i-th residual vector
   {
      hipMemcpy(rre_d,rhsre[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rim_d,rhsim[i],szrhs,hipMemcpyHostToDevice);

      for(int j=0; j<=i; j++)  // multiply mat[j] with sol[i-j]
      {
         int idx=0;
         for(int i1=0; i1<dim; i1++)
            for(int j1=0; j1<dim; j1++)
            {
               Are_h[idx]   = matre[j][i1][j1];
               Aim_h[idx++] = matim[j][i1][j1];
            }
      
         hipMemcpy(Are_d,Are_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aim_d,Aim_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(xre_d,solre[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xim_d,solim[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(Are_d,Are_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aim_d,Aim_h,szmat,hipMemcpyHostToDevice);

         if(vrblvl > 0)
            cout << "GPU_cmplx_linear_residue launches " << nbt
                 << " thread blocks in step " << i << ", " << j << endl;

         hipEvent_t start,stop;       // to measure time spent by kernels 
         hipEventCreate(&start);
         hipEventCreate(&stop);
         float milliseconds;

         hipEventRecord(start);
         cmplx_bals_tail<<<nbt,szt>>>
            (dim,szt,Are_d,Aim_d,xre_d,xim_d,rre_d,rim_d);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *lapms += milliseconds;
         flopcount_cmplx_bals_tail(dim,add,mul);
      }
      hipMemcpy(resvecre[i],rre_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecim[i],rim_d,szrhs,hipMemcpyDeviceToHost);
   }
   if(vrblvl > 1)
   {
      for(int i=0; i<degp1; i++) 
      {
         cout << "Solution vector " << i << " :" << endl;
         for(int j=0; j<dim; j++)
            cout << solre[i][j] << "  " << solim[i][j] << endl;

         cout << "Residual vector " << i << " :" << endl;
         for(int j=0; j<dim; j++)
            cout << resvecre[i][j] << "  " << resvecim[i][j] << endl;
      }
   }
   *resmax = 0.0;
   for(int i=0; i<degp1; i++)
   {
      double *rire = resvecre[i];
      double *riim = resvecim[i];

      for(int j=0; j<dim; j++)
         if(abs(rire[j]) + abs(riim[j]) > *resmax)
            *resmax = abs(rire[j]) + abs(riim[j]);
   }
   free(Are_h); free(Aim_h);

   hipFree(rre_d); hipFree(rim_d);
   hipFree(xre_d); hipFree(xim_d);
   hipFree(Are_d); hipFree(Aim_d);
}
