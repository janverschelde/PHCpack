#include "hip/hip_runtime.h"
// The file dbl8_tail_kernels.cu defines the functions with prototypes in
// the file dbl8_tail_kernels.h.

#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <cmath>
#include <hip/hip_vector_types.h>
#ifdef gpufun
#include "double_double_gpufun.cu"
#include "quad_double_gpufun.cu"
#include "octo_double_gpufun.cu"
#endif
#include "dbl8_tail_kernels.h"
#include "dbl_bals_flopcounts.h"

using namespace std;

__global__ void dbl8_bals_tail
 ( int ncols, int szt,
   double *Ahihihi, double *Alohihi, double *Ahilohi, double *Alolohi,
   double *Ahihilo, double *Alohilo, double *Ahilolo, double *Alololo,
   double *xhihihi, double *xlohihi, double *xhilohi, double *xlolohi,
   double *xhihilo, double *xlohilo, double *xhilolo, double *xlololo,
   double *bhihihi, double *blohihi, double *bhilohi, double *blolohi,
   double *bhihilo, double *blohilo, double *bhilolo, double *blololo )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int idx = bdx*szt + tdx; // thread tdx updates b[idx]

   double Ajhihihi;                 // register for Ahihihi[idx][j]
   double Ajlohihi;                 // register for Alohihi[idx][j]
   double Ajhilohi;                 // register for Ahilohi[idx][j]
   double Ajlolohi;                 // register for Alolohi[idx][j]
   double Ajhihilo;                 // register for Ahihilo[idx][j]
   double Ajlohilo;                 // register for Alohilo[idx][j]
   double Ajhilolo;                 // register for Ahilolo[idx][j]
   double Ajlololo;                 // register for Alololo[idx][j]
   double xjhihihi;                 // register for xhihihi[j]
   double xjlohihi;                 // register for xlohihi[j]
   double xjhilohi;                 // register for xhilohi[j]
   double xjlolohi;                 // register for xlolohi[j]
   double xjhihilo;                 // register for xhihilo[j]
   double xjlohilo;                 // register for xlohilo[j]
   double xjhilolo;                 // register for xhilolo[j]
   double xjlololo;                 // register for xlololo[j]
   double bihihihi = bhihihi[idx];  // register for bhihihi[idx]
   double bilohihi = blohihi[idx];  // register for blohihi[idx]
   double bihilohi = bhilohi[idx];  // register for bhilohi[idx]
   double bilolohi = blolohi[idx];  // register for blolohi[idx]
   double bihihilo = bhihilo[idx];  // register for bhihilo[idx]
   double bilohilo = blohilo[idx];  // register for blohilo[idx]
   double bihilolo = bhilolo[idx];  // register for bhilolo[idx]
   double bilololo = blololo[idx];  // register for blololo[idx]
   double acchihihi,acclohihi,acchilohi,acclolohi;
   double acchihilo,acclohilo,acchilolo,acclololo;

   int offset = idx*ncols;

   for(int j=0; j<ncols; j++)
   {
      Ajhihihi = Ahihihi[offset+j];
      Ajlohihi = Alohihi[offset+j];
      Ajhilohi = Ahilohi[offset+j];
      Ajlolohi = Alolohi[offset+j];
      Ajhihilo = Ahihilo[offset+j];
      Ajlohilo = Alohilo[offset+j];
      Ajhilolo = Ahilolo[offset+j];
      Ajlololo = Alololo[offset+j];
      xjhihihi = xhihihi[j];
      xjlohihi = xlohihi[j];
      xjhilohi = xhilohi[j];
      xjlolohi = xlolohi[j];
      xjhihilo = xhihilo[j];
      xjlohilo = xlohilo[j];
      xjhilolo = xhilolo[j];
      xjlololo = xlololo[j];
      // bi = bi - Aj*xj;
      odg_mul(Ajhihihi,Ajlohihi,Ajhilohi,Ajlolohi,
              Ajhihilo,Ajlohilo,Ajhilolo,Ajlololo,
              xjhihihi,xjlohihi,xjhilohi,xjlolohi,
              xjhihilo,xjlohilo,xjhilolo,xjlololo,
              &acchihihi,&acclohihi,&acchilohi,&acclolohi,
              &acchihilo,&acclohilo,&acchilolo,&acclololo);
      odg_dec(&bihihihi,&bilohihi,&bihilohi,&bilolohi,
              &bihihilo,&bilohilo,&bihilolo,&bilololo,
              acchihihi,acclohihi,acchilohi,acclolohi,
              acchihilo,acclohilo,acchilolo,acclololo);
   }
   bhihihi[idx] = bihihihi;
   blohihi[idx] = bilohihi;
   bhilohi[idx] = bihilohi;
   blolohi[idx] = bilolohi;
   bhihilo[idx] = bihihilo;
   blohilo[idx] = bilohilo;
   bhilolo[idx] = bihilolo;
   blololo[idx] = bilololo;
}

__global__ void cmplx8_bals_tail
 ( int ncols, int szt,
   double *Arehihihi, double *Arelohihi, double *Arehilohi, double *Arelolohi,
   double *Arehihilo, double *Arelohilo, double *Arehilolo, double *Arelololo,
   double *Aimhihihi, double *Aimlohihi, double *Aimhilohi, double *Aimlolohi,
   double *Aimhihilo, double *Aimlohilo, double *Aimhilolo, double *Aimlololo,
   double *xrehihihi, double *xrelohihi, double *xrehilohi, double *xrelolohi,
   double *xrehihilo, double *xrelohilo, double *xrehilolo, double *xrelololo,
   double *ximhihihi, double *ximlohihi, double *ximhilohi, double *ximlolohi, 
   double *ximhihilo, double *ximlohilo, double *ximhilolo, double *ximlololo, 
   double *brehihihi, double *brelohihi, double *brehilohi, double *brelolohi,
   double *brehihilo, double *brelohilo, double *brehilolo, double *brelololo,
   double *bimhihihi, double *bimlohihi, double *bimhilohi, double *bimlolohi,
   double *bimhihilo, double *bimlohilo, double *bimhilolo, double *bimlololo )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int idx = bdx*szt + tdx; // thread tdx updates b[idx]

   double Ajrehihihi;             // register for Arehihihi[idx][j]
   double Ajrelohihi;             // register for Arelohihi[idx][j]
   double Ajrehilohi;             // register for Arehilohi[idx][j]
   double Ajrelolohi;             // register for Arelolohi[idx][j]
   double Ajrehihilo;             // register for Arehihilo[idx][j]
   double Ajrelohilo;             // register for Arelohilo[idx][j]
   double Ajrehilolo;             // register for Arehilolo[idx][j]
   double Ajrelololo;             // register for Arelololo[idx][j]
   double Ajimhihihi;             // register for Aimhihihi[idx][j]
   double Ajimlohihi;             // register for Aimlohihi[idx][j]
   double Ajimhilohi;             // register for Aimhilohi[idx][j]
   double Ajimlolohi;             // register for Aimlolohi[idx][j]
   double Ajimhihilo;             // register for Aimhihilo[idx][j]
   double Ajimlohilo;             // register for Aimlohilo[idx][j]
   double Ajimhilolo;             // register for Aimhilolo[idx][j]
   double Ajimlololo;             // register for Aimlololo[idx][j]
   double xjrehihihi;             // register for xrehihihi[j]
   double xjrelohihi;             // register for xrelohihi[j]
   double xjrehilohi;             // register for xrehilohi[j]
   double xjrelolohi;             // register for xrelolohi[j]
   double xjrehihilo;             // register for xrehihilo[j]
   double xjrelohilo;             // register for xrelohilo[j]
   double xjrehilolo;             // register for xrehilolo[j]
   double xjrelololo;             // register for xrelololo[j]
   double xjimhihihi;             // register for ximhihihi[j]
   double xjimlohihi;             // register for ximlohihi[j]
   double xjimhilohi;             // register for ximhilohi[j]
   double xjimlolohi;             // register for ximlolohi[j]
   double xjimhihilo;             // register for ximhihilo[j]
   double xjimlohilo;             // register for ximlohilo[j]
   double xjimhilolo;             // register for ximhilolo[j]
   double xjimlololo;             // register for ximlololo[j]
   double birehihihi = brehihihi[idx];  // register for brehihihi[idx]
   double birelohihi = brelohihi[idx];  // register for brelohihi[idx]
   double birehilohi = brehilohi[idx];  // register for brehilohi[idx]
   double birelolohi = brelolohi[idx];  // register for brelolohi[idx]
   double birehihilo = brehihilo[idx];  // register for brehihilo[idx]
   double birelohilo = brelohilo[idx];  // register for brelohilo[idx]
   double birehilolo = brehilolo[idx];  // register for brehilolo[idx]
   double birelololo = brelololo[idx];  // register for brelololo[idx]
   double biimhihihi = bimhihihi[idx];  // register for bimhihihi[idx]
   double biimlohihi = bimlohihi[idx];  // register for bimlohihi[idx]
   double biimhilohi = bimhilohi[idx];  // register for bimhilohi[idx]
   double biimlolohi = bimlolohi[idx];  // register for bimlolohi[idx]
   double biimhihilo = bimhihilo[idx];  // register for bimhihilo[idx]
   double biimlohilo = bimlohilo[idx];  // register for bimlohilo[idx]
   double biimhilolo = bimhilolo[idx];  // register for bimhilolo[idx]
   double biimlololo = bimlololo[idx];  // register for bimlololo[idx]
   double acchihihi,acclohihi,acchilohi,acclolohi;
   double acchihilo,acclohilo,acchilolo,acclololo;

   int offset = idx*ncols;

   for(int j=0; j<ncols; j++)
   {
      Ajrehihihi = Arehihihi[offset+j];
      Ajrelohihi = Arelohihi[offset+j];
      Ajrehilohi = Arehilohi[offset+j];
      Ajrelolohi = Arelolohi[offset+j];
      Ajrehihilo = Arehihilo[offset+j];
      Ajrelohilo = Arelohilo[offset+j];
      Ajrehilolo = Arehilolo[offset+j];
      Ajrelololo = Arelololo[offset+j];
      Ajimhihihi = Aimhihihi[offset+j];
      Ajimlohihi = Aimlohihi[offset+j];
      Ajimhilohi = Aimhilohi[offset+j];
      Ajimlolohi = Aimlolohi[offset+j];
      Ajimhihilo = Aimhihilo[offset+j];
      Ajimlohilo = Aimlohilo[offset+j];
      Ajimhilolo = Aimhilolo[offset+j];
      Ajimlololo = Aimlololo[offset+j];
      xjrehihihi = xrehihihi[j];
      xjrelohihi = xrelohihi[j];
      xjrehilohi = xrehilohi[j];
      xjrelolohi = xrelolohi[j];
      xjrehihilo = xrehihilo[j];
      xjrelohilo = xrelohilo[j];
      xjrehilolo = xrehilolo[j];
      xjrelololo = xrelololo[j];
      xjimhihihi = ximhihihi[j];
      xjimlohihi = ximlohihi[j];
      xjimhilohi = ximhilohi[j];
      xjimlolohi = ximlolohi[j];
      xjimhihilo = ximhihilo[j];
      xjimlohilo = ximlohilo[j];
      xjimhilolo = ximhilolo[j];
      xjimlololo = ximlololo[j];
      // bi = bi - Aj*xj;
      // zre = Ajre*xjre - Ajim*xjim;
      // bire = bire - zre;
      odg_mul(Ajrehihihi,Ajrelohihi,Ajrehilohi,Ajrelolohi,
              Ajrehihilo,Ajrelohilo,Ajrehilolo,Ajrelololo,
              xjrehihihi,xjrelohihi,xjrehilohi,xjrelolohi,
              xjrehihilo,xjrelohilo,xjrehilolo,xjrelololo,
              &acchihihi,&acclohihi,&acchilohi,&acclolohi,
              &acchihilo,&acclohilo,&acchilolo,&acclololo);
      odg_dec(&birehihihi,&birelohihi,&birehilohi,&birelolohi,
              &birehihilo,&birelohilo,&birehilolo,&birelololo,
              acchihihi,acclohihi,acchilohi,acclolohi,
              acchihilo,acclohilo,acchilolo,acclololo);
      odg_mul(Ajimhihihi,Ajimlohihi,Ajimhilohi,Ajimlolohi,
              Ajimhihilo,Ajimlohilo,Ajimhilolo,Ajimlololo,
              xjimhihihi,xjimlohihi,xjimhilohi,xjimlolohi,
              xjimhihilo,xjimlohilo,xjimhilolo,xjimlololo,
              &acchihihi,&acclohihi,&acchilohi,&acclolohi,
              &acchihilo,&acclohilo,&acchilolo,&acclololo);
      odg_inc(&birehihihi,&birelohihi,&birehilohi,&birelolohi,
              &birehihilo,&birelohilo,&birehilolo,&birelololo,
              acchihihi,acclohihi,acchilohi,acclolohi,
              acchihilo,acclohilo,acchilolo,acclololo);
      // zim = Ajre*xjim + Ajim*xjre;
      // biim = biim - zim;
      odg_mul(Ajrehihihi,Ajrelohihi,Ajrehilohi,Ajrelolohi,
              Ajrehihilo,Ajrelohilo,Ajrehilolo,Ajrelololo,
              xjimhihihi,xjimlohihi,xjimhilohi,xjimlolohi,
              xjimhihilo,xjimlohilo,xjimhilolo,xjimlololo,
              &acchihihi,&acclohihi,&acchilohi,&acclolohi,
              &acchihilo,&acclohilo,&acchilolo,&acclololo);
      odg_dec(&biimhihihi,&biimlohihi,&biimhilohi,&biimlolohi,
              &biimhihilo,&biimlohilo,&biimhilolo,&biimlololo,
              acchihihi,acclohihi,acchilohi,acclolohi,
              acchihilo,acclohilo,acchilolo,acclololo);
      odg_mul(Ajimhihihi,Ajimlohihi,Ajimhilohi,Ajimlolohi,
              Ajimhihilo,Ajimlohilo,Ajimhilolo,Ajimlololo,
              xjrehihihi,xjrelohihi,xjrehilohi,xjrelolohi,
              xjrehihilo,xjrelohilo,xjrehilolo,xjrelololo,
              &acchihihi,&acclohihi,&acchilohi,&acclolohi,
              &acchihilo,&acclohilo,&acchilolo,&acclololo);
      odg_dec(&biimhihihi,&biimlohihi,&biimhilohi,&biimlolohi,
              &biimhihilo,&biimlohilo,&biimhilolo,&biimlololo,
              acchihihi,acclohihi,acchilohi,acclolohi,
              acchihilo,acclohilo,acchilolo,acclololo);
   }
   brehihihi[idx] = birehihihi;
   brelohihi[idx] = birelohihi;
   brehilohi[idx] = birehilohi;
   brelolohi[idx] = birelolohi;
   brehihilo[idx] = birehihilo;
   brelohilo[idx] = birelohilo;
   brehilolo[idx] = birehilolo;
   brelololo[idx] = birelololo;
   bimhihihi[idx] = biimhihihi;
   bimlohihi[idx] = biimlohihi;
   bimhilohi[idx] = biimhilohi;
   bimlolohi[idx] = biimlolohi;
   bimhihilo[idx] = biimhihilo;
   bimlohilo[idx] = biimlohilo;
   bimhilolo[idx] = biimhilolo;
   bimlololo[idx] = biimlololo;
}

void write_dbl8_balsflops ( int ctype, int ncols, float lapsms )
{
   cout << fixed << setprecision(3);
   cout << "Time spent for b = b - A*x : " << lapsms
        << " milliseconds." << endl;

   long long int flopcnt;
   if(ctype == 0)
      flopcnt = 270*ncols*ncols + 1742*ncols*ncols;
      // as many + as * in one inner product
   else
      flopcnt = 4*270*ncols*ncols + 4*1742*ncols*ncols;
      // for complex *: 2 ops for +, 6 for *, which is 8 in total

   cout << "    Total number of floating-point operations : "
        << flopcnt << endl;

   long long int bytecnt;

   if(ctype == 0)
      bytecnt = 8*ncols*ncols;
   else
      bytecnt = 16*ncols*ncols;

   cout << "    Total number of bytes : " << bytecnt << endl;

   double intensity = ((double) flopcnt)/bytecnt;
   cout << "     Arithmetic intensity : "
        << scientific << setprecision(3) << intensity
        << " #flops/#bytes" << endl;

   double kernflops = 1000.0*((double) flopcnt)/lapsms;
   // double wallflops = ((double) flopcnt)/timelapsed;
   const int gigacnt = pow(2.0,30);

   cout << "Kernel Time Flops : "
        << scientific << setprecision(3) << kernflops;
   cout << fixed << setprecision(3)
        << " = " << kernflops/gigacnt << " Gigaflops" << endl;
/*
   cout << " Wall Clock Flops : "
        << scientific << setprecision(3) << wallflops;
   cout << fixed << setprecision(3)
        << " = " << wallflops/gigacnt << " Gigaflops" << endl;
 */
}

void GPU_dbl8_bals_tail
 ( int nrows, int ncols, int szt, int nbt, int degp1, int stage,
   double ***mathihihi, double ***matlohihi,
   double ***mathilohi, double ***matlolohi,
   double ***mathihilo, double ***matlohilo,
   double ***mathilolo, double ***matlololo,
   double **rhshihihi, double **rhslohihi,
   double **rhshilohi, double **rhslolohi,
   double **rhshihilo, double **rhslohilo,
   double **rhshilolo, double **rhslololo,
   double **solhihihi, double **sollohihi,
   double **solhilohi, double **sollolohi,
   double **solhihilo, double **sollohilo,
   double **solhilolo, double **sollololo,
   double *totupdlapsedms, int vrblvl )
{
   if(vrblvl > 1)
   {
      cout << "GPU_dbl8_bals_tail input blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : "
                 << rhshihihi[k][i] << "  " << rhslohihi[k][i] << "  "
                 << rhshilohi[k][i] << "  " << rhslolohi[k][i] << endl
                 << "  "
                 << rhshihilo[k][i] << "  " << rhslohilo[k][i] << "  "
                 << rhshilolo[k][i] << "  " << rhslololo[k][i] << endl;
      }
   }
   double *bhihihi_d;
   double *blohihi_d;
   double *bhilohi_d;
   double *blolohi_d;
   double *bhihilo_d;
   double *blohilo_d;
   double *bhilolo_d;
   double *blololo_d;
   const size_t szrhs = nrows*sizeof(double);
   hipMalloc((void**)&bhihihi_d,szrhs);
   hipMalloc((void**)&blohihi_d,szrhs);
   hipMalloc((void**)&bhilohi_d,szrhs);
   hipMalloc((void**)&blolohi_d,szrhs);
   hipMalloc((void**)&bhihilo_d,szrhs);
   hipMalloc((void**)&blohilo_d,szrhs);
   hipMalloc((void**)&bhilolo_d,szrhs);
   hipMalloc((void**)&blololo_d,szrhs);

   double *xhihihi_d;
   double *xlohihi_d;
   double *xhilohi_d;
   double *xlolohi_d;
   double *xhihilo_d;
   double *xlohilo_d;
   double *xhilolo_d;
   double *xlololo_d;
   const size_t szsol = ncols*sizeof(double);
   hipMalloc((void**)&xhihihi_d,szsol);
   hipMalloc((void**)&xlohihi_d,szsol);
   hipMalloc((void**)&xhilohi_d,szsol);
   hipMalloc((void**)&xlolohi_d,szsol);
   hipMalloc((void**)&xhihilo_d,szsol);
   hipMalloc((void**)&xlohilo_d,szsol);
   hipMalloc((void**)&xhilolo_d,szsol);
   hipMalloc((void**)&xlololo_d,szsol);
   hipMemcpy(xhihihi_d,solhihihi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xlohihi_d,sollohihi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xhilohi_d,solhilohi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xlolohi_d,sollolohi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xhihilo_d,solhihilo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xlohilo_d,sollohilo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xhilolo_d,solhilolo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xlololo_d,sollololo[stage-1],szsol,hipMemcpyHostToDevice);

   double *Ahihihi_d;
   double *Alohihi_d;
   double *Ahilohi_d;
   double *Alolohi_d;
   double *Ahihilo_d;
   double *Alohilo_d;
   double *Ahilolo_d;
   double *Alololo_d;
   const size_t szmat = nrows*ncols*sizeof(double);
   hipMalloc((void**)&Ahihihi_d,szmat);
   hipMalloc((void**)&Alohihi_d,szmat);
   hipMalloc((void**)&Ahilohi_d,szmat);
   hipMalloc((void**)&Alolohi_d,szmat);
   hipMalloc((void**)&Ahihilo_d,szmat);
   hipMalloc((void**)&Alohilo_d,szmat);
   hipMalloc((void**)&Ahilolo_d,szmat);
   hipMalloc((void**)&Alololo_d,szmat);

   double *Ahihihi_h = new double[szmat];
   double *Alohihi_h = new double[szmat];
   double *Ahilohi_h = new double[szmat];
   double *Alolohi_h = new double[szmat];
   double *Ahihilo_h = new double[szmat];
   double *Alohilo_h = new double[szmat];
   double *Ahilolo_h = new double[szmat];
   double *Alololo_h = new double[szmat];

   for(int k=stage; k<degp1; k++)
   {
      if(vrblvl > 1)
         cout << "GPU_dbl8_bals_tail launches " << nbt
              << " thread blocks in step " << k-stage << endl;

      int idx=0;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++)
         {
            Ahihihi_h[idx]   = mathihihi[k-stage+1][i][j];
            Alohihi_h[idx]   = matlohihi[k-stage+1][i][j];
            Ahilohi_h[idx]   = mathilohi[k-stage+1][i][j];
            Alolohi_h[idx]   = matlolohi[k-stage+1][i][j];
            Ahihilo_h[idx]   = mathihilo[k-stage+1][i][j];
            Alohilo_h[idx]   = matlohilo[k-stage+1][i][j];
            Ahilolo_h[idx]   = mathilolo[k-stage+1][i][j];
            Alololo_h[idx++] = matlololo[k-stage+1][i][j];
         }

      hipMemcpy(bhihihi_d,rhshihihi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(blohihi_d,rhslohihi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bhilohi_d,rhshilohi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(blolohi_d,rhslolohi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bhihilo_d,rhshihilo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(blohilo_d,rhslohilo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bhilolo_d,rhshilolo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(blololo_d,rhslololo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(Ahihihi_d,Ahihihi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Alohihi_d,Alohihi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Ahilohi_d,Ahilohi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Alolohi_d,Alolohi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Ahihilo_d,Ahihilo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Alohilo_d,Alohilo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Ahilolo_d,Ahilolo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Alololo_d,Alololo_h,szmat,hipMemcpyHostToDevice);

      if(vrblvl > 1)
         cout << "nbt = " << nbt << ", szt = " << szt
              << ", ncols = " << ncols << endl;

      hipEvent_t start,stop;       // to measure time spent by kernels 
      hipEventCreate(&start);
      hipEventCreate(&stop);
      float milliseconds;

      hipEventRecord(start);
      dbl8_bals_tail<<<nbt,szt>>>
          (ncols,szt,
           Ahihihi_d,Alohihi_d,Ahilohi_d,Alolohi_d,
           Ahihilo_d,Alohilo_d,Ahilolo_d,Alololo_d,
           xhihihi_d,xlohihi_d,xhilohi_d,xlolohi_d,
           xhihilo_d,xlohilo_d,xhilolo_d,xlololo_d,
           bhihihi_d,blohihi_d,bhilohi_d,blolohi_d,
           bhihilo_d,blohilo_d,bhilolo_d,blololo_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);

      *totupdlapsedms += milliseconds;

      if(vrblvl > 0) write_dbl8_balsflops(0,ncols,milliseconds);
      
      if(vrblvl > 1)
         cout << "copying block " << k << " of right hand side ..." << endl;

      hipMemcpy(rhshihihi[k],bhihihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhslohihi[k],blohihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhshilohi[k],bhilohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhslolohi[k],blolohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhshihilo[k],bhihilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhslohilo[k],blohilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhshilolo[k],bhilolo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhslololo[k],blololo_d,szrhs,hipMemcpyDeviceToHost);
   }
   free(Ahihihi_h); free(Alohihi_h); free(Ahilohi_h); free(Alolohi_h);
   free(Ahihilo_h); free(Alohilo_h); free(Ahilolo_h); free(Alololo_h);

   if(vrblvl > 1)
   {
      cout << "GPU_dbl8_bals_tail copied blocks of rhs :" << endl;

      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : "
                 << rhshihihi[k][i] << "  " << rhslohihi[k][i] << "  "
                 << rhshilohi[k][i] << "  " << rhslolohi[k][i] << endl
                 << "  "
                 << rhshihilo[k][i] << "  " << rhslohilo[k][i] << "  "
                 << rhshilolo[k][i] << "  " << rhslololo[k][i] << endl;
      }
   }
   hipFree(bhihihi_d); hipFree(blohihi_d);
   hipFree(bhilohi_d); hipFree(blolohi_d);
   hipFree(bhihilo_d); hipFree(blohilo_d);
   hipFree(bhilolo_d); hipFree(blololo_d);
   hipFree(xhihihi_d); hipFree(xlohihi_d);
   hipFree(xhilohi_d); hipFree(xlolohi_d);
   hipFree(xhihilo_d); hipFree(xlohilo_d);
   hipFree(xhilolo_d); hipFree(xlololo_d);
   hipFree(Ahihihi_d); hipFree(Alohihi_d);
   hipFree(Ahilohi_d); hipFree(Alolohi_d);
   hipFree(Ahihilo_d); hipFree(Alohilo_d);
   hipFree(Ahilolo_d); hipFree(Alololo_d);
}

void GPU_cmplx8_bals_tail
 ( int nrows, int ncols, int szt, int nbt, int degp1, int stage,
   double ***matrehihihi, double ***matrelohihi,
   double ***matrehilohi, double ***matrelolohi,
   double ***matrehihilo, double ***matrelohilo,
   double ***matrehilolo, double ***matrelololo,
   double ***matimhihihi, double ***matimlohihi,
   double ***matimhilohi, double ***matimlolohi,
   double ***matimhihilo, double ***matimlohilo,
   double ***matimhilolo, double ***matimlololo,
   double **rhsrehihihi, double **rhsrelohihi,
   double **rhsrehilohi, double **rhsrelolohi,
   double **rhsrehihilo, double **rhsrelohilo,
   double **rhsrehilolo, double **rhsrelololo,
   double **rhsimhihihi, double **rhsimlohihi,
   double **rhsimhilohi, double **rhsimlolohi,
   double **rhsimhihilo, double **rhsimlohilo,
   double **rhsimhilolo, double **rhsimlololo,
   double **solrehihihi, double **solrelohihi,
   double **solrehilohi, double **solrelolohi,
   double **solrehihilo, double **solrelohilo,
   double **solrehilolo, double **solrelololo,
   double **solimhihihi, double **solimlohihi,
   double **solimhilohi, double **solimlolohi,
   double **solimhihilo, double **solimlohilo,
   double **solimhilolo, double **solimlololo,
   double *totupdlapsedms, int vrblvl )
{
   if(vrblvl > 1)
   {
      cout << "GPU_cmplx8_bals_tail input blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : "
                 << rhsrehihihi[k][i] << "  " << rhsrelohihi[k][i] << endl
                 << "  "
                 << rhsrehilohi[k][i] << "  " << rhsrelolohi[k][i] << endl
                 << "  "
                 << rhsrehihilo[k][i] << "  " << rhsrelohilo[k][i] << endl
                 << "  "
                 << rhsrehilolo[k][i] << "  " << rhsrelololo[k][i] << endl
                 << "  "
                 << rhsimhihihi[k][i] << "  " << rhsimlohihi[k][i] << endl
                 << "  "
                 << rhsimhilohi[k][i] << "  " << rhsimlolohi[k][i] << endl
                 << "  "
                 << rhsimhihilo[k][i] << "  " << rhsimlohilo[k][i] << endl
                 << "  "
                 << rhsimhilolo[k][i] << "  " << rhsimlololo[k][i] << endl;
      }
   }
   double *brehihihi_d;
   double *brelohihi_d;
   double *brehilohi_d;
   double *brelolohi_d;
   double *brehihilo_d;
   double *brelohilo_d;
   double *brehilolo_d;
   double *brelololo_d;
   double *bimhihihi_d;
   double *bimlohihi_d;
   double *bimhilohi_d;
   double *bimlolohi_d;
   double *bimhihilo_d;
   double *bimlohilo_d;
   double *bimhilolo_d;
   double *bimlololo_d;
   const size_t szrhs = nrows*sizeof(double);
   hipMalloc((void**)&brehihihi_d,szrhs);
   hipMalloc((void**)&brelohihi_d,szrhs);
   hipMalloc((void**)&brehilohi_d,szrhs);
   hipMalloc((void**)&brelolohi_d,szrhs);
   hipMalloc((void**)&brehihilo_d,szrhs);
   hipMalloc((void**)&brelohilo_d,szrhs);
   hipMalloc((void**)&brehilolo_d,szrhs);
   hipMalloc((void**)&brelololo_d,szrhs);
   hipMalloc((void**)&bimhihihi_d,szrhs);
   hipMalloc((void**)&bimlohihi_d,szrhs);
   hipMalloc((void**)&bimhilohi_d,szrhs);
   hipMalloc((void**)&bimlolohi_d,szrhs);
   hipMalloc((void**)&bimhihilo_d,szrhs);
   hipMalloc((void**)&bimlohilo_d,szrhs);
   hipMalloc((void**)&bimhilolo_d,szrhs);
   hipMalloc((void**)&bimlololo_d,szrhs);

   double *xrehihihi_d;
   double *xrelohihi_d;
   double *xrehilohi_d;
   double *xrelolohi_d;
   double *xrehihilo_d;
   double *xrelohilo_d;
   double *xrehilolo_d;
   double *xrelololo_d;
   double *ximhihihi_d;
   double *ximlohihi_d;
   double *ximhilohi_d;
   double *ximlolohi_d;
   double *ximhihilo_d;
   double *ximlohilo_d;
   double *ximhilolo_d;
   double *ximlololo_d;
   const size_t szsol = ncols*sizeof(double);
   hipMalloc((void**)&xrehihihi_d,szsol);
   hipMalloc((void**)&xrelohihi_d,szsol);
   hipMalloc((void**)&xrehilohi_d,szsol);
   hipMalloc((void**)&xrelolohi_d,szsol);
   hipMalloc((void**)&xrehihilo_d,szsol);
   hipMalloc((void**)&xrelohilo_d,szsol);
   hipMalloc((void**)&xrehilolo_d,szsol);
   hipMalloc((void**)&xrelololo_d,szsol);
   hipMalloc((void**)&ximhihihi_d,szsol);
   hipMalloc((void**)&ximlohihi_d,szsol);
   hipMalloc((void**)&ximhilohi_d,szsol);
   hipMalloc((void**)&ximlolohi_d,szsol);
   hipMalloc((void**)&ximhihilo_d,szsol);
   hipMalloc((void**)&ximlohilo_d,szsol);
   hipMalloc((void**)&ximhilolo_d,szsol);
   hipMalloc((void**)&ximlololo_d,szsol);
   hipMemcpy(xrehihihi_d,solrehihihi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xrelohihi_d,solrelohihi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xrehilohi_d,solrehilohi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xrelolohi_d,solrelolohi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xrehihilo_d,solrehihilo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xrelohilo_d,solrelohilo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xrehilolo_d,solrehilolo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(xrelololo_d,solrelololo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(ximhihihi_d,solimhihihi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(ximlohihi_d,solimlohihi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(ximhilohi_d,solimhilohi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(ximlolohi_d,solimlolohi[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(ximhihilo_d,solimhihilo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(ximlohilo_d,solimlohilo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(ximhilolo_d,solimhilolo[stage-1],szsol,hipMemcpyHostToDevice);
   hipMemcpy(ximlololo_d,solimlololo[stage-1],szsol,hipMemcpyHostToDevice);

   double *Arehihihi_d;
   double *Arelohihi_d;
   double *Arehilohi_d;
   double *Arelolohi_d;
   double *Arehihilo_d;
   double *Arelohilo_d;
   double *Arehilolo_d;
   double *Arelololo_d;
   double *Aimhihihi_d;
   double *Aimlohihi_d;
   double *Aimhilohi_d;
   double *Aimlolohi_d;
   double *Aimhihilo_d;
   double *Aimlohilo_d;
   double *Aimhilolo_d;
   double *Aimlololo_d;
   const size_t szmat = nrows*ncols*sizeof(double);
   hipMalloc((void**)&Arehihihi_d,szmat);
   hipMalloc((void**)&Arelohihi_d,szmat);
   hipMalloc((void**)&Arehilohi_d,szmat);
   hipMalloc((void**)&Arelolohi_d,szmat);
   hipMalloc((void**)&Arehihilo_d,szmat);
   hipMalloc((void**)&Arelohilo_d,szmat);
   hipMalloc((void**)&Arehilolo_d,szmat);
   hipMalloc((void**)&Arelololo_d,szmat);
   hipMalloc((void**)&Aimhihihi_d,szmat);
   hipMalloc((void**)&Aimlohihi_d,szmat);
   hipMalloc((void**)&Aimhilohi_d,szmat);
   hipMalloc((void**)&Aimlolohi_d,szmat);
   hipMalloc((void**)&Aimhihilo_d,szmat);
   hipMalloc((void**)&Aimlohilo_d,szmat);
   hipMalloc((void**)&Aimhilolo_d,szmat);
   hipMalloc((void**)&Aimlololo_d,szmat);

   double *Arehihihi_h = new double[szmat];
   double *Arelohihi_h = new double[szmat];
   double *Arehilohi_h = new double[szmat];
   double *Arelolohi_h = new double[szmat];
   double *Arehihilo_h = new double[szmat];
   double *Arelohilo_h = new double[szmat];
   double *Arehilolo_h = new double[szmat];
   double *Arelololo_h = new double[szmat];
   double *Aimhihihi_h = new double[szmat];
   double *Aimlohihi_h = new double[szmat];
   double *Aimhilohi_h = new double[szmat];
   double *Aimlolohi_h = new double[szmat];
   double *Aimhihilo_h = new double[szmat];
   double *Aimlohilo_h = new double[szmat];
   double *Aimhilolo_h = new double[szmat];
   double *Aimlololo_h = new double[szmat];

   for(int k=stage; k<degp1; k++)
   {
      if(vrblvl > 1)
         cout << "GPU_cmplx8_bals_tail launches " << nbt
              << " thread blocks in step " << k-stage << endl;

      int idx=0;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++)
         {
            Arehihihi_h[idx]   = matrehihihi[k-stage+1][i][j];
            Arelohihi_h[idx]   = matrelohihi[k-stage+1][i][j];
            Arehilohi_h[idx]   = matrehilohi[k-stage+1][i][j];
            Arelolohi_h[idx]   = matrelolohi[k-stage+1][i][j];
            Arehihilo_h[idx]   = matrehihilo[k-stage+1][i][j];
            Arelohilo_h[idx]   = matrelohilo[k-stage+1][i][j];
            Arehilolo_h[idx]   = matrehilolo[k-stage+1][i][j];
            Arelololo_h[idx]   = matrelololo[k-stage+1][i][j];
            Aimhihihi_h[idx]   = matimhihihi[k-stage+1][i][j];
            Aimlohihi_h[idx]   = matimlohihi[k-stage+1][i][j];
            Aimhilohi_h[idx]   = matimhilohi[k-stage+1][i][j];
            Aimlolohi_h[idx]   = matimlolohi[k-stage+1][i][j];
            Aimhihilo_h[idx]   = matimhihilo[k-stage+1][i][j];
            Aimlohilo_h[idx]   = matimlohilo[k-stage+1][i][j];
            Aimhilolo_h[idx]   = matimhilolo[k-stage+1][i][j];
            Aimlololo_h[idx++] = matimlololo[k-stage+1][i][j];
         }
      
      hipMemcpy(brehihihi_d,rhsrehihihi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(brelohihi_d,rhsrelohihi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(brehilohi_d,rhsrehilohi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(brelolohi_d,rhsrelolohi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(brehihilo_d,rhsrehihilo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(brelohilo_d,rhsrelohilo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(brehilolo_d,rhsrehilolo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(brelololo_d,rhsrelololo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bimhihihi_d,rhsimhihihi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bimlohihi_d,rhsimlohihi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bimhilohi_d,rhsimhilohi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bimlolohi_d,rhsimlolohi[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bimhihilo_d,rhsimhihilo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bimlohilo_d,rhsimlohilo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bimhilolo_d,rhsimhilolo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(bimlololo_d,rhsimlololo[k],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(Arehihihi_d,Arehihihi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Arelohihi_d,Arelohihi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Arehilohi_d,Arehilohi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Arelolohi_d,Arelolohi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Arehihilo_d,Arehihilo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Arelohilo_d,Arelohilo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Arehilolo_d,Arehilolo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Arelololo_d,Arelololo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Aimhihihi_d,Aimhihihi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Aimlohihi_d,Aimlohihi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Aimhilohi_d,Aimhilohi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Aimlolohi_d,Aimlolohi_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Aimhihilo_d,Aimhihilo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Aimlohilo_d,Aimlohilo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Aimhilolo_d,Aimhilolo_h,szmat,hipMemcpyHostToDevice);
      hipMemcpy(Aimlololo_d,Aimlololo_h,szmat,hipMemcpyHostToDevice);

      if(vrblvl > 1)
         cout << "nbt = " << nbt << ", szt = " << szt
              << ", ncols = " << ncols << endl;

      hipEvent_t start,stop;       // to measure time spent by kernels 
      hipEventCreate(&start);
      hipEventCreate(&stop);
      float milliseconds;

      hipEventRecord(start);
      cmplx8_bals_tail<<<nbt,szt>>>
         (ncols,szt,Arehihihi_d,Arelohihi_d,Arehilohi_d,Arelolohi_d,
                    Arehihilo_d,Arelohilo_d,Arehilolo_d,Arelololo_d,
                    Aimhihihi_d,Aimlohihi_d,Aimhilohi_d,Aimlolohi_d,
                    Aimhihilo_d,Aimlohilo_d,Aimhilolo_d,Aimlololo_d,
          xrehihihi_d,xrelohihi_d,xrehilohi_d,xrelolohi_d,
          xrehihilo_d,xrelohilo_d,xrehilolo_d,xrelololo_d,
          ximhihihi_d,ximlohihi_d,ximhilohi_d,ximlolohi_d,
          ximhihilo_d,ximlohilo_d,ximhilolo_d,ximlololo_d,
          brehihihi_d,brelohihi_d,brehilohi_d,brelolohi_d,
          brehihilo_d,brelohilo_d,brehilolo_d,brelololo_d,
          bimhihihi_d,bimlohihi_d,bimhilohi_d,bimlolohi_d,
          bimhihilo_d,bimlohilo_d,bimhilolo_d,bimlololo_d);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds,start,stop);

      *totupdlapsedms += milliseconds;

      if(vrblvl > 0) write_dbl8_balsflops(1,ncols,milliseconds);
      
      if(vrblvl > 1)
         cout << "copying block " << k << " of right hand side ..." << endl;

      hipMemcpy(rhsrehihihi[k],brehihihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsrelohihi[k],brelohihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsrehilohi[k],brehilohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsrelolohi[k],brelolohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsrehihilo[k],brehihilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsrelohilo[k],brelohilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsrehilolo[k],brehilolo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsrelololo[k],brelololo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsimhihihi[k],bimhihihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsimlohihi[k],bimlohihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsimhilohi[k],bimhilohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsimlolohi[k],bimlolohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsimhihilo[k],bimhihilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsimlohilo[k],bimlohilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsimhilolo[k],bimhilolo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(rhsimlololo[k],bimlololo_d,szrhs,hipMemcpyDeviceToHost);
   }
   free(Arehihihi_h); free(Aimhihihi_h);
   free(Arelohihi_h); free(Aimlohihi_h);
   free(Arehilohi_h); free(Aimhilohi_h);
   free(Arelolohi_h); free(Aimlolohi_h);
   free(Arehihilo_h); free(Aimhihilo_h);
   free(Arelohilo_h); free(Aimlohilo_h);
   free(Arehilolo_h); free(Aimhilolo_h);
   free(Arelololo_h); free(Aimlololo_h);

   if(vrblvl > 1)
   {
      cout << "GPU_cmplx8_bals_tail copied blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : "
                 << rhsrehihihi[k][i] << "  " << rhsrelohihi[k][i] << endl
                 << "  "
                 << rhsrehilohi[k][i] << "  " << rhsrelolohi[k][i] << endl
                 << "  "
                 << rhsrehihilo[k][i] << "  " << rhsrelohilo[k][i] << endl
                 << "  "
                 << rhsrehilolo[k][i] << "  " << rhsrelololo[k][i] << endl
                 << "  "
                 << rhsimhihihi[k][i] << "  " << rhsimlohihi[k][i] << endl
                 << "  "
                 << rhsimhilohi[k][i] << "  " << rhsimlolohi[k][i] << endl
                 << "  "
                 << rhsimhihilo[k][i] << "  " << rhsimlohilo[k][i] << endl
                 << "  "
                 << rhsimhilolo[k][i] << "  " << rhsimlololo[k][i] << endl;
      }
   }
   hipFree(brehihihi_d); hipFree(brelohihi_d);
   hipFree(brehilohi_d); hipFree(brelolohi_d);
   hipFree(brehihilo_d); hipFree(brelohilo_d);
   hipFree(brehilolo_d); hipFree(brelololo_d);
   hipFree(bimhihihi_d); hipFree(bimlohihi_d);
   hipFree(bimhilohi_d); hipFree(bimlolohi_d);
   hipFree(bimhihilo_d); hipFree(bimlohilo_d);
   hipFree(bimhilolo_d); hipFree(bimlololo_d);
   hipFree(xrehihihi_d); hipFree(xrelohihi_d);
   hipFree(xrehilohi_d); hipFree(xrelolohi_d);
   hipFree(xrehihilo_d); hipFree(xrelohilo_d);
   hipFree(xrehilolo_d); hipFree(xrelololo_d);
   hipFree(ximhihihi_d); hipFree(ximlohihi_d);
   hipFree(ximhilohi_d); hipFree(ximlolohi_d);
   hipFree(ximhihilo_d); hipFree(ximlohilo_d);
   hipFree(ximhilolo_d); hipFree(ximlololo_d);
   hipFree(Arehihihi_d); hipFree(Arelohihi_d);
   hipFree(Arehilohi_d); hipFree(Arelolohi_d);
   hipFree(Arehihilo_d); hipFree(Arelohilo_d);
   hipFree(Arehilolo_d); hipFree(Arelololo_d);
   hipFree(Aimhihihi_d); hipFree(Aimlohihi_d);
   hipFree(Aimhilohi_d); hipFree(Aimlolohi_d);
   hipFree(Aimhihilo_d); hipFree(Aimlohilo_d);
   hipFree(Aimhilolo_d); hipFree(Aimlololo_d);
}

void GPU_dbl8_linear_residue
 ( int dim, int degp1, int szt, int nbt, int tailidx,
   double ***mathihihi, double ***matlohihi,
   double ***mathilohi, double ***matlolohi,
   double ***mathihilo, double ***matlohilo,
   double ***mathilolo, double ***matlololo,
   double **rhshihihi, double **rhslohihi,
   double **rhshilohi, double **rhslolohi,
   double **rhshihilo, double **rhslohilo,
   double **rhshilolo, double **rhslololo,
   double **solhihihi, double **sollohihi,
   double **solhilohi, double **sollolohi,
   double **solhihilo, double **sollohilo,
   double **solhilolo, double **sollololo,
   double **resvechihihi, double **resveclohihi,
   double **resvechilohi, double **resveclolohi,
   double **resvechihilo, double **resveclohilo,
   double **resvechilolo, double **resveclololo,
   double *resmaxhihihi, double *resmaxlohihi,
   double *resmaxhilohi, double *resmaxlolohi,
   double *resmaxhihilo, double *resmaxlohilo,
   double *resmaxhilolo, double *resmaxlololo,
   double *totreslapsedms, long long int *add, long long int *mul,
   int vrblvl )
{
   double *rhihihi_d;
   double *rlohihi_d;
   double *rhilohi_d;
   double *rlolohi_d;
   double *rhihilo_d;
   double *rlohilo_d;
   double *rhilolo_d;
   double *rlololo_d;
   const size_t szrhs = dim*sizeof(double);
   hipMalloc((void**)&rhihihi_d,szrhs);
   hipMalloc((void**)&rlohihi_d,szrhs);
   hipMalloc((void**)&rhilohi_d,szrhs);
   hipMalloc((void**)&rlolohi_d,szrhs);
   hipMalloc((void**)&rhihilo_d,szrhs);
   hipMalloc((void**)&rlohilo_d,szrhs);
   hipMalloc((void**)&rhilolo_d,szrhs);
   hipMalloc((void**)&rlololo_d,szrhs);

   double *xhihihi_d;
   double *xlohihi_d;
   double *xhilohi_d;
   double *xlolohi_d;
   double *xhihilo_d;
   double *xlohilo_d;
   double *xhilolo_d;
   double *xlololo_d;
   const size_t szsol = dim*sizeof(double);
   hipMalloc((void**)&xhihihi_d,szsol);
   hipMalloc((void**)&xlohihi_d,szsol);
   hipMalloc((void**)&xhilohi_d,szsol);
   hipMalloc((void**)&xlolohi_d,szsol);
   hipMalloc((void**)&xhihilo_d,szsol);
   hipMalloc((void**)&xlohilo_d,szsol);
   hipMalloc((void**)&xhilolo_d,szsol);
   hipMalloc((void**)&xlololo_d,szsol);

   double *Ahihihi_d;
   double *Alohihi_d;
   double *Ahilohi_d;
   double *Alolohi_d;
   double *Ahihilo_d;
   double *Alohilo_d;
   double *Ahilolo_d;
   double *Alololo_d;
   const size_t szmat = dim*dim*sizeof(double);
   hipMalloc((void**)&Ahihihi_d,szmat);
   hipMalloc((void**)&Alohihi_d,szmat);
   hipMalloc((void**)&Ahilohi_d,szmat);
   hipMalloc((void**)&Alolohi_d,szmat);
   hipMalloc((void**)&Ahihilo_d,szmat);
   hipMalloc((void**)&Alohilo_d,szmat);
   hipMalloc((void**)&Ahilolo_d,szmat);
   hipMalloc((void**)&Alololo_d,szmat);

   double *Ahihihi_h = new double[dim*dim];
   double *Alohihi_h = new double[dim*dim];
   double *Ahilohi_h = new double[dim*dim];
   double *Alolohi_h = new double[dim*dim];
   double *Ahihilo_h = new double[dim*dim];
   double *Alohilo_h = new double[dim*dim];
   double *Ahilolo_h = new double[dim*dim];
   double *Alololo_h = new double[dim*dim];

   *add = 0; // initialize number of additions
   *mul = 0; // initialize number of multiplications

   if(vrblvl > 0)
      cout << "GPU_dbl8_linear_residue for deg+1 : " << degp1 << endl;

   for(int i=tailidx; i<degp1; i++)  // compute i-th residual vector
   {
      hipMemcpy(rhihihi_d,rhshihihi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rlohihi_d,rhslohihi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rhilohi_d,rhshilohi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rlolohi_d,rhslolohi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rhihilo_d,rhshihilo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rlohilo_d,rhslohilo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rhilolo_d,rhshilolo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rlololo_d,rhslololo[i],szrhs,hipMemcpyHostToDevice);

      for(int j=0; j<=(i-tailidx); j++)  // multiply mat[j] with sol[i-j]
      {
         int idx=0;
         for(int i1=0; i1<dim; i1++)
            for(int j1=0; j1<dim; j1++)
            {
               Ahihihi_h[idx]   = mathihihi[j][i1][j1];
               Alohihi_h[idx]   = matlohihi[j][i1][j1];
               Ahilohi_h[idx]   = mathilohi[j][i1][j1];
               Alolohi_h[idx]   = matlolohi[j][i1][j1];
               Ahihilo_h[idx]   = mathihilo[j][i1][j1];
               Alohilo_h[idx]   = matlohilo[j][i1][j1];
               Ahilolo_h[idx]   = mathilolo[j][i1][j1];
               Alololo_h[idx++] = matlololo[j][i1][j1];
            }
      
         hipMemcpy(Ahihihi_d,Ahihihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Alohihi_d,Alohihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Ahilohi_d,Ahilohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Alolohi_d,Alolohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Ahihilo_d,Ahihilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Alohilo_d,Alohilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Ahilolo_d,Ahilolo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Alololo_d,Alololo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(xhihihi_d,solhihihi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xlohihi_d,sollohihi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xhilohi_d,solhilohi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xlolohi_d,sollolohi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xhihilo_d,solhihilo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xlohilo_d,sollohilo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xhilolo_d,solhilolo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xlololo_d,sollololo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(Ahihihi_d,Ahihihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Alohihi_d,Alohihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Ahilohi_d,Ahilohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Alolohi_d,Alolohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Ahihilo_d,Ahihilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Alohilo_d,Alohilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Ahilolo_d,Ahilolo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Alololo_d,Alololo_h,szmat,hipMemcpyHostToDevice);

         if(vrblvl > 1)
            cout << "GPU_dbl8_linear_residue launches " << nbt
                 << " thread blocks in step " << i << ", " << j << endl;

         hipEvent_t start,stop;       // to measure time spent by kernels 
         hipEventCreate(&start);
         hipEventCreate(&stop);
         float milliseconds;

         hipEventRecord(start);
         dbl8_bals_tail<<<nbt,szt>>>
            (dim,szt,Ahihihi_d,Alohihi_d,Ahilohi_d,Alolohi_d,
                     Ahihilo_d,Alohilo_d,Ahilolo_d,Alololo_d,
                     xhihihi_d,xlohihi_d,xhilohi_d,xlolohi_d,
                     xhihilo_d,xlohilo_d,xhilolo_d,xlololo_d,
                     rhihihi_d,rlohihi_d,rhilohi_d,rlolohi_d,
                     rhihilo_d,rlohilo_d,rhilolo_d,rlololo_d);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *totreslapsedms += milliseconds;
         flopcount_dbl_bals_tail(dim,add,mul);

         if(vrblvl > 0) write_dbl8_balsflops(0,dim,milliseconds);
      }
      hipMemcpy(resvechihihi[i],rhihihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resveclohihi[i],rlohihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvechilohi[i],rhilohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resveclolohi[i],rlolohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvechihilo[i],rhihilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resveclohilo[i],rlohilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvechilolo[i],rhilolo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resveclololo[i],rlololo_d,szrhs,hipMemcpyDeviceToHost);
   }
   if(vrblvl > 1)
   {
      for(int i=tailidx; i<degp1; i++)
      {
         cout << "Solution vector " << i << " :" << endl;
         for(int j=0; j<dim; j++)
         {
            cout << solhihihi[i][j] << "  " << sollohihi[i][j] << endl;
            cout << solhilohi[i][j] << "  " << sollolohi[i][j] << endl;
            cout << solhihilo[i][j] << "  " << sollohilo[i][j] << endl;
            cout << solhilolo[i][j] << "  " << sollololo[i][j] << endl;
         }
         cout << "Residual vector " << i << " :" << endl;
         for(int j=0; j<dim; j++)
         {
            cout << resvechihihi[i][j] << "  " << resveclohihi[i][j] << endl;
            cout << resvechilohi[i][j] << "  " << resveclolohi[i][j] << endl;
            cout << resvechihilo[i][j] << "  " << resveclohilo[i][j] << endl;
            cout << resvechilolo[i][j] << "  " << resveclololo[i][j] << endl;
         }
      }
   }
   *resmaxhihihi = 0.0; *resmaxlohihi = 0.0;
   *resmaxhilohi = 0.0; *resmaxlolohi = 0.0;
   *resmaxhihilo = 0.0; *resmaxlohilo = 0.0;
   *resmaxhilolo = 0.0; *resmaxlololo = 0.0;
   
   for(int i=tailidx; i<degp1; i++)
   {
      double *rihihihi = resvechihihi[i];
      double *rilohihi = resveclohihi[i];
      double *rihilohi = resvechilohi[i];
      double *rilolohi = resveclolohi[i];
      double *rihihilo = resvechihilo[i];
      double *rilohilo = resveclohilo[i];
      double *rihilolo = resvechilolo[i];
      double *rilololo = resveclololo[i];

      for(int j=0; j<dim; j++)
         if(abs(rihihihi[j]) > *resmaxhihihi)
         {
            *resmaxhihihi = abs(rihihihi[j]);
            *resmaxlohihi = abs(rilohihi[j]);
            *resmaxhilohi = abs(rihilohi[j]);
            *resmaxlolohi = abs(rilolohi[j]);
            *resmaxhihilo = abs(rihihilo[j]);
            *resmaxlohilo = abs(rilohilo[j]);
            *resmaxhilolo = abs(rihilolo[j]);
            *resmaxlololo = abs(rilololo[j]);
         }
   }
   free(Ahihihi_h); free(Alohihi_h); free(Ahilohi_h); free(Alolohi_h);
   free(Ahihilo_h); free(Alohilo_h); free(Ahilolo_h); free(Alololo_h);

   hipFree(rhihihi_d); hipFree(rlohihi_d);
   hipFree(rhilohi_d); hipFree(rlolohi_d);
   hipFree(rhihilo_d); hipFree(rlohilo_d);
   hipFree(rhilolo_d); hipFree(rlololo_d);
   hipFree(xhihihi_d); hipFree(xlohihi_d);
   hipFree(xhilohi_d); hipFree(xlolohi_d);
   hipFree(xhihilo_d); hipFree(xlohilo_d);
   hipFree(xhilolo_d); hipFree(xlololo_d);
   hipFree(Ahihihi_d); hipFree(Alohihi_d);
   hipFree(Ahilohi_d); hipFree(Alolohi_d);
   hipFree(Ahihilo_d); hipFree(Alohilo_d);
   hipFree(Ahilolo_d); hipFree(Alololo_d);
}

void GPU_cmplx8_linear_residue
 ( int dim, int degp1, int szt, int nbt, int tailidx,
   double ***matrehihihi, double ***matrelohihi,
   double ***matrehilohi, double ***matrelolohi,
   double ***matrehihilo, double ***matrelohilo,
   double ***matrehilolo, double ***matrelololo,
   double ***matimhihihi, double ***matimlohihi,
   double ***matimhilohi, double ***matimlolohi,
   double ***matimhihilo, double ***matimlohilo,
   double ***matimhilolo, double ***matimlololo,
   double **rhsrehihihi, double **rhsrelohihi,
   double **rhsrehilohi, double **rhsrelolohi,
   double **rhsrehihilo, double **rhsrelohilo,
   double **rhsrehilolo, double **rhsrelololo,
   double **rhsimhihihi, double **rhsimlohihi, 
   double **rhsimhilohi, double **rhsimlolohi, 
   double **rhsimhihilo, double **rhsimlohilo, 
   double **rhsimhilolo, double **rhsimlololo, 
   double **solrehihihi, double **solrelohihi,
   double **solrehilohi, double **solrelolohi,
   double **solrehihilo, double **solrelohilo,
   double **solrehilolo, double **solrelololo,
   double **solimhihihi, double **solimlohihi,
   double **solimhilohi, double **solimlolohi,
   double **solimhihilo, double **solimlohilo,
   double **solimhilolo, double **solimlololo,
   double **resvecrehihihi, double **resvecrelohihi,
   double **resvecrehilohi, double **resvecrelolohi,
   double **resvecrehihilo, double **resvecrelohilo,
   double **resvecrehilolo, double **resvecrelololo,
   double **resvecimhihihi, double **resvecimlohihi,
   double **resvecimhilohi, double **resvecimlolohi,
   double **resvecimhihilo, double **resvecimlohilo,
   double **resvecimhilolo, double **resvecimlololo,
   double *resmaxhihihi, double *resmaxlohihi,
   double *resmaxhilohi, double *resmaxlolohi,
   double *resmaxhihilo, double *resmaxlohilo,
   double *resmaxhilolo, double *resmaxlololo,
   double *totreslapsedms, long long int *add, long long int *mul,
   int vrblvl )
{
   double *rrehihihi_d;
   double *rrelohihi_d;
   double *rrehilohi_d;
   double *rrelolohi_d;
   double *rrehihilo_d;
   double *rrelohilo_d;
   double *rrehilolo_d;
   double *rrelololo_d;
   double *rimhihihi_d;
   double *rimlohihi_d;
   double *rimhilohi_d;
   double *rimlolohi_d;
   double *rimhihilo_d;
   double *rimlohilo_d;
   double *rimhilolo_d;
   double *rimlololo_d;
   const size_t szrhs = dim*sizeof(double);
   hipMalloc((void**)&rrehihihi_d,szrhs);
   hipMalloc((void**)&rrelohihi_d,szrhs);
   hipMalloc((void**)&rrehilohi_d,szrhs);
   hipMalloc((void**)&rrelolohi_d,szrhs);
   hipMalloc((void**)&rrehihilo_d,szrhs);
   hipMalloc((void**)&rrelohilo_d,szrhs);
   hipMalloc((void**)&rrehilolo_d,szrhs);
   hipMalloc((void**)&rrelololo_d,szrhs);
   hipMalloc((void**)&rimhihihi_d,szrhs);
   hipMalloc((void**)&rimlohihi_d,szrhs);
   hipMalloc((void**)&rimhilohi_d,szrhs);
   hipMalloc((void**)&rimlolohi_d,szrhs);
   hipMalloc((void**)&rimhihilo_d,szrhs);
   hipMalloc((void**)&rimlohilo_d,szrhs);
   hipMalloc((void**)&rimhilolo_d,szrhs);
   hipMalloc((void**)&rimlololo_d,szrhs);

   double *xrehihihi_d;
   double *xrelohihi_d;
   double *xrehilohi_d;
   double *xrelolohi_d;
   double *xrehihilo_d;
   double *xrelohilo_d;
   double *xrehilolo_d;
   double *xrelololo_d;
   double *ximhihihi_d;
   double *ximlohihi_d;
   double *ximhilohi_d;
   double *ximlolohi_d;
   double *ximhihilo_d;
   double *ximlohilo_d;
   double *ximhilolo_d;
   double *ximlololo_d;
   const size_t szsol = dim*sizeof(double);
   hipMalloc((void**)&xrehihihi_d,szsol);
   hipMalloc((void**)&xrelohihi_d,szsol);
   hipMalloc((void**)&xrehilohi_d,szsol);
   hipMalloc((void**)&xrelolohi_d,szsol);
   hipMalloc((void**)&xrehihilo_d,szsol);
   hipMalloc((void**)&xrelohilo_d,szsol);
   hipMalloc((void**)&xrehilolo_d,szsol);
   hipMalloc((void**)&xrelololo_d,szsol);
   hipMalloc((void**)&ximhihihi_d,szsol);
   hipMalloc((void**)&ximlohihi_d,szsol);
   hipMalloc((void**)&ximhilohi_d,szsol);
   hipMalloc((void**)&ximlolohi_d,szsol);
   hipMalloc((void**)&ximhihilo_d,szsol);
   hipMalloc((void**)&ximlohilo_d,szsol);
   hipMalloc((void**)&ximhilolo_d,szsol);
   hipMalloc((void**)&ximlololo_d,szsol);

   double *Arehihihi_d;
   double *Arelohihi_d;
   double *Arehilohi_d;
   double *Arelolohi_d;
   double *Arehihilo_d;
   double *Arelohilo_d;
   double *Arehilolo_d;
   double *Arelololo_d;
   double *Aimhihihi_d;
   double *Aimlohihi_d;
   double *Aimhilohi_d;
   double *Aimlolohi_d;
   double *Aimhihilo_d;
   double *Aimlohilo_d;
   double *Aimhilolo_d;
   double *Aimlololo_d;
   const size_t szmat = dim*dim*sizeof(double);
   hipMalloc((void**)&Arehihihi_d,szmat);
   hipMalloc((void**)&Arelohihi_d,szmat);
   hipMalloc((void**)&Arehilohi_d,szmat);
   hipMalloc((void**)&Arelolohi_d,szmat);
   hipMalloc((void**)&Arehihilo_d,szmat);
   hipMalloc((void**)&Arelohilo_d,szmat);
   hipMalloc((void**)&Arehilolo_d,szmat);
   hipMalloc((void**)&Arelololo_d,szmat);
   hipMalloc((void**)&Aimhihihi_d,szmat);
   hipMalloc((void**)&Aimlohihi_d,szmat);
   hipMalloc((void**)&Aimhilohi_d,szmat);
   hipMalloc((void**)&Aimlolohi_d,szmat);
   hipMalloc((void**)&Aimhihilo_d,szmat);
   hipMalloc((void**)&Aimlohilo_d,szmat);
   hipMalloc((void**)&Aimhilolo_d,szmat);
   hipMalloc((void**)&Aimlololo_d,szmat);

   double *Arehihihi_h = new double[dim*dim];
   double *Arelohihi_h = new double[dim*dim];
   double *Arehilohi_h = new double[dim*dim];
   double *Arelolohi_h = new double[dim*dim];
   double *Arehihilo_h = new double[dim*dim];
   double *Arelohilo_h = new double[dim*dim];
   double *Arehilolo_h = new double[dim*dim];
   double *Arelololo_h = new double[dim*dim];
   double *Aimhihihi_h = new double[dim*dim];
   double *Aimlohihi_h = new double[dim*dim];
   double *Aimhilohi_h = new double[dim*dim];
   double *Aimlolohi_h = new double[dim*dim];
   double *Aimhihilo_h = new double[dim*dim];
   double *Aimlohilo_h = new double[dim*dim];
   double *Aimhilolo_h = new double[dim*dim];
   double *Aimlololo_h = new double[dim*dim];

   *add = 0; // initialize number of additions
   *mul = 0; // initialize number of multiplications

   if(vrblvl > 0)
      cout << "GPU_cmplx8_linear_residue for deg+1 : " << degp1 << endl;

   for(int i=tailidx; i<degp1; i++)  // compute i-th residual vector
   {
      hipMemcpy(rrehihihi_d,rhsrehihihi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rrelohihi_d,rhsrelohihi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rrehilohi_d,rhsrehilohi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rrelolohi_d,rhsrelolohi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rrehihilo_d,rhsrehihilo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rrelohilo_d,rhsrelohilo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rrehilolo_d,rhsrehilolo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rrelololo_d,rhsrelololo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rimhihihi_d,rhsimhihihi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rimlohihi_d,rhsimlohihi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rimhilohi_d,rhsimhilohi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rimlolohi_d,rhsimlolohi[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rimhihilo_d,rhsimhihilo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rimlohilo_d,rhsimlohilo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rimhilolo_d,rhsimhilolo[i],szrhs,hipMemcpyHostToDevice);
      hipMemcpy(rimlololo_d,rhsimlololo[i],szrhs,hipMemcpyHostToDevice);

      for(int j=0; j<=(i-tailidx); j++)  // multiply mat[j] with sol[i-j]
      {
         int idx=0;
         for(int i1=0; i1<dim; i1++)
            for(int j1=0; j1<dim; j1++)
            {
               Arehihihi_h[idx]   = matrehihihi[j][i1][j1];
               Arelohihi_h[idx]   = matrelohihi[j][i1][j1];
               Arehilohi_h[idx]   = matrehilohi[j][i1][j1];
               Arelolohi_h[idx]   = matrelolohi[j][i1][j1];
               Arehihilo_h[idx]   = matrehihilo[j][i1][j1];
               Arelohilo_h[idx]   = matrelohilo[j][i1][j1];
               Arehilolo_h[idx]   = matrehilolo[j][i1][j1];
               Arelololo_h[idx]   = matrelololo[j][i1][j1];
               Aimhihihi_h[idx]   = matimhihihi[j][i1][j1];
               Aimlohihi_h[idx]   = matimlohihi[j][i1][j1];
               Aimhilohi_h[idx]   = matimhilohi[j][i1][j1];
               Aimlolohi_h[idx]   = matimlolohi[j][i1][j1];
               Aimhihilo_h[idx]   = matimhihilo[j][i1][j1];
               Aimlohilo_h[idx]   = matimlohilo[j][i1][j1];
               Aimhilolo_h[idx]   = matimhilolo[j][i1][j1];
               Aimlololo_h[idx++] = matimlololo[j][i1][j1];
            }
      
         hipMemcpy(Arehihihi_d,Arehihihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arelohihi_d,Arelohihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arehilohi_d,Arehilohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arelolohi_d,Arelolohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arehihilo_d,Arehihilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arelohilo_d,Arelohilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arehilolo_d,Arehilolo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arelololo_d,Arelololo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimhihihi_d,Aimhihihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimlohihi_d,Aimlohihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimhilohi_d,Aimhilohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimlolohi_d,Aimlolohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimhihilo_d,Aimhihilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimlohilo_d,Aimlohilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimhilolo_d,Aimhilolo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimlololo_d,Aimlololo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(xrehihihi_d,solrehihihi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xrelohihi_d,solrelohihi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xrehilohi_d,solrehilohi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xrelolohi_d,solrelolohi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xrehihilo_d,solrehihilo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xrelohilo_d,solrelohilo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xrehilolo_d,solrehilolo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(xrelololo_d,solrelololo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(ximhihihi_d,solimhihihi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(ximlohihi_d,solimlohihi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(ximhilohi_d,solimhilohi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(ximlolohi_d,solimlolohi[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(ximhihilo_d,solimhihilo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(ximlohilo_d,solimlohilo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(ximhilolo_d,solimhilolo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(ximlololo_d,solimlololo[i-j],szsol,hipMemcpyHostToDevice);
         hipMemcpy(Arehihihi_d,Arehihihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arelohihi_d,Arelohihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arehilohi_d,Arehilohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arelolohi_d,Arelolohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arehihilo_d,Arehihilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arelohilo_d,Arelohilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arehilolo_d,Arehilolo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Arelololo_d,Arelololo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimhihihi_d,Aimhihihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimlohihi_d,Aimlohihi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimhilohi_d,Aimhilohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimlolohi_d,Aimlolohi_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimhihilo_d,Aimhihilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimlohilo_d,Aimlohilo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimhilolo_d,Aimhilolo_h,szmat,hipMemcpyHostToDevice);
         hipMemcpy(Aimlololo_d,Aimlololo_h,szmat,hipMemcpyHostToDevice);

         if(vrblvl > 1)
            cout << "GPU_cmplx8_linear_residue launches " << nbt
                 << " thread blocks in step " << i << ", " << j << endl;

         hipEvent_t start,stop;       // to measure time spent by kernels 
         hipEventCreate(&start);
         hipEventCreate(&stop);
         float milliseconds;

         hipEventRecord(start);
         cmplx8_bals_tail<<<nbt,szt>>>
            (dim,szt,Arehihihi_d,Arelohihi_d,Arehilohi_d,Arelolohi_d,
                     Arehihilo_d,Arelohilo_d,Arehilolo_d,Arelololo_d,
                     Aimhihihi_d,Aimlohihi_d,Aimhilohi_d,Aimlolohi_d,
                     Aimhihilo_d,Aimlohilo_d,Aimhilolo_d,Aimlololo_d,
                     xrehihihi_d,xrelohihi_d,xrehilohi_d,xrelolohi_d,
                     xrehihilo_d,xrelohilo_d,xrehilolo_d,xrelololo_d,
                     ximhihihi_d,ximlohihi_d,ximhilohi_d,ximlolohi_d,
                     ximhihilo_d,ximlohilo_d,ximhilolo_d,ximlololo_d,
                     rrehihihi_d,rrelohihi_d,rrehilohi_d,rrelolohi_d,
                     rrehihilo_d,rrelohilo_d,rrehilolo_d,rrelololo_d,
                     rimhihihi_d,rimlohihi_d,rimhilohi_d,rimlolohi_d,
                     rimhihilo_d,rimlohilo_d,rimhilolo_d,rimlololo_d);
         hipEventRecord(stop);
         hipEventSynchronize(stop);
         hipEventElapsedTime(&milliseconds,start,stop);
         *totreslapsedms += milliseconds;
         flopcount_cmplx_bals_tail(dim,add,mul);

         if(vrblvl > 0) write_dbl8_balsflops(1,dim,milliseconds);
      }
      hipMemcpy(resvecrehihihi[i],rrehihihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecrelohihi[i],rrelohihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecrehilohi[i],rrehilohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecrelolohi[i],rrelolohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecrehihilo[i],rrehihilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecrelohilo[i],rrelohilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecrehilolo[i],rrehilolo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecrelololo[i],rrelololo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecimhihihi[i],rimhihihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecimlohihi[i],rimlohihi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecimhilohi[i],rimhilohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecimlolohi[i],rimlolohi_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecimhihilo[i],rimhihilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecimlohilo[i],rimlohilo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecimhilolo[i],rimhilolo_d,szrhs,hipMemcpyDeviceToHost);
      hipMemcpy(resvecimlololo[i],rimlololo_d,szrhs,hipMemcpyDeviceToHost);
   }
   if(vrblvl > 1)
   {
      for(int i=tailidx; i<degp1; i++)
      {
         cout << "Solution vector " << i << " :" << endl;
         for(int j=0; j<dim; j++)
            cout << solrehihihi[i][j] << "  " << solrelohihi[i][j] << endl
                 << "  "
                 << solrehilohi[i][j] << "  " << solrelolohi[i][j] << endl
                 << "  "
                 << solrehihilo[i][j] << "  " << solrelohilo[i][j] << endl
                 << "  "
                 << solrehilolo[i][j] << "  " << solrelololo[i][j] << endl
                 << "  "
                 << solimhihihi[i][j] << "  " << solimlohihi[i][j] << endl
                 << "  "
                 << solimhilohi[i][j] << "  " << solimlolohi[i][j] << endl
                 << "  "
                 << solimhihilo[i][j] << "  " << solimlohilo[i][j] << endl
                 << "  "
                 << solimhilolo[i][j] << "  " << solimlololo[i][j] << endl;

         cout << "Residual vector " << i << " :" << endl;
         for(int j=0; j<dim; j++)
            cout << resvecrehihihi[i][j] << "  "
                 << resvecrelohihi[i][j] << endl << "  "
                 << resvecrehilohi[i][j] << "  "
                 << resvecrelolohi[i][j] << endl << "  "
                 << resvecrehihilo[i][j] << "  "
                 << resvecrelohilo[i][j] << endl << "  "
                 << resvecrehilolo[i][j] << "  "
                 << resvecrelololo[i][j] << endl << "  "
                 << resvecimhihihi[i][j] << "  "
                 << resvecimlohihi[i][j] << endl << "  "
                 << resvecimhilohi[i][j] << "  "
                 << resvecimlolohi[i][j] << endl << "  "
                 << resvecimhihilo[i][j] << "  "
                 << resvecimlohilo[i][j] << endl << "  "
                 << resvecimhilolo[i][j] << "  "
                 << resvecimlololo[i][j] << endl;
      }
   }
   *resmaxhihihi = 0.0; *resmaxlohihi = 0.0;
   *resmaxhilohi = 0.0; *resmaxlolohi = 0.0;
   *resmaxhihilo = 0.0; *resmaxlohilo = 0.0;
   *resmaxhilolo = 0.0; *resmaxlololo = 0.0;

   for(int i=tailidx; i<degp1; i++)
   {
      double *rirehihihi = resvecrehihihi[i];
      double *rirelohihi = resvecrelohihi[i];
      double *rirehilohi = resvecrehilohi[i];
      double *rirelolohi = resvecrelolohi[i];
      double *rirehihilo = resvecrehihilo[i];
      double *rirelohilo = resvecrelohilo[i];
      double *rirehilolo = resvecrehilolo[i];
      double *rirelololo = resvecrelololo[i];
      double *riimhihihi = resvecimhihihi[i];
      double *riimlohihi = resvecimlohihi[i];
      double *riimhilohi = resvecimhilohi[i];
      double *riimlolohi = resvecimlolohi[i];
      double *riimhihilo = resvecimhihilo[i];
      double *riimlohilo = resvecimlohilo[i];
      double *riimhilolo = resvecimhilolo[i];
      double *riimlololo = resvecimlololo[i];

      for(int j=0; j<dim; j++)
         if(abs(rirehihihi[j]) + abs(riimhihihi[j]) > *resmaxhihihi)
         {
            *resmaxhihihi = abs(rirehihihi[j]) + abs(riimhihihi[j]);
            *resmaxlohihi = abs(rirelohihi[j]) + abs(riimlohihi[j]);
            *resmaxhilohi = abs(rirehilohi[j]) + abs(riimhilohi[j]);
            *resmaxlolohi = abs(rirelolohi[j]) + abs(riimlolohi[j]);
            *resmaxhihilo = abs(rirehihilo[j]) + abs(riimhihilo[j]);
            *resmaxlohilo = abs(rirelohilo[j]) + abs(riimlohilo[j]);
            *resmaxhilolo = abs(rirehilolo[j]) + abs(riimhilolo[j]);
            *resmaxlololo = abs(rirelololo[j]) + abs(riimlololo[j]);
         }
   }
   free(Arehihihi_h); free(Arelohihi_h); free(Arehilohi_h); free(Arelolohi_h);
   free(Arehihilo_h); free(Arelohilo_h); free(Arehilolo_h); free(Arelololo_h);
   free(Aimhihihi_h); free(Aimlohihi_h); free(Aimhilohi_h); free(Aimlolohi_h);
   free(Aimhihilo_h); free(Aimlohilo_h); free(Aimhilolo_h); free(Aimlololo_h);

   hipFree(rrehihihi_d); hipFree(rrelohihi_d);
   hipFree(rrehilohi_d); hipFree(rrelolohi_d);
   hipFree(rrehihilo_d); hipFree(rrelohilo_d);
   hipFree(rrehilolo_d); hipFree(rrelololo_d);
   hipFree(rimhihihi_d); hipFree(rimlohihi_d);
   hipFree(rimhilohi_d); hipFree(rimlolohi_d);
   hipFree(rimhihilo_d); hipFree(rimlohilo_d);
   hipFree(rimhilolo_d); hipFree(rimlololo_d);
   hipFree(xrehihihi_d); hipFree(xrelohihi_d);
   hipFree(xrehilohi_d); hipFree(xrelolohi_d);
   hipFree(xrehihilo_d); hipFree(xrelohilo_d);
   hipFree(xrehilolo_d); hipFree(xrelololo_d);
   hipFree(ximhihihi_d); hipFree(ximlohihi_d);
   hipFree(ximhilohi_d); hipFree(ximlolohi_d);
   hipFree(ximhihilo_d); hipFree(ximlohilo_d);
   hipFree(ximhilolo_d); hipFree(ximlololo_d);
   hipFree(Arehihihi_d); hipFree(Arelohihi_d);
   hipFree(Arehilohi_d); hipFree(Arelolohi_d);
   hipFree(Arehihilo_d); hipFree(Arelohilo_d);
   hipFree(Arehilolo_d); hipFree(Arelololo_d);
   hipFree(Aimhihihi_d); hipFree(Aimlohihi_d);
   hipFree(Aimhilohi_d); hipFree(Aimlolohi_d);
   hipFree(Aimhihilo_d); hipFree(Aimlohilo_d);
   hipFree(Aimhilolo_d); hipFree(Aimlololo_d);
}
