#include "hip/hip_runtime.h"
// The file dbl_bals_kernels.cu defines the functions with prototypes in
// the file dbl_bals_kernels.h.

#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <cmath>
#include <hip/hip_vector_types.h>
#include "dbl_baqr_kernels.h"
#include "dbl_tabs_kernels.h"
#include "dbl_bals_kernels.h"

using namespace std;

__global__ void dbl_bals_tail
 ( int ncols, int szt, double *A, double *x, double *b )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int idx = bdx*szt + tdx; // thread tdx updates b[idx]

   double Aj;           // register for A[idx][j]
   double xj;           // register for hold x[j]
   double bi = b[idx];  // register for b[idx]

   int offset = idx*ncols;

   for(int j=0; j<ncols; j++)
   {
      Aj = A[offset+j];
      xj = x[j];
      bi = bi - Aj*xj;
   }
   b[idx] = bi;
}

__global__ void dbl_bals_qtb
 ( int ncols, int szt, double *Qt, double *b, double *r )
{
   const int bdx = blockIdx.x;
   const int tdx = threadIdx.x;
   const int idx = bdx*szt + tdx; // thread tdx updates b[idx]

   double Qj;           // register for Q^T[idx][j]
   double bj;           // register for hold b[j]
   double ri = 0.0;     // register for result, b[idx]

   int offset = idx*ncols;

   for(int j=0; j<ncols; j++)
   {
      Qj = Qt[offset+j];
      bj = b[j];
      ri = ri + Qj*bj;
   }
   r[idx] = ri;
}

void GPU_dbl_bals_head
 ( int nrows, int ncols, int szt, int nbt,
   double **A, double **Q, double **R, double *b, double *x, bool verbose )
{
   double qrtimelapsed_d;
   double houselapsedms,RTvlapsedms,tileRlapsedms,vb2Wlapsedms;
   double WYTlapsedms,QWYTlapsedms,Qaddlapsedms;
   double YWTlapsedms,YWTClapsedms,Raddlapsedms;
   long long int qraddcnt = 0;
   long long int qrmulcnt = 0;
   long long int qrdivcnt = 0;
   long long int sqrtcnt = 0;

   if(verbose > 0) 
      cout << "-> GPU computes the blocked Householder QR ..." << endl;

   GPU_dbl_blocked_houseqr
      (nrows,ncols,szt,nbt,A,Q,R,
       &houselapsedms,&RTvlapsedms,&tileRlapsedms,&vb2Wlapsedms,
       &WYTlapsedms,&QWYTlapsedms,&Qaddlapsedms,
       &YWTlapsedms,&YWTClapsedms,&Raddlapsedms,&qrtimelapsed_d,
       &qraddcnt,&qrmulcnt,&qrdivcnt,&sqrtcnt,verbose);

   double bstimelapsed_d;
   double elapsedms,invlapsed,mullapsed,sublapsed;
   long long int bsaddcnt = 0;
   long long int bsmulcnt = 0;
   long long int bsdivcnt = 0;

   if(verbose > 0)
      cout << "-> GPU solves an upper triangular system ..." << endl;

   if(verbose > 0)
   {
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++)
            cout << "R[" << i << "][" << j << "] : " << R[i][j] << endl;

      for(int i=0; i<nrows; i++)
         cout << "b[" << i << "] : " << b[i] << endl;
   }

   GPU_dbl_upper_tiled_solver
      (ncols,szt,nbt,R,b,x,
       &invlapsed,&mullapsed,&sublapsed,&elapsedms,&bstimelapsed_d,
       &bsaddcnt,&bsmulcnt,&bsdivcnt);
}

void GPU_dbl_bals_tail
 ( int nrows, int ncols, int szt, int nbt, int degp1, int stage,
   double ***mat, double **rhs, double **sol, bool verbose )
{
   if(verbose)
   {
      cout << "GPU_dbl_bals_tail input blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : " << rhs[k][i] << endl;
      }
   }

   double *b_d;
   const size_t szrhs = nrows*sizeof(double);
   hipMalloc((void**)&b_d,szrhs);

   double *x_d;
   const size_t szsol = ncols*sizeof(double);
   hipMalloc((void**)&x_d,szsol);
   hipMemcpy(x_d,&sol[stage-1],szsol,hipMemcpyHostToDevice);

   double *A_d;
   const size_t szmat = nrows*ncols*sizeof(double);
   hipMalloc((void**)&A_d,szmat);

   double *A_h = new double[szmat];

   for(int k=stage; k<degp1; k++)
   {
      if(verbose)
         cout << "GPU_dbl_bals_tail launches " << nbt
              << " thread blocks in step " << k-stage << endl;

      int idx=0;
      for(int i=0; i<nrows; i++)
         for(int j=0; j<ncols; j++) A_h[idx++] = mat[k][i][j];
      
      hipMemcpy(b_d,&(rhs[k][0]),szrhs,hipMemcpyHostToDevice);
      hipMemcpy(A_d,A_h,szmat,hipMemcpyHostToDevice);

      if(verbose)
         cout << "nbt = " << nbt << ", szt = " << szt
              << ", ncols = " << ncols << endl;

      dbl_bals_tail<<<nbt,szt>>>(ncols,szt,A_d,x_d,b_d);
      
      if(verbose)
         cout << "copying block " << k << " of right hand side ..." << endl;

      hipMemcpy(&(rhs[k][0]),b_d,szrhs,hipMemcpyDeviceToHost);
   }
   free(A_h);

   if(verbose)
   {
      cout << "GPU_dbl_bals_tail copied blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : " << rhs[k][i] << endl;
      }
   }
}

void GPU_dbl_bals_qtb
 ( int ncols, int szt, int nbt, double **Q, double *b, bool verbose )
{
   double *b_d;
   const size_t szrhs = ncols*sizeof(double);
   hipMalloc((void**)&b_d,szrhs);

   double *r_d;
   const size_t szsol = ncols*sizeof(double);
   hipMalloc((void**)&r_d,szsol);

   double *Qt_d;
   const size_t szmat = ncols*ncols*sizeof(double);
   hipMalloc((void**)&Qt_d,szmat);

   double *Qt_h = new double[szmat];

   int idx=0;
   for(int i=0; i<ncols; i++)
      for(int j=0; j<ncols; j++) Qt_h[idx++] = Q[j][i];

   hipMemcpy(b_d,b,szrhs,hipMemcpyHostToDevice);
   hipMemcpy(Qt_d,Qt_h,szmat,hipMemcpyHostToDevice);

   dbl_bals_qtb<<<nbt,szt>>>(ncols,szt,Qt_d,b_d,r_d);

   hipMemcpy(b,r_d,szrhs,hipMemcpyDeviceToHost);

   free(Qt_h);
}

void GPU_dbl_bals_solve
 ( int dim, int degp1, int szt, int nbt,
   double ***mat, double **rhs, double **sol, int vrblvl )
{
   const int nrows = dim;
   const int ncols = dim;
   const bool bvrb = (vrblvl > 0);

   double **A = new double*[nrows];
   double **Q = new double*[nrows];
   double **R = new double*[nrows];

   double *b = new double[nrows];
   double *x = new double[ncols];

   if(vrblvl)
   {
      cout << "GPU_dbl_bals_solve blocks of rhs :" << endl;
      for(int k=0; k<degp1; k++)
      {
         for(int i=0; i<nrows; i++)
            cout << "rhs[" << k << "][" << i << "] : " << rhs[k][i] << endl;
      }
   }

   for(int i=0; i<nrows; i++)
   {
      A[i] = new double[ncols];
      for(int j=0; j<ncols; j++) A[i][j] = mat[0][i][j];
      b[i] = rhs[0][i];
      Q[i] = new double[nrows];
      R[i] = new double[ncols];
      for(int j=0; j<ncols; j++) R[i][j] = mat[0][i][j];
   }

   GPU_dbl_bals_head(nrows,ncols,szt,nbt,A,Q,R,b,x,bvrb);

   for(int j=0; j<ncols; j++) sol[0][j] = x[j];

   for(int stage=1; stage<degp1; stage++)
   {
      if(vrblvl > 0)
         cout << "stage " << stage << " in solve tail ..." << endl;

      GPU_dbl_bals_tail(nrows,ncols,szt,nbt,degp1,stage,mat,rhs,sol,bvrb);

      if(vrblvl)
      {
         cout << "blocks of rhs before assignment :" << endl;
         for(int k=0; k<degp1; k++)
         {
            for(int i=0; i<nrows; i++)
               cout << "rhs[" << k << "][" << i << "] : " << rhs[k][i] << endl;
         }
      }

      for(int i=0; i<nrows; i++) 
      {
         cout << "assigning component " << i
              << ", stage = " << stage << endl;
         b[i] = rhs[stage][i];
      }
      double bstimelapsed_d;
      double elapsedms,invlapsed,mullapsed,sublapsed;
      long long int bsaddcnt = 0;
      long long int bsmulcnt = 0;
      long long int bsdivcnt = 0;

      if(bvrb > 0)
         cout << "-> GPU multiplies rhs with Q^T ..." << endl;

      GPU_dbl_bals_qtb(ncols,szt,nbt,Q,b,bvrb);

      if(bvrb > 0)
         cout << "-> GPU solves an upper triangular system ..." << endl;

      GPU_dbl_upper_tiled_solver
         (ncols,szt,nbt,R,b,x,
          &invlapsed,&mullapsed,&sublapsed,&elapsedms,&bstimelapsed_d,
          &bsaddcnt,&bsmulcnt,&bsdivcnt);

      for(int j=0; j<ncols; j++) sol[stage][j] = x[j];
   }

   for(int i=0; i<nrows; i++)
   {
      free(A[i]); free(Q[i]); free(R[i]);
   }
   free(A); free(Q); free(R); free(b); free(x);
}
